#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/limits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

////////////////////////////////////////////////////////////
// centeredGradient

namespace tvl1flow
{
    __global__ void centeredGradientKernel(const PtrStepSzf src, PtrStepf dx, PtrStepf dy)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= src.cols || y >= src.rows)
            return;

        dx(y, x) = 0.5f * (src(y, ::min(x + 1, src.cols - 1)) - src(y, ::max(x - 1, 0)));
        dy(y, x) = 0.5f * (src(::min(y + 1, src.rows - 1), x) - src(::max(y - 1, 0), x));
    }

    void centeredGradient(PtrStepSzf src, PtrStepSzf dx, PtrStepSzf dy, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

        centeredGradientKernel<<<grid, block, 0, stream>>>(src, dx, dy);
        cudaSafeCall( hipGetLastError() );

        if (!stream)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

////////////////////////////////////////////////////////////
// warpBackward

namespace tvl1flow
{
    static __device__ __forceinline__ float bicubicCoeff(float x_)
    {
        float x = fabsf(x_);
        if (x <= 1.0f)
        {
            return x * x * (1.5f * x - 2.5f) + 1.0f;
        }
        else if (x < 2.0f)
        {
            return x * (x * (-0.5f * x + 2.5f) - 4.0f) + 2.0f;
        }
        else
        {
            return 0.0f;
        }
    }

    texture<float, hipTextureType2D, hipReadModeElementType> tex_I1 (false, hipFilterModePoint, hipAddressModeClamp);
    texture<float, hipTextureType2D, hipReadModeElementType> tex_I1x(false, hipFilterModePoint, hipAddressModeClamp);
    texture<float, hipTextureType2D, hipReadModeElementType> tex_I1y(false, hipFilterModePoint, hipAddressModeClamp);

    __global__ void warpBackwardKernel(const PtrStepSzf I0, const PtrStepf u1, const PtrStepf u2, PtrStepf I1w, PtrStepf I1wx, PtrStepf I1wy, PtrStepf grad, PtrStepf rho)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= I0.cols || y >= I0.rows)
            return;

        const float u1Val = u1(y, x);
        const float u2Val = u2(y, x);

        const float wx = x + u1Val;
        const float wy = y + u2Val;

        const int xmin = ::ceilf(wx - 2.0f);
        const int xmax = ::floorf(wx + 2.0f);

        const int ymin = ::ceilf(wy - 2.0f);
        const int ymax = ::floorf(wy + 2.0f);

        float sum  = 0.0f;
        float sumx = 0.0f;
        float sumy = 0.0f;
        float wsum = 0.0f;

        for (int cy = ymin; cy <= ymax; ++cy)
        {
            for (int cx = xmin; cx <= xmax; ++cx)
            {
                const float w = bicubicCoeff(wx - cx) * bicubicCoeff(wy - cy);

                sum  += w * tex2D(tex_I1 , cx, cy);
                sumx += w * tex2D(tex_I1x, cx, cy);
                sumy += w * tex2D(tex_I1y, cx, cy);

                wsum += w;
            }
        }

        const float coeff = 1.0f / wsum;

        const float I1wVal  = sum  * coeff;
        const float I1wxVal = sumx * coeff;
        const float I1wyVal = sumy * coeff;

        I1w(y, x)  = I1wVal;
        I1wx(y, x) = I1wxVal;
        I1wy(y, x) = I1wyVal;

        const float Ix2 = I1wxVal * I1wxVal;
        const float Iy2 = I1wyVal * I1wyVal;

        // store the |Grad(I1)|^2
        grad(y, x) = Ix2 + Iy2;

        // compute the constant part of the rho function
        const float I0Val = I0(y, x);
        rho(y, x) = I1wVal - I1wxVal * u1Val - I1wyVal * u2Val - I0Val;
    }

    void warpBackward(PtrStepSzf I0, PtrStepSzf I1, PtrStepSzf I1x, PtrStepSzf I1y,
                      PtrStepSzf u1, PtrStepSzf u2, PtrStepSzf I1w, PtrStepSzf I1wx,
                      PtrStepSzf I1wy, PtrStepSzf grad, PtrStepSzf rho,
                      hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(I0.cols, block.x), divUp(I0.rows, block.y));

        bindTexture(&tex_I1 , I1);
        bindTexture(&tex_I1x, I1x);
        bindTexture(&tex_I1y, I1y);

        warpBackwardKernel<<<grid, block, 0, stream>>>(I0, u1, u2, I1w, I1wx, I1wy, grad, rho);
        cudaSafeCall( hipGetLastError() );

        if (!stream)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

////////////////////////////////////////////////////////////
// estimateU

namespace tvl1flow
{
    __device__ float divergence(const PtrStepf& v1, const PtrStepf& v2, int y, int x)
    {
        if (x > 0 && y > 0)
        {
            const float v1x = v1(y, x) - v1(y, x - 1);
            const float v2y = v2(y, x) - v2(y - 1, x);
            return v1x + v2y;
        }
        else
        {
            if (y > 0)
                return v1(y, 0) + v2(y, 0) - v2(y - 1, 0);
            else
            {
                if (x > 0)
                    return v1(0, x) - v1(0, x - 1) + v2(0, x);
                else
                    return v1(0, 0) + v2(0, 0);
            }
        }
    }

    __global__ void estimateUKernel(const PtrStepSzf I1wx, const PtrStepf I1wy,
                              const PtrStepf grad, const PtrStepf rho_c,
                              const PtrStepf p11, const PtrStepf p12,
                              const PtrStepf p21, const PtrStepf p22,
                              const PtrStepf p31, const PtrStepf p32,
                              PtrStepf u1, PtrStepf u2, PtrStepf u3, PtrStepf error,
                              const float l_t, const float theta, const float gamma, const bool calcError)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= I1wx.cols || y >= I1wx.rows)
            return;

        const float I1wxVal = I1wx(y, x);
        const float I1wyVal = I1wy(y, x);
        const float gradVal = grad(y, x);
        const float u1OldVal = u1(y, x);
        const float u2OldVal = u2(y, x);
        const float u3OldVal = gamma ? u3(y, x) : 0;

        const float rho = rho_c(y, x) + (I1wxVal * u1OldVal + I1wyVal * u2OldVal + gamma * u3OldVal);

        // estimate the values of the variable (v1, v2) (thresholding operator TH)

        float d1 = 0.0f;
        float d2 = 0.0f;
        float d3 = 0.0f;

        if (rho < -l_t * gradVal)
        {
            d1 = l_t * I1wxVal;
            d2 = l_t * I1wyVal;
            if (gamma)
                d3 = l_t * gamma;
        }
        else if (rho > l_t * gradVal)
        {
            d1 = -l_t * I1wxVal;
            d2 = -l_t * I1wyVal;
            if (gamma)
                d3 = -l_t * gamma;
        }
        else if (gradVal > numeric_limits<float>::epsilon())
        {
            const float fi = -rho / gradVal;
            d1 = fi * I1wxVal;
            d2 = fi * I1wyVal;
            if (gamma)
                d3 = fi * gamma;
        }

        const float v1 = u1OldVal + d1;
        const float v2 = u2OldVal + d2;
        const float v3 = u3OldVal + d3;

        // compute the divergence of the dual variable (p1, p2)

        const float div_p1 = divergence(p11, p12, y, x);
        const float div_p2 = divergence(p21, p22, y, x);
        const float div_p3 = gamma ? divergence(p31, p32, y, x) : 0;

        // estimate the values of the optical flow (u1, u2)

        const float u1NewVal = v1 + theta * div_p1;
        const float u2NewVal = v2 + theta * div_p2;
        const float u3NewVal = gamma ? v3 + theta * div_p3 : 0;

        u1(y, x) = u1NewVal;
        u2(y, x) = u2NewVal;
        if (gamma)
            u3(y, x) = u3NewVal;

        if (calcError)
        {
            const float n1 = (u1OldVal - u1NewVal) * (u1OldVal - u1NewVal);
            const float n2 = (u2OldVal - u2NewVal) * (u2OldVal - u2NewVal);
            error(y, x) = n1 + n2;
        }
    }

    void estimateU(PtrStepSzf I1wx, PtrStepSzf I1wy,
                   PtrStepSzf grad, PtrStepSzf rho_c,
                   PtrStepSzf p11, PtrStepSzf p12, PtrStepSzf p21, PtrStepSzf p22, PtrStepSzf p31, PtrStepSzf p32,
                   PtrStepSzf u1, PtrStepSzf u2, PtrStepSzf u3, PtrStepSzf error,
                   float l_t, float theta, float gamma, bool calcError,
                   hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(I1wx.cols, block.x), divUp(I1wx.rows, block.y));

        estimateUKernel<<<grid, block, 0, stream>>>(I1wx, I1wy, grad, rho_c, p11, p12, p21, p22, p31, p32, u1, u2, u3, error, l_t, theta, gamma, calcError);
        cudaSafeCall( hipGetLastError() );

        if (!stream)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

////////////////////////////////////////////////////////////
// estimateDualVariables

namespace tvl1flow
{
    __global__ void estimateDualVariablesKernel(const PtrStepSzf u1, const PtrStepf u2, const PtrStepSzf u3,
                                                PtrStepf p11, PtrStepf p12, PtrStepf p21, PtrStepf p22, PtrStepf p31, PtrStepf p32, const float taut, const float gamma)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= u1.cols || y >= u1.rows)
            return;

        const float u1x = u1(y, ::min(x + 1, u1.cols - 1)) - u1(y, x);
        const float u1y = u1(::min(y + 1, u1.rows - 1), x) - u1(y, x);

        const float u2x = u2(y, ::min(x + 1, u1.cols - 1)) - u2(y, x);
        const float u2y = u2(::min(y + 1, u1.rows - 1), x) - u2(y, x);

        const float u3x = gamma ? u3(y, ::min(x + 1, u1.cols - 1)) - u3(y, x) : 0;
        const float u3y = gamma ? u3(::min(y + 1, u1.rows - 1), x) - u3(y, x) : 0;

        const float g1 = ::hypotf(u1x, u1y);
        const float g2 = ::hypotf(u2x, u2y);
        const float g3 = gamma ? ::hypotf(u3x, u3y) : 0;

        const float ng1 = 1.0f + taut * g1;
        const float ng2 = 1.0f + taut * g2;
        const float ng3 = gamma ? 1.0f + taut * g3 : 0;

        p11(y, x) = (p11(y, x) + taut * u1x) / ng1;
        p12(y, x) = (p12(y, x) + taut * u1y) / ng1;
        p21(y, x) = (p21(y, x) + taut * u2x) / ng2;
        p22(y, x) = (p22(y, x) + taut * u2y) / ng2;
        if (gamma)
        {
            p31(y, x) = (p31(y, x) + taut * u3x) / ng3;
            p32(y, x) = (p32(y, x) + taut * u3y) / ng3;
        }
    }

    void estimateDualVariables(PtrStepSzf u1, PtrStepSzf u2, PtrStepSzf u3,
                               PtrStepSzf p11, PtrStepSzf p12, PtrStepSzf p21, PtrStepSzf p22, PtrStepSzf p31, PtrStepSzf p32,
                               float taut, float gamma,
                               hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(u1.cols, block.x), divUp(u1.rows, block.y));

        estimateDualVariablesKernel<<<grid, block, 0, stream>>>(u1, u2, u3, p11, p12, p21, p22, p31, p32, taut, gamma);
        cudaSafeCall( hipGetLastError() );

        if (!stream)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

#endif // !defined CUDA_DISABLER
