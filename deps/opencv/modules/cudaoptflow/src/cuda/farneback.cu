#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"

#define tx threadIdx.x
#define ty threadIdx.y
#define bx blockIdx.x
#define by blockIdx.y
#define bdx blockDim.x
#define bdy blockDim.y

#define BORDER_SIZE 5
#define MAX_KSIZE_HALF 100

namespace cv { namespace cuda { namespace device { namespace optflow_farneback
{
    __constant__ float c_g[8];
    __constant__ float c_xg[8];
    __constant__ float c_xxg[8];
    __constant__ float c_ig11, c_ig03, c_ig33, c_ig55;


    template <int polyN>
    __global__ void polynomialExpansion(
            const int height, const int width, const PtrStepf src, PtrStepf dst)
    {
        const int y = by * bdy + ty;
        const int x = bx * (bdx - 2*polyN) + tx - polyN;

        if (y < height)
        {
            extern __shared__ float smem[];
            volatile float *row = smem + tx;
            int xWarped = ::min(::max(x, 0), width - 1);

            row[0] = src(y, xWarped) * c_g[0];
            row[bdx] = 0.f;
            row[2*bdx] = 0.f;

            for (int k = 1; k <= polyN; ++k)
            {
                float t0 = src(::max(y - k, 0), xWarped);
                float t1 = src(::min(y + k, height - 1), xWarped);

                row[0] += c_g[k] * (t0 + t1);
                row[bdx] += c_xg[k] * (t1 - t0);
                row[2*bdx] += c_xxg[k] * (t0 + t1);
            }

            __syncthreads();

            if (tx >= polyN && tx + polyN < bdx && x < width)
            {
                float b1 = c_g[0] * row[0];
                float b3 = c_g[0] * row[bdx];
                float b5 = c_g[0] * row[2*bdx];
                float b2 = 0, b4 = 0, b6 = 0;

                for (int k = 1; k <= polyN; ++k)
                {
                    b1 += (row[k] + row[-k]) * c_g[k];
                    b4 += (row[k] + row[-k]) * c_xxg[k];
                    b2 += (row[k] - row[-k]) * c_xg[k];
                    b3 += (row[k + bdx] + row[-k + bdx]) * c_g[k];
                    b6 += (row[k + bdx] - row[-k + bdx]) * c_xg[k];
                    b5 += (row[k + 2*bdx] + row[-k + 2*bdx]) * c_g[k];
                }

                dst(y, xWarped) = b3*c_ig11;
                dst(height + y, xWarped) = b2*c_ig11;
                dst(2*height + y, xWarped) = b1*c_ig03 + b5*c_ig33;
                dst(3*height + y, xWarped) = b1*c_ig03 + b4*c_ig33;
                dst(4*height + y, xWarped) = b6*c_ig55;
            }
        }
    }


    void setPolynomialExpansionConsts(
            int polyN, const float *g, const float *xg, const float *xxg,
            float ig11, float ig03, float ig33, float ig55)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_g), g, (polyN + 1) * sizeof(*g)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_xg), xg, (polyN + 1) * sizeof(*xg)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_xxg), xxg, (polyN + 1) * sizeof(*xxg)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ig11), &ig11, sizeof(ig11)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ig03), &ig03, sizeof(ig03)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ig33), &ig33, sizeof(ig33)));
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ig55), &ig55, sizeof(ig55)));
    }


    void polynomialExpansionGpu(const PtrStepSzf &src, int polyN, PtrStepSzf dst, hipStream_t stream)
    {
        dim3 block(256);
        dim3 grid(divUp(src.cols, block.x - 2*polyN), src.rows);
        int smem = 3 * block.x * sizeof(float);

        if (polyN == 5)
            polynomialExpansion<5><<<grid, block, smem, stream>>>(src.rows, src.cols, src, dst);
        else if (polyN == 7)
            polynomialExpansion<7><<<grid, block, smem, stream>>>(src.rows, src.cols, src, dst);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    __constant__ float c_border[BORDER_SIZE + 1];

    __global__ void updateMatrices(
            const int height, const int width, const PtrStepf flowx, const PtrStepf flowy,
            const PtrStepf R0, const PtrStepf R1, PtrStepf M)
    {
        const int y = by * bdy + ty;
        const int x = bx * bdx + tx;

        if (y < height && x < width)
        {
            float dx = flowx(y, x);
            float dy = flowy(y, x);
            float fx = x + dx;
            float fy = y + dy;

            int x1 = floorf(fx);
            int y1 = floorf(fy);
            fx -= x1; fy -= y1;

            float r2, r3, r4, r5, r6;

            if (x1 >= 0 && y1 >= 0 && x1 < width - 1 && y1 < height - 1)
            {
                float a00 = (1.f - fx) * (1.f - fy);
                float a01 = fx * (1.f - fy);
                float a10 = (1.f - fx) * fy;
                float a11 = fx * fy;

                r2 = a00 * R1(y1, x1) +
                     a01 * R1(y1, x1 + 1) +
                     a10 * R1(y1 + 1, x1) +
                     a11 * R1(y1 + 1, x1 + 1);

                r3 = a00 * R1(height + y1, x1) +
                     a01 * R1(height + y1, x1 + 1) +
                     a10 * R1(height + y1 + 1, x1) +
                     a11 * R1(height + y1 + 1, x1 + 1);

                r4 = a00 * R1(2*height + y1, x1) +
                     a01 * R1(2*height + y1, x1 + 1) +
                     a10 * R1(2*height + y1 + 1, x1) +
                     a11 * R1(2*height + y1 + 1, x1 + 1);

                r5 = a00 * R1(3*height + y1, x1) +
                     a01 * R1(3*height + y1, x1 + 1) +
                     a10 * R1(3*height + y1 + 1, x1) +
                     a11 * R1(3*height + y1 + 1, x1 + 1);

                r6 = a00 * R1(4*height + y1, x1) +
                     a01 * R1(4*height + y1, x1 + 1) +
                     a10 * R1(4*height + y1 + 1, x1) +
                     a11 * R1(4*height + y1 + 1, x1 + 1);

                r4 = (R0(2*height + y, x) + r4) * 0.5f;
                r5 = (R0(3*height + y, x) + r5) * 0.5f;
                r6 = (R0(4*height + y, x) + r6) * 0.25f;
            }
            else
            {
                r2 = r3 = 0.f;
                r4 = R0(2*height + y, x);
                r5 = R0(3*height + y, x);
                r6 = R0(4*height + y, x) * 0.5f;
            }

            r2 = (R0(y, x) - r2) * 0.5f;
            r3 = (R0(height + y, x) - r3) * 0.5f;

            r2 += r4*dy + r6*dx;
            r3 += r6*dy + r5*dx;

            float scale =
                    c_border[::min(x, BORDER_SIZE)] *
                    c_border[::min(y, BORDER_SIZE)] *
                    c_border[::min(width - x - 1, BORDER_SIZE)] *
                    c_border[::min(height - y - 1, BORDER_SIZE)];

            r2 *= scale; r3 *= scale; r4 *= scale;
            r5 *= scale; r6 *= scale;

            M(y, x) = r4*r4 + r6*r6;
            M(height + y, x) = (r4 + r5)*r6;
            M(2*height + y, x) = r5*r5 + r6*r6;
            M(3*height + y, x) = r4*r2 + r6*r3;
            M(4*height + y, x) = r6*r2 + r5*r3;
        }
    }


    void setUpdateMatricesConsts()
    {
        static const float border[BORDER_SIZE + 1] = {0.14f, 0.14f, 0.4472f, 0.4472f, 0.4472f, 1.f};
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_border), border, (BORDER_SIZE + 1) * sizeof(*border)));
    }


    void updateMatricesGpu(
            const PtrStepSzf flowx, const PtrStepSzf flowy, const PtrStepSzf R0, const PtrStepSzf R1,
            PtrStepSzf M, hipStream_t stream)
    {
        dim3 block(32, 8);
        dim3 grid(divUp(flowx.cols, block.x), divUp(flowx.rows, block.y));

        updateMatrices<<<grid, block, 0, stream>>>(flowx.rows, flowx.cols, flowx, flowy, R0, R1, M);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    __global__ void updateFlow(
            const int height, const int width, const PtrStepf M, PtrStepf flowx, PtrStepf flowy)
    {
        const int y = by * bdy + ty;
        const int x = bx * bdx + tx;

        if (y < height && x < width)
        {
            float g11 = M(y, x);
            float g12 = M(height + y, x);
            float g22 = M(2*height + y, x);
            float h1 = M(3*height + y, x);
            float h2 = M(4*height + y, x);

            float detInv = 1.f / (g11*g22 - g12*g12 + 1e-3f);

            flowx(y, x) = (g11*h2 - g12*h1) * detInv;
            flowy(y, x) = (g22*h1 - g12*h2) * detInv;
        }
    }


    void updateFlowGpu(const PtrStepSzf M, PtrStepSzf flowx, PtrStepSzf flowy, hipStream_t stream)
    {
        dim3 block(32, 8);
        dim3 grid(divUp(flowx.cols, block.x), divUp(flowx.rows, block.y));

        updateFlow<<<grid, block, 0, stream>>>(flowx.rows, flowx.cols, M, flowx, flowy);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    /*__global__ void boxFilter(
            const int height, const int width, const PtrStepf src,
            const int ksizeHalf, const float boxAreaInv, PtrStepf dst)
    {
        const int y = by * bdy + ty;
        const int x = bx * bdx + tx;

        extern __shared__ float smem[];
        volatile float *row = smem + ty * (bdx + 2*ksizeHalf);

        if (y < height)
        {
            // Vertical pass
            for (int i = tx; i < bdx + 2*ksizeHalf; i += bdx)
            {
                int xExt = int(bx * bdx) + i - ksizeHalf;
                xExt = ::min(::max(xExt, 0), width - 1);

                row[i] = src(y, xExt);
                for (int j = 1; j <= ksizeHalf; ++j)
                    row[i] += src(::max(y - j, 0), xExt) + src(::min(y + j, height - 1), xExt);
            }

            if (x < width)
            {
                __syncthreads();

                // Horizontal passs
                row += tx + ksizeHalf;
                float res = row[0];
                for (int i = 1; i <= ksizeHalf; ++i)
                    res += row[-i] + row[i];
                dst(y, x) = res * boxAreaInv;
            }
        }
    }


    void boxFilterGpu(const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, hipStream_t stream)
    {
        dim3 block(256);
        dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));
        int smem = (block.x + 2*ksizeHalf) * block.y * sizeof(float);

        float boxAreaInv = 1.f / ((1 + 2*ksizeHalf) * (1 + 2*ksizeHalf));
        boxFilter<<<grid, block, smem, stream>>>(src.rows, src.cols, src, ksizeHalf, boxAreaInv, dst);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }*/


    __global__ void boxFilter5(
            const int height, const int width, const PtrStepf src,
            const int ksizeHalf, const float boxAreaInv, PtrStepf dst)
    {
        const int y = by * bdy + ty;
        const int x = bx * bdx + tx;

        extern __shared__ float smem[];

        const int smw = bdx + 2*ksizeHalf; // shared memory "width"
        volatile float *row = smem + 5 * ty * smw;

        if (y < height)
        {
            // Vertical pass
            for (int i = tx; i < bdx + 2*ksizeHalf; i += bdx)
            {
                int xExt = int(bx * bdx) + i - ksizeHalf;
                xExt = ::min(::max(xExt, 0), width - 1);

                #pragma unroll
                for (int k = 0; k < 5; ++k)
                    row[k*smw + i] = src(k*height + y, xExt);

                for (int j = 1; j <= ksizeHalf; ++j)
                    #pragma unroll
                    for (int k = 0; k < 5; ++k)
                        row[k*smw + i] +=
                                src(k*height + ::max(y - j, 0), xExt) +
                                src(k*height + ::min(y + j, height - 1), xExt);
            }

            if (x < width)
            {
                __syncthreads();

                // Horizontal passs

                row += tx + ksizeHalf;
                float res[5];

                #pragma unroll
                for (int k = 0; k < 5; ++k)
                    res[k] = row[k*smw];

                for (int i = 1; i <= ksizeHalf; ++i)
                    #pragma unroll
                    for (int k = 0; k < 5; ++k)
                        res[k] += row[k*smw - i] + row[k*smw + i];

                #pragma unroll
                for (int k = 0; k < 5; ++k)
                    dst(k*height + y, x) = res[k] * boxAreaInv;
            }
        }
    }


    void boxFilter5Gpu(const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, hipStream_t stream)
    {
        int height = src.rows / 5;
        int width = src.cols;

        dim3 block(256);
        dim3 grid(divUp(width, block.x), divUp(height, block.y));
        int smem = (block.x + 2*ksizeHalf) * 5 * block.y * sizeof(float);

        float boxAreaInv = 1.f / ((1 + 2*ksizeHalf) * (1 + 2*ksizeHalf));
        boxFilter5<<<grid, block, smem, stream>>>(height, width, src, ksizeHalf, boxAreaInv, dst);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void boxFilter5Gpu_CC11(const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, hipStream_t stream)
    {
        int height = src.rows / 5;
        int width = src.cols;

        dim3 block(128);
        dim3 grid(divUp(width, block.x), divUp(height, block.y));
        int smem = (block.x + 2*ksizeHalf) * 5 * block.y * sizeof(float);

        float boxAreaInv = 1.f / ((1 + 2*ksizeHalf) * (1 + 2*ksizeHalf));
        boxFilter5<<<grid, block, smem, stream>>>(height, width, src, ksizeHalf, boxAreaInv, dst);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    __constant__ float c_gKer[MAX_KSIZE_HALF + 1];

    template <typename Border>
    __global__ void gaussianBlur(
            const int height, const int width, const PtrStepf src, const int ksizeHalf,
            const Border b, PtrStepf dst)
    {
        const int y = by * bdy + ty;
        const int x = bx * bdx + tx;

        extern __shared__ float smem[];
        volatile float *row = smem + ty * (bdx + 2*ksizeHalf);

        if (y < height)
        {
            // Vertical pass
            for (int i = tx; i < bdx + 2*ksizeHalf; i += bdx)
            {
                int xExt = int(bx * bdx) + i - ksizeHalf;
                xExt = b.idx_col(xExt);
                row[i] = src(y, xExt) * c_gKer[0];
                for (int j = 1; j <= ksizeHalf; ++j)
                    row[i] +=
                            (src(b.idx_row_low(y - j), xExt) +
                             src(b.idx_row_high(y + j), xExt)) * c_gKer[j];
            }

            if (x < width)
            {
                __syncthreads();

                // Horizontal pass
                row += tx + ksizeHalf;
                float res = row[0] * c_gKer[0];
                for (int i = 1; i <= ksizeHalf; ++i)
                    res += (row[-i] + row[i]) * c_gKer[i];
                dst(y, x) = res;
            }
        }
    }


    void setGaussianBlurKernel(const float *gKer, int ksizeHalf)
    {
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_gKer), gKer, (ksizeHalf + 1) * sizeof(*gKer)));
    }


    template <typename Border>
    void gaussianBlurCaller(const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, hipStream_t stream)
    {
        int height = src.rows;
        int width = src.cols;

        dim3 block(256);
        dim3 grid(divUp(width, block.x), divUp(height, block.y));
        int smem = (block.x + 2*ksizeHalf) * block.y * sizeof(float);
        Border b(height, width);

        gaussianBlur<<<grid, block, smem, stream>>>(height, width, src, ksizeHalf, b, dst);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void gaussianBlurGpu(
            const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, int borderMode, hipStream_t stream)
    {
        typedef void (*caller_t)(const PtrStepSzf, int, PtrStepSzf, hipStream_t);

        static const caller_t callers[] =
        {
            0 /*gaussianBlurCaller<BrdConstant<float> >*/,
            gaussianBlurCaller<BrdReplicate<float> >,
            0 /*gaussianBlurCaller<BrdReflect<float> >*/,
            0 /*gaussianBlurCaller<BrdWrap<float> >*/,
            gaussianBlurCaller<BrdReflect101<float> >
        };

        callers[borderMode](src, ksizeHalf, dst, stream);
    }


    template <typename Border>
    __global__ void gaussianBlur5(
            const int height, const int width, const PtrStepf src, const int ksizeHalf,
            const Border b, PtrStepf dst)
    {
        const int y = by * bdy + ty;
        const int x = bx * bdx + tx;

        extern __shared__ float smem[];

        const int smw = bdx + 2*ksizeHalf; // shared memory "width"
        volatile float *row = smem + 5 * ty * smw;

        if (y < height)
        {
            // Vertical pass
            for (int i = tx; i < bdx + 2*ksizeHalf; i += bdx)
            {
                int xExt = int(bx * bdx) + i - ksizeHalf;
                xExt = b.idx_col(xExt);

                #pragma unroll
                for (int k = 0; k < 5; ++k)
                    row[k*smw + i] = src(k*height + y, xExt) * c_gKer[0];

                for (int j = 1; j <= ksizeHalf; ++j)
                    #pragma unroll
                    for (int k = 0; k < 5; ++k)
                        row[k*smw + i] +=
                                (src(k*height + b.idx_row_low(y - j), xExt) +
                                 src(k*height + b.idx_row_high(y + j), xExt)) * c_gKer[j];
            }

            if (x < width)
            {
                __syncthreads();

                // Horizontal pass

                row += tx + ksizeHalf;
                float res[5];

                #pragma unroll
                for (int k = 0; k < 5; ++k)
                    res[k] = row[k*smw] * c_gKer[0];

                for (int i = 1; i <= ksizeHalf; ++i)
                    #pragma unroll
                    for (int k = 0; k < 5; ++k)
                        res[k] += (row[k*smw - i] + row[k*smw + i]) * c_gKer[i];

                #pragma unroll
                for (int k = 0; k < 5; ++k)
                    dst(k*height + y, x) = res[k];
            }
        }
    }


    template <typename Border, int blockDimX>
    void gaussianBlur5Caller(
            const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, hipStream_t stream)
    {
        int height = src.rows / 5;
        int width = src.cols;

        dim3 block(blockDimX);
        dim3 grid(divUp(width, block.x), divUp(height, block.y));
        int smem = (block.x + 2*ksizeHalf) * 5 * block.y * sizeof(float);
        Border b(height, width);

        gaussianBlur5<<<grid, block, smem, stream>>>(height, width, src, ksizeHalf, b, dst);

        cudaSafeCall(hipGetLastError());

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    void gaussianBlur5Gpu(
            const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, int borderMode, hipStream_t stream)
    {
        typedef void (*caller_t)(const PtrStepSzf, int, PtrStepSzf, hipStream_t);

        static const caller_t callers[] =
        {
            0 /*gaussianBlur5Caller<BrdConstant<float>,256>*/,
            gaussianBlur5Caller<BrdReplicate<float>,256>,
            0 /*gaussianBlur5Caller<BrdReflect<float>,256>*/,
            0 /*gaussianBlur5Caller<BrdWrap<float>,256>*/,
            gaussianBlur5Caller<BrdReflect101<float>,256>
        };

        callers[borderMode](src, ksizeHalf, dst, stream);
    }

    void gaussianBlur5Gpu_CC11(
            const PtrStepSzf src, int ksizeHalf, PtrStepSzf dst, int borderMode, hipStream_t stream)
    {
        typedef void (*caller_t)(const PtrStepSzf, int, PtrStepSzf, hipStream_t);

        static const caller_t callers[] =
        {
            0 /*gaussianBlur5Caller<BrdConstant<float>,128>*/,
            gaussianBlur5Caller<BrdReplicate<float>,128>,
            0 /*gaussianBlur5Caller<BrdReflect<float>,128>*/,
            0 /*gaussianBlur5Caller<BrdWrap<float>,128>*/,
            gaussianBlur5Caller<BrdReflect101<float>,128>
        };

        callers[borderMode](src, ksizeHalf, dst, stream);
    }

}}}} // namespace cv { namespace cuda { namespace cudev { namespace optflow_farneback


#endif /* CUDA_DISABLER */
