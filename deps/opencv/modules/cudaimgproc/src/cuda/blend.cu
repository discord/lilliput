#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace blend
    {
        template <typename T>
        __global__ void blendLinearKernel(int rows, int cols, int cn, const PtrStep<T> img1, const PtrStep<T> img2,
                                          const PtrStepf weights1, const PtrStepf weights2, PtrStep<T> result)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < rows && x < cols)
            {
                int x_ = x / cn;
                float w1 = weights1.ptr(y)[x_];
                float w2 = weights2.ptr(y)[x_];
                T p1 = img1.ptr(y)[x];
                T p2 = img2.ptr(y)[x];
                result.ptr(y)[x] = (p1 * w1 + p2 * w2) / (w1 + w2 + 1e-5f);
            }
        }

        template <typename T>
        void blendLinearCaller(int rows, int cols, int cn, PtrStep<T> img1, PtrStep<T> img2, PtrStepf weights1, PtrStepf weights2, PtrStep<T> result, hipStream_t stream)
        {
            dim3 threads(16, 16);
            dim3 grid(divUp(cols * cn, threads.x), divUp(rows, threads.y));

            blendLinearKernel<<<grid, threads, 0, stream>>>(rows, cols * cn, cn, img1, img2, weights1, weights2, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }

        template void blendLinearCaller<uchar>(int, int, int, PtrStep<uchar>, PtrStep<uchar>, PtrStepf, PtrStepf, PtrStep<uchar>, hipStream_t stream);
        template void blendLinearCaller<float>(int, int, int, PtrStep<float>, PtrStep<float>, PtrStepf, PtrStepf, PtrStep<float>, hipStream_t stream);


        __global__ void blendLinearKernel8UC4(int rows, int cols, const PtrStepb img1, const PtrStepb img2,
                                              const PtrStepf weights1, const PtrStepf weights2, PtrStepb result)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < rows && x < cols)
            {
                float w1 = weights1.ptr(y)[x];
                float w2 = weights2.ptr(y)[x];
                float sum_inv = 1.f / (w1 + w2 + 1e-5f);
                w1 *= sum_inv;
                w2 *= sum_inv;
                uchar4 p1 = ((const uchar4*)img1.ptr(y))[x];
                uchar4 p2 = ((const uchar4*)img2.ptr(y))[x];
                ((uchar4*)result.ptr(y))[x] = make_uchar4(p1.x * w1 + p2.x * w2, p1.y * w1 + p2.y * w2,
                                                          p1.z * w1 + p2.z * w2, p1.w * w1 + p2.w * w2);
            }
        }

        void blendLinearCaller8UC4(int rows, int cols, PtrStepb img1, PtrStepb img2, PtrStepf weights1, PtrStepf weights2, PtrStepb result, hipStream_t stream)
        {
            dim3 threads(16, 16);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            blendLinearKernel8UC4<<<grid, threads, 0, stream>>>(rows, cols, img1, img2, weights1, weights2, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    } // namespace blend
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
