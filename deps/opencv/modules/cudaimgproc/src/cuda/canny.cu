#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/utility.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace canny
{
    struct L1 : binary_function<int, int, float>
    {
        __device__ __forceinline__ float operator ()(int x, int y) const
        {
            return ::abs(x) + ::abs(y);
        }

        __host__ __device__ __forceinline__ L1() {}
        __host__ __device__ __forceinline__ L1(const L1&) {}
    };
    struct L2 : binary_function<int, int, float>
    {
        __device__ __forceinline__ float operator ()(int x, int y) const
        {
            return ::sqrtf(x * x + y * y);
        }

        __host__ __device__ __forceinline__ L2() {}
        __host__ __device__ __forceinline__ L2(const L2&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<canny::L1> : DefaultTransformFunctorTraits<canny::L1>
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits<canny::L2> : DefaultTransformFunctorTraits<canny::L2>
    {
        enum { smart_shift = 4 };
    };
}}}

namespace canny
{
    texture<uchar, hipTextureType2D, hipReadModeElementType> tex_src(false, hipFilterModePoint, hipAddressModeClamp);
    struct SrcTex
    {
        int xoff;
        int yoff;
        __host__ SrcTex(int _xoff, int _yoff) : xoff(_xoff), yoff(_yoff) {}

        __device__ __forceinline__ int operator ()(int y, int x) const
        {
            return tex2D(tex_src, x + xoff, y + yoff);
        }
    };

    template <class Norm> __global__
    void calcMagnitudeKernel(const SrcTex src, PtrStepi dx, PtrStepi dy, PtrStepSzf mag, const Norm norm)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y >= mag.rows || x >= mag.cols)
            return;

        int dxVal = (src(y - 1, x + 1) + 2 * src(y, x + 1) + src(y + 1, x + 1)) - (src(y - 1, x - 1) + 2 * src(y, x - 1) + src(y + 1, x - 1));
        int dyVal = (src(y + 1, x - 1) + 2 * src(y + 1, x) + src(y + 1, x + 1)) - (src(y - 1, x - 1) + 2 * src(y - 1, x) + src(y - 1, x + 1));

        dx(y, x) = dxVal;
        dy(y, x) = dyVal;

        mag(y, x) = norm(dxVal, dyVal);
    }

    void calcMagnitude(PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzi dx, PtrStepSzi dy, PtrStepSzf mag, bool L2Grad, hipStream_t stream)
    {
        const dim3 block(16, 16);
        const dim3 grid(divUp(mag.cols, block.x), divUp(mag.rows, block.y));

        bindTexture(&tex_src, srcWhole);
        SrcTex src(xoff, yoff);

        if (L2Grad)
        {
            L2 norm;
            calcMagnitudeKernel<<<grid, block, 0, stream>>>(src, dx, dy, mag, norm);
        }
        else
        {
            L1 norm;
            calcMagnitudeKernel<<<grid, block, 0, stream>>>(src, dx, dy, mag, norm);
        }

        cudaSafeCall( hipGetLastError() );

        if (stream == NULL)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void calcMagnitude(PtrStepSzi dx, PtrStepSzi dy, PtrStepSzf mag, bool L2Grad, hipStream_t stream)
    {
        if (L2Grad)
        {
            L2 norm;
            transform(dx, dy, mag, norm, WithOutMask(), stream);
        }
        else
        {
            L1 norm;
            transform(dx, dy, mag, norm, WithOutMask(), stream);
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    texture<float, hipTextureType2D, hipReadModeElementType> tex_mag(false, hipFilterModePoint, hipAddressModeClamp);

    __global__ void calcMapKernel(const PtrStepSzi dx, const PtrStepi dy, PtrStepi map, const float low_thresh, const float high_thresh)
    {
        const int CANNY_SHIFT = 15;
        const int TG22 = (int)(0.4142135623730950488016887242097*(1<<CANNY_SHIFT) + 0.5);

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x == 0 || x >= dx.cols - 1 || y == 0 || y >= dx.rows - 1)
            return;

        int dxVal = dx(y, x);
        int dyVal = dy(y, x);

        const int s = (dxVal ^ dyVal) < 0 ? -1 : 1;
        const float m = tex2D(tex_mag, x, y);

        dxVal = ::abs(dxVal);
        dyVal = ::abs(dyVal);

        // 0 - the pixel can not belong to an edge
        // 1 - the pixel might belong to an edge
        // 2 - the pixel does belong to an edge
        int edge_type = 0;

        if (m > low_thresh)
        {
            const int tg22x = dxVal * TG22;
            const int tg67x = tg22x + ((dxVal + dxVal) << CANNY_SHIFT);

            dyVal <<= CANNY_SHIFT;

            if (dyVal < tg22x)
            {
                if (m > tex2D(tex_mag, x - 1, y) && m >= tex2D(tex_mag, x + 1, y))
                    edge_type = 1 + (int)(m > high_thresh);
            }
            else if(dyVal > tg67x)
            {
                if (m > tex2D(tex_mag, x, y - 1) && m >= tex2D(tex_mag, x, y + 1))
                    edge_type = 1 + (int)(m > high_thresh);
            }
            else
            {
                if (m > tex2D(tex_mag, x - s, y - 1) && m >= tex2D(tex_mag, x + s, y + 1))
                    edge_type = 1 + (int)(m > high_thresh);
            }
        }

        map(y, x) = edge_type;
    }

    void calcMap(PtrStepSzi dx, PtrStepSzi dy, PtrStepSzf mag, PtrStepSzi map, float low_thresh, float high_thresh, hipStream_t stream)
    {
        const dim3 block(16, 16);
        const dim3 grid(divUp(dx.cols, block.x), divUp(dx.rows, block.y));

        bindTexture(&tex_mag, mag);

        calcMapKernel<<<grid, block, 0, stream>>>(dx, dy, map, low_thresh, high_thresh);
        cudaSafeCall( hipGetLastError() );

        if (stream == NULL)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __device__ int counter = 0;

    __device__ __forceinline__ bool checkIdx(int y, int x, int rows, int cols)
    {
        return (y >= 0) && (y < rows) && (x >= 0) && (x < cols);
    }

    __global__ void edgesHysteresisLocalKernel(PtrStepSzi map, short2* st)
    {
        __shared__ volatile int smem[18][18];

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        smem[threadIdx.y + 1][threadIdx.x + 1] = checkIdx(y, x, map.rows, map.cols) ? map(y, x) : 0;
        if (threadIdx.y == 0)
            smem[0][threadIdx.x + 1] = checkIdx(y - 1, x, map.rows, map.cols) ? map(y - 1, x) : 0;
        if (threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][threadIdx.x + 1] = checkIdx(y + 1, x, map.rows, map.cols) ? map(y + 1, x) : 0;
        if (threadIdx.x == 0)
            smem[threadIdx.y + 1][0] = checkIdx(y, x - 1, map.rows, map.cols) ? map(y, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1)
            smem[threadIdx.y + 1][blockDim.x + 1] = checkIdx(y, x + 1, map.rows, map.cols) ? map(y, x + 1) : 0;
        if (threadIdx.x == 0 && threadIdx.y == 0)
            smem[0][0] = checkIdx(y - 1, x - 1, map.rows, map.cols) ? map(y - 1, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0)
            smem[0][blockDim.x + 1] = checkIdx(y - 1, x + 1, map.rows, map.cols) ? map(y - 1, x + 1) : 0;
        if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][0] = checkIdx(y + 1, x - 1, map.rows, map.cols) ? map(y + 1, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][blockDim.x + 1] = checkIdx(y + 1, x + 1, map.rows, map.cols) ? map(y + 1, x + 1) : 0;

        __syncthreads();

        if (x >= map.cols || y >= map.rows)
            return;

        int n;

        #pragma unroll
        for (int k = 0; k < 16; ++k)
        {
            n = 0;

            if (smem[threadIdx.y + 1][threadIdx.x + 1] == 1)
            {
                n += smem[threadIdx.y    ][threadIdx.x    ] == 2;
                n += smem[threadIdx.y    ][threadIdx.x + 1] == 2;
                n += smem[threadIdx.y    ][threadIdx.x + 2] == 2;

                n += smem[threadIdx.y + 1][threadIdx.x    ] == 2;
                n += smem[threadIdx.y + 1][threadIdx.x + 2] == 2;

                n += smem[threadIdx.y + 2][threadIdx.x    ] == 2;
                n += smem[threadIdx.y + 2][threadIdx.x + 1] == 2;
                n += smem[threadIdx.y + 2][threadIdx.x + 2] == 2;
            }

            __syncthreads();

            if (n > 0)
                smem[threadIdx.y + 1][threadIdx.x + 1] = 2;

            __syncthreads();
        }

        const int e = smem[threadIdx.y + 1][threadIdx.x + 1];

        map(y, x) = e;

        n = 0;

        if (e == 2)
        {
            n += smem[threadIdx.y    ][threadIdx.x    ] == 1;
            n += smem[threadIdx.y    ][threadIdx.x + 1] == 1;
            n += smem[threadIdx.y    ][threadIdx.x + 2] == 1;

            n += smem[threadIdx.y + 1][threadIdx.x    ] == 1;
            n += smem[threadIdx.y + 1][threadIdx.x + 2] == 1;

            n += smem[threadIdx.y + 2][threadIdx.x    ] == 1;
            n += smem[threadIdx.y + 2][threadIdx.x + 1] == 1;
            n += smem[threadIdx.y + 2][threadIdx.x + 2] == 1;
        }

        if (n > 0)
        {
            const int ind =  ::atomicAdd(&counter, 1);
            st[ind] = make_short2(x, y);
        }
    }

    void edgesHysteresisLocal(PtrStepSzi map, short2* st1, hipStream_t stream)
    {
        void* counter_ptr;
        cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(counter)) );

        cudaSafeCall( hipMemsetAsync(counter_ptr, 0, sizeof(int), stream) );

        const dim3 block(16, 16);
        const dim3 grid(divUp(map.cols, block.x), divUp(map.rows, block.y));

        edgesHysteresisLocalKernel<<<grid, block, 0, stream>>>(map, st1);
        cudaSafeCall( hipGetLastError() );

        if (stream == NULL)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __constant__ int c_dx[8] = {-1,  0,  1, -1, 1, -1, 0, 1};
    __constant__ int c_dy[8] = {-1, -1, -1,  0, 0,  1, 1, 1};

    __global__ void edgesHysteresisGlobalKernel(PtrStepSzi map, short2* st1, short2* st2, const int count)
    {
        const int stack_size = 512;

        __shared__ int s_counter;
        __shared__ int s_ind;
        __shared__ short2 s_st[stack_size];

        if (threadIdx.x == 0)
            s_counter = 0;

        __syncthreads();

        int ind = blockIdx.y * gridDim.x + blockIdx.x;

        if (ind >= count)
            return;

        short2 pos = st1[ind];

        if (threadIdx.x < 8)
        {
            pos.x += c_dx[threadIdx.x];
            pos.y += c_dy[threadIdx.x];

            if (pos.x > 0 && pos.x < map.cols - 1 && pos.y > 0 && pos.y < map.rows - 1 && map(pos.y, pos.x) == 1)
            {
                map(pos.y, pos.x) = 2;

                ind = Emulation::smem::atomicAdd(&s_counter, 1);

                s_st[ind] = pos;
            }
        }

        __syncthreads();

        while (s_counter > 0 && s_counter <= stack_size - blockDim.x)
        {
            const int subTaskIdx = threadIdx.x >> 3;
            const int portion = ::min(s_counter, blockDim.x >> 3);

            if (subTaskIdx < portion)
                pos = s_st[s_counter - 1 - subTaskIdx];

            __syncthreads();

            if (threadIdx.x == 0)
                s_counter -= portion;

            __syncthreads();

            if (subTaskIdx < portion)
            {
                pos.x += c_dx[threadIdx.x & 7];
                pos.y += c_dy[threadIdx.x & 7];

                if (pos.x > 0 && pos.x < map.cols - 1 && pos.y > 0 && pos.y < map.rows - 1 && map(pos.y, pos.x) == 1)
                {
                    map(pos.y, pos.x) = 2;

                    ind = Emulation::smem::atomicAdd(&s_counter, 1);

                    s_st[ind] = pos;
                }
            }

            __syncthreads();
        }

        if (s_counter > 0)
        {
            if (threadIdx.x == 0)
            {
                s_ind = ::atomicAdd(&counter, s_counter);

                if (s_ind + s_counter > map.cols * map.rows)
                    s_counter = 0;
            }

            __syncthreads();

            ind = s_ind;

            for (int i = threadIdx.x; i < s_counter; i += blockDim.x)
                st2[ind + i] = s_st[i];
        }
    }

    void edgesHysteresisGlobal(PtrStepSzi map, short2* st1, short2* st2, hipStream_t stream)
    {
        void* counter_ptr;
        cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(canny::counter)) );

        int count;
        cudaSafeCall( hipMemcpyAsync(&count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost, stream) );
        cudaSafeCall( hipStreamSynchronize(stream) );

        while (count > 0)
        {
            cudaSafeCall( hipMemsetAsync(counter_ptr, 0, sizeof(int), stream) );

            const dim3 block(128);
            const dim3 grid(::min(count, 65535u), divUp(count, 65535), 1);

            edgesHysteresisGlobalKernel<<<grid, block, 0, stream>>>(map, st1, st2, count);
            cudaSafeCall( hipGetLastError() );

            if (stream == NULL)
                cudaSafeCall( hipDeviceSynchronize() );

            cudaSafeCall( hipMemcpyAsync(&count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            cudaSafeCall( hipStreamSynchronize(stream) );

            count = min(count, map.cols * map.rows);

            //std::swap(st1, st2);
            short2* tmp = st1;
            st1 = st2;
            st2 = tmp;
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    struct GetEdges : unary_function<int, uchar>
    {
        __device__ __forceinline__ uchar operator ()(int e) const
        {
            return (uchar)(-(e >> 1));
        }

        __host__ __device__ __forceinline__ GetEdges() {}
        __host__ __device__ __forceinline__ GetEdges(const GetEdges&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<canny::GetEdges> : DefaultTransformFunctorTraits<canny::GetEdges>
    {
        enum { smart_shift = 4 };
    };
}}}

namespace canny
{
    void getEdges(PtrStepSzi map, PtrStepSzb dst, hipStream_t stream)
    {
        transform(map, dst, GetEdges(), WithOutMask(), stream);
    }
}

#endif /* CUDA_DISABLER */
