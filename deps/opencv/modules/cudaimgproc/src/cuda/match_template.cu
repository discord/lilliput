#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_math.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace match_template
    {
        __device__ __forceinline__ float sum(float v) { return v; }
        __device__ __forceinline__ float sum(float2 v) { return v.x + v.y; }
        __device__ __forceinline__ float sum(float3 v) { return v.x + v.y + v.z; }
        __device__ __forceinline__ float sum(float4 v) { return v.x + v.y + v.z + v.w; }

        __device__ __forceinline__ float first(float v) { return v; }
        __device__ __forceinline__ float first(float2 v) { return v.x; }
        __device__ __forceinline__ float first(float3 v) { return v.x; }
        __device__ __forceinline__ float first(float4 v) { return v.x; }

        __device__ __forceinline__ float mul(float a, float b) { return a * b; }
        __device__ __forceinline__ float2 mul(float2 a, float2 b) { return make_float2(a.x * b.x, a.y * b.y); }
        __device__ __forceinline__ float3 mul(float3 a, float3 b) { return make_float3(a.x * b.x, a.y * b.y, a.z * b.z); }
        __device__ __forceinline__ float4 mul(float4 a, float4 b) { return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w); }

        __device__ __forceinline__ float mul(uchar a, uchar b) { return a * b; }
        __device__ __forceinline__ float2 mul(uchar2 a, uchar2 b) { return make_float2(a.x * b.x, a.y * b.y); }
        __device__ __forceinline__ float3 mul(uchar3 a, uchar3 b) { return make_float3(a.x * b.x, a.y * b.y, a.z * b.z); }
        __device__ __forceinline__ float4 mul(uchar4 a, uchar4 b) { return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w); }

        __device__ __forceinline__ float sub(float a, float b) { return a - b; }
        __device__ __forceinline__ float2 sub(float2 a, float2 b) { return make_float2(a.x - b.x, a.y - b.y); }
        __device__ __forceinline__ float3 sub(float3 a, float3 b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
        __device__ __forceinline__ float4 sub(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }

        __device__ __forceinline__ float sub(uchar a, uchar b) { return a - b; }
        __device__ __forceinline__ float2 sub(uchar2 a, uchar2 b) { return make_float2(a.x - b.x, a.y - b.y); }
        __device__ __forceinline__ float3 sub(uchar3 a, uchar3 b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
        __device__ __forceinline__ float4 sub(uchar4 a, uchar4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }

        //////////////////////////////////////////////////////////////////////
        // Naive_CCORR

        template <typename T, int cn>
        __global__ void matchTemplateNaiveKernel_CCORR(int w, int h, const PtrStepb image, const PtrStepb templ, PtrStepSzf result)
        {
            typedef typename TypeVec<T, cn>::vec_type Type;
            typedef typename TypeVec<float, cn>::vec_type Typef;

            int x = blockDim.x * blockIdx.x + threadIdx.x;
            int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                Typef res = VecTraits<Typef>::all(0);

                for (int i = 0; i < h; ++i)
                {
                    const Type* image_ptr = (const Type*)image.ptr(y + i);
                    const Type* templ_ptr = (const Type*)templ.ptr(i);
                    for (int j = 0; j < w; ++j)
                        res = res + mul(image_ptr[x + j], templ_ptr[j]);
                }

                result.ptr(y)[x] = sum(res);
            }
        }

        template <typename T, int cn>
        void matchTemplateNaive_CCORR(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, hipStream_t stream)
        {
            const dim3 threads(32, 8);
            const dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplateNaiveKernel_CCORR<T, cn><<<grid, threads, 0, stream>>>(templ.cols, templ.rows, image, templ, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void matchTemplateNaive_CCORR_32F(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, int cn, hipStream_t stream)
        {
            typedef void (*caller_t)(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, hipStream_t stream);

            static const caller_t callers[] =
            {
                0, matchTemplateNaive_CCORR<float, 1>, matchTemplateNaive_CCORR<float, 2>, matchTemplateNaive_CCORR<float, 3>, matchTemplateNaive_CCORR<float, 4>
            };

            callers[cn](image, templ, result, stream);
        }


        void matchTemplateNaive_CCORR_8U(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, int cn, hipStream_t stream)
        {
            typedef void (*caller_t)(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, hipStream_t stream);

            static const caller_t callers[] =
            {
                0, matchTemplateNaive_CCORR<uchar, 1>, matchTemplateNaive_CCORR<uchar, 2>, matchTemplateNaive_CCORR<uchar, 3>, matchTemplateNaive_CCORR<uchar, 4>
            };

            callers[cn](image, templ, result, stream);
        }

        //////////////////////////////////////////////////////////////////////
        // Naive_SQDIFF

        template <typename T, int cn>
        __global__ void matchTemplateNaiveKernel_SQDIFF(int w, int h, const PtrStepb image, const PtrStepb templ, PtrStepSzf result)
        {
            typedef typename TypeVec<T, cn>::vec_type Type;
            typedef typename TypeVec<float, cn>::vec_type Typef;

            int x = blockDim.x * blockIdx.x + threadIdx.x;
            int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                Typef res = VecTraits<Typef>::all(0);
                Typef delta;

                for (int i = 0; i < h; ++i)
                {
                    const Type* image_ptr = (const Type*)image.ptr(y + i);
                    const Type* templ_ptr = (const Type*)templ.ptr(i);
                    for (int j = 0; j < w; ++j)
                    {
                        delta = sub(image_ptr[x + j], templ_ptr[j]);
                        res = res + delta * delta;
                    }
                }

                result.ptr(y)[x] = sum(res);
            }
        }

        template <typename T, int cn>
        void matchTemplateNaive_SQDIFF(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, hipStream_t stream)
        {
            const dim3 threads(32, 8);
            const dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplateNaiveKernel_SQDIFF<T, cn><<<grid, threads, 0, stream>>>(templ.cols, templ.rows, image, templ, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void matchTemplateNaive_SQDIFF_32F(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, int cn, hipStream_t stream)
        {
            typedef void (*caller_t)(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, hipStream_t stream);

            static const caller_t callers[] =
            {
                0, matchTemplateNaive_SQDIFF<float, 1>, matchTemplateNaive_SQDIFF<float, 2>, matchTemplateNaive_SQDIFF<float, 3>, matchTemplateNaive_SQDIFF<float, 4>
            };

            callers[cn](image, templ, result, stream);
        }

        void matchTemplateNaive_SQDIFF_8U(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, int cn, hipStream_t stream)
        {
            typedef void (*caller_t)(const PtrStepSzb image, const PtrStepSzb templ, PtrStepSzf result, hipStream_t stream);

            static const caller_t callers[] =
            {
                0, matchTemplateNaive_SQDIFF<uchar, 1>, matchTemplateNaive_SQDIFF<uchar, 2>, matchTemplateNaive_SQDIFF<uchar, 3>, matchTemplateNaive_SQDIFF<uchar, 4>
            };

            callers[cn](image, templ, result, stream);
        }

        //////////////////////////////////////////////////////////////////////
        // Prepared_SQDIFF

        template <int cn>
        __global__ void matchTemplatePreparedKernel_SQDIFF_8U(int w, int h, const PtrStep<double> image_sqsum, double templ_sqsum, PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sqsum_ = (float)(
                        (image_sqsum.ptr(y + h)[(x + w) * cn] - image_sqsum.ptr(y)[(x + w) * cn]) -
                        (image_sqsum.ptr(y + h)[x * cn] - image_sqsum.ptr(y)[x * cn]));
                float ccorr = result.ptr(y)[x];
                result.ptr(y)[x] = image_sqsum_ - 2.f * ccorr + templ_sqsum;
            }
        }

        template <int cn>
        void matchTemplatePrepared_SQDIFF_8U(int w, int h, const PtrStepSz<double> image_sqsum, double templ_sqsum, PtrStepSzf result, hipStream_t stream)
        {
            const dim3 threads(32, 8);
            const dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplatePreparedKernel_SQDIFF_8U<cn><<<grid, threads, 0, stream>>>(w, h, image_sqsum, templ_sqsum, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void matchTemplatePrepared_SQDIFF_8U(int w, int h, const PtrStepSz<double> image_sqsum, double templ_sqsum, PtrStepSzf result, int cn,
                                             hipStream_t stream)
        {
            typedef void (*caller_t)(int w, int h, const PtrStepSz<double> image_sqsum, double templ_sqsum, PtrStepSzf result, hipStream_t stream);

            static const caller_t callers[] =
            {
                0, matchTemplatePrepared_SQDIFF_8U<1>, matchTemplatePrepared_SQDIFF_8U<2>, matchTemplatePrepared_SQDIFF_8U<3>, matchTemplatePrepared_SQDIFF_8U<4>
            };

            callers[cn](w, h, image_sqsum, templ_sqsum, result, stream);
        }

        //////////////////////////////////////////////////////////////////////
        // Prepared_SQDIFF_NORMED

        // normAcc* are accurate normalization routines which make CUDA matchTemplate
        // consistent with CPU one

        __device__ float normAcc(float num, float denum)
        {
            if (::fabs(num) < denum)
                return num / denum;
            if (::fabs(num) < denum * 1.125f)
                return num > 0 ? 1 : -1;
            return 0;
        }


        __device__ float normAcc_SQDIFF(float num, float denum)
        {
            if (::fabs(num) < denum)
                return num / denum;
            if (::fabs(num) < denum * 1.125f)
                return num > 0 ? 1 : -1;
            return 1;
        }


        template <int cn>
        __global__ void matchTemplatePreparedKernel_SQDIFF_NORMED_8U(
                int w, int h, const PtrStep<double> image_sqsum,
                double templ_sqsum, PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sqsum_ = (float)(
                        (image_sqsum.ptr(y + h)[(x + w) * cn] - image_sqsum.ptr(y)[(x + w) * cn]) -
                        (image_sqsum.ptr(y + h)[x * cn] - image_sqsum.ptr(y)[x * cn]));
                float ccorr = result.ptr(y)[x];
                result.ptr(y)[x] = normAcc_SQDIFF(image_sqsum_ - 2.f * ccorr + templ_sqsum,
                                                  sqrtf(image_sqsum_ * templ_sqsum));
            }
        }

        template <int cn>
        void matchTemplatePrepared_SQDIFF_NORMED_8U(int w, int h, const PtrStepSz<double> image_sqsum, double templ_sqsum,
                                                    PtrStepSzf result, hipStream_t stream)
        {
            const dim3 threads(32, 8);
            const dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplatePreparedKernel_SQDIFF_NORMED_8U<cn><<<grid, threads, 0, stream>>>(w, h, image_sqsum, templ_sqsum, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }


        void matchTemplatePrepared_SQDIFF_NORMED_8U(int w, int h, const PtrStepSz<double> image_sqsum, double templ_sqsum,
                                                    PtrStepSzf result, int cn, hipStream_t stream)
        {
            typedef void (*caller_t)(int w, int h, const PtrStepSz<double> image_sqsum, double templ_sqsum, PtrStepSzf result, hipStream_t stream);
            static const caller_t callers[] =
            {
                0, matchTemplatePrepared_SQDIFF_NORMED_8U<1>, matchTemplatePrepared_SQDIFF_NORMED_8U<2>, matchTemplatePrepared_SQDIFF_NORMED_8U<3>, matchTemplatePrepared_SQDIFF_NORMED_8U<4>
            };

            callers[cn](w, h, image_sqsum, templ_sqsum, result, stream);
        }

        //////////////////////////////////////////////////////////////////////
        // Prepared_CCOFF

        __global__ void matchTemplatePreparedKernel_CCOFF_8U(int w, int h, float templ_sum_scale, const PtrStep<int> image_sum, PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_ = (float)(
                        (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                        (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
                float ccorr = result.ptr(y)[x];
                result.ptr(y)[x] = ccorr - image_sum_ * templ_sum_scale;
            }
        }

        void matchTemplatePrepared_CCOFF_8U(int w, int h, const PtrStepSz<int> image_sum, int templ_sum, PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplatePreparedKernel_CCOFF_8U<<<grid, threads, 0, stream>>>(w, h, (float)templ_sum / (w * h), image_sum, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }



        __global__ void matchTemplatePreparedKernel_CCOFF_8UC2(
                int w, int h, float templ_sum_scale_r, float templ_sum_scale_g,
                const PtrStep<int> image_sum_r,
                const PtrStep<int> image_sum_g,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_r_ = (float)(
                        (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                        (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
                float image_sum_g_ = (float)(
                        (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                        (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
                float ccorr = result.ptr(y)[x];
                result.ptr(y)[x] = ccorr - image_sum_r_ * templ_sum_scale_r
                                         - image_sum_g_ * templ_sum_scale_g;
            }
        }

        void matchTemplatePrepared_CCOFF_8UC2(
                int w, int h,
                const PtrStepSz<int> image_sum_r,
                const PtrStepSz<int> image_sum_g,
                int templ_sum_r, int templ_sum_g,
                PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplatePreparedKernel_CCOFF_8UC2<<<grid, threads, 0, stream>>>(
                    w, h, (float)templ_sum_r / (w * h), (float)templ_sum_g / (w * h),
                    image_sum_r, image_sum_g, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }



        __global__ void matchTemplatePreparedKernel_CCOFF_8UC3(
                int w, int h,
                float templ_sum_scale_r,
                float templ_sum_scale_g,
                float templ_sum_scale_b,
                const PtrStep<int> image_sum_r,
                const PtrStep<int> image_sum_g,
                const PtrStep<int> image_sum_b,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_r_ = (float)(
                        (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                        (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
                float image_sum_g_ = (float)(
                        (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                        (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
                float image_sum_b_ = (float)(
                        (image_sum_b.ptr(y + h)[x + w] - image_sum_b.ptr(y)[x + w]) -
                        (image_sum_b.ptr(y + h)[x] - image_sum_b.ptr(y)[x]));
                float ccorr = result.ptr(y)[x];
                result.ptr(y)[x] = ccorr - image_sum_r_ * templ_sum_scale_r
                                         - image_sum_g_ * templ_sum_scale_g
                                         - image_sum_b_ * templ_sum_scale_b;
            }
        }

        void matchTemplatePrepared_CCOFF_8UC3(
                int w, int h,
                const PtrStepSz<int> image_sum_r,
                const PtrStepSz<int> image_sum_g,
                const PtrStepSz<int> image_sum_b,
                int templ_sum_r,
                int templ_sum_g,
                int templ_sum_b,
                PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplatePreparedKernel_CCOFF_8UC3<<<grid, threads, 0, stream>>>(
                    w, h,
                    (float)templ_sum_r / (w * h),
                    (float)templ_sum_g / (w * h),
                    (float)templ_sum_b / (w * h),
                    image_sum_r, image_sum_g, image_sum_b, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }



        __global__ void matchTemplatePreparedKernel_CCOFF_8UC4(
                int w, int h,
                float templ_sum_scale_r,
                float templ_sum_scale_g,
                float templ_sum_scale_b,
                float templ_sum_scale_a,
                const PtrStep<int> image_sum_r,
                const PtrStep<int> image_sum_g,
                const PtrStep<int> image_sum_b,
                const PtrStep<int> image_sum_a,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_r_ = (float)(
                        (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                        (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
                float image_sum_g_ = (float)(
                        (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                        (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
                float image_sum_b_ = (float)(
                        (image_sum_b.ptr(y + h)[x + w] - image_sum_b.ptr(y)[x + w]) -
                        (image_sum_b.ptr(y + h)[x] - image_sum_b.ptr(y)[x]));
                float image_sum_a_ = (float)(
                        (image_sum_a.ptr(y + h)[x + w] - image_sum_a.ptr(y)[x + w]) -
                        (image_sum_a.ptr(y + h)[x] - image_sum_a.ptr(y)[x]));
                float ccorr = result.ptr(y)[x];
                result.ptr(y)[x] = ccorr - image_sum_r_ * templ_sum_scale_r
                                         - image_sum_g_ * templ_sum_scale_g
                                         - image_sum_b_ * templ_sum_scale_b
                                         - image_sum_a_ * templ_sum_scale_a;
            }
        }

        void matchTemplatePrepared_CCOFF_8UC4(
                int w, int h,
                const PtrStepSz<int> image_sum_r,
                const PtrStepSz<int> image_sum_g,
                const PtrStepSz<int> image_sum_b,
                const PtrStepSz<int> image_sum_a,
                int templ_sum_r,
                int templ_sum_g,
                int templ_sum_b,
                int templ_sum_a,
                PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            matchTemplatePreparedKernel_CCOFF_8UC4<<<grid, threads, 0, stream>>>(
                    w, h,
                    (float)templ_sum_r / (w * h),
                    (float)templ_sum_g / (w * h),
                    (float)templ_sum_b / (w * h),
                    (float)templ_sum_a / (w * h),
                    image_sum_r, image_sum_g, image_sum_b, image_sum_a,
                    result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        //////////////////////////////////////////////////////////////////////
        // Prepared_CCOFF_NORMED

        __global__ void matchTemplatePreparedKernel_CCOFF_NORMED_8U(
                int w, int h, float weight,
                float templ_sum_scale, float templ_sqsum_scale,
                const PtrStep<int> image_sum,
                const PtrStep<double> image_sqsum,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float ccorr = result.ptr(y)[x];
                float image_sum_ = (float)(
                        (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                        (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
                float image_sqsum_ = (float)(
                        (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                        (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
                result.ptr(y)[x] = normAcc(ccorr - image_sum_ * templ_sum_scale,
                                           sqrtf(templ_sqsum_scale * (image_sqsum_ - weight * image_sum_ * image_sum_)));
            }
        }

        void matchTemplatePrepared_CCOFF_NORMED_8U(
                    int w, int h, const PtrStepSz<int> image_sum,
                    const PtrStepSz<double> image_sqsum,
                    int templ_sum, double templ_sqsum,
                    PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            float weight = 1.f / (w * h);
            float templ_sum_scale = templ_sum * weight;
            float templ_sqsum_scale = templ_sqsum - weight * templ_sum * templ_sum;

            matchTemplatePreparedKernel_CCOFF_NORMED_8U<<<grid, threads, 0, stream>>>(
                    w, h, weight, templ_sum_scale, templ_sqsum_scale,
                    image_sum, image_sqsum, result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }



        __global__ void matchTemplatePreparedKernel_CCOFF_NORMED_8UC2(
                int w, int h, float weight,
                float templ_sum_scale_r, float templ_sum_scale_g,
                float templ_sqsum_scale,
                const PtrStep<int> image_sum_r, const PtrStep<double> image_sqsum_r,
                const PtrStep<int> image_sum_g, const PtrStep<double> image_sqsum_g,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_r_ = (float)(
                        (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                        (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
                float image_sqsum_r_ = (float)(
                        (image_sqsum_r.ptr(y + h)[x + w] - image_sqsum_r.ptr(y)[x + w]) -
                        (image_sqsum_r.ptr(y + h)[x] - image_sqsum_r.ptr(y)[x]));
                float image_sum_g_ = (float)(
                        (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                        (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
                float image_sqsum_g_ = (float)(
                        (image_sqsum_g.ptr(y + h)[x + w] - image_sqsum_g.ptr(y)[x + w]) -
                        (image_sqsum_g.ptr(y + h)[x] - image_sqsum_g.ptr(y)[x]));

                float num = result.ptr(y)[x] - image_sum_r_ * templ_sum_scale_r
                                             - image_sum_g_ * templ_sum_scale_g;
                float denum = sqrtf(templ_sqsum_scale * (image_sqsum_r_ - weight * image_sum_r_ * image_sum_r_
                                                         + image_sqsum_g_ - weight * image_sum_g_ * image_sum_g_));
                result.ptr(y)[x] = normAcc(num, denum);
            }
        }

        void matchTemplatePrepared_CCOFF_NORMED_8UC2(
                    int w, int h,
                    const PtrStepSz<int> image_sum_r, const PtrStepSz<double> image_sqsum_r,
                    const PtrStepSz<int> image_sum_g, const PtrStepSz<double> image_sqsum_g,
                    int templ_sum_r, double templ_sqsum_r,
                    int templ_sum_g, double templ_sqsum_g,
                    PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            float weight = 1.f / (w * h);
            float templ_sum_scale_r = templ_sum_r * weight;
            float templ_sum_scale_g = templ_sum_g * weight;
            float templ_sqsum_scale = templ_sqsum_r - weight * templ_sum_r * templ_sum_r
                                       + templ_sqsum_g - weight * templ_sum_g * templ_sum_g;

            matchTemplatePreparedKernel_CCOFF_NORMED_8UC2<<<grid, threads, 0, stream>>>(
                    w, h, weight,
                    templ_sum_scale_r, templ_sum_scale_g,
                    templ_sqsum_scale,
                    image_sum_r, image_sqsum_r,
                    image_sum_g, image_sqsum_g,
                    result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }



        __global__ void matchTemplatePreparedKernel_CCOFF_NORMED_8UC3(
                int w, int h, float weight,
                float templ_sum_scale_r, float templ_sum_scale_g, float templ_sum_scale_b,
                float templ_sqsum_scale,
                const PtrStep<int> image_sum_r, const PtrStep<double> image_sqsum_r,
                const PtrStep<int> image_sum_g, const PtrStep<double> image_sqsum_g,
                const PtrStep<int> image_sum_b, const PtrStep<double> image_sqsum_b,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_r_ = (float)(
                        (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                        (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
                float image_sqsum_r_ = (float)(
                        (image_sqsum_r.ptr(y + h)[x + w] - image_sqsum_r.ptr(y)[x + w]) -
                        (image_sqsum_r.ptr(y + h)[x] - image_sqsum_r.ptr(y)[x]));
                float image_sum_g_ = (float)(
                        (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                        (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
                float image_sqsum_g_ = (float)(
                        (image_sqsum_g.ptr(y + h)[x + w] - image_sqsum_g.ptr(y)[x + w]) -
                        (image_sqsum_g.ptr(y + h)[x] - image_sqsum_g.ptr(y)[x]));
                float image_sum_b_ = (float)(
                        (image_sum_b.ptr(y + h)[x + w] - image_sum_b.ptr(y)[x + w]) -
                        (image_sum_b.ptr(y + h)[x] - image_sum_b.ptr(y)[x]));
                float image_sqsum_b_ = (float)(
                        (image_sqsum_b.ptr(y + h)[x + w] - image_sqsum_b.ptr(y)[x + w]) -
                        (image_sqsum_b.ptr(y + h)[x] - image_sqsum_b.ptr(y)[x]));

                float num = result.ptr(y)[x] - image_sum_r_ * templ_sum_scale_r
                                             - image_sum_g_ * templ_sum_scale_g
                                             - image_sum_b_ * templ_sum_scale_b;
                float denum = sqrtf(templ_sqsum_scale * (image_sqsum_r_ - weight * image_sum_r_ * image_sum_r_
                                                         + image_sqsum_g_ - weight * image_sum_g_ * image_sum_g_
                                                         + image_sqsum_b_ - weight * image_sum_b_ * image_sum_b_));
                result.ptr(y)[x] = normAcc(num, denum);
            }
        }

        void matchTemplatePrepared_CCOFF_NORMED_8UC3(
                    int w, int h,
                    const PtrStepSz<int> image_sum_r, const PtrStepSz<double> image_sqsum_r,
                    const PtrStepSz<int> image_sum_g, const PtrStepSz<double> image_sqsum_g,
                    const PtrStepSz<int> image_sum_b, const PtrStepSz<double> image_sqsum_b,
                    int templ_sum_r, double templ_sqsum_r,
                    int templ_sum_g, double templ_sqsum_g,
                    int templ_sum_b, double templ_sqsum_b,
                    PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            float weight = 1.f / (w * h);
            float templ_sum_scale_r = templ_sum_r * weight;
            float templ_sum_scale_g = templ_sum_g * weight;
            float templ_sum_scale_b = templ_sum_b * weight;
            float templ_sqsum_scale = templ_sqsum_r - weight * templ_sum_r * templ_sum_r
                                      + templ_sqsum_g - weight * templ_sum_g * templ_sum_g
                                      + templ_sqsum_b - weight * templ_sum_b * templ_sum_b;

            matchTemplatePreparedKernel_CCOFF_NORMED_8UC3<<<grid, threads, 0, stream>>>(
                    w, h, weight,
                    templ_sum_scale_r, templ_sum_scale_g, templ_sum_scale_b,
                    templ_sqsum_scale,
                    image_sum_r, image_sqsum_r,
                    image_sum_g, image_sqsum_g,
                    image_sum_b, image_sqsum_b,
                    result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }



        __global__ void matchTemplatePreparedKernel_CCOFF_NORMED_8UC4(
                int w, int h, float weight,
                float templ_sum_scale_r, float templ_sum_scale_g, float templ_sum_scale_b,
                float templ_sum_scale_a, float templ_sqsum_scale,
                const PtrStep<int> image_sum_r, const PtrStep<double> image_sqsum_r,
                const PtrStep<int> image_sum_g, const PtrStep<double> image_sqsum_g,
                const PtrStep<int> image_sum_b, const PtrStep<double> image_sqsum_b,
                const PtrStep<int> image_sum_a, const PtrStep<double> image_sqsum_a,
                PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sum_r_ = (float)(
                        (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                        (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
                float image_sqsum_r_ = (float)(
                        (image_sqsum_r.ptr(y + h)[x + w] - image_sqsum_r.ptr(y)[x + w]) -
                        (image_sqsum_r.ptr(y + h)[x] - image_sqsum_r.ptr(y)[x]));
                float image_sum_g_ = (float)(
                        (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                        (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
                float image_sqsum_g_ = (float)(
                        (image_sqsum_g.ptr(y + h)[x + w] - image_sqsum_g.ptr(y)[x + w]) -
                        (image_sqsum_g.ptr(y + h)[x] - image_sqsum_g.ptr(y)[x]));
                float image_sum_b_ = (float)(
                        (image_sum_b.ptr(y + h)[x + w] - image_sum_b.ptr(y)[x + w]) -
                        (image_sum_b.ptr(y + h)[x] - image_sum_b.ptr(y)[x]));
                float image_sqsum_b_ = (float)(
                        (image_sqsum_b.ptr(y + h)[x + w] - image_sqsum_b.ptr(y)[x + w]) -
                        (image_sqsum_b.ptr(y + h)[x] - image_sqsum_b.ptr(y)[x]));
                float image_sum_a_ = (float)(
                        (image_sum_a.ptr(y + h)[x + w] - image_sum_a.ptr(y)[x + w]) -
                        (image_sum_a.ptr(y + h)[x] - image_sum_a.ptr(y)[x]));
                float image_sqsum_a_ = (float)(
                        (image_sqsum_a.ptr(y + h)[x + w] - image_sqsum_a.ptr(y)[x + w]) -
                        (image_sqsum_a.ptr(y + h)[x] - image_sqsum_a.ptr(y)[x]));

                float num = result.ptr(y)[x] - image_sum_r_ * templ_sum_scale_r - image_sum_g_ * templ_sum_scale_g
                                             - image_sum_b_ * templ_sum_scale_b - image_sum_a_ * templ_sum_scale_a;
                float denum = sqrtf(templ_sqsum_scale * (image_sqsum_r_ - weight * image_sum_r_ * image_sum_r_
                                                         + image_sqsum_g_ - weight * image_sum_g_ * image_sum_g_
                                                         + image_sqsum_b_ - weight * image_sum_b_ * image_sum_b_
                                                         + image_sqsum_a_ - weight * image_sum_a_ * image_sum_a_));
                result.ptr(y)[x] = normAcc(num, denum);
            }
        }

        void matchTemplatePrepared_CCOFF_NORMED_8UC4(
                    int w, int h,
                    const PtrStepSz<int> image_sum_r, const PtrStepSz<double> image_sqsum_r,
                    const PtrStepSz<int> image_sum_g, const PtrStepSz<double> image_sqsum_g,
                    const PtrStepSz<int> image_sum_b, const PtrStepSz<double> image_sqsum_b,
                    const PtrStepSz<int> image_sum_a, const PtrStepSz<double> image_sqsum_a,
                    int templ_sum_r, double templ_sqsum_r,
                    int templ_sum_g, double templ_sqsum_g,
                    int templ_sum_b, double templ_sqsum_b,
                    int templ_sum_a, double templ_sqsum_a,
                    PtrStepSzf result, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            float weight = 1.f / (w * h);
            float templ_sum_scale_r = templ_sum_r * weight;
            float templ_sum_scale_g = templ_sum_g * weight;
            float templ_sum_scale_b = templ_sum_b * weight;
            float templ_sum_scale_a = templ_sum_a * weight;
            float templ_sqsum_scale = templ_sqsum_r - weight * templ_sum_r * templ_sum_r
                                      + templ_sqsum_g - weight * templ_sum_g * templ_sum_g
                                      + templ_sqsum_b - weight * templ_sum_b * templ_sum_b
                                      + templ_sqsum_a - weight * templ_sum_a * templ_sum_a;

            matchTemplatePreparedKernel_CCOFF_NORMED_8UC4<<<grid, threads, 0, stream>>>(
                    w, h, weight,
                    templ_sum_scale_r, templ_sum_scale_g, templ_sum_scale_b, templ_sum_scale_a,
                    templ_sqsum_scale,
                    image_sum_r, image_sqsum_r,
                    image_sum_g, image_sqsum_g,
                    image_sum_b, image_sqsum_b,
                    image_sum_a, image_sqsum_a,
                    result);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        //////////////////////////////////////////////////////////////////////
        // normalize

        template <int cn>
        __global__ void normalizeKernel_8U(
                int w, int h, const PtrStep<double> image_sqsum,
                double templ_sqsum, PtrStepSzf result)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                float image_sqsum_ = (float)(
                        (image_sqsum.ptr(y + h)[(x + w) * cn] - image_sqsum.ptr(y)[(x + w) * cn]) -
                        (image_sqsum.ptr(y + h)[x * cn] - image_sqsum.ptr(y)[x * cn]));
                result.ptr(y)[x] = normAcc(result.ptr(y)[x], sqrtf(image_sqsum_ * templ_sqsum));
            }
        }

        void normalize_8U(int w, int h, const PtrStepSz<double> image_sqsum,
                          double templ_sqsum, PtrStepSzf result, int cn, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            switch (cn)
            {
            case 1:
                normalizeKernel_8U<1><<<grid, threads, 0, stream>>>(w, h, image_sqsum, templ_sqsum, result);
                break;
            case 2:
                normalizeKernel_8U<2><<<grid, threads, 0, stream>>>(w, h, image_sqsum, templ_sqsum, result);
                break;
            case 3:
                normalizeKernel_8U<3><<<grid, threads, 0, stream>>>(w, h, image_sqsum, templ_sqsum, result);
                break;
            case 4:
                normalizeKernel_8U<4><<<grid, threads, 0, stream>>>(w, h, image_sqsum, templ_sqsum, result);
                break;
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        //////////////////////////////////////////////////////////////////////
        // extractFirstChannel

        template <int cn>
        __global__ void extractFirstChannel_32F(const PtrStepb image, PtrStepSzf result)
        {
            typedef typename TypeVec<float, cn>::vec_type Typef;

            int x = blockDim.x * blockIdx.x + threadIdx.x;
            int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < result.cols && y < result.rows)
            {
                Typef val = ((const Typef*)image.ptr(y))[x];
                result.ptr(y)[x] = first(val);
            }
        }

        void extractFirstChannel_32F(const PtrStepSzb image, PtrStepSzf result, int cn, hipStream_t stream)
        {
            dim3 threads(32, 8);
            dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

            switch (cn)
            {
            case 1:
                extractFirstChannel_32F<1><<<grid, threads, 0, stream>>>(image, result);
                break;
            case 2:
                extractFirstChannel_32F<2><<<grid, threads, 0, stream>>>(image, result);
                break;
            case 3:
                extractFirstChannel_32F<3><<<grid, threads, 0, stream>>>(image, result);
                break;
            case 4:
                extractFirstChannel_32F<4><<<grid, threads, 0, stream>>>(image, result);
                break;
            }
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    } //namespace match_template
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
