#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace hough
    {
        __device__ int g_counter;

        template <int PIXELS_PER_THREAD>
        __global__ void buildPointList(const PtrStepSzb src, unsigned int* list)
        {
            __shared__ unsigned int s_queues[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_qsize[4];
            __shared__ int s_globStart[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (threadIdx.x == 0)
                s_qsize[threadIdx.y] = 0;
            __syncthreads();

            if (y < src.rows)
            {
                // fill the queue
                const uchar* srcRow = src.ptr(y);
                for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < src.cols; ++i, xx += blockDim.x)
                {
                    if (srcRow[xx])
                    {
                        const unsigned int val = (y << 16) | xx;
                        const int qidx = Emulation::smem::atomicAdd(&s_qsize[threadIdx.y], 1);
                        s_queues[threadIdx.y][qidx] = val;
                    }
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int totalSize = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_globStart[i] = totalSize;
                    totalSize += s_qsize[i];
                }

                // calculate the offset in the global list
                const int globalOffset = atomicAdd(&g_counter, totalSize);
                for (int i = 0; i < blockDim.y; ++i)
                    s_globStart[i] += globalOffset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_qsize[threadIdx.y];
            int gidx = s_globStart[threadIdx.y] + threadIdx.x;
            for(int i = threadIdx.x; i < qsize; i += blockDim.x, gidx += blockDim.x)
                list[gidx] = s_queues[threadIdx.y][i];
        }

        int buildPointList_gpu(PtrStepSzb src, unsigned int* list)
        {
            const int PIXELS_PER_THREAD = 16;

            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(src.cols, block.x * PIXELS_PER_THREAD), divUp(src.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildPointList<PIXELS_PER_THREAD>), hipFuncCachePreferShared) );

            buildPointList<PIXELS_PER_THREAD><<<grid, block>>>(src, list);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }
    }
}}}

#endif /* CUDA_DISABLER */
