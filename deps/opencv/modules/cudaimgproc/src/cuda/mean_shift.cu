#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        texture<uchar4, 2> tex_meanshift;

        __device__ short2 do_mean_shift(int x0, int y0, unsigned char* out,
                                        size_t out_step, int cols, int rows,
                                        int sp, int sr, int maxIter, float eps)
        {
            int isr2 = sr*sr;
            uchar4 c = tex2D(tex_meanshift, x0, y0 );

            // iterate meanshift procedure
            for( int iter = 0; iter < maxIter; iter++ )
            {
                int count = 0;
                int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
                float icount;

                //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
                int minx = x0-sp;
                int miny = y0-sp;
                int maxx = x0+sp;
                int maxy = y0+sp;

                for( int y = miny; y <= maxy; y++)
                {
                    int rowCount = 0;
                    for( int x = minx; x <= maxx; x++ )
                    {
                        uchar4 t = tex2D( tex_meanshift, x, y );

                        int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                        if( norm2 <= isr2 )
                        {
                            s0 += t.x; s1 += t.y; s2 += t.z;
                            sx += x; rowCount++;
                        }
                    }
                    count += rowCount;
                    sy += y*rowCount;
                }

                if( count == 0 )
                    break;

                icount = 1.f/count;
                int x1 = __float2int_rz(sx*icount);
                int y1 = __float2int_rz(sy*icount);
                s0 = __float2int_rz(s0*icount);
                s1 = __float2int_rz(s1*icount);
                s2 = __float2int_rz(s2*icount);

                int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

                bool stopFlag = (x0 == x1 && y0 == y1) || (::abs(x1-x0) + ::abs(y1-y0) + norm2 <= eps);

                x0 = x1; y0 = y1;
                c.x = s0; c.y = s1; c.z = s2;

                if( stopFlag )
                    break;
            }

            int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 4 * sizeof(uchar);
            *(uchar4*)(out + base) = c;

            return make_short2((short)x0, (short)y0);
        }

        __global__ void meanshift_kernel(unsigned char* out, size_t out_step, int cols, int rows, int sp, int sr, int maxIter, float eps )
        {
            int x0 = blockIdx.x * blockDim.x + threadIdx.x;
            int y0 = blockIdx.y * blockDim.y + threadIdx.y;

            if( x0 < cols && y0 < rows )
                do_mean_shift(x0, y0, out, out_step, cols, rows, sp, sr, maxIter, eps);
        }

        void meanShiftFiltering_gpu(const PtrStepSzb& src, PtrStepSzb dst, int sp, int sr, int maxIter, float eps, hipStream_t stream)
        {
            dim3 grid(1, 1, 1);
            dim3 threads(32, 8, 1);
            grid.x = divUp(src.cols, threads.x);
            grid.y = divUp(src.rows, threads.y);

            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
            cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

            meanshift_kernel<<< grid, threads, 0, stream >>>( dst.data, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void meanshiftproc_kernel(unsigned char* outr, size_t outrstep,
                                             unsigned char* outsp, size_t outspstep,
                                             int cols, int rows,
                                             int sp, int sr, int maxIter, float eps)
        {
            int x0 = blockIdx.x * blockDim.x + threadIdx.x;
            int y0 = blockIdx.y * blockDim.y + threadIdx.y;

            if( x0 < cols && y0 < rows )
            {
                int basesp = (blockIdx.y * blockDim.y + threadIdx.y) * outspstep + (blockIdx.x * blockDim.x + threadIdx.x) * 2 * sizeof(short);
                *(short2*)(outsp + basesp) = do_mean_shift(x0, y0, outr, outrstep, cols, rows, sp, sr, maxIter, eps);
            }
        }

        void meanShiftProc_gpu(const PtrStepSzb& src, PtrStepSzb dstr, PtrStepSzb dstsp, int sp, int sr, int maxIter, float eps, hipStream_t stream)
        {
            dim3 grid(1, 1, 1);
            dim3 threads(32, 8, 1);
            grid.x = divUp(src.cols, threads.x);
            grid.y = divUp(src.rows, threads.y);

            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
            cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

            meanshiftproc_kernel<<< grid, threads, 0, stream >>>( dstr.data, dstr.step, dstsp.data, dstsp.step, dstr.cols, dstr.rows, sp, sr, maxIter, eps );
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}

#endif
