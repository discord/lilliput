#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include <thrust/execution_policy.h>
namespace cv { namespace cuda { namespace device
{
    namespace gfft
    {
        texture<float, hipTextureType2D, hipReadModeElementType> eigTex(0, hipFilterModePoint, hipAddressModeClamp);

        __device__ int g_counter = 0;

        template <class Mask> __global__ void findCorners(float threshold, const Mask mask, float2* corners, int max_count, int rows, int cols)
        {
            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (i > 0 && i < rows - 1 && j > 0 && j < cols - 1 && mask(i, j))
            {
                float val = tex2D(eigTex, j, i);

                if (val > threshold)
                {
                    float maxVal = val;

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i - 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j    , i - 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i - 1), maxVal);

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i), maxVal);

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i + 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j    , i + 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i + 1), maxVal);

                    if (val == maxVal)
                    {
                        const int ind = ::atomicAdd(&g_counter, 1);

                        if (ind < max_count)
                            corners[ind] = make_float2(j, i);
                    }
                }
            }
        }

        int findCorners_gpu(PtrStepSzf eig, float threshold, PtrStepSzb mask, float2* corners, int max_count, hipStream_t stream)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemsetAsync(counter_ptr, 0, sizeof(int), stream) );

            bindTexture(&eigTex, eig);

            dim3 block(16, 16);
            dim3 grid(divUp(eig.cols, block.x), divUp(eig.rows, block.y));

            if (mask.data)
                findCorners<<<grid, block, 0, stream>>>(threshold, SingleMask(mask), corners, max_count, eig.rows, eig.cols);
            else
                findCorners<<<grid, block, 0, stream>>>(threshold, WithOutMask(), corners, max_count, eig.rows, eig.cols);

            cudaSafeCall( hipGetLastError() );

            int count;
            cudaSafeCall( hipMemcpyAsync(&count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            if (stream)
                cudaSafeCall(hipStreamSynchronize(stream));
            else
                cudaSafeCall( hipDeviceSynchronize() );
            return std::min(count, max_count);
        }

        class EigGreater
        {
        public:
            __device__ __forceinline__ bool operator()(float2 a, float2 b) const
            {
                return tex2D(eigTex, a.x, a.y) > tex2D(eigTex, b.x, b.y);
            }
        };


        void sortCorners_gpu(PtrStepSzf eig, float2* corners, int count, hipStream_t stream)
        {
            bindTexture(&eigTex, eig);

            thrust::device_ptr<float2> ptr(corners);
#if THRUST_VERSION >= 100802
            if (stream)
                thrust::sort(thrust::cuda::par(ThrustAllocator::getAllocator()).on(stream), ptr, ptr + count, EigGreater());
            else
                thrust::sort(thrust::cuda::par(ThrustAllocator::getAllocator()), ptr, ptr + count, EigGreater());
#else
            thrust::sort(ptr, ptr + count, EigGreater());
#endif
        }
    } // namespace optical_flow
}}}


#endif /* CUDA_DISABLER */
