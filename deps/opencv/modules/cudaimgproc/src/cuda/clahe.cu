#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/scan.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace clahe
{
    __global__ void calcLutKernel(const PtrStepb src, PtrStepb lut,
                                  const int2 tileSize, const int tilesX,
                                  const int clipLimit, const float lutScale)
    {
        __shared__ int smem[512];

        const int tx = blockIdx.x;
        const int ty = blockIdx.y;
        const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        smem[tid] = 0;
        __syncthreads();

        for (int i = threadIdx.y; i < tileSize.y; i += blockDim.y)
        {
            const uchar* srcPtr = src.ptr(ty * tileSize.y + i) + tx * tileSize.x;
            for (int j = threadIdx.x; j < tileSize.x; j += blockDim.x)
            {
                const int data = srcPtr[j];
                Emulation::smem::atomicAdd(&smem[data], 1);
            }
        }

        __syncthreads();

        int tHistVal = smem[tid];

        __syncthreads();

        if (clipLimit > 0)
        {
            // clip histogram bar

            int clipped = 0;
            if (tHistVal > clipLimit)
            {
                clipped = tHistVal - clipLimit;
                tHistVal = clipLimit;
            }

            // find number of overall clipped samples

            reduce<256>(smem, clipped, tid, plus<int>());

            // broadcast evaluated value

            __shared__ int totalClipped;

            if (tid == 0)
                totalClipped = clipped;
            __syncthreads();

            // redistribute clipped samples evenly

            int redistBatch = totalClipped / 256;
            tHistVal += redistBatch;

            int residual = totalClipped - redistBatch * 256;
            if (tid < residual)
                ++tHistVal;
        }

        const int lutVal = blockScanInclusive<256>(tHistVal, smem, tid);

        lut(ty * tilesX + tx, tid) = saturate_cast<uchar>(__float2int_rn(lutScale * lutVal));
    }

    void calcLut(PtrStepSzb src, PtrStepb lut, int tilesX, int tilesY, int2 tileSize, int clipLimit, float lutScale, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(tilesX, tilesY);

        calcLutKernel<<<grid, block, 0, stream>>>(src, lut, tileSize, tilesX, clipLimit, lutScale);

        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    __global__ void tranformKernel(const PtrStepSzb src, PtrStepb dst, const PtrStepb lut, const int2 tileSize, const int tilesX, const int tilesY)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= src.cols || y >= src.rows)
            return;

        const float tyf = (static_cast<float>(y) / tileSize.y) - 0.5f;
        int ty1 = __float2int_rd(tyf);
        int ty2 = ty1 + 1;
        const float ya = tyf - ty1;
        ty1 = ::max(ty1, 0);
        ty2 = ::min(ty2, tilesY - 1);

        const float txf = (static_cast<float>(x) / tileSize.x) - 0.5f;
        int tx1 = __float2int_rd(txf);
        int tx2 = tx1 + 1;
        const float xa = txf - tx1;
        tx1 = ::max(tx1, 0);
        tx2 = ::min(tx2, tilesX - 1);

        const int srcVal = src(y, x);

        float res = 0;

        res += lut(ty1 * tilesX + tx1, srcVal) * ((1.0f - xa) * (1.0f - ya));
        res += lut(ty1 * tilesX + tx2, srcVal) * ((xa) * (1.0f - ya));
        res += lut(ty2 * tilesX + tx1, srcVal) * ((1.0f - xa) * (ya));
        res += lut(ty2 * tilesX + tx2, srcVal) * ((xa) * (ya));

        dst(y, x) = saturate_cast<uchar>(res);
    }

    void transform(PtrStepSzb src, PtrStepSzb dst, PtrStepb lut, int tilesX, int tilesY, int2 tileSize, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

        cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(tranformKernel), hipFuncCachePreferL1) );

        tranformKernel<<<grid, block, 0, stream>>>(src, dst, lut, tileSize, tilesX, tilesY);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

#endif // CUDA_DISABLER
