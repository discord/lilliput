#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/transform.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace hist
{
    __global__ void histogram256Kernel(const uchar* src, int cols, int rows, size_t step, int* hist)
    {
        __shared__ int shist[256];

        const int y = blockIdx.x * blockDim.y + threadIdx.y;
        const int tid = threadIdx.y * blockDim.x + threadIdx.x;

        shist[tid] = 0;
        __syncthreads();

        if (y < rows)
        {
            const unsigned int* rowPtr = (const unsigned int*) (src + y * step);

            const int cols_4 = cols / 4;
            for (int x = threadIdx.x; x < cols_4; x += blockDim.x)
            {
                unsigned int data = rowPtr[x];

                Emulation::smem::atomicAdd(&shist[(data >>  0) & 0xFFU], 1);
                Emulation::smem::atomicAdd(&shist[(data >>  8) & 0xFFU], 1);
                Emulation::smem::atomicAdd(&shist[(data >> 16) & 0xFFU], 1);
                Emulation::smem::atomicAdd(&shist[(data >> 24) & 0xFFU], 1);
            }

            if (cols % 4 != 0 && threadIdx.x == 0)
            {
                for (int x = cols_4 * 4; x < cols; ++x)
                {
                    unsigned int data = ((const uchar*)rowPtr)[x];
                    Emulation::smem::atomicAdd(&shist[data], 1);
                }
            }
        }

        __syncthreads();

        const int histVal = shist[tid];
        if (histVal > 0)
            ::atomicAdd(hist + tid, histVal);
    }

    void histogram256(PtrStepSzb src, int* hist, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(src.rows, block.y));

        histogram256Kernel<<<grid, block, 0, stream>>>(src.data, src.cols, src.rows, src.step, hist);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

/////////////////////////////////////////////////////////////////////////

namespace hist
{
    __device__ __forceinline__ void histEvenInc(int* shist, uint data, int binSize, int lowerLevel, int upperLevel)
    {
        if (data >= lowerLevel && data <= upperLevel)
        {
            const uint ind = (data - lowerLevel) / binSize;
            Emulation::smem::atomicAdd(shist + ind, 1);
        }
    }

    __global__ void histEven8u(const uchar* src, const size_t step, const int rows, const int cols,
                               int* hist, const int binCount, const int binSize, const int lowerLevel, const int upperLevel)
    {
        extern __shared__ int shist[];

        const int y = blockIdx.x * blockDim.y + threadIdx.y;
        const int tid = threadIdx.y * blockDim.x + threadIdx.x;

        if (tid < binCount)
            shist[tid] = 0;

        __syncthreads();

        if (y < rows)
        {
            const uchar* rowPtr = src + y * step;
            const uint* rowPtr4 = (uint*) rowPtr;

            const int cols_4 = cols / 4;
            for (int x = threadIdx.x; x < cols_4; x += blockDim.x)
            {
                const uint data = rowPtr4[x];

                histEvenInc(shist, (data >>  0) & 0xFFU, binSize, lowerLevel, upperLevel);
                histEvenInc(shist, (data >>  8) & 0xFFU, binSize, lowerLevel, upperLevel);
                histEvenInc(shist, (data >> 16) & 0xFFU, binSize, lowerLevel, upperLevel);
                histEvenInc(shist, (data >> 24) & 0xFFU, binSize, lowerLevel, upperLevel);
            }

            if (cols % 4 != 0 && threadIdx.x == 0)
            {
                for (int x = cols_4 * 4; x < cols; ++x)
                {
                    const uchar data = rowPtr[x];
                    histEvenInc(shist, data, binSize, lowerLevel, upperLevel);
                }
            }
        }

        __syncthreads();

        if (tid < binCount)
        {
            const int histVal = shist[tid];

            if (histVal > 0)
                ::atomicAdd(hist + tid, histVal);
        }
    }

    void histEven8u(PtrStepSzb src, int* hist, int binCount, int lowerLevel, int upperLevel, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(src.rows, block.y));

        const int binSize = divUp(upperLevel - lowerLevel, binCount);

        const size_t smem_size = binCount * sizeof(int);

        histEven8u<<<grid, block, smem_size, stream>>>(src.data, src.step, src.rows, src.cols, hist, binCount, binSize, lowerLevel, upperLevel);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

/////////////////////////////////////////////////////////////////////////

namespace hist
{
    __constant__ int c_lut[256];

    struct EqualizeHist : unary_function<uchar, uchar>
    {
        float scale;

        __host__ EqualizeHist(float _scale) : scale(_scale) {}

        __device__ __forceinline__ uchar operator ()(uchar val) const
        {
            const int lut = c_lut[val];
            return __float2int_rn(scale * lut);
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<hist::EqualizeHist> : DefaultTransformFunctorTraits<hist::EqualizeHist>
    {
        enum { smart_shift = 4 };
    };
}}}

namespace hist
{
    void equalizeHist(PtrStepSzb src, PtrStepSzb dst, const int* lut, hipStream_t stream)
    {
        if (stream == 0)
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_lut), lut, 256 * sizeof(int), 0, hipMemcpyDeviceToDevice) );
        else
            cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_lut), lut, 256 * sizeof(int), 0, hipMemcpyDeviceToDevice, stream) );

        const float scale = 255.0f / (src.cols * src.rows);

        device::transform(src, dst, EqualizeHist(scale), WithOutMask(), stream);
    }
}

#endif /* CUDA_DISABLER */
