#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    struct ShiftMap
    {
        typedef int2 value_type;
        typedef int index_type;

        int top;
        int left;

        __device__ __forceinline__ int2 operator ()(int y, int x) const
        {
            return make_int2(x - left, y - top);
        }
    };

    struct ShiftMapSz : ShiftMap
    {
        int rows, cols;
    };
}

namespace cv { namespace cudev {

template <> struct PtrTraits<ShiftMapSz> : PtrTraitsBase<ShiftMapSz, ShiftMap>
{
};

}}

namespace
{
    template <typename T, int cn>
    void copyMakeBorderImpl(const GpuMat& src, GpuMat& dst, int top, int left, int borderMode, cv::Scalar borderValue, Stream& stream)
    {
        typedef typename MakeVec<T, cn>::type src_type;

        cv::Scalar_<T> borderValue_ = borderValue;
        const src_type brdVal = VecTraits<src_type>::make(borderValue_.val);

        ShiftMapSz map;
        map.top = top;
        map.left = left;
        map.rows = dst.rows;
        map.cols = dst.cols;

        switch (borderMode)
        {
        case cv::BORDER_CONSTANT:
            gridCopy(remapPtr(brdConstant(globPtr<src_type>(src), brdVal), map), globPtr<src_type>(dst), stream);
            break;
        case cv::BORDER_REPLICATE:
            gridCopy(remapPtr(brdReplicate(globPtr<src_type>(src)), map), globPtr<src_type>(dst), stream);
            break;
        case cv::BORDER_REFLECT:
            gridCopy(remapPtr(brdReflect(globPtr<src_type>(src)), map), globPtr<src_type>(dst), stream);
            break;
        case cv::BORDER_WRAP:
            gridCopy(remapPtr(brdWrap(globPtr<src_type>(src)), map), globPtr<src_type>(dst), stream);
            break;
        case cv::BORDER_REFLECT_101:
            gridCopy(remapPtr(brdReflect101(globPtr<src_type>(src)), map), globPtr<src_type>(dst), stream);
            break;
        };
    }
}

void cv::cuda::copyMakeBorder(InputArray _src, OutputArray _dst, int top, int bottom, int left, int right, int borderType, Scalar value, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, GpuMat& dst, int top, int left, int borderMode, cv::Scalar borderValue, Stream& stream);
    static const func_t funcs[6][4] =
    {
        {    copyMakeBorderImpl<uchar , 1>  ,     copyMakeBorderImpl<uchar , 2>  ,     copyMakeBorderImpl<uchar , 3>  ,     copyMakeBorderImpl<uchar , 4>  },
        {0 /*copyMakeBorderImpl<schar , 1>*/, 0 /*copyMakeBorderImpl<schar , 2>*/, 0 /*copyMakeBorderImpl<schar , 3>*/, 0 /*copyMakeBorderImpl<schar , 4>*/},
        {    copyMakeBorderImpl<ushort, 1>  , 0 /*copyMakeBorderImpl<ushort, 2>*/,     copyMakeBorderImpl<ushort, 3>  ,     copyMakeBorderImpl<ushort, 4>  },
        {    copyMakeBorderImpl<short , 1>  , 0 /*copyMakeBorderImpl<short , 2>*/,     copyMakeBorderImpl<short , 3>  ,     copyMakeBorderImpl<short , 4>  },
        {0 /*copyMakeBorderImpl<int   , 1>*/, 0 /*copyMakeBorderImpl<int   , 2>*/, 0 /*copyMakeBorderImpl<int   , 3>*/, 0 /*copyMakeBorderImpl<int   , 4>*/},
        {    copyMakeBorderImpl<float , 1>  , 0 /*copyMakeBorderImpl<float , 2>*/,     copyMakeBorderImpl<float , 3>  ,     copyMakeBorderImpl<float  ,4>  }
    };

    GpuMat src = getInputMat(_src, stream);

    const int depth = src.depth();
    const int cn = src.channels();

    CV_Assert( depth <= CV_32F && cn <= 4 );
    CV_Assert( borderType == BORDER_REFLECT_101 || borderType == BORDER_REPLICATE || borderType == BORDER_CONSTANT || borderType == BORDER_REFLECT || borderType == BORDER_WRAP );

    GpuMat dst = getOutputMat(_dst, src.rows + top + bottom, src.cols + left + right, src.type(), stream);

    const func_t func = funcs[depth][cn - 1];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src, dst, top, left, borderType, value, stream);

    syncOutput(dst, _dst, stream);
}

#endif
