#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };
}

//////////////////////////////////////////////////////////////////////////////
/// abs

namespace
{
    template <typename T>
    void absMat(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), abs_func<T>(), stream);
    }
}

void cv::cuda::abs(InputArray _src, OutputArray _dst, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        absMat<uchar>,
        absMat<schar>,
        absMat<ushort>,
        absMat<short>,
        absMat<int>,
        absMat<float>,
        absMat<double>
    };

    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.depth() <= CV_64F );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);

    funcs[src.depth()](src.reshape(1), dst.reshape(1), stream);

    syncOutput(dst, _dst, stream);
}

//////////////////////////////////////////////////////////////////////////////
/// sqr

namespace
{
    template <typename T> struct SqrOp : unary_function<T, T>
    {
        __device__ __forceinline__ T operator ()(T x) const
        {
            return cudev::saturate_cast<T>(x * x);
        }
    };

    template <typename T>
    void sqrMat(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), SqrOp<T>(), stream);
    }
}

void cv::cuda::sqr(InputArray _src, OutputArray _dst, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        sqrMat<uchar>,
        sqrMat<schar>,
        sqrMat<ushort>,
        sqrMat<short>,
        sqrMat<int>,
        sqrMat<float>,
        sqrMat<double>
    };

    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.depth() <= CV_64F );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);

    funcs[src.depth()](src.reshape(1), dst.reshape(1), stream);

    syncOutput(dst, _dst, stream);
}

//////////////////////////////////////////////////////////////////////////////
/// sqrt

namespace
{
    template <typename T>
    void sqrtMat(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), sqrt_func<T>(), stream);
    }
}

void cv::cuda::sqrt(InputArray _src, OutputArray _dst, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        sqrtMat<uchar>,
        sqrtMat<schar>,
        sqrtMat<ushort>,
        sqrtMat<short>,
        sqrtMat<int>,
        sqrtMat<float>,
        sqrtMat<double>
    };

    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.depth() <= CV_64F );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);

    funcs[src.depth()](src.reshape(1), dst.reshape(1), stream);

    syncOutput(dst, _dst, stream);
}

////////////////////////////////////////////////////////////////////////
/// exp

namespace
{
    template <typename T> struct ExpOp : unary_function<T, T>
    {
        __device__ __forceinline__ T operator ()(T x) const
        {
            exp_func<T> f;
            return cudev::saturate_cast<T>(f(x));
        }
    };

    template <typename T>
    void expMat(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), ExpOp<T>(), stream);
    }
}

void cv::cuda::exp(InputArray _src, OutputArray _dst, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        expMat<uchar>,
        expMat<schar>,
        expMat<ushort>,
        expMat<short>,
        expMat<int>,
        expMat<float>,
        expMat<double>
    };

    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.depth() <= CV_64F );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);

    funcs[src.depth()](src.reshape(1), dst.reshape(1), stream);

    syncOutput(dst, _dst, stream);
}

////////////////////////////////////////////////////////////////////////
// log

namespace
{
    template <typename T>
    void logMat(const GpuMat& src, const GpuMat& dst, Stream& stream)
    {
        gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), log_func<T>(), stream);
    }
}

void cv::cuda::log(InputArray _src, OutputArray _dst, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        logMat<uchar>,
        logMat<schar>,
        logMat<ushort>,
        logMat<short>,
        logMat<int>,
        logMat<float>,
        logMat<double>
    };

    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.depth() <= CV_64F );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);

    funcs[src.depth()](src.reshape(1), dst.reshape(1), stream);

    syncOutput(dst, _dst, stream);
}

////////////////////////////////////////////////////////////////////////
// pow

namespace
{
    template<typename T, bool Signed = numeric_limits<T>::is_signed> struct PowOp : unary_function<T, T>
    {
        float power;

        __device__ __forceinline__ T operator()(T e) const
        {
            return cudev::saturate_cast<T>(__powf((float)e, power));
        }
    };
    template<typename T> struct PowOp<T, true> : unary_function<T, T>
    {
        float power;

        __device__ __forceinline__ T operator()(T e) const
        {
            T res = cudev::saturate_cast<T>(__powf((float)e, power));

            if ((e < 0) && (1 & static_cast<int>(power)))
                res *= -1;

            return res;
        }
    };
    template<> struct PowOp<float> : unary_function<float, float>
    {
        float power;

        __device__ __forceinline__ float operator()(float e) const
        {
            return __powf(::fabs(e), power);
        }
    };
    template<> struct PowOp<double> : unary_function<double, double>
    {
        double power;

        __device__ __forceinline__ double operator()(double e) const
        {
            return ::pow(::fabs(e), power);
        }
    };

    template<typename T>
    void powMat(const GpuMat& src, double power, const GpuMat& dst, Stream& stream)
    {
        PowOp<T> op;
        op.power = static_cast<typename LargerType<T, float>::type>(power);

        gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), op, stream);
    }
}

void cv::cuda::pow(InputArray _src, double power, OutputArray _dst, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& src, double power, const GpuMat& dst, Stream& stream);
    static const func_t funcs[] =
    {
        powMat<uchar>,
        powMat<schar>,
        powMat<ushort>,
        powMat<short>,
        powMat<int>,
        powMat<float>,
        powMat<double>
    };

    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.depth() <= CV_64F );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);

    funcs[src.depth()](src.reshape(1), power, dst.reshape(1), stream);

    syncOutput(dst, _dst, stream);
}

#endif
