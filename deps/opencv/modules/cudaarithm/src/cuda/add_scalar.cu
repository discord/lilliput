#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void addScalar(const GpuMat& src, cv::Scalar val, bool, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int);

namespace
{
    template <typename SrcType, typename ScalarType, typename DstType> struct AddScalarOp : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            return saturate_cast<DstType>(saturate_cast<ScalarType>(a) + val);
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename SrcType, typename ScalarDepth, typename DstType>
    void addScalarImpl(const GpuMat& src, cv::Scalar value, GpuMat& dst, const GpuMat& mask, Stream& stream)
    {
        typedef typename MakeVec<ScalarDepth, VecTraits<SrcType>::cn>::type ScalarType;

        cv::Scalar_<ScalarDepth> value_ = value;

        AddScalarOp<SrcType, ScalarType, DstType> op;
        op.val = VecTraits<ScalarType>::make(value_.val);

        if (mask.data)
            gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, globPtr<uchar>(mask), stream);
        else
            gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, stream);
    }
}

void addScalar(const GpuMat& src, cv::Scalar val, bool, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src, cv::Scalar val, GpuMat& dst, const GpuMat& mask, Stream& stream);
    static const func_t funcs[7][7][4] =
    {
        {
            {addScalarImpl<uchar, float, uchar>, addScalarImpl<uchar2, float, uchar2>, addScalarImpl<uchar3, float, uchar3>, addScalarImpl<uchar4, float, uchar4>},
            {addScalarImpl<uchar, float, schar>, addScalarImpl<uchar2, float, char2>, addScalarImpl<uchar3, float, char3>, addScalarImpl<uchar4, float, char4>},
            {addScalarImpl<uchar, float, ushort>, addScalarImpl<uchar2, float, ushort2>, addScalarImpl<uchar3, float, ushort3>, addScalarImpl<uchar4, float, ushort4>},
            {addScalarImpl<uchar, float, short>, addScalarImpl<uchar2, float, short2>, addScalarImpl<uchar3, float, short3>, addScalarImpl<uchar4, float, short4>},
            {addScalarImpl<uchar, float, int>, addScalarImpl<uchar2, float, int2>, addScalarImpl<uchar3, float, int3>, addScalarImpl<uchar4, float, int4>},
            {addScalarImpl<uchar, float, float>, addScalarImpl<uchar2, float, float2>, addScalarImpl<uchar3, float, float3>, addScalarImpl<uchar4, float, float4>},
            {addScalarImpl<uchar, double, double>, addScalarImpl<uchar2, double, double2>, addScalarImpl<uchar3, double, double3>, addScalarImpl<uchar4, double, double4>}
        },
        {
            {addScalarImpl<schar, float, uchar>, addScalarImpl<char2, float, uchar2>, addScalarImpl<char3, float, uchar3>, addScalarImpl<char4, float, uchar4>},
            {addScalarImpl<schar, float, schar>, addScalarImpl<char2, float, char2>, addScalarImpl<char3, float, char3>, addScalarImpl<char4, float, char4>},
            {addScalarImpl<schar, float, ushort>, addScalarImpl<char2, float, ushort2>, addScalarImpl<char3, float, ushort3>, addScalarImpl<char4, float, ushort4>},
            {addScalarImpl<schar, float, short>, addScalarImpl<char2, float, short2>, addScalarImpl<char3, float, short3>, addScalarImpl<char4, float, short4>},
            {addScalarImpl<schar, float, int>, addScalarImpl<char2, float, int2>, addScalarImpl<char3, float, int3>, addScalarImpl<char4, float, int4>},
            {addScalarImpl<schar, float, float>, addScalarImpl<char2, float, float2>, addScalarImpl<char3, float, float3>, addScalarImpl<char4, float, float4>},
            {addScalarImpl<schar, double, double>, addScalarImpl<char2, double, double2>, addScalarImpl<char3, double, double3>, addScalarImpl<char4, double, double4>}
        },
        {
            {0 /*addScalarImpl<ushort, float, uchar>*/, 0 /*addScalarImpl<ushort2, float, uchar2>*/, 0 /*addScalarImpl<ushort3, float, uchar3>*/, 0 /*addScalarImpl<ushort4, float, uchar4>*/},
            {0 /*addScalarImpl<ushort, float, schar>*/, 0 /*addScalarImpl<ushort2, float, char2>*/, 0 /*addScalarImpl<ushort3, float, char3>*/, 0 /*addScalarImpl<ushort4, float, char4>*/},
            {addScalarImpl<ushort, float, ushort>, addScalarImpl<ushort2, float, ushort2>, addScalarImpl<ushort3, float, ushort3>, addScalarImpl<ushort4, float, ushort4>},
            {addScalarImpl<ushort, float, short>, addScalarImpl<ushort2, float, short2>, addScalarImpl<ushort3, float, short3>, addScalarImpl<ushort4, float, short4>},
            {addScalarImpl<ushort, float, int>, addScalarImpl<ushort2, float, int2>, addScalarImpl<ushort3, float, int3>, addScalarImpl<ushort4, float, int4>},
            {addScalarImpl<ushort, float, float>, addScalarImpl<ushort2, float, float2>, addScalarImpl<ushort3, float, float3>, addScalarImpl<ushort4, float, float4>},
            {addScalarImpl<ushort, double, double>, addScalarImpl<ushort2, double, double2>, addScalarImpl<ushort3, double, double3>, addScalarImpl<ushort4, double, double4>}
        },
        {
            {0 /*addScalarImpl<short, float, uchar>*/, 0 /*addScalarImpl<short2, float, uchar2>*/, 0 /*addScalarImpl<short3, float, uchar3>*/, 0 /*addScalarImpl<short4, float, uchar4>*/},
            {0 /*addScalarImpl<short, float, schar>*/, 0 /*addScalarImpl<short2, float, char2>*/, 0 /*addScalarImpl<short3, float, char3>*/, 0 /*addScalarImpl<short4, float, char4>*/},
            {addScalarImpl<short, float, ushort>, addScalarImpl<short2, float, ushort2>, addScalarImpl<short3, float, ushort3>, addScalarImpl<short4, float, ushort4>},
            {addScalarImpl<short, float, short>, addScalarImpl<short2, float, short2>, addScalarImpl<short3, float, short3>, addScalarImpl<short4, float, short4>},
            {addScalarImpl<short, float, int>, addScalarImpl<short2, float, int2>, addScalarImpl<short3, float, int3>, addScalarImpl<short4, float, int4>},
            {addScalarImpl<short, float, float>, addScalarImpl<short2, float, float2>, addScalarImpl<short3, float, float3>, addScalarImpl<short4, float, float4>},
            {addScalarImpl<short, double, double>, addScalarImpl<short2, double, double2>, addScalarImpl<short3, double, double3>, addScalarImpl<short4, double, double4>}
        },
        {
            {0 /*addScalarImpl<int, float, uchar>*/, 0 /*addScalarImpl<int2, float, uchar2>*/, 0 /*addScalarImpl<int3, float, uchar3>*/, 0 /*addScalarImpl<int4, float, uchar4>*/},
            {0 /*addScalarImpl<int, float, schar>*/, 0 /*addScalarImpl<int2, float, char2>*/, 0 /*addScalarImpl<int3, float, char3>*/, 0 /*addScalarImpl<int4, float, char4>*/},
            {0 /*addScalarImpl<int, float, ushort>*/, 0 /*addScalarImpl<int2, float, ushort2>*/, 0 /*addScalarImpl<int3, float, ushort3>*/, 0 /*addScalarImpl<int4, float, ushort4>*/},
            {0 /*addScalarImpl<int, float, short>*/, 0 /*addScalarImpl<int2, float, short2>*/, 0 /*addScalarImpl<int3, float, short3>*/, 0 /*addScalarImpl<int4, float, short4>*/},
            {addScalarImpl<int, float, int>, addScalarImpl<int2, float, int2>, addScalarImpl<int3, float, int3>, addScalarImpl<int4, float, int4>},
            {addScalarImpl<int, float, float>, addScalarImpl<int2, float, float2>, addScalarImpl<int3, float, float3>, addScalarImpl<int4, float, float4>},
            {addScalarImpl<int, double, double>, addScalarImpl<int2, double, double2>, addScalarImpl<int3, double, double3>, addScalarImpl<int4, double, double4>}
        },
        {
            {0 /*addScalarImpl<float, float, uchar>*/, 0 /*addScalarImpl<float2, float, uchar2>*/, 0 /*addScalarImpl<float3, float, uchar3>*/, 0 /*addScalarImpl<float4, float, uchar4>*/},
            {0 /*addScalarImpl<float, float, schar>*/, 0 /*addScalarImpl<float2, float, char2>*/, 0 /*addScalarImpl<float3, float, char3>*/, 0 /*addScalarImpl<float4, float, char4>*/},
            {0 /*addScalarImpl<float, float, ushort>*/, 0 /*addScalarImpl<float2, float, ushort2>*/, 0 /*addScalarImpl<float3, float, ushort3>*/, 0 /*addScalarImpl<float4, float, ushort4>*/},
            {0 /*addScalarImpl<float, float, short>*/, 0 /*addScalarImpl<float2, float, short2>*/, 0 /*addScalarImpl<float3, float, short3>*/, 0 /*addScalarImpl<float4, float, short4>*/},
            {0 /*addScalarImpl<float, float, int>*/, 0 /*addScalarImpl<float2, float, int2>*/, 0 /*addScalarImpl<float3, float, int3>*/, 0 /*addScalarImpl<float4, float, int4>*/},
            {addScalarImpl<float, float, float>, addScalarImpl<float2, float, float2>, addScalarImpl<float3, float, float3>, addScalarImpl<float4, float, float4>},
            {addScalarImpl<float, double, double>, addScalarImpl<float2, double, double2>, addScalarImpl<float3, double, double3>, addScalarImpl<float4, double, double4>}
        },
        {
            {0 /*addScalarImpl<double, double, uchar>*/, 0 /*addScalarImpl<double2, double, uchar2>*/, 0 /*addScalarImpl<double3, double, uchar3>*/, 0 /*addScalarImpl<double4, double, uchar4>*/},
            {0 /*addScalarImpl<double, double, schar>*/, 0 /*addScalarImpl<double2, double, char2>*/, 0 /*addScalarImpl<double3, double, char3>*/, 0 /*addScalarImpl<double4, double, char4>*/},
            {0 /*addScalarImpl<double, double, ushort>*/, 0 /*addScalarImpl<double2, double, ushort2>*/, 0 /*addScalarImpl<double3, double, ushort3>*/, 0 /*addScalarImpl<double4, double, ushort4>*/},
            {0 /*addScalarImpl<double, double, short>*/, 0 /*addScalarImpl<double2, double, short2>*/, 0 /*addScalarImpl<double3, double, short3>*/, 0 /*addScalarImpl<double4, double, short4>*/},
            {0 /*addScalarImpl<double, double, int>*/, 0 /*addScalarImpl<double2, double, int2>*/, 0 /*addScalarImpl<double3, double, int3>*/, 0 /*addScalarImpl<double4, double, int4>*/},
            {0 /*addScalarImpl<double, double, float>*/, 0 /*addScalarImpl<double2, double, float2>*/, 0 /*addScalarImpl<double3, double, float3>*/, 0 /*addScalarImpl<double4, double, float4>*/},
            {addScalarImpl<double, double, double>, addScalarImpl<double2, double, double2>, addScalarImpl<double3, double, double3>, addScalarImpl<double4, double, double4>}
        }
    };

    const int sdepth = src.depth();
    const int ddepth = dst.depth();
    const int cn = src.channels();

    CV_DbgAssert( sdepth <= CV_64F && ddepth <= CV_64F && cn <= 4 );

    const func_t func = funcs[sdepth][ddepth][cn - 1];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src, val, dst, mask, stream);
}

#endif
