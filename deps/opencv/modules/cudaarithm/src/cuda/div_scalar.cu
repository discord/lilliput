#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void divScalar(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat& mask, double scale, Stream& stream, int);

namespace
{
    template <typename T, int cn> struct SafeDiv;
    template <typename T> struct SafeDiv<T, 1>
    {
        __device__ __forceinline__ static T op(T a, T b)
        {
            return b != 0 ? a / b : 0;
        }
    };
    template <typename T> struct SafeDiv<T, 2>
    {
        __device__ __forceinline__ static T op(const T& a, const T& b)
        {
            T res;

            res.x = b.x != 0 ? a.x / b.x : 0;
            res.y = b.y != 0 ? a.y / b.y : 0;

            return res;
        }
    };
    template <typename T> struct SafeDiv<T, 3>
    {
        __device__ __forceinline__ static T op(const T& a, const T& b)
        {
            T res;

            res.x = b.x != 0 ? a.x / b.x : 0;
            res.y = b.y != 0 ? a.y / b.y : 0;
            res.z = b.z != 0 ? a.z / b.z : 0;

            return res;
        }
    };
    template <typename T> struct SafeDiv<T, 4>
    {
        __device__ __forceinline__ static T op(const T& a, const T& b)
        {
            T res;

            res.x = b.x != 0 ? a.x / b.x : 0;
            res.y = b.y != 0 ? a.y / b.y : 0;
            res.z = b.z != 0 ? a.z / b.z : 0;
            res.w = b.w != 0 ? a.w / b.w : 0;

            return res;
        }
    };

    template <typename SrcType, typename ScalarType, typename DstType> struct DivScalarOp : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            return saturate_cast<DstType>(SafeDiv<ScalarType, VecTraits<ScalarType>::cn>::op(saturate_cast<ScalarType>(a), val));
        }
    };

    template <typename SrcType, typename ScalarType, typename DstType> struct DivScalarOpInv : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            return saturate_cast<DstType>(SafeDiv<ScalarType, VecTraits<ScalarType>::cn>::op(val, saturate_cast<ScalarType>(a)));
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename SrcType, typename ScalarDepth, typename DstType>
    void divScalarImpl(const GpuMat& src, cv::Scalar value, bool inv, GpuMat& dst, Stream& stream)
    {
        typedef typename MakeVec<ScalarDepth, VecTraits<SrcType>::cn>::type ScalarType;

        cv::Scalar_<ScalarDepth> value_ = value;

        if (inv)
        {
            DivScalarOpInv<SrcType, ScalarType, DstType> op;
            op.val = VecTraits<ScalarType>::make(value_.val);

            gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, stream);
        }
        else
        {
            DivScalarOp<SrcType, ScalarType, DstType> op;
            op.val = VecTraits<ScalarType>::make(value_.val);

            gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, stream);
        }
    }
}

void divScalar(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat&, double scale, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, Stream& stream);
    static const func_t funcs[7][7][4] =
    {
        {
            {divScalarImpl<uchar, float, uchar>, divScalarImpl<uchar2, float, uchar2>, divScalarImpl<uchar3, float, uchar3>, divScalarImpl<uchar4, float, uchar4>},
            {divScalarImpl<uchar, float, schar>, divScalarImpl<uchar2, float, char2>, divScalarImpl<uchar3, float, char3>, divScalarImpl<uchar4, float, char4>},
            {divScalarImpl<uchar, float, ushort>, divScalarImpl<uchar2, float, ushort2>, divScalarImpl<uchar3, float, ushort3>, divScalarImpl<uchar4, float, ushort4>},
            {divScalarImpl<uchar, float, short>, divScalarImpl<uchar2, float, short2>, divScalarImpl<uchar3, float, short3>, divScalarImpl<uchar4, float, short4>},
            {divScalarImpl<uchar, float, int>, divScalarImpl<uchar2, float, int2>, divScalarImpl<uchar3, float, int3>, divScalarImpl<uchar4, float, int4>},
            {divScalarImpl<uchar, float, float>, divScalarImpl<uchar2, float, float2>, divScalarImpl<uchar3, float, float3>, divScalarImpl<uchar4, float, float4>},
            {divScalarImpl<uchar, double, double>, divScalarImpl<uchar2, double, double2>, divScalarImpl<uchar3, double, double3>, divScalarImpl<uchar4, double, double4>}
        },
        {
            {divScalarImpl<schar, float, uchar>, divScalarImpl<char2, float, uchar2>, divScalarImpl<char3, float, uchar3>, divScalarImpl<char4, float, uchar4>},
            {divScalarImpl<schar, float, schar>, divScalarImpl<char2, float, char2>, divScalarImpl<char3, float, char3>, divScalarImpl<char4, float, char4>},
            {divScalarImpl<schar, float, ushort>, divScalarImpl<char2, float, ushort2>, divScalarImpl<char3, float, ushort3>, divScalarImpl<char4, float, ushort4>},
            {divScalarImpl<schar, float, short>, divScalarImpl<char2, float, short2>, divScalarImpl<char3, float, short3>, divScalarImpl<char4, float, short4>},
            {divScalarImpl<schar, float, int>, divScalarImpl<char2, float, int2>, divScalarImpl<char3, float, int3>, divScalarImpl<char4, float, int4>},
            {divScalarImpl<schar, float, float>, divScalarImpl<char2, float, float2>, divScalarImpl<char3, float, float3>, divScalarImpl<char4, float, float4>},
            {divScalarImpl<schar, double, double>, divScalarImpl<char2, double, double2>, divScalarImpl<char3, double, double3>, divScalarImpl<char4, double, double4>}
        },
        {
            {0 /*divScalarImpl<ushort, float, uchar>*/, 0 /*divScalarImpl<ushort2, float, uchar2>*/, 0 /*divScalarImpl<ushort3, float, uchar3>*/, 0 /*divScalarImpl<ushort4, float, uchar4>*/},
            {0 /*divScalarImpl<ushort, float, schar>*/, 0 /*divScalarImpl<ushort2, float, char2>*/, 0 /*divScalarImpl<ushort3, float, char3>*/, 0 /*divScalarImpl<ushort4, float, char4>*/},
            {divScalarImpl<ushort, float, ushort>, divScalarImpl<ushort2, float, ushort2>, divScalarImpl<ushort3, float, ushort3>, divScalarImpl<ushort4, float, ushort4>},
            {divScalarImpl<ushort, float, short>, divScalarImpl<ushort2, float, short2>, divScalarImpl<ushort3, float, short3>, divScalarImpl<ushort4, float, short4>},
            {divScalarImpl<ushort, float, int>, divScalarImpl<ushort2, float, int2>, divScalarImpl<ushort3, float, int3>, divScalarImpl<ushort4, float, int4>},
            {divScalarImpl<ushort, float, float>, divScalarImpl<ushort2, float, float2>, divScalarImpl<ushort3, float, float3>, divScalarImpl<ushort4, float, float4>},
            {divScalarImpl<ushort, double, double>, divScalarImpl<ushort2, double, double2>, divScalarImpl<ushort3, double, double3>, divScalarImpl<ushort4, double, double4>}
        },
        {
            {0 /*divScalarImpl<short, float, uchar>*/, 0 /*divScalarImpl<short2, float, uchar2>*/, 0 /*divScalarImpl<short3, float, uchar3>*/, 0 /*divScalarImpl<short4, float, uchar4>*/},
            {0 /*divScalarImpl<short, float, schar>*/, 0 /*divScalarImpl<short2, float, char2>*/, 0 /*divScalarImpl<short3, float, char3>*/, 0 /*divScalarImpl<short4, float, char4>*/},
            {divScalarImpl<short, float, ushort>, divScalarImpl<short2, float, ushort2>, divScalarImpl<short3, float, ushort3>, divScalarImpl<short4, float, ushort4>},
            {divScalarImpl<short, float, short>, divScalarImpl<short2, float, short2>, divScalarImpl<short3, float, short3>, divScalarImpl<short4, float, short4>},
            {divScalarImpl<short, float, int>, divScalarImpl<short2, float, int2>, divScalarImpl<short3, float, int3>, divScalarImpl<short4, float, int4>},
            {divScalarImpl<short, float, float>, divScalarImpl<short2, float, float2>, divScalarImpl<short3, float, float3>, divScalarImpl<short4, float, float4>},
            {divScalarImpl<short, double, double>, divScalarImpl<short2, double, double2>, divScalarImpl<short3, double, double3>, divScalarImpl<short4, double, double4>}
        },
        {
            {0 /*divScalarImpl<int, float, uchar>*/, 0 /*divScalarImpl<int2, float, uchar2>*/, 0 /*divScalarImpl<int3, float, uchar3>*/, 0 /*divScalarImpl<int4, float, uchar4>*/},
            {0 /*divScalarImpl<int, float, schar>*/, 0 /*divScalarImpl<int2, float, char2>*/, 0 /*divScalarImpl<int3, float, char3>*/, 0 /*divScalarImpl<int4, float, char4>*/},
            {0 /*divScalarImpl<int, float, ushort>*/, 0 /*divScalarImpl<int2, float, ushort2>*/, 0 /*divScalarImpl<int3, float, ushort3>*/, 0 /*divScalarImpl<int4, float, ushort4>*/},
            {0 /*divScalarImpl<int, float, short>*/, 0 /*divScalarImpl<int2, float, short2>*/, 0 /*divScalarImpl<int3, float, short3>*/, 0 /*divScalarImpl<int4, float, short4>*/},
            {divScalarImpl<int, float, int>, divScalarImpl<int2, float, int2>, divScalarImpl<int3, float, int3>, divScalarImpl<int4, float, int4>},
            {divScalarImpl<int, float, float>, divScalarImpl<int2, float, float2>, divScalarImpl<int3, float, float3>, divScalarImpl<int4, float, float4>},
            {divScalarImpl<int, double, double>, divScalarImpl<int2, double, double2>, divScalarImpl<int3, double, double3>, divScalarImpl<int4, double, double4>}
        },
        {
            {0 /*divScalarImpl<float, float, uchar>*/, 0 /*divScalarImpl<float2, float, uchar2>*/, 0 /*divScalarImpl<float3, float, uchar3>*/, 0 /*divScalarImpl<float4, float, uchar4>*/},
            {0 /*divScalarImpl<float, float, schar>*/, 0 /*divScalarImpl<float2, float, char2>*/, 0 /*divScalarImpl<float3, float, char3>*/, 0 /*divScalarImpl<float4, float, char4>*/},
            {0 /*divScalarImpl<float, float, ushort>*/, 0 /*divScalarImpl<float2, float, ushort2>*/, 0 /*divScalarImpl<float3, float, ushort3>*/, 0 /*divScalarImpl<float4, float, ushort4>*/},
            {0 /*divScalarImpl<float, float, short>*/, 0 /*divScalarImpl<float2, float, short2>*/, 0 /*divScalarImpl<float3, float, short3>*/, 0 /*divScalarImpl<float4, float, short4>*/},
            {0 /*divScalarImpl<float, float, int>*/, 0 /*divScalarImpl<float2, float, int2>*/, 0 /*divScalarImpl<float3, float, int3>*/, 0 /*divScalarImpl<float4, float, int4>*/},
            {divScalarImpl<float, float, float>, divScalarImpl<float2, float, float2>, divScalarImpl<float3, float, float3>, divScalarImpl<float4, float, float4>},
            {divScalarImpl<float, double, double>, divScalarImpl<float2, double, double2>, divScalarImpl<float3, double, double3>, divScalarImpl<float4, double, double4>}
        },
        {
            {0 /*divScalarImpl<double, double, uchar>*/, 0 /*divScalarImpl<double2, double, uchar2>*/, 0 /*divScalarImpl<double3, double, uchar3>*/, 0 /*divScalarImpl<double4, double, uchar4>*/},
            {0 /*divScalarImpl<double, double, schar>*/, 0 /*divScalarImpl<double2, double, char2>*/, 0 /*divScalarImpl<double3, double, char3>*/, 0 /*divScalarImpl<double4, double, char4>*/},
            {0 /*divScalarImpl<double, double, ushort>*/, 0 /*divScalarImpl<double2, double, ushort2>*/, 0 /*divScalarImpl<double3, double, ushort3>*/, 0 /*divScalarImpl<double4, double, ushort4>*/},
            {0 /*divScalarImpl<double, double, short>*/, 0 /*divScalarImpl<double2, double, short2>*/, 0 /*divScalarImpl<double3, double, short3>*/, 0 /*divScalarImpl<double4, double, short4>*/},
            {0 /*divScalarImpl<double, double, int>*/, 0 /*divScalarImpl<double2, double, int2>*/, 0 /*divScalarImpl<double3, double, int3>*/, 0 /*divScalarImpl<double4, double, int4>*/},
            {0 /*divScalarImpl<double, double, float>*/, 0 /*divScalarImpl<double2, double, float2>*/, 0 /*divScalarImpl<double3, double, float3>*/, 0 /*divScalarImpl<double4, double, float4>*/},
            {divScalarImpl<double, double, double>, divScalarImpl<double2, double, double2>, divScalarImpl<double3, double, double3>, divScalarImpl<double4, double, double4>}
        }
    };

    const int sdepth = src.depth();
    const int ddepth = dst.depth();
    const int cn = src.channels();

    CV_DbgAssert( sdepth <= CV_64F && ddepth <= CV_64F && cn <= 4 );

    if (inv)
    {
        val[0] *= scale;
        val[1] *= scale;
        val[2] *= scale;
        val[3] *= scale;
    }
    else
    {
        val[0] /= scale;
        val[1] /= scale;
        val[2] /= scale;
        val[3] /= scale;
    }

    const func_t func = funcs[sdepth][ddepth][cn - 1];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src, val, inv, dst, stream);
}

#endif
