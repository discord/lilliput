#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

//////////////////////////////////////////////////////////////////////////////
// mulSpectrums

namespace
{
    __device__ __forceinline__ float real(const float2& val)
    {
        return val.x;
    }

    __device__ __forceinline__ float imag(const float2& val)
    {
        return val.y;
    }

    __device__ __forceinline__ float2 cmul(const float2& a, const float2& b)
    {
        return make_float2((real(a) * real(b)) - (imag(a) * imag(b)),
                           (real(a) * imag(b)) + (imag(a) * real(b)));
    }

    __device__ __forceinline__ float2 conj(const float2& a)
    {
        return make_float2(real(a), -imag(a));
    }

    struct comlex_mul : binary_function<float2, float2, float2>
    {
        __device__ __forceinline__ float2 operator ()(const float2& a, const float2& b) const
        {
            return cmul(a, b);
        }
    };

    struct comlex_mul_conj : binary_function<float2, float2, float2>
    {
        __device__ __forceinline__ float2 operator ()(const float2& a, const float2& b) const
        {
            return cmul(a, conj(b));
        }
    };

    struct comlex_mul_scale : binary_function<float2, float2, float2>
    {
        float scale;

        __device__ __forceinline__ float2 operator ()(const float2& a, const float2& b) const
        {
            return scale * cmul(a, b);
        }
    };

    struct comlex_mul_conj_scale : binary_function<float2, float2, float2>
    {
        float scale;

        __device__ __forceinline__ float2 operator ()(const float2& a, const float2& b) const
        {
            return scale * cmul(a, conj(b));
        }
    };
}

void cv::cuda::mulSpectrums(InputArray _src1, InputArray _src2, OutputArray _dst, int flags, bool conjB, Stream& stream)
{
    (void) flags;

    GpuMat src1 = getInputMat(_src1, stream);
    GpuMat src2 = getInputMat(_src2, stream);

    CV_Assert( src1.type() == src2.type() && src1.type() == CV_32FC2 );
    CV_Assert( src1.size() == src2.size() );

    GpuMat dst = getOutputMat(_dst, src1.size(), CV_32FC2, stream);

    if (conjB)
        gridTransformBinary(globPtr<float2>(src1), globPtr<float2>(src2), globPtr<float2>(dst), comlex_mul_conj(), stream);
    else
        gridTransformBinary(globPtr<float2>(src1), globPtr<float2>(src2), globPtr<float2>(dst), comlex_mul(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::mulAndScaleSpectrums(InputArray _src1, InputArray _src2, OutputArray _dst, int flags, float scale, bool conjB, Stream& stream)
{
    (void) flags;

    GpuMat src1 = getInputMat(_src1, stream);
    GpuMat src2 = getInputMat(_src2, stream);

    CV_Assert( src1.type() == src2.type() && src1.type() == CV_32FC2);
    CV_Assert( src1.size() == src2.size() );

    GpuMat dst = getOutputMat(_dst, src1.size(), CV_32FC2, stream);

    if (conjB)
    {
        comlex_mul_conj_scale op;
        op.scale = scale;
        gridTransformBinary(globPtr<float2>(src1), globPtr<float2>(src2), globPtr<float2>(dst), op, stream);
    }
    else
    {
        comlex_mul_scale op;
        op.scale = scale;
        gridTransformBinary(globPtr<float2>(src1), globPtr<float2>(src2), globPtr<float2>(dst), op, stream);
    }

    syncOutput(dst, _dst, stream);
}

#endif
