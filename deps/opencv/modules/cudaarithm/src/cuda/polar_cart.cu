#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

void cv::cuda::magnitude(InputArray _x, InputArray _y, OutputArray _dst, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat dst = getOutputMat(_dst, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(dst.reshape(1));

    gridTransformBinary(xc, yc, magc, magnitude_func<float>(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::magnitudeSqr(InputArray _x, InputArray _y, OutputArray _dst, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat dst = getOutputMat(_dst, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(dst.reshape(1));

    gridTransformBinary(xc, yc, magc, magnitude_sqr_func<float>(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::phase(InputArray _x, InputArray _y, OutputArray _dst, bool angleInDegrees, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat dst = getOutputMat(_dst, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> anglec(dst.reshape(1));

    if (angleInDegrees)
        gridTransformBinary(xc, yc, anglec, direction_func<float, true>(), stream);
    else
        gridTransformBinary(xc, yc, anglec, direction_func<float, false>(), stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::cartToPolar(InputArray _x, InputArray _y, OutputArray _mag, OutputArray _angle, bool angleInDegrees, Stream& stream)
{
    GpuMat x = getInputMat(_x, stream);
    GpuMat y = getInputMat(_y, stream);

    CV_Assert( x.depth() == CV_32F );
    CV_Assert( y.type() == x.type() && y.size() == x.size() );

    GpuMat mag = getOutputMat(_mag, x.size(), CV_32FC1, stream);
    GpuMat angle = getOutputMat(_angle, x.size(), CV_32FC1, stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(mag.reshape(1));
    GpuMat_<float> anglec(angle.reshape(1));

    if (angleInDegrees)
    {
        gridTransformTuple(zipPtr(xc, yc),
                           tie(magc, anglec),
                           make_tuple(
                               binaryTupleAdapter<0, 1>(magnitude_func<float>()),
                               binaryTupleAdapter<0, 1>(direction_func<float, true>())),
                           stream);
    }
    else
    {
        gridTransformTuple(zipPtr(xc, yc),
                           tie(magc, anglec),
                           make_tuple(
                               binaryTupleAdapter<0, 1>(magnitude_func<float>()),
                               binaryTupleAdapter<0, 1>(direction_func<float, false>())),
                           stream);
    }

    syncOutput(mag, _mag, stream);
    syncOutput(angle, _angle, stream);
}

namespace
{
    template <bool useMag>
    __global__ void polarToCartImpl(const GlobPtr<float> mag, const GlobPtr<float> angle, GlobPtr<float> xmat, GlobPtr<float> ymat, const float scale, const int rows, const int cols)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= cols || y >= rows)
            return;

        const float mag_val = useMag ? mag(y, x) : 1.0f;
        const float angle_val = angle(y, x);

        float sin_a, cos_a;
        ::sincosf(scale * angle_val, &sin_a, &cos_a);

        xmat(y, x) = mag_val * cos_a;
        ymat(y, x) = mag_val * sin_a;
    }
}

void cv::cuda::polarToCart(InputArray _mag, InputArray _angle, OutputArray _x, OutputArray _y, bool angleInDegrees, Stream& _stream)
{
    GpuMat mag = getInputMat(_mag, _stream);
    GpuMat angle = getInputMat(_angle, _stream);

    CV_Assert( angle.depth() == CV_32F );
    CV_Assert( mag.empty() || (mag.type() == angle.type() && mag.size() == angle.size()) );

    GpuMat x = getOutputMat(_x, angle.size(), CV_32FC1, _stream);
    GpuMat y = getOutputMat(_y, angle.size(), CV_32FC1, _stream);

    GpuMat_<float> xc(x.reshape(1));
    GpuMat_<float> yc(y.reshape(1));
    GpuMat_<float> magc(mag.reshape(1));
    GpuMat_<float> anglec(angle.reshape(1));

    const dim3 block(32, 8);
    const dim3 grid(divUp(anglec.cols, block.x), divUp(anglec.rows, block.y));

    const float scale = angleInDegrees ? (CV_PI_F / 180.0f) : 1.0f;

    hipStream_t stream = StreamAccessor::getStream(_stream);

    if (magc.empty())
        polarToCartImpl<false><<<grid, block, 0, stream>>>(shrinkPtr(magc), shrinkPtr(anglec), shrinkPtr(xc), shrinkPtr(yc), scale, anglec.rows, anglec.cols);
    else
        polarToCartImpl<true><<<grid, block, 0, stream>>>(shrinkPtr(magc), shrinkPtr(anglec), shrinkPtr(xc), shrinkPtr(yc), scale, anglec.rows, anglec.cols);

    CV_CUDEV_SAFE_CALL( hipGetLastError() );

    syncOutput(x, _x, _stream);
    syncOutput(y, _y, _stream);

    if (stream == 0)
        CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

#endif
