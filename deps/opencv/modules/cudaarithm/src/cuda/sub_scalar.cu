#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void subScalar(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int);

namespace
{
    template <typename SrcType, typename ScalarType, typename DstType> struct SubScalarOp : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            return saturate_cast<DstType>(saturate_cast<ScalarType>(a) - val);
        }
    };

    template <typename SrcType, typename ScalarType, typename DstType> struct SubScalarOpInv : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            return saturate_cast<DstType>(val - saturate_cast<ScalarType>(a));
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename SrcType, typename ScalarDepth, typename DstType>
    void subScalarImpl(const GpuMat& src, cv::Scalar value, bool inv, GpuMat& dst, const GpuMat& mask, Stream& stream)
    {
        typedef typename MakeVec<ScalarDepth, VecTraits<SrcType>::cn>::type ScalarType;

        cv::Scalar_<ScalarDepth> value_ = value;

        if (inv)
        {
            SubScalarOpInv<SrcType, ScalarType, DstType> op;
            op.val = VecTraits<ScalarType>::make(value_.val);

            if (mask.data)
                gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, globPtr<uchar>(mask), stream);
            else
                gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, stream);
        }
        else
        {
            SubScalarOp<SrcType, ScalarType, DstType> op;
            op.val = VecTraits<ScalarType>::make(value_.val);

            if (mask.data)
                gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, globPtr<uchar>(mask), stream);
            else
                gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, stream);
        }
    }
}

void subScalar(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat& mask, Stream& stream);
    static const func_t funcs[7][7][4] =
    {
        {
            {subScalarImpl<uchar, float, uchar>, subScalarImpl<uchar2, float, uchar2>, subScalarImpl<uchar3, float, uchar3>, subScalarImpl<uchar4, float, uchar4>},
            {subScalarImpl<uchar, float, schar>, subScalarImpl<uchar2, float, char2>, subScalarImpl<uchar3, float, char3>, subScalarImpl<uchar4, float, char4>},
            {subScalarImpl<uchar, float, ushort>, subScalarImpl<uchar2, float, ushort2>, subScalarImpl<uchar3, float, ushort3>, subScalarImpl<uchar4, float, ushort4>},
            {subScalarImpl<uchar, float, short>, subScalarImpl<uchar2, float, short2>, subScalarImpl<uchar3, float, short3>, subScalarImpl<uchar4, float, short4>},
            {subScalarImpl<uchar, float, int>, subScalarImpl<uchar2, float, int2>, subScalarImpl<uchar3, float, int3>, subScalarImpl<uchar4, float, int4>},
            {subScalarImpl<uchar, float, float>, subScalarImpl<uchar2, float, float2>, subScalarImpl<uchar3, float, float3>, subScalarImpl<uchar4, float, float4>},
            {subScalarImpl<uchar, double, double>, subScalarImpl<uchar2, double, double2>, subScalarImpl<uchar3, double, double3>, subScalarImpl<uchar4, double, double4>}
        },
        {
            {subScalarImpl<schar, float, uchar>, subScalarImpl<char2, float, uchar2>, subScalarImpl<char3, float, uchar3>, subScalarImpl<char4, float, uchar4>},
            {subScalarImpl<schar, float, schar>, subScalarImpl<char2, float, char2>, subScalarImpl<char3, float, char3>, subScalarImpl<char4, float, char4>},
            {subScalarImpl<schar, float, ushort>, subScalarImpl<char2, float, ushort2>, subScalarImpl<char3, float, ushort3>, subScalarImpl<char4, float, ushort4>},
            {subScalarImpl<schar, float, short>, subScalarImpl<char2, float, short2>, subScalarImpl<char3, float, short3>, subScalarImpl<char4, float, short4>},
            {subScalarImpl<schar, float, int>, subScalarImpl<char2, float, int2>, subScalarImpl<char3, float, int3>, subScalarImpl<char4, float, int4>},
            {subScalarImpl<schar, float, float>, subScalarImpl<char2, float, float2>, subScalarImpl<char3, float, float3>, subScalarImpl<char4, float, float4>},
            {subScalarImpl<schar, double, double>, subScalarImpl<char2, double, double2>, subScalarImpl<char3, double, double3>, subScalarImpl<char4, double, double4>}
        },
        {
            {0 /*subScalarImpl<ushort, float, uchar>*/, 0 /*subScalarImpl<ushort2, float, uchar2>*/, 0 /*subScalarImpl<ushort3, float, uchar3>*/, 0 /*subScalarImpl<ushort4, float, uchar4>*/},
            {0 /*subScalarImpl<ushort, float, schar>*/, 0 /*subScalarImpl<ushort2, float, char2>*/, 0 /*subScalarImpl<ushort3, float, char3>*/, 0 /*subScalarImpl<ushort4, float, char4>*/},
            {subScalarImpl<ushort, float, ushort>, subScalarImpl<ushort2, float, ushort2>, subScalarImpl<ushort3, float, ushort3>, subScalarImpl<ushort4, float, ushort4>},
            {subScalarImpl<ushort, float, short>, subScalarImpl<ushort2, float, short2>, subScalarImpl<ushort3, float, short3>, subScalarImpl<ushort4, float, short4>},
            {subScalarImpl<ushort, float, int>, subScalarImpl<ushort2, float, int2>, subScalarImpl<ushort3, float, int3>, subScalarImpl<ushort4, float, int4>},
            {subScalarImpl<ushort, float, float>, subScalarImpl<ushort2, float, float2>, subScalarImpl<ushort3, float, float3>, subScalarImpl<ushort4, float, float4>},
            {subScalarImpl<ushort, double, double>, subScalarImpl<ushort2, double, double2>, subScalarImpl<ushort3, double, double3>, subScalarImpl<ushort4, double, double4>}
        },
        {
            {0 /*subScalarImpl<short, float, uchar>*/, 0 /*subScalarImpl<short2, float, uchar2>*/, 0 /*subScalarImpl<short3, float, uchar3>*/, 0 /*subScalarImpl<short4, float, uchar4>*/},
            {0 /*subScalarImpl<short, float, schar>*/, 0 /*subScalarImpl<short2, float, char2>*/, 0 /*subScalarImpl<short3, float, char3>*/, 0 /*subScalarImpl<short4, float, char4>*/},
            {subScalarImpl<short, float, ushort>, subScalarImpl<short2, float, ushort2>, subScalarImpl<short3, float, ushort3>, subScalarImpl<short4, float, ushort4>},
            {subScalarImpl<short, float, short>, subScalarImpl<short2, float, short2>, subScalarImpl<short3, float, short3>, subScalarImpl<short4, float, short4>},
            {subScalarImpl<short, float, int>, subScalarImpl<short2, float, int2>, subScalarImpl<short3, float, int3>, subScalarImpl<short4, float, int4>},
            {subScalarImpl<short, float, float>, subScalarImpl<short2, float, float2>, subScalarImpl<short3, float, float3>, subScalarImpl<short4, float, float4>},
            {subScalarImpl<short, double, double>, subScalarImpl<short2, double, double2>, subScalarImpl<short3, double, double3>, subScalarImpl<short4, double, double4>}
        },
        {
            {0 /*subScalarImpl<int, float, uchar>*/, 0 /*subScalarImpl<int2, float, uchar2>*/, 0 /*subScalarImpl<int3, float, uchar3>*/, 0 /*subScalarImpl<int4, float, uchar4>*/},
            {0 /*subScalarImpl<int, float, schar>*/, 0 /*subScalarImpl<int2, float, char2>*/, 0 /*subScalarImpl<int3, float, char3>*/, 0 /*subScalarImpl<int4, float, char4>*/},
            {0 /*subScalarImpl<int, float, ushort>*/, 0 /*subScalarImpl<int2, float, ushort2>*/, 0 /*subScalarImpl<int3, float, ushort3>*/, 0 /*subScalarImpl<int4, float, ushort4>*/},
            {0 /*subScalarImpl<int, float, short>*/, 0 /*subScalarImpl<int2, float, short2>*/, 0 /*subScalarImpl<int3, float, short3>*/, 0 /*subScalarImpl<int4, float, short4>*/},
            {subScalarImpl<int, float, int>, subScalarImpl<int2, float, int2>, subScalarImpl<int3, float, int3>, subScalarImpl<int4, float, int4>},
            {subScalarImpl<int, float, float>, subScalarImpl<int2, float, float2>, subScalarImpl<int3, float, float3>, subScalarImpl<int4, float, float4>},
            {subScalarImpl<int, double, double>, subScalarImpl<int2, double, double2>, subScalarImpl<int3, double, double3>, subScalarImpl<int4, double, double4>}
        },
        {
            {0 /*subScalarImpl<float, float, uchar>*/, 0 /*subScalarImpl<float2, float, uchar2>*/, 0 /*subScalarImpl<float3, float, uchar3>*/, 0 /*subScalarImpl<float4, float, uchar4>*/},
            {0 /*subScalarImpl<float, float, schar>*/, 0 /*subScalarImpl<float2, float, char2>*/, 0 /*subScalarImpl<float3, float, char3>*/, 0 /*subScalarImpl<float4, float, char4>*/},
            {0 /*subScalarImpl<float, float, ushort>*/, 0 /*subScalarImpl<float2, float, ushort2>*/, 0 /*subScalarImpl<float3, float, ushort3>*/, 0 /*subScalarImpl<float4, float, ushort4>*/},
            {0 /*subScalarImpl<float, float, short>*/, 0 /*subScalarImpl<float2, float, short2>*/, 0 /*subScalarImpl<float3, float, short3>*/, 0 /*subScalarImpl<float4, float, short4>*/},
            {0 /*subScalarImpl<float, float, int>*/, 0 /*subScalarImpl<float2, float, int2>*/, 0 /*subScalarImpl<float3, float, int3>*/, 0 /*subScalarImpl<float4, float, int4>*/},
            {subScalarImpl<float, float, float>, subScalarImpl<float2, float, float2>, subScalarImpl<float3, float, float3>, subScalarImpl<float4, float, float4>},
            {subScalarImpl<float, double, double>, subScalarImpl<float2, double, double2>, subScalarImpl<float3, double, double3>, subScalarImpl<float4, double, double4>}
        },
        {
            {0 /*subScalarImpl<double, double, uchar>*/, 0 /*subScalarImpl<double2, double, uchar2>*/, 0 /*subScalarImpl<double3, double, uchar3>*/, 0 /*subScalarImpl<double4, double, uchar4>*/},
            {0 /*subScalarImpl<double, double, schar>*/, 0 /*subScalarImpl<double2, double, char2>*/, 0 /*subScalarImpl<double3, double, char3>*/, 0 /*subScalarImpl<double4, double, char4>*/},
            {0 /*subScalarImpl<double, double, ushort>*/, 0 /*subScalarImpl<double2, double, ushort2>*/, 0 /*subScalarImpl<double3, double, ushort3>*/, 0 /*subScalarImpl<double4, double, ushort4>*/},
            {0 /*subScalarImpl<double, double, short>*/, 0 /*subScalarImpl<double2, double, short2>*/, 0 /*subScalarImpl<double3, double, short3>*/, 0 /*subScalarImpl<double4, double, short4>*/},
            {0 /*subScalarImpl<double, double, int>*/, 0 /*subScalarImpl<double2, double, int2>*/, 0 /*subScalarImpl<double3, double, int3>*/, 0 /*subScalarImpl<double4, double, int4>*/},
            {0 /*subScalarImpl<double, double, float>*/, 0 /*subScalarImpl<double2, double, float2>*/, 0 /*subScalarImpl<double3, double, float3>*/, 0 /*subScalarImpl<double4, double, float4>*/},
            {subScalarImpl<double, double, double>, subScalarImpl<double2, double, double2>, subScalarImpl<double3, double, double3>, subScalarImpl<double4, double, double4>}
        }
    };

    const int sdepth = src.depth();
    const int ddepth = dst.depth();
    const int cn = src.channels();

    CV_DbgAssert( sdepth <= CV_64F && ddepth <= CV_64F && cn <= 4 );

    const func_t func = funcs[sdepth][ddepth][cn - 1];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src, val, inv, dst, mask, stream);
}

#endif
