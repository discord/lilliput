#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void mulScalar(const GpuMat& src, cv::Scalar val, bool, GpuMat& dst, const GpuMat& mask, double scale, Stream& stream, int);

namespace
{
    template <typename SrcType, typename ScalarType, typename DstType> struct MulScalarOp : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            return saturate_cast<DstType>(saturate_cast<ScalarType>(a) * val);
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename SrcType, typename ScalarDepth, typename DstType>
    void mulScalarImpl(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream)
    {
        typedef typename MakeVec<ScalarDepth, VecTraits<SrcType>::cn>::type ScalarType;

        cv::Scalar_<ScalarDepth> value_ = value;

        MulScalarOp<SrcType, ScalarType, DstType> op;
        op.val = VecTraits<ScalarType>::make(value_.val);

        gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<DstType>(dst), op, stream);
    }
}

void mulScalar(const GpuMat& src, cv::Scalar val, bool, GpuMat& dst, const GpuMat&, double scale, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src, cv::Scalar val, GpuMat& dst, Stream& stream);
    static const func_t funcs[7][7][4] =
    {
        {
            {mulScalarImpl<uchar, float, uchar>, mulScalarImpl<uchar2, float, uchar2>, mulScalarImpl<uchar3, float, uchar3>, mulScalarImpl<uchar4, float, uchar4>},
            {mulScalarImpl<uchar, float, schar>, mulScalarImpl<uchar2, float, char2>, mulScalarImpl<uchar3, float, char3>, mulScalarImpl<uchar4, float, char4>},
            {mulScalarImpl<uchar, float, ushort>, mulScalarImpl<uchar2, float, ushort2>, mulScalarImpl<uchar3, float, ushort3>, mulScalarImpl<uchar4, float, ushort4>},
            {mulScalarImpl<uchar, float, short>, mulScalarImpl<uchar2, float, short2>, mulScalarImpl<uchar3, float, short3>, mulScalarImpl<uchar4, float, short4>},
            {mulScalarImpl<uchar, float, int>, mulScalarImpl<uchar2, float, int2>, mulScalarImpl<uchar3, float, int3>, mulScalarImpl<uchar4, float, int4>},
            {mulScalarImpl<uchar, float, float>, mulScalarImpl<uchar2, float, float2>, mulScalarImpl<uchar3, float, float3>, mulScalarImpl<uchar4, float, float4>},
            {mulScalarImpl<uchar, double, double>, mulScalarImpl<uchar2, double, double2>, mulScalarImpl<uchar3, double, double3>, mulScalarImpl<uchar4, double, double4>}
        },
        {
            {mulScalarImpl<schar, float, uchar>, mulScalarImpl<char2, float, uchar2>, mulScalarImpl<char3, float, uchar3>, mulScalarImpl<char4, float, uchar4>},
            {mulScalarImpl<schar, float, schar>, mulScalarImpl<char2, float, char2>, mulScalarImpl<char3, float, char3>, mulScalarImpl<char4, float, char4>},
            {mulScalarImpl<schar, float, ushort>, mulScalarImpl<char2, float, ushort2>, mulScalarImpl<char3, float, ushort3>, mulScalarImpl<char4, float, ushort4>},
            {mulScalarImpl<schar, float, short>, mulScalarImpl<char2, float, short2>, mulScalarImpl<char3, float, short3>, mulScalarImpl<char4, float, short4>},
            {mulScalarImpl<schar, float, int>, mulScalarImpl<char2, float, int2>, mulScalarImpl<char3, float, int3>, mulScalarImpl<char4, float, int4>},
            {mulScalarImpl<schar, float, float>, mulScalarImpl<char2, float, float2>, mulScalarImpl<char3, float, float3>, mulScalarImpl<char4, float, float4>},
            {mulScalarImpl<schar, double, double>, mulScalarImpl<char2, double, double2>, mulScalarImpl<char3, double, double3>, mulScalarImpl<char4, double, double4>}
        },
        {
            {0 /*mulScalarImpl<ushort, float, uchar>*/, 0 /*mulScalarImpl<ushort2, float, uchar2>*/, 0 /*mulScalarImpl<ushort3, float, uchar3>*/, 0 /*mulScalarImpl<ushort4, float, uchar4>*/},
            {0 /*mulScalarImpl<ushort, float, schar>*/, 0 /*mulScalarImpl<ushort2, float, char2>*/, 0 /*mulScalarImpl<ushort3, float, char3>*/, 0 /*mulScalarImpl<ushort4, float, char4>*/},
            {mulScalarImpl<ushort, float, ushort>, mulScalarImpl<ushort2, float, ushort2>, mulScalarImpl<ushort3, float, ushort3>, mulScalarImpl<ushort4, float, ushort4>},
            {mulScalarImpl<ushort, float, short>, mulScalarImpl<ushort2, float, short2>, mulScalarImpl<ushort3, float, short3>, mulScalarImpl<ushort4, float, short4>},
            {mulScalarImpl<ushort, float, int>, mulScalarImpl<ushort2, float, int2>, mulScalarImpl<ushort3, float, int3>, mulScalarImpl<ushort4, float, int4>},
            {mulScalarImpl<ushort, float, float>, mulScalarImpl<ushort2, float, float2>, mulScalarImpl<ushort3, float, float3>, mulScalarImpl<ushort4, float, float4>},
            {mulScalarImpl<ushort, double, double>, mulScalarImpl<ushort2, double, double2>, mulScalarImpl<ushort3, double, double3>, mulScalarImpl<ushort4, double, double4>}
        },
        {
            {0 /*mulScalarImpl<short, float, uchar>*/, 0 /*mulScalarImpl<short2, float, uchar2>*/, 0 /*mulScalarImpl<short3, float, uchar3>*/, 0 /*mulScalarImpl<short4, float, uchar4>*/},
            {0 /*mulScalarImpl<short, float, schar>*/, 0 /*mulScalarImpl<short2, float, char2>*/, 0 /*mulScalarImpl<short3, float, char3>*/, 0 /*mulScalarImpl<short4, float, char4>*/},
            {mulScalarImpl<short, float, ushort>, mulScalarImpl<short2, float, ushort2>, mulScalarImpl<short3, float, ushort3>, mulScalarImpl<short4, float, ushort4>},
            {mulScalarImpl<short, float, short>, mulScalarImpl<short2, float, short2>, mulScalarImpl<short3, float, short3>, mulScalarImpl<short4, float, short4>},
            {mulScalarImpl<short, float, int>, mulScalarImpl<short2, float, int2>, mulScalarImpl<short3, float, int3>, mulScalarImpl<short4, float, int4>},
            {mulScalarImpl<short, float, float>, mulScalarImpl<short2, float, float2>, mulScalarImpl<short3, float, float3>, mulScalarImpl<short4, float, float4>},
            {mulScalarImpl<short, double, double>, mulScalarImpl<short2, double, double2>, mulScalarImpl<short3, double, double3>, mulScalarImpl<short4, double, double4>}
        },
        {
            {0 /*mulScalarImpl<int, float, uchar>*/, 0 /*mulScalarImpl<int2, float, uchar2>*/, 0 /*mulScalarImpl<int3, float, uchar3>*/, 0 /*mulScalarImpl<int4, float, uchar4>*/},
            {0 /*mulScalarImpl<int, float, schar>*/, 0 /*mulScalarImpl<int2, float, char2>*/, 0 /*mulScalarImpl<int3, float, char3>*/, 0 /*mulScalarImpl<int4, float, char4>*/},
            {0 /*mulScalarImpl<int, float, ushort>*/, 0 /*mulScalarImpl<int2, float, ushort2>*/, 0 /*mulScalarImpl<int3, float, ushort3>*/, 0 /*mulScalarImpl<int4, float, ushort4>*/},
            {0 /*mulScalarImpl<int, float, short>*/, 0 /*mulScalarImpl<int2, float, short2>*/, 0 /*mulScalarImpl<int3, float, short3>*/, 0 /*mulScalarImpl<int4, float, short4>*/},
            {mulScalarImpl<int, float, int>, mulScalarImpl<int2, float, int2>, mulScalarImpl<int3, float, int3>, mulScalarImpl<int4, float, int4>},
            {mulScalarImpl<int, float, float>, mulScalarImpl<int2, float, float2>, mulScalarImpl<int3, float, float3>, mulScalarImpl<int4, float, float4>},
            {mulScalarImpl<int, double, double>, mulScalarImpl<int2, double, double2>, mulScalarImpl<int3, double, double3>, mulScalarImpl<int4, double, double4>}
        },
        {
            {0 /*mulScalarImpl<float, float, uchar>*/, 0 /*mulScalarImpl<float2, float, uchar2>*/, 0 /*mulScalarImpl<float3, float, uchar3>*/, 0 /*mulScalarImpl<float4, float, uchar4>*/},
            {0 /*mulScalarImpl<float, float, schar>*/, 0 /*mulScalarImpl<float2, float, char2>*/, 0 /*mulScalarImpl<float3, float, char3>*/, 0 /*mulScalarImpl<float4, float, char4>*/},
            {0 /*mulScalarImpl<float, float, ushort>*/, 0 /*mulScalarImpl<float2, float, ushort2>*/, 0 /*mulScalarImpl<float3, float, ushort3>*/, 0 /*mulScalarImpl<float4, float, ushort4>*/},
            {0 /*mulScalarImpl<float, float, short>*/, 0 /*mulScalarImpl<float2, float, short2>*/, 0 /*mulScalarImpl<float3, float, short3>*/, 0 /*mulScalarImpl<float4, float, short4>*/},
            {0 /*mulScalarImpl<float, float, int>*/, 0 /*mulScalarImpl<float2, float, int2>*/, 0 /*mulScalarImpl<float3, float, int3>*/, 0 /*mulScalarImpl<float4, float, int4>*/},
            {mulScalarImpl<float, float, float>, mulScalarImpl<float2, float, float2>, mulScalarImpl<float3, float, float3>, mulScalarImpl<float4, float, float4>},
            {mulScalarImpl<float, double, double>, mulScalarImpl<float2, double, double2>, mulScalarImpl<float3, double, double3>, mulScalarImpl<float4, double, double4>}
        },
        {
            {0 /*mulScalarImpl<double, double, uchar>*/, 0 /*mulScalarImpl<double2, double, uchar2>*/, 0 /*mulScalarImpl<double3, double, uchar3>*/, 0 /*mulScalarImpl<double4, double, uchar4>*/},
            {0 /*mulScalarImpl<double, double, schar>*/, 0 /*mulScalarImpl<double2, double, char2>*/, 0 /*mulScalarImpl<double3, double, char3>*/, 0 /*mulScalarImpl<double4, double, char4>*/},
            {0 /*mulScalarImpl<double, double, ushort>*/, 0 /*mulScalarImpl<double2, double, ushort2>*/, 0 /*mulScalarImpl<double3, double, ushort3>*/, 0 /*mulScalarImpl<double4, double, ushort4>*/},
            {0 /*mulScalarImpl<double, double, short>*/, 0 /*mulScalarImpl<double2, double, short2>*/, 0 /*mulScalarImpl<double3, double, short3>*/, 0 /*mulScalarImpl<double4, double, short4>*/},
            {0 /*mulScalarImpl<double, double, int>*/, 0 /*mulScalarImpl<double2, double, int2>*/, 0 /*mulScalarImpl<double3, double, int3>*/, 0 /*mulScalarImpl<double4, double, int4>*/},
            {0 /*mulScalarImpl<double, double, float>*/, 0 /*mulScalarImpl<double2, double, float2>*/, 0 /*mulScalarImpl<double3, double, float3>*/, 0 /*mulScalarImpl<double4, double, float4>*/},
            {mulScalarImpl<double, double, double>, mulScalarImpl<double2, double, double2>, mulScalarImpl<double3, double, double3>, mulScalarImpl<double4, double, double4>}
        }
    };

    const int sdepth = src.depth();
    const int ddepth = dst.depth();
    const int cn = src.channels();

    CV_DbgAssert( sdepth <= CV_64F && ddepth <= CV_64F && cn <= 4 );

    val[0] *= scale;
    val[1] *= scale;
    val[2] *= scale;
    val[3] *= scale;

    const func_t func = funcs[sdepth][ddepth][cn - 1];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src, val, dst, stream);
}

#endif
