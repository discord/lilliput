#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                          License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 2013, OpenCV Foundation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "test_precomp.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;
using namespace cvtest;

////////////////////////////////////////////////////////////////////////////////
// SqrtTest

template <typename T>
class SqrtTest : public ::testing::Test
{
public:
    void test_gpumat()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src = randomMat(size, type);

        GpuMat_<T> d_src(src);

        GpuMat_<T> dst = sqrt_(d_src);

        Mat dst_gold;
        cv::sqrt(src, dst_gold);

        EXPECT_MAT_NEAR(dst_gold, dst, 1e-4);
    }

    void test_expr()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src1 = randomMat(size, type);
        Mat src2 = randomMat(size, type);

        GpuMat_<T> d_src1(src1), d_src2(src2);

        GpuMat_<T> dst = sqrt_(d_src1 * d_src2);

        Mat dst_gold;
        cv::multiply(src1, src2, dst_gold);
        cv::sqrt(dst_gold, dst_gold);

        EXPECT_MAT_NEAR(dst_gold, dst, 1e-4);
    }
};

TYPED_TEST_CASE(SqrtTest, float);

TYPED_TEST(SqrtTest, GpuMat)
{
    SqrtTest<TypeParam>::test_gpumat();
}

TYPED_TEST(SqrtTest, Expr)
{
    SqrtTest<TypeParam>::test_expr();
}

////////////////////////////////////////////////////////////////////////////////
// MagnitudeTest

template <typename T>
class MagnitudeTest : public ::testing::Test
{
public:
    void test_accuracy()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src1 = randomMat(size, type);
        Mat src2 = randomMat(size, type);

        GpuMat_<T> d_src1(src1), d_src2(src2);

        GpuMat_<T> dst1 = hypot_(d_src1, d_src2);
        GpuMat_<T> dst2 = magnitude_(d_src1, d_src2);
        GpuMat_<T> dst3 = sqrt_(sqr_(d_src1) + sqr_(d_src2));

        EXPECT_MAT_NEAR(dst1, dst2, 1e-4);
        EXPECT_MAT_NEAR(dst2, dst3, 0.0);
    }
};

TYPED_TEST_CASE(MagnitudeTest, float);

TYPED_TEST(MagnitudeTest, Accuracy)
{
    MagnitudeTest<TypeParam>::test_accuracy();
}

////////////////////////////////////////////////////////////////////////////////
// PowTest

template <typename T>
class PowTest : public ::testing::Test
{
public:
    void test_accuracy()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src = randomMat(size, type);

        GpuMat_<T> d_src(src);

        GpuMat_<T> dst1 = pow_(d_src, 0.5);
        GpuMat_<T> dst2 = sqrt_(d_src);

        EXPECT_MAT_NEAR(dst1, dst2, 1e-5);
    }
};

TYPED_TEST_CASE(PowTest, float);

TYPED_TEST(PowTest, Accuracy)
{
    PowTest<TypeParam>::test_accuracy();
}
