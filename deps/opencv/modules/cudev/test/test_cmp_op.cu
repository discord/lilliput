#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                          License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 2013, OpenCV Foundation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "test_precomp.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;
using namespace cvtest;

typedef ::testing::Types<uchar, ushort, short, int, float> AllTypes;

////////////////////////////////////////////////////////////////////////////////
// LessTest

template <typename T>
class LessTest : public ::testing::Test
{
public:
    void test_gpumat_gpumat()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src1 = randomMat(size, type);
        Mat src2 = randomMat(size, type);

        GpuMat_<T> d_src1(src1), d_src2(src2);

        GpuMat_<uchar> dst = (d_src1 < d_src2) * 255;

        Mat dst_gold;
        cv::compare(src1, src2, dst_gold, CMP_LT);

        EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
    }
};

TYPED_TEST_CASE(LessTest, AllTypes);

TYPED_TEST(LessTest, GpuMat_GpuMat)
{
    LessTest<TypeParam>::test_gpumat_gpumat();
}

////////////////////////////////////////////////////////////////////////////////
// MinTest

template <typename T>
class MinTest : public ::testing::Test
{
public:
    void test_gpumat_gpumat()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src1 = randomMat(size, type);
        Mat src2 = randomMat(size, type);

        GpuMat_<T> d_src1(src1), d_src2(src2);

        GpuMat_<T> dst = min_(d_src1, d_src2);

        Mat dst_gold;
        cv::min(src1, src2, dst_gold);

        EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
    }
};

TYPED_TEST_CASE(MinTest, AllTypes);

TYPED_TEST(MinTest, GpuMat_GpuMat)
{
    MinTest<TypeParam>::test_gpumat_gpumat();
}

////////////////////////////////////////////////////////////////////////////////
// ThreshBinaryTest

typedef ::testing::Types<uchar, short, float> ThreshTypes;

template <typename T>
class ThreshBinaryTest : public ::testing::Test
{
public:
    void test_gpumat()
    {
        const Size size = randomSize(100, 400);
        const int type = DataType<T>::type;

        Mat src = randomMat(size, type);

        GpuMat_<T> d_src(src);

        GpuMat_<T> dst = threshBinary_(d_src, 128, 0);

        Mat dst_gold;
        cv::threshold(src, dst_gold, 128, 0, THRESH_BINARY);

        EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
    }
};

TYPED_TEST_CASE(ThreshBinaryTest, ThreshTypes);

TYPED_TEST(ThreshBinaryTest, GpuMat)
{
    ThreshBinaryTest<TypeParam>::test_gpumat();
}
