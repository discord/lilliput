#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                          License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 2013, OpenCV Foundation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "test_precomp.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;
using namespace cvtest;

// remap

enum { HALF_SIZE=0, UPSIDE_DOWN, REFLECTION_X, REFLECTION_BOTH };

static void generateMap(Mat& mapx, Mat& mapy, int remapMode)
{
    for (int j = 0; j < mapx.rows; ++j)
    {
        for (int i = 0; i < mapx.cols; ++i)
        {
            switch (remapMode)
            {
            case HALF_SIZE:
                if (i > mapx.cols*0.25 && i < mapx.cols*0.75 && j > mapx.rows*0.25 && j < mapx.rows*0.75)
                {
                    mapx.at<float>(j,i) = 2.f * (i - mapx.cols * 0.25f) + 0.5f;
                    mapy.at<float>(j,i) = 2.f * (j - mapx.rows * 0.25f) + 0.5f;
                }
                else
                {
                    mapx.at<float>(j,i) = 0.f;
                    mapy.at<float>(j,i) = 0.f;
                }
                break;
            case UPSIDE_DOWN:
                mapx.at<float>(j,i) = static_cast<float>(i);
                mapy.at<float>(j,i) = static_cast<float>(mapx.rows - j);
                break;
            case REFLECTION_X:
                mapx.at<float>(j,i) = static_cast<float>(mapx.cols - i);
                mapy.at<float>(j,i) = static_cast<float>(j);
                break;
            case REFLECTION_BOTH:
                mapx.at<float>(j,i) = static_cast<float>(mapx.cols - i);
                mapy.at<float>(j,i) = static_cast<float>(mapx.rows - j);
                break;
            } // end of switch
        }
    }
}

static void test_remap(int remapMode)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_32FC1, 0, 1);

    Mat mapx(size, CV_32FC1);
    Mat mapy(size, CV_32FC1);
    generateMap(mapx, mapy, remapMode);

    GpuMat_<float> d_src(src);
    GpuMat_<float> d_mapx(mapx);
    GpuMat_<float> d_mapy(mapy);

    GpuMat_<float> dst = remap_(interNearest(brdReplicate(d_src)), d_mapx, d_mapy);

    Mat dst_gold;
    cv::remap(src, dst_gold, mapx, mapy, INTER_NEAREST, BORDER_REPLICATE);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

TEST(Remap, HALF_SIZE)
{
    test_remap(HALF_SIZE);
}

TEST(Remap, UPSIDE_DOWN)
{
    test_remap(UPSIDE_DOWN);
}

TEST(Remap, REFLECTION_X)
{
    test_remap(REFLECTION_X);
}

TEST(Remap, REFLECTION_BOTH)
{
    test_remap(REFLECTION_BOTH);
}

// resize

TEST(Resize, Upscale)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_32FC1, 0, 1);

    GpuMat_<float> d_src(src);
    Texture<float> tex_src(d_src);

    GpuMat_<float> dst1 = resize_(interCubic(tex_src), 2, 2);

    Mat mapx(size.height * 2, size.width * 2, CV_32FC1);
    Mat mapy(size.height * 2, size.width * 2, CV_32FC1);

    for (int y = 0; y < mapx.rows; ++y)
    {
        for (int x = 0; x < mapx.cols; ++x)
        {
            mapx.at<float>(y, x) = static_cast<float>(x / 2);
            mapy.at<float>(y, x) = static_cast<float>(y / 2);
        }
    }

    GpuMat_<float> d_mapx(mapx);
    GpuMat_<float> d_mapy(mapy);

    GpuMat_<float> dst2 = remap_(interCubic(brdReplicate(d_src)), d_mapx, d_mapy);

    EXPECT_MAT_NEAR(dst1, dst2, 0.0);
}

TEST(Resize, Downscale)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_32FC1, 0, 1);
    const float fx = 1.0f / 3.0f;
    const float fy = 1.0f / 3.0f;

    GpuMat_<float> d_src(src);
    Texture<float> tex_src(d_src);

    GpuMat_<float> dst1 = resize_(interArea(tex_src, Size(3, 3)), fx, fy);

    Mat mapx(cv::saturate_cast<int>(size.height * fy), cv::saturate_cast<int>(size.width * fx), CV_32FC1);
    Mat mapy(cv::saturate_cast<int>(size.height * fy), cv::saturate_cast<int>(size.width * fx), CV_32FC1);

    for (int y = 0; y < mapx.rows; ++y)
    {
        for (int x = 0; x < mapx.cols; ++x)
        {
            mapx.at<float>(y, x) = x / fx;
            mapy.at<float>(y, x) = y / fy;
        }
    }

    GpuMat_<float> d_mapx(mapx);
    GpuMat_<float> d_mapy(mapy);

    GpuMat_<float> dst2 = remap_(interArea(brdReplicate(d_src), Size(3, 3)), d_mapx, d_mapy);

    EXPECT_MAT_NEAR(dst1, dst2, 0.0);
}

// warpAffine & warpPerspective

Mat createAffineTransfomMatrix(Size srcSize, float angle, bool perspective)
{
    cv::Mat M(perspective ? 3 : 2, 3, CV_32FC1);

    {
        M.at<float>(0, 0) = std::cos(angle); M.at<float>(0, 1) = -std::sin(angle); M.at<float>(0, 2) = static_cast<float>(srcSize.width / 2);
        M.at<float>(1, 0) = std::sin(angle); M.at<float>(1, 1) =  std::cos(angle); M.at<float>(1, 2) = 0.0f;
    }
    if (perspective)
    {
        M.at<float>(2, 0) = 0.0f           ; M.at<float>(2, 1) =  0.0f           ; M.at<float>(2, 2) = 1.0f;
    }

    return M;
}

TEST(WarpAffine, Rotation)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_32FC1, 0, 1);
    Mat M = createAffineTransfomMatrix(size, static_cast<float>(CV_PI / 4), false);

    GpuMat_<float> d_src(src);
    GpuMat_<float> d_M;
    createContinuous(M.size(), M.type(), d_M);
    d_M.upload(M);

    GpuMat_<float> dst = warpAffine_(interNearest(brdConstant(d_src)), size, d_M);

    Mat dst_gold;
    cv::warpAffine(src, dst_gold, M, size, INTER_NEAREST | WARP_INVERSE_MAP);

    EXPECT_MAT_SIMILAR(dst_gold, dst, 1e-3);
}

TEST(WarpPerspective, Rotation)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_32FC1, 0, 1);
    Mat M = createAffineTransfomMatrix(size, static_cast<float>(CV_PI / 4), true);

    GpuMat_<float> d_src(src);
    GpuMat_<float> d_M;
    createContinuous(M.size(), M.type(), d_M);
    d_M.upload(M);

    GpuMat_<float> dst = warpPerspective_(interNearest(brdConstant(d_src)), size, d_M);

    Mat dst_gold;
    cv::warpPerspective(src, dst_gold, M, size, INTER_NEAREST | WARP_INVERSE_MAP);

    EXPECT_MAT_SIMILAR(dst_gold, dst, 1e-3);
}
