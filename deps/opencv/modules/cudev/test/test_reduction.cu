#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                          License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Copyright (C) 2013, OpenCV Foundation, all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "test_precomp.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;
using namespace cvtest;

TEST(Sum, GpuMat)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<float> dst = sum_(d_src);
    float res;
    dst.download(_OutputArray(&res, 1));

    Scalar dst_gold = cv::sum(src);

    ASSERT_FLOAT_EQ(static_cast<float>(dst_gold[0]), res);
}

TEST(Sum, Expr)
{
    const Size size = randomSize(100, 400);

    Mat src1 = randomMat(size, CV_32FC1, 0, 1);
    Mat src2 = randomMat(size, CV_32FC1, 0, 1);

    GpuMat_<float> d_src1(src1), d_src2(src2);

    GpuMat_<float> dst = sum_(abs_(d_src1 - d_src2));
    float res;
    dst.download(_OutputArray(&res, 1));

    Scalar dst_gold = cv::norm(src1, src2, NORM_L1);

    ASSERT_FLOAT_EQ(static_cast<float>(dst_gold[0]), res);
}

TEST(MinVal, GpuMat)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<float> dst = minVal_(d_src);
    float res;
    dst.download(_OutputArray(&res, 1));

    double res_gold;
    cv::minMaxLoc(src, &res_gold, 0);

    ASSERT_FLOAT_EQ(static_cast<float>(res_gold), res);
}

TEST(MaxVal, Expr)
{
    const Size size = randomSize(100, 400);

    Mat src1 = randomMat(size, CV_32SC1);
    Mat src2 = randomMat(size, CV_32SC1);

    GpuMat_<int> d_src1(src1), d_src2(src2);

    GpuMat_<float> dst = maxVal_(abs_(d_src1 - d_src2));
    float res;
    dst.download(_OutputArray(&res, 1));

    double res_gold = cv::norm(src1, src2, NORM_INF);

    ASSERT_FLOAT_EQ(static_cast<float>(res_gold), res);
}

TEST(MinMaxVal, GpuMat)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<float> dst = minMaxVal_(d_src);
    float res[2];
    dst.download(Mat(1, 2, CV_32FC1, res));

    double res_gold[2];
    cv::minMaxLoc(src, &res_gold[0], &res_gold[1]);

    ASSERT_FLOAT_EQ(static_cast<float>(res_gold[0]), res[0]);
    ASSERT_FLOAT_EQ(static_cast<float>(res_gold[1]), res[1]);
}

TEST(NonZeroCount, Accuracy)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1, 0, 5);

    GpuMat_<uchar> d_src(src);

    GpuMat_<int> dst1 = countNonZero_(d_src);
    GpuMat_<int> dst2 = sum_(cvt_<int>(d_src) != 0);

    EXPECT_MAT_NEAR(dst1, dst2, 0.0);
}

TEST(ReduceToRow, Sum)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<int> dst = reduceToRow_<Sum<int> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 0, REDUCE_SUM, CV_32S);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

TEST(ReduceToRow, Avg)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<float> dst = reduceToRow_<Avg<float> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 0, REDUCE_AVG, CV_32F);

    EXPECT_MAT_NEAR(dst_gold, dst, 1e-4);
}

TEST(ReduceToRow, Min)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<uchar> dst = reduceToRow_<Min<uchar> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 0, REDUCE_MIN);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

TEST(ReduceToRow, Max)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<uchar> dst = reduceToRow_<Max<uchar> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 0, REDUCE_MAX);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

TEST(ReduceToColumn, Sum)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<int> dst = reduceToColumn_<Sum<int> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 1, REDUCE_SUM, CV_32S);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

TEST(ReduceToColumn, Avg)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<float> dst = reduceToColumn_<Avg<float> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 1, REDUCE_AVG, CV_32F);

    EXPECT_MAT_NEAR(dst_gold, dst, 1e-4);
}

TEST(ReduceToColumn, Min)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<uchar> dst = reduceToColumn_<Min<uchar> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 1, REDUCE_MIN);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

TEST(ReduceToColumn, Max)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<uchar> dst = reduceToColumn_<Max<uchar> >(d_src);

    Mat dst_gold;
    cv::reduce(src, dst_gold, 1, REDUCE_MAX);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}

static void calcHistGold(const cv::Mat& src, cv::Mat& hist)
{
    hist.create(1, 256, CV_32SC1);
    hist.setTo(cv::Scalar::all(0));

    int* hist_row = hist.ptr<int>();
    for (int y = 0; y < src.rows; ++y)
    {
        const uchar* src_row = src.ptr(y);

        for (int x = 0; x < src.cols; ++x)
            ++hist_row[src_row[x]];
    }
}

TEST(Histogram, GpuMat)
{
    const Size size = randomSize(100, 400);

    Mat src = randomMat(size, CV_8UC1);

    GpuMat_<uchar> d_src(src);

    GpuMat_<int> dst = histogram_<256>(d_src);

    Mat dst_gold;
    calcHistGold(src, dst_gold);

    EXPECT_MAT_NEAR(dst_gold, dst, 0.0);
}
