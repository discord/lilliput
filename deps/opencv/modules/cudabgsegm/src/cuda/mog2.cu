#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/limits.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace mog2
    {
        ///////////////////////////////////////////////////////////////
        // Utility

        __device__ __forceinline__ float cvt(uchar val)
        {
            return val;
        }
        __device__ __forceinline__ float3 cvt(const uchar3& val)
        {
            return make_float3(val.x, val.y, val.z);
        }
        __device__ __forceinline__ float4 cvt(const uchar4& val)
        {
            return make_float4(val.x, val.y, val.z, val.w);
        }

        __device__ __forceinline__ float sqr(float val)
        {
            return val * val;
        }
        __device__ __forceinline__ float sqr(const float3& val)
        {
            return val.x * val.x + val.y * val.y + val.z * val.z;
        }
        __device__ __forceinline__ float sqr(const float4& val)
        {
            return val.x * val.x + val.y * val.y + val.z * val.z;
        }

        __device__ __forceinline__ float sum(float val)
        {
            return val;
        }
        __device__ __forceinline__ float sum(const float3& val)
        {
            return val.x + val.y + val.z;
        }
        __device__ __forceinline__ float sum(const float4& val)
        {
            return val.x + val.y + val.z;
        }

        template <class Ptr2D>
        __device__ __forceinline__ void swap(Ptr2D& ptr, int x, int y, int k, int rows)
        {
            typename Ptr2D::elem_type val = ptr(k * rows + y, x);
            ptr(k * rows + y, x) = ptr((k + 1) * rows + y, x);
            ptr((k + 1) * rows + y, x) = val;
        }

        ///////////////////////////////////////////////////////////////
        // MOG2

        __constant__ int           c_nmixtures;
        __constant__ float         c_Tb;
        __constant__ float         c_TB;
        __constant__ float         c_Tg;
        __constant__ float         c_varInit;
        __constant__ float         c_varMin;
        __constant__ float         c_varMax;
        __constant__ float         c_tau;
        __constant__ unsigned char c_shadowVal;

        void loadConstants(int nmixtures, float Tb, float TB, float Tg, float varInit, float varMin, float varMax, float tau, unsigned char shadowVal)
        {
            varMin = ::fminf(varMin, varMax);
            varMax = ::fmaxf(varMin, varMax);

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_nmixtures), &nmixtures, sizeof(int)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Tb), &Tb, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_TB), &TB, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Tg), &Tg, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_varInit), &varInit, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_varMin), &varMin, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_varMax), &varMax, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_tau), &tau, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_shadowVal), &shadowVal, sizeof(unsigned char)) );
        }

        template <bool detectShadows, typename SrcT, typename WorkT>
        __global__ void mog2(const PtrStepSz<SrcT> frame, PtrStepb fgmask, PtrStepb modesUsed,
                             PtrStepf gmm_weight, PtrStepf gmm_variance, PtrStep<WorkT> gmm_mean,
                             const float alphaT, const float alpha1, const float prune)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= frame.cols || y >= frame.rows)
                return;

            WorkT pix = cvt(frame(y, x));

            //calculate distances to the modes (+ sort)
            //here we need to go in descending order!!!

            bool background = false; // true - the pixel classified as background

            //internal:

            bool fitsPDF = false; //if it remains zero a new GMM mode will be added

            int nmodes = modesUsed(y, x);
            int nNewModes = nmodes; //current number of modes in GMM

            float totalWeight = 0.0f;

            //go through all modes

            for (int mode = 0; mode < nmodes; ++mode)
            {
                //need only weight if fit is found
                float weight = alpha1 * gmm_weight(mode * frame.rows + y, x) + prune;
                int swap_count = 0;
                //fit not found yet
                if (!fitsPDF)
                {
                    //check if it belongs to some of the remaining modes
                    float var = gmm_variance(mode * frame.rows + y, x);

                    WorkT mean = gmm_mean(mode * frame.rows + y, x);

                    //calculate difference and distance
                    WorkT diff = mean - pix;
                    float dist2 = sqr(diff);

                    //background? - Tb - usually larger than Tg
                    if (totalWeight < c_TB && dist2 < c_Tb * var)
                        background = true;

                    //check fit
                    if (dist2 < c_Tg * var)
                    {
                        //belongs to the mode
                        fitsPDF = true;

                        //update distribution

                        //update weight
                        weight += alphaT;
                        float k = alphaT / weight;

                        //update mean
                        gmm_mean(mode * frame.rows + y, x) = mean - k * diff;

                        //update variance
                        float varnew = var + k * (dist2 - var);

                        //limit the variance
                        varnew = ::fmaxf(varnew, c_varMin);
                        varnew = ::fminf(varnew, c_varMax);

                        gmm_variance(mode * frame.rows + y, x) = varnew;

                        //sort
                        //all other weights are at the same place and
                        //only the matched (iModes) is higher -> just find the new place for it

                        for (int i = mode; i > 0; --i)
                        {
                            //check one up
                            if (weight < gmm_weight((i - 1) * frame.rows + y, x))
                                break;

                            swap_count++;
                            //swap one up
                            swap(gmm_weight, x, y, i - 1, frame.rows);
                            swap(gmm_variance, x, y, i - 1, frame.rows);
                            swap(gmm_mean, x, y, i - 1, frame.rows);
                        }

                        //belongs to the mode - bFitsPDF becomes 1
                    }
                } // !fitsPDF

                //check prune
                if (weight < -prune)
                {
                    weight = 0.0f;
                    nmodes--;
                }

                gmm_weight((mode - swap_count) * frame.rows + y, x) = weight; //update weight by the calculated value
                totalWeight += weight;
            }

            //renormalize weights

            totalWeight = 1.f / totalWeight;
            for (int mode = 0; mode < nmodes; ++mode)
                gmm_weight(mode * frame.rows + y, x) *= totalWeight;

            nmodes = nNewModes;

            //make new mode if needed and exit

            if (!fitsPDF)
            {
                // replace the weakest or add a new one
                int mode = nmodes == c_nmixtures ? c_nmixtures - 1 : nmodes++;

                if (nmodes == 1)
                    gmm_weight(mode * frame.rows + y, x) = 1.f;
                else
                {
                    gmm_weight(mode * frame.rows + y, x) = alphaT;

                    // renormalize all other weights

                    for (int i = 0; i < nmodes - 1; ++i)
                        gmm_weight(i * frame.rows + y, x) *= alpha1;
                }

                // init

                gmm_mean(mode * frame.rows + y, x) = pix;
                gmm_variance(mode * frame.rows + y, x) = c_varInit;

                //sort
                //find the new place for it

                for (int i = nmodes - 1; i > 0; --i)
                {
                    // check one up
                    if (alphaT < gmm_weight((i - 1) * frame.rows + y, x))
                        break;

                    //swap one up
                    swap(gmm_weight, x, y, i - 1, frame.rows);
                    swap(gmm_variance, x, y, i - 1, frame.rows);
                    swap(gmm_mean, x, y, i - 1, frame.rows);
                }
            }

            //set the number of modes
            modesUsed(y, x) = nmodes;

            bool isShadow = false;
            if (detectShadows && !background)
            {
                float tWeight = 0.0f;

                // check all the components  marked as background:
                for (int mode = 0; mode < nmodes; ++mode)
                {
                    WorkT mean = gmm_mean(mode * frame.rows + y, x);

                    WorkT pix_mean = pix * mean;

                    float numerator = sum(pix_mean);
                    float denominator = sqr(mean);

                    // no division by zero allowed
                    if (denominator == 0)
                        break;

                    // if tau < a < 1 then also check the color distortion
                    if (numerator <= denominator && numerator >= c_tau * denominator)
                    {
                        float a = numerator / denominator;

                        WorkT dD = a * mean - pix;

                        if (sqr(dD) < c_Tb * gmm_variance(mode * frame.rows + y, x) * a * a)
                        {
                            isShadow = true;
                            break;
                        }
                    };

                    tWeight += gmm_weight(mode * frame.rows + y, x);
                    if (tWeight > c_TB)
                        break;
                }
            }

            fgmask(y, x) = background ? 0 : isShadow ? c_shadowVal : 255;
        }

        template <typename SrcT, typename WorkT>
        void mog2_caller(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzf variance, PtrStepSzb mean,
                         float alphaT, float prune, bool detectShadows, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            const float alpha1 = 1.0f - alphaT;

            if (detectShadows)
            {
                cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(mog2<true), SrcT, WorkT>, hipFuncCachePreferL1) );

                mog2<true, SrcT, WorkT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask, modesUsed,
                                                                    weight, variance, (PtrStepSz<WorkT>) mean,
                                                                    alphaT, alpha1, prune);
            }
            else
            {
                cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(mog2<false), SrcT, WorkT>, hipFuncCachePreferL1) );

                mog2<false, SrcT, WorkT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask, modesUsed,
                                                                    weight, variance, (PtrStepSz<WorkT>) mean,
                                                                    alphaT, alpha1, prune);
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void mog2_gpu(PtrStepSzb frame, int cn, PtrStepSzb fgmask, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzf variance, PtrStepSzb mean,
                      float alphaT, float prune, bool detectShadows, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzf variance, PtrStepSzb mean, float alphaT, float prune, bool detectShadows, hipStream_t stream);

            static const func_t funcs[] =
            {
                0, mog2_caller<uchar, float>, 0, mog2_caller<uchar3, float3>, mog2_caller<uchar4, float4>
            };

            funcs[cn](frame, fgmask, modesUsed, weight, variance, mean, alphaT, prune, detectShadows, stream);
        }

        template <typename WorkT, typename OutT>
        __global__ void getBackgroundImage2(const PtrStepSzb modesUsed, const PtrStepf gmm_weight, const PtrStep<WorkT> gmm_mean, PtrStep<OutT> dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= modesUsed.cols || y >= modesUsed.rows)
                return;

            int nmodes = modesUsed(y, x);

            WorkT meanVal = VecTraits<WorkT>::all(0.0f);
            float totalWeight = 0.0f;

            for (int mode = 0; mode < nmodes; ++mode)
            {
                float weight = gmm_weight(mode * modesUsed.rows + y, x);

                WorkT mean = gmm_mean(mode * modesUsed.rows + y, x);
                meanVal = meanVal + weight * mean;

                totalWeight += weight;

                if(totalWeight > c_TB)
                    break;
            }

            meanVal = meanVal * (1.f / totalWeight);

            dst(y, x) = saturate_cast<OutT>(meanVal);
        }

        template <typename WorkT, typename OutT>
        void getBackgroundImage2_caller(PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(modesUsed.cols, block.x), divUp(modesUsed.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(getBackgroundImage2<WorkT), OutT>, hipFuncCachePreferL1) );

            getBackgroundImage2<WorkT, OutT><<<grid, block, 0, stream>>>(modesUsed, weight, (PtrStepSz<WorkT>) mean, (PtrStepSz<OutT>) dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void getBackgroundImage2_gpu(int cn, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, hipStream_t stream);

            static const func_t funcs[] =
            {
                0, getBackgroundImage2_caller<float, uchar>, 0, getBackgroundImage2_caller<float3, uchar3>, getBackgroundImage2_caller<float4, uchar4>
            };

            funcs[cn](modesUsed, weight, mean, dst, stream);
        }
    }
}}}


#endif /* CUDA_DISABLER */
