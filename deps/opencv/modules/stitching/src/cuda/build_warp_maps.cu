#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        // TODO use intrinsics like __sinf and so on

        namespace build_warp_maps
        {

            __constant__ float ck_rinv[9];
            __constant__ float cr_kinv[9];
            __constant__ float ct[3];
            __constant__ float cscale;
        }


        class PlaneMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                float x_ = u / cscale - ct[0];
                float y_ = v / cscale - ct[1];

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * (1 - ct[2]);
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * (1 - ct[2]);
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * (1 - ct[2]);

                x /= z;
                y /= z;
            }
        };


        class CylindricalMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                u /= cscale;
                float x_ = ::sinf(u);
                float y_ = v / cscale;
                float z_ = ::cosf(u);

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * z_;
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * z_;
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * z_;

                if (z > 0) { x /= z; y /= z; }
                else x = y = -1;
            }
        };


        class SphericalMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                v /= cscale;
                u /= cscale;

                float sinv = ::sinf(v);
                float x_ = sinv * ::sinf(u);
                float y_ = -::cosf(v);
                float z_ = sinv * ::cosf(u);

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * z_;
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * z_;
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * z_;

                if (z > 0) { x /= z; y /= z; }
                else x = y = -1;
            }
        };


        template <typename Mapper>
        __global__ void buildWarpMapsKernel(int tl_u, int tl_v, int cols, int rows,
                                            PtrStepf map_x, PtrStepf map_y)
        {
            int du = blockIdx.x * blockDim.x + threadIdx.x;
            int dv = blockIdx.y * blockDim.y + threadIdx.y;
            if (du < cols && dv < rows)
            {
                float u = tl_u + du;
                float v = tl_v + dv;
                float x, y;
                Mapper::mapBackward(u, v, x, y);
                map_x.ptr(dv)[du] = x;
                map_y.ptr(dv)[du] = y;
            }
        }


        void buildWarpPlaneMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                const float k_rinv[9], const float r_kinv[9], const float t[3],
                                float scale, hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ct), t, 3*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<PlaneMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void buildWarpCylindricalMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                      const float k_rinv[9], const float r_kinv[9], float scale,
                                      hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<CylindricalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void buildWarpSphericalMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                    const float k_rinv[9], const float r_kinv[9], float scale,
                                    hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<SphericalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev {


#endif /* CUDA_DISABLER */
