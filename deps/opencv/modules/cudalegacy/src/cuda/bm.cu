#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace optflowbm
{
    texture<uchar, hipTextureType2D, hipReadModeElementType> tex_prev(false, hipFilterModePoint, hipAddressModeClamp);
    texture<uchar, hipTextureType2D, hipReadModeElementType> tex_curr(false, hipFilterModePoint, hipAddressModeClamp);

    __device__ int cmpBlocks(int X1, int Y1, int X2, int Y2, int2 blockSize)
    {
        int s = 0;

        for (int y = 0; y < blockSize.y; ++y)
        {
            for (int x = 0; x < blockSize.x; ++x)
                s += ::abs(tex2D(tex_prev, X1 + x, Y1 + y) - tex2D(tex_curr, X2 + x, Y2 + y));
        }

        return s;
    }

    __global__ void calcOptFlowBM(PtrStepSzf velx, PtrStepf vely, const int2 blockSize, const int2 shiftSize, const bool usePrevious,
                                  const int maxX, const int maxY, const int acceptLevel, const int escapeLevel,
                                  const short2* ss, const int ssCount)
    {
        const int j = blockIdx.x * blockDim.x + threadIdx.x;
        const int i = blockIdx.y * blockDim.y + threadIdx.y;

        if (i >= velx.rows || j >= velx.cols)
            return;

        const int X1 = j * shiftSize.x;
        const int Y1 = i * shiftSize.y;

        const int offX = usePrevious ? __float2int_rn(velx(i, j)) : 0;
        const int offY = usePrevious ? __float2int_rn(vely(i, j)) : 0;

        int X2 = X1 + offX;
        int Y2 = Y1 + offY;

        int dist = numeric_limits<int>::max();

        if (0 <= X2 && X2 <= maxX && 0 <= Y2 && Y2 <= maxY)
            dist = cmpBlocks(X1, Y1, X2, Y2, blockSize);

        int countMin = 1;
        int sumx = offX;
        int sumy = offY;

        if (dist > acceptLevel)
        {
            // do brute-force search
            for (int k = 0; k < ssCount; ++k)
            {
                const short2 ssVal = ss[k];

                const int dx = offX + ssVal.x;
                const int dy = offY + ssVal.y;

                X2 = X1 + dx;
                Y2 = Y1 + dy;

                if (0 <= X2 && X2 <= maxX && 0 <= Y2 && Y2 <= maxY)
                {
                    const int tmpDist = cmpBlocks(X1, Y1, X2, Y2, blockSize);
                    if (tmpDist < acceptLevel)
                    {
                        sumx = dx;
                        sumy = dy;
                        countMin = 1;
                        break;
                    }

                    if (tmpDist < dist)
                    {
                        dist = tmpDist;
                        sumx = dx;
                        sumy = dy;
                        countMin = 1;
                    }
                    else if (tmpDist == dist)
                    {
                        sumx += dx;
                        sumy += dy;
                        countMin++;
                    }
                }
            }

            if (dist > escapeLevel)
            {
                sumx = offX;
                sumy = offY;
                countMin = 1;
            }
        }

        velx(i, j) = static_cast<float>(sumx) / countMin;
        vely(i, j) = static_cast<float>(sumy) / countMin;
    }

    void calc(PtrStepSzb prev, PtrStepSzb curr, PtrStepSzf velx, PtrStepSzf vely, int2 blockSize, int2 shiftSize, bool usePrevious,
              int maxX, int maxY, int acceptLevel, int escapeLevel, const short2* ss, int ssCount, hipStream_t stream)
    {
        bindTexture(&tex_prev, prev);
        bindTexture(&tex_curr, curr);

        const dim3 block(32, 8);
        const dim3 grid(divUp(velx.cols, block.x), divUp(vely.rows, block.y));

        calcOptFlowBM<<<grid, block, 0, stream>>>(velx, vely, blockSize, shiftSize, usePrevious,
                                                  maxX, maxY, acceptLevel,  escapeLevel, ss, ssCount);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

#endif // !defined CUDA_DISABLER
