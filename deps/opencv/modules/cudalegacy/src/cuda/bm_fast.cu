#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace optflowbm_fast
{
    enum
    {
        CTA_SIZE = 128,

        TILE_COLS = 128,
        TILE_ROWS = 32,

        STRIDE = CTA_SIZE
    };

    template <typename T> __device__ __forceinline__ int calcDist(T a, T b)
    {
        return ::abs(a - b);
    }

    template <class T> struct FastOptFlowBM
    {

        int search_radius;
        int block_radius;

        int search_window;
        int block_window;

        PtrStepSz<T> I0;
        PtrStep<T> I1;

        mutable PtrStepi buffer;

        FastOptFlowBM(int search_window_, int block_window_,
                      PtrStepSz<T> I0_, PtrStepSz<T> I1_,
                      PtrStepi buffer_) :
            search_radius(search_window_ / 2), block_radius(block_window_ / 2),
            search_window(search_window_), block_window(block_window_),
            I0(I0_), I1(I1_),
            buffer(buffer_)
        {
        }

        __device__ __forceinline__ void initSums_BruteForce(int i, int j, int* dist_sums, PtrStepi& col_sums, PtrStepi& up_col_sums) const
        {
            for (int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
            {
                dist_sums[index] = 0;

                for (int tx = 0; tx < block_window; ++tx)
                    col_sums(tx, index) = 0;

                int y = index / search_window;
                int x = index - y * search_window;

                int ay = i;
                int ax = j;

                int by = i + y - search_radius;
                int bx = j + x - search_radius;

                for (int tx = -block_radius; tx <= block_radius; ++tx)
                {
                    int col_sum = 0;
                    for (int ty = -block_radius; ty <= block_radius; ++ty)
                    {
                        int dist = calcDist(I0(ay + ty, ax + tx), I1(by + ty, bx + tx));

                        dist_sums[index] += dist;
                        col_sum += dist;
                    }

                    col_sums(tx + block_radius, index) = col_sum;
                }

                up_col_sums(j, index) = col_sums(block_window - 1, index);
            }
        }

        __device__ __forceinline__ void shiftRight_FirstRow(int i, int j, int first, int* dist_sums, PtrStepi& col_sums, PtrStepi& up_col_sums) const
        {
            for (int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
            {
                int y = index / search_window;
                int x = index - y * search_window;

                int ay = i;
                int ax = j + block_radius;

                int by = i + y - search_radius;
                int bx = j + x - search_radius + block_radius;

                int col_sum = 0;

                for (int ty = -block_radius; ty <= block_radius; ++ty)
                    col_sum += calcDist(I0(ay + ty, ax), I1(by + ty, bx));

                dist_sums[index] += col_sum - col_sums(first, index);

                col_sums(first, index) = col_sum;
                up_col_sums(j, index) = col_sum;
            }
        }

        __device__ __forceinline__ void shiftRight_UpSums(int i, int j, int first, int* dist_sums, PtrStepi& col_sums, PtrStepi& up_col_sums) const
        {
            int ay = i;
            int ax = j + block_radius;

            T a_up   = I0(ay - block_radius - 1, ax);
            T a_down = I0(ay + block_radius, ax);

            for(int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
            {
                int y = index / search_window;
                int x = index - y * search_window;

                int by = i + y - search_radius;
                int bx = j + x - search_radius + block_radius;

                T b_up   = I1(by - block_radius - 1, bx);
                T b_down = I1(by + block_radius, bx);

                int col_sum = up_col_sums(j, index) + calcDist(a_down, b_down) - calcDist(a_up, b_up);

                dist_sums[index] += col_sum  - col_sums(first, index);
                col_sums(first, index) = col_sum;
                up_col_sums(j, index) = col_sum;
            }
        }

        __device__ __forceinline__ void convolve_window(int i, int j, const int* dist_sums, float& velx, float& vely) const
        {
            int bestDist = numeric_limits<int>::max();
            int bestInd = -1;

            for (int index = threadIdx.x; index < search_window * search_window; index += STRIDE)
            {
                int curDist = dist_sums[index];
                if (curDist < bestDist)
                {
                    bestDist = curDist;
                    bestInd = index;
                }
            }

            __shared__ int cta_dist_buffer[CTA_SIZE];
            __shared__ int cta_ind_buffer[CTA_SIZE];

            reduceKeyVal<CTA_SIZE>(cta_dist_buffer, bestDist, cta_ind_buffer, bestInd, threadIdx.x, less<int>());

            if (threadIdx.x == 0)
            {
                int y = bestInd / search_window;
                int x = bestInd - y * search_window;

                velx = x - search_radius;
                vely = y - search_radius;
            }
        }

        __device__ __forceinline__ void operator()(PtrStepf velx, PtrStepf vely) const
        {
            int tbx = blockIdx.x * TILE_COLS;
            int tby = blockIdx.y * TILE_ROWS;

            int tex = ::min(tbx + TILE_COLS, I0.cols);
            int tey = ::min(tby + TILE_ROWS, I0.rows);

            PtrStepi col_sums;
            col_sums.data = buffer.ptr(I0.cols + blockIdx.x * block_window) + blockIdx.y * search_window * search_window;
            col_sums.step = buffer.step;

            PtrStepi up_col_sums;
            up_col_sums.data = buffer.data + blockIdx.y * search_window * search_window;
            up_col_sums.step = buffer.step;

            extern __shared__ int dist_sums[]; //search_window * search_window

            int first = 0;

            for (int i = tby; i < tey; ++i)
            {
                for (int j = tbx; j < tex; ++j)
                {
                    __syncthreads();

                    if (j == tbx)
                    {
                        initSums_BruteForce(i, j, dist_sums, col_sums, up_col_sums);
                        first = 0;
                    }
                    else
                    {
                        if (i == tby)
                          shiftRight_FirstRow(i, j, first, dist_sums, col_sums, up_col_sums);
                        else
                          shiftRight_UpSums(i, j, first, dist_sums, col_sums, up_col_sums);

                        first = (first + 1) % block_window;
                    }

                    __syncthreads();

                    convolve_window(i, j, dist_sums, velx(i, j), vely(i, j));
                }
            }
        }

    };

    template<typename T> __global__ void optflowbm_fast_kernel(const FastOptFlowBM<T> fbm, PtrStepf velx, PtrStepf vely)
    {
        fbm(velx, vely);
    }

    void get_buffer_size(int src_cols, int src_rows, int search_window, int block_window, int& buffer_cols, int& buffer_rows)
    {
        dim3 grid(divUp(src_cols, TILE_COLS), divUp(src_rows, TILE_ROWS));

        buffer_cols = search_window * search_window * grid.y;
        buffer_rows = src_cols + block_window * grid.x;
    }

    template <typename T>
    void calc(PtrStepSzb I0, PtrStepSzb I1, PtrStepSzf velx, PtrStepSzf vely, PtrStepi buffer, int search_window, int block_window, hipStream_t stream)
    {
        FastOptFlowBM<T> fbm(search_window, block_window, I0, I1, buffer);

        dim3 block(CTA_SIZE, 1);
        dim3 grid(divUp(I0.cols, TILE_COLS), divUp(I0.rows, TILE_ROWS));

        size_t smem = search_window * search_window * sizeof(int);

        optflowbm_fast_kernel<<<grid, block, smem, stream>>>(fbm, velx, vely);
        cudaSafeCall ( hipGetLastError () );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void calc<uchar>(PtrStepSzb I0, PtrStepSzb I1, PtrStepSzf velx, PtrStepSzf vely, PtrStepi buffer, int search_window, int block_window, hipStream_t stream);
}

#endif // !defined CUDA_DISABLER
