#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "fgd.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace fgd
{
    ////////////////////////////////////////////////////////////////////////////
    // calcDiffHistogram

    const unsigned int UINT_BITS = 32U;
    const int LOG_WARP_SIZE = 5;
    const int WARP_SIZE = 1 << LOG_WARP_SIZE;
#if (__CUDA_ARCH__ < 120)
    const unsigned int TAG_MASK = (1U << (UINT_BITS - LOG_WARP_SIZE)) - 1U;
#endif

    const int MERGE_THREADBLOCK_SIZE = 256;

    __device__ __forceinline__ void addByte(unsigned int* s_WarpHist_, unsigned int data, unsigned int threadTag)
    {
        #if (__CUDA_ARCH__ < 120)
            volatile unsigned int* s_WarpHist = s_WarpHist_;
            unsigned int count;
            do
            {
                count = s_WarpHist[data] & TAG_MASK;
                count = threadTag | (count + 1);
                s_WarpHist[data] = count;
            } while (s_WarpHist[data] != count);
        #else
            atomicInc(s_WarpHist_ + data, (unsigned int)(-1));
        #endif
    }


    template <typename PT, typename CT>
    __global__ void calcPartialHistogram(const PtrStepSz<PT> prevFrame, const PtrStep<CT> curFrame, unsigned int* partialBuf0, unsigned int* partialBuf1, unsigned int* partialBuf2)
    {
#if (__CUDA_ARCH__ < 200)
        const int HISTOGRAM_WARP_COUNT = 4;
#else
        const int HISTOGRAM_WARP_COUNT = 6;
#endif
        const int HISTOGRAM_THREADBLOCK_SIZE = HISTOGRAM_WARP_COUNT * WARP_SIZE;
        const int HISTOGRAM_THREADBLOCK_MEMORY = HISTOGRAM_WARP_COUNT * HISTOGRAM_BIN_COUNT;

        //Per-warp subhistogram storage
        __shared__ unsigned int s_Hist0[HISTOGRAM_THREADBLOCK_MEMORY];
        __shared__ unsigned int s_Hist1[HISTOGRAM_THREADBLOCK_MEMORY];
        __shared__ unsigned int s_Hist2[HISTOGRAM_THREADBLOCK_MEMORY];

        //Clear shared memory storage for current threadblock before processing
        #pragma unroll
        for (int i = 0; i < (HISTOGRAM_THREADBLOCK_MEMORY / HISTOGRAM_THREADBLOCK_SIZE); ++i)
        {
           s_Hist0[threadIdx.x + i * HISTOGRAM_THREADBLOCK_SIZE] = 0;
           s_Hist1[threadIdx.x + i * HISTOGRAM_THREADBLOCK_SIZE] = 0;
           s_Hist2[threadIdx.x + i * HISTOGRAM_THREADBLOCK_SIZE] = 0;
        }
        __syncthreads();

        const unsigned int warpId = threadIdx.x >> LOG_WARP_SIZE;

        unsigned int* s_WarpHist0 = s_Hist0 + warpId * HISTOGRAM_BIN_COUNT;
        unsigned int* s_WarpHist1 = s_Hist1 + warpId * HISTOGRAM_BIN_COUNT;
        unsigned int* s_WarpHist2 = s_Hist2 + warpId * HISTOGRAM_BIN_COUNT;

        const unsigned int tag = threadIdx.x << (UINT_BITS - LOG_WARP_SIZE);
        const int dataCount = prevFrame.rows * prevFrame.cols;
        for (unsigned int pos = blockIdx.x * HISTOGRAM_THREADBLOCK_SIZE + threadIdx.x; pos < dataCount; pos += HISTOGRAM_THREADBLOCK_SIZE * PARTIAL_HISTOGRAM_COUNT)
        {
            const unsigned int y = pos / prevFrame.cols;
            const unsigned int x = pos % prevFrame.cols;

            PT prevVal = prevFrame(y, x);
            CT curVal = curFrame(y, x);

            int3 diff = make_int3(
                ::abs(curVal.x - prevVal.x),
                ::abs(curVal.y - prevVal.y),
                ::abs(curVal.z - prevVal.z)
            );

            addByte(s_WarpHist0, diff.x, tag);
            addByte(s_WarpHist1, diff.y, tag);
            addByte(s_WarpHist2, diff.z, tag);
        }
        __syncthreads();

        //Merge per-warp histograms into per-block and write to global memory
        for (unsigned int bin = threadIdx.x; bin < HISTOGRAM_BIN_COUNT; bin += HISTOGRAM_THREADBLOCK_SIZE)
        {
            unsigned int sum0 = 0;
            unsigned int sum1 = 0;
            unsigned int sum2 = 0;

            #pragma unroll
            for (int i = 0; i < HISTOGRAM_WARP_COUNT; ++i)
            {
                #if (__CUDA_ARCH__ < 120)
                    sum0 += s_Hist0[bin + i * HISTOGRAM_BIN_COUNT] & TAG_MASK;
                    sum1 += s_Hist1[bin + i * HISTOGRAM_BIN_COUNT] & TAG_MASK;
                    sum2 += s_Hist2[bin + i * HISTOGRAM_BIN_COUNT] & TAG_MASK;
                #else
                    sum0 += s_Hist0[bin + i * HISTOGRAM_BIN_COUNT];
                    sum1 += s_Hist1[bin + i * HISTOGRAM_BIN_COUNT];
                    sum2 += s_Hist2[bin + i * HISTOGRAM_BIN_COUNT];
                #endif
            }

            partialBuf0[blockIdx.x * HISTOGRAM_BIN_COUNT + bin] = sum0;
            partialBuf1[blockIdx.x * HISTOGRAM_BIN_COUNT + bin] = sum1;
            partialBuf2[blockIdx.x * HISTOGRAM_BIN_COUNT + bin] = sum2;
        }
    }

    __global__ void mergeHistogram(const unsigned int* partialBuf0, const unsigned int* partialBuf1, const unsigned int* partialBuf2, unsigned int* hist0, unsigned int* hist1, unsigned int* hist2)
    {
        unsigned int sum0 = 0;
        unsigned int sum1 = 0;
        unsigned int sum2 = 0;

        #pragma unroll
        for (unsigned int i = threadIdx.x; i < PARTIAL_HISTOGRAM_COUNT; i += MERGE_THREADBLOCK_SIZE)
        {
            sum0 += partialBuf0[blockIdx.x + i * HISTOGRAM_BIN_COUNT];
            sum1 += partialBuf1[blockIdx.x + i * HISTOGRAM_BIN_COUNT];
            sum2 += partialBuf2[blockIdx.x + i * HISTOGRAM_BIN_COUNT];
        }

        __shared__ unsigned int data0[MERGE_THREADBLOCK_SIZE];
        __shared__ unsigned int data1[MERGE_THREADBLOCK_SIZE];
        __shared__ unsigned int data2[MERGE_THREADBLOCK_SIZE];

        plus<unsigned int> op;
        reduce<MERGE_THREADBLOCK_SIZE>(smem_tuple(data0, data1, data2), thrust::tie(sum0, sum1, sum2), threadIdx.x, thrust::make_tuple(op, op, op));

        if(threadIdx.x == 0)
        {
            hist0[blockIdx.x] = sum0;
            hist1[blockIdx.x] = sum1;
            hist2[blockIdx.x] = sum2;
        }
    }

    template <typename PT, typename CT>
    void calcDiffHistogram_gpu(PtrStepSzb prevFrame, PtrStepSzb curFrame,
                               unsigned int* hist0, unsigned int* hist1, unsigned int* hist2,
                               unsigned int* partialBuf0, unsigned int* partialBuf1, unsigned int* partialBuf2,
                               bool cc20, hipStream_t stream)
    {
        const int HISTOGRAM_WARP_COUNT = cc20 ? 6 : 4;
        const int HISTOGRAM_THREADBLOCK_SIZE = HISTOGRAM_WARP_COUNT * WARP_SIZE;

        calcPartialHistogram<PT, CT><<<PARTIAL_HISTOGRAM_COUNT, HISTOGRAM_THREADBLOCK_SIZE, 0, stream>>>(
                (PtrStepSz<PT>)prevFrame, (PtrStepSz<CT>)curFrame, partialBuf0, partialBuf1, partialBuf2);
        cudaSafeCall( hipGetLastError() );

        mergeHistogram<<<HISTOGRAM_BIN_COUNT, MERGE_THREADBLOCK_SIZE, 0, stream>>>(partialBuf0, partialBuf1, partialBuf2, hist0, hist1, hist2);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void calcDiffHistogram_gpu<uchar3, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, unsigned int* hist0, unsigned int* hist1, unsigned int* hist2, unsigned int* partialBuf0, unsigned int* partialBuf1, unsigned int* partialBuf2, bool cc20, hipStream_t stream);
    template void calcDiffHistogram_gpu<uchar3, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, unsigned int* hist0, unsigned int* hist1, unsigned int* hist2, unsigned int* partialBuf0, unsigned int* partialBuf1, unsigned int* partialBuf2, bool cc20, hipStream_t stream);
    template void calcDiffHistogram_gpu<uchar4, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, unsigned int* hist0, unsigned int* hist1, unsigned int* hist2, unsigned int* partialBuf0, unsigned int* partialBuf1, unsigned int* partialBuf2, bool cc20, hipStream_t stream);
    template void calcDiffHistogram_gpu<uchar4, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, unsigned int* hist0, unsigned int* hist1, unsigned int* hist2, unsigned int* partialBuf0, unsigned int* partialBuf1, unsigned int* partialBuf2, bool cc20, hipStream_t stream);

    /////////////////////////////////////////////////////////////////////////
    // calcDiffThreshMask

    template <typename PT, typename CT>
    __global__ void calcDiffThreshMask(const PtrStepSz<PT> prevFrame, const PtrStep<CT> curFrame, uchar3 bestThres, PtrStepb changeMask)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        if (y > prevFrame.rows || x > prevFrame.cols)
            return;

        PT prevVal = prevFrame(y, x);
        CT curVal = curFrame(y, x);

        int3 diff = make_int3(
            ::abs(curVal.x - prevVal.x),
            ::abs(curVal.y - prevVal.y),
            ::abs(curVal.z - prevVal.z)
        );

        if (diff.x > bestThres.x || diff.y > bestThres.y || diff.z > bestThres.z)
            changeMask(y, x) = 255;
    }

    template <typename PT, typename CT>
    void calcDiffThreshMask_gpu(PtrStepSzb prevFrame, PtrStepSzb curFrame, uchar3 bestThres, PtrStepSzb changeMask, hipStream_t stream)
    {
        dim3 block(32, 8);
        dim3 grid(divUp(prevFrame.cols, block.x), divUp(prevFrame.rows, block.y));

        calcDiffThreshMask<PT, CT><<<grid, block, 0, stream>>>((PtrStepSz<PT>)prevFrame, (PtrStepSz<CT>)curFrame, bestThres, changeMask);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void calcDiffThreshMask_gpu<uchar3, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, uchar3 bestThres, PtrStepSzb changeMask, hipStream_t stream);
    template void calcDiffThreshMask_gpu<uchar3, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, uchar3 bestThres, PtrStepSzb changeMask, hipStream_t stream);
    template void calcDiffThreshMask_gpu<uchar4, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, uchar3 bestThres, PtrStepSzb changeMask, hipStream_t stream);
    template void calcDiffThreshMask_gpu<uchar4, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, uchar3 bestThres, PtrStepSzb changeMask, hipStream_t stream);

    /////////////////////////////////////////////////////////////////////////
    // bgfgClassification

    __constant__ BGPixelStat c_stat;

    void setBGPixelStat(const BGPixelStat& stat)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_stat), &stat, sizeof(BGPixelStat)) );
    }

    template <typename T> struct Output;
    template <> struct Output<uchar3>
    {
        static __device__ __forceinline__ uchar3 make(uchar v0, uchar v1, uchar v2)
        {
            return make_uchar3(v0, v1, v2);
        }
    };
    template <> struct Output<uchar4>
    {
        static __device__ __forceinline__ uchar4 make(uchar v0, uchar v1, uchar v2)
        {
            return make_uchar4(v0, v1, v2, 255);
        }
    };

    template <typename PT, typename CT, typename OT>
    __global__ void bgfgClassification(const PtrStepSz<PT> prevFrame, const PtrStep<CT> curFrame,
                                       const PtrStepb Ftd, const PtrStepb Fbd, PtrStepb foreground,
                                       int deltaC, int deltaCC, float alpha2, int N1c, int N1cc)
    {
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        const int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i > prevFrame.rows || j > prevFrame.cols)
            return;

        if (Fbd(i, j) || Ftd(i, j))
        {
            float Pb  = 0.0f;
            float Pv  = 0.0f;
            float Pvb = 0.0f;

            int val = 0;

            // Is it a motion pixel?
            if (Ftd(i, j))
            {
                if (!c_stat.is_trained_dyn_model(i, j))
                    val = 1;
                else
                {
                    PT prevVal = prevFrame(i, j);
                    CT curVal = curFrame(i, j);

                    // Compare with stored CCt vectors:
                    for (int k = 0; k < N1cc && c_stat.PV_CC(i, j, k) > alpha2; ++k)
                    {
                        OT v1 = c_stat.V1_CC<OT>(i, j, k);
                        OT v2 = c_stat.V2_CC<OT>(i, j, k);

                        if (::abs(v1.x - prevVal.x) <= deltaCC &&
                            ::abs(v1.y - prevVal.y) <= deltaCC &&
                            ::abs(v1.z - prevVal.z) <= deltaCC &&
                            ::abs(v2.x - curVal.x) <= deltaCC &&
                            ::abs(v2.y - curVal.y) <= deltaCC &&
                            ::abs(v2.z - curVal.z) <= deltaCC)
                        {
                            Pv += c_stat.PV_CC(i, j, k);
                            Pvb += c_stat.PVB_CC(i, j, k);
                        }
                    }

                    Pb = c_stat.Pbcc(i, j);
                    if (2 * Pvb * Pb <= Pv)
                        val = 1;
                }
            }
            else if(c_stat.is_trained_st_model(i, j))
            {
                CT curVal = curFrame(i, j);

                // Compare with stored Ct vectors:
                for (int k = 0; k < N1c && c_stat.PV_C(i, j, k) > alpha2; ++k)
                {
                    OT v = c_stat.V_C<OT>(i, j, k);

                    if (::abs(v.x - curVal.x) <= deltaC &&
                        ::abs(v.y - curVal.y) <= deltaC &&
                        ::abs(v.z - curVal.z) <= deltaC)
                    {
                        Pv += c_stat.PV_C(i, j, k);
                        Pvb += c_stat.PVB_C(i, j, k);
                    }
                }
                Pb = c_stat.Pbc(i, j);
                if (2 * Pvb * Pb <= Pv)
                    val = 1;
            }

            // Update foreground:
            foreground(i, j) = static_cast<uchar>(val);
        } // end if( change detection...
    }

    template <typename PT, typename CT, typename OT>
    void bgfgClassification_gpu(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground,
                                int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream)
    {
        dim3 block(32, 8);
        dim3 grid(divUp(prevFrame.cols, block.x), divUp(prevFrame.rows, block.y));

        cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(bgfgClassification<PT), CT, OT>, hipFuncCachePreferL1) );

        bgfgClassification<PT, CT, OT><<<grid, block, 0, stream>>>((PtrStepSz<PT>)prevFrame, (PtrStepSz<CT>)curFrame,
                                                                   Ftd, Fbd, foreground,
                                                                   deltaC, deltaCC, alpha2, N1c, N1cc);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void bgfgClassification_gpu<uchar3, uchar3, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar3, uchar3, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar3, uchar4, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar3, uchar4, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar4, uchar3, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar4, uchar3, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar4, uchar4, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);
    template void bgfgClassification_gpu<uchar4, uchar4, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, int deltaC, int deltaCC, float alpha2, int N1c, int N1cc, hipStream_t stream);

    ////////////////////////////////////////////////////////////////////////////
    // updateBackgroundModel

    template <typename PT, typename CT, typename OT, class PrevFramePtr2D, class CurFramePtr2D, class FtdPtr2D, class FbdPtr2D>
    __global__ void updateBackgroundModel(int cols, int rows, const PrevFramePtr2D prevFrame, const CurFramePtr2D curFrame, const FtdPtr2D Ftd, const FbdPtr2D Fbd,
                                          PtrStepb foreground, PtrStep<OT> background,
                                          int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T)
    {
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        const int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i > rows || j > cols)
            return;

        const float MIN_PV = 1e-10f;

        const uchar is_trained_dyn_model = c_stat.is_trained_dyn_model(i, j);
        if (Ftd(i, j) || !is_trained_dyn_model)
        {
            const float alpha = is_trained_dyn_model ? alpha2 : alpha3;

            float Pbcc = c_stat.Pbcc(i, j);

            //update Pb
            Pbcc *= (1.0f - alpha);
            if (!foreground(i, j))
            {
                Pbcc += alpha;
            }

            int min_dist = numeric_limits<int>::max();
            int indx = -1;

            PT prevVal = prevFrame(i, j);
            CT curVal = curFrame(i, j);

            // Find best Vi match:
            for (int k = 0; k < N2cc; ++k)
            {
                float PV_CC = c_stat.PV_CC(i, j, k);
                if (!PV_CC)
                    break;

                if (PV_CC < MIN_PV)
                {
                    c_stat.PV_CC(i, j, k) = 0;
                    c_stat.PVB_CC(i, j, k) = 0;
                    continue;
                }

                c_stat.PV_CC(i, j, k) = PV_CC * (1.0f - alpha);
                c_stat.PVB_CC(i, j, k) = c_stat.PVB_CC(i, j, k) * (1.0f - alpha);

                OT v1 = c_stat.V1_CC<OT>(i, j, k);

                int3 val1 = make_int3(
                    ::abs(v1.x - prevVal.x),
                    ::abs(v1.y - prevVal.y),
                    ::abs(v1.z - prevVal.z)
                );

                OT v2 = c_stat.V2_CC<OT>(i, j, k);

                int3 val2 = make_int3(
                    ::abs(v2.x - curVal.x),
                    ::abs(v2.y - curVal.y),
                    ::abs(v2.z - curVal.z)
                );

                int dist = val1.x + val1.y + val1.z + val2.x + val2.y + val2.z;

                if (dist < min_dist &&
                    val1.x <= deltaCC && val1.y <= deltaCC && val1.z <= deltaCC &&
                    val2.x <= deltaCC && val2.y <= deltaCC && val2.z <= deltaCC)
                {
                    min_dist = dist;
                    indx = k;
                }
            }

            if (indx < 0)
            {
                // Replace N2th elem in the table by new feature:
                indx = N2cc - 1;
                c_stat.PV_CC(i, j, indx) = alpha;
                c_stat.PVB_CC(i, j, indx) = alpha;

                //udate Vt
                c_stat.V1_CC<OT>(i, j, indx) = Output<OT>::make(prevVal.x, prevVal.y, prevVal.z);
                c_stat.V2_CC<OT>(i, j, indx) = Output<OT>::make(curVal.x, curVal.y, curVal.z);
            }
            else
            {
                // Update:
                c_stat.PV_CC(i, j, indx) += alpha;

                if (!foreground(i, j))
                {
                    c_stat.PVB_CC(i, j, indx) += alpha;
                }
            }

            //re-sort CCt table by Pv
            const float PV_CC_indx = c_stat.PV_CC(i, j, indx);
            const float PVB_CC_indx = c_stat.PVB_CC(i, j, indx);
            const OT V1_CC_indx = c_stat.V1_CC<OT>(i, j, indx);
            const OT V2_CC_indx = c_stat.V2_CC<OT>(i, j, indx);
            for (int k = 0; k < indx; ++k)
            {
                if (c_stat.PV_CC(i, j, k) <= PV_CC_indx)
                {
                    //shift elements
                    float Pv_tmp1;
                    float Pv_tmp2 = PV_CC_indx;

                    float Pvb_tmp1;
                    float Pvb_tmp2 = PVB_CC_indx;

                    OT v1_tmp1;
                    OT v1_tmp2 = V1_CC_indx;

                    OT v2_tmp1;
                    OT v2_tmp2 = V2_CC_indx;

                    for (int l = k; l <= indx; ++l)
                    {
                        Pv_tmp1 = c_stat.PV_CC(i, j, l);
                        c_stat.PV_CC(i, j, l) = Pv_tmp2;
                        Pv_tmp2 = Pv_tmp1;

                        Pvb_tmp1 = c_stat.PVB_CC(i, j, l);
                        c_stat.PVB_CC(i, j, l) = Pvb_tmp2;
                        Pvb_tmp2 = Pvb_tmp1;

                        v1_tmp1 = c_stat.V1_CC<OT>(i, j, l);
                        c_stat.V1_CC<OT>(i, j, l) = v1_tmp2;
                        v1_tmp2 = v1_tmp1;

                        v2_tmp1 = c_stat.V2_CC<OT>(i, j, l);
                        c_stat.V2_CC<OT>(i, j, l) = v2_tmp2;
                        v2_tmp2 = v2_tmp1;
                    }

                    break;
                }
            }

            float sum1 = 0.0f;
            float sum2 = 0.0f;

            //check "once-off" changes
            for (int k = 0; k < N1cc; ++k)
            {
                const float PV_CC = c_stat.PV_CC(i, j, k);
                if (!PV_CC)
                    break;

                sum1 += PV_CC;
                sum2 += c_stat.PVB_CC(i, j, k);
            }

            if (sum1 > T)
                c_stat.is_trained_dyn_model(i, j) = 1;

            float diff = sum1 - Pbcc * sum2;

            // Update stat table:
            if (diff > T)
            {
                //new BG features are discovered
                for (int k = 0; k < N1cc; ++k)
                {
                    const float PV_CC = c_stat.PV_CC(i, j, k);
                    if (!PV_CC)
                        break;

                    c_stat.PVB_CC(i, j, k) = (PV_CC - Pbcc * c_stat.PVB_CC(i, j, k)) / (1.0f - Pbcc);
                }
            }

            c_stat.Pbcc(i, j) = Pbcc;
        }

        // Handle "stationary" pixel:
        if (!Ftd(i, j))
        {
            const float alpha = c_stat.is_trained_st_model(i, j) ? alpha2 : alpha3;

            float Pbc = c_stat.Pbc(i, j);

            //update Pb
            Pbc *= (1.0f - alpha);
            if (!foreground(i, j))
            {
                Pbc += alpha;
            }

            int min_dist = numeric_limits<int>::max();
            int indx = -1;

            CT curVal = curFrame(i, j);

            //find best Vi match
            for (int k = 0; k < N2c; ++k)
            {
                float PV_C = c_stat.PV_C(i, j, k);

                if (PV_C < MIN_PV)
                {
                    c_stat.PV_C(i, j, k) = 0;
                    c_stat.PVB_C(i, j, k) = 0;
                    continue;
                }

                // Exponential decay of memory
                c_stat.PV_C(i, j, k) = PV_C * (1.0f - alpha);
                c_stat.PVB_C(i, j, k) = c_stat.PVB_C(i, j, k) * (1.0f - alpha);

                OT v = c_stat.V_C<OT>(i, j, k);
                int3 val = make_int3(
                    ::abs(v.x - curVal.x),
                    ::abs(v.y - curVal.y),
                    ::abs(v.z - curVal.z)
                );

                int dist = val.x + val.y + val.z;

                if (dist < min_dist && val.x <= deltaC && val.y <= deltaC && val.z <= deltaC)
                {
                    min_dist = dist;
                    indx = k;
                }
            }

            if (indx < 0)
            {
                //N2th elem in the table is replaced by a new features
                indx = N2c - 1;

                c_stat.PV_C(i, j, indx) = alpha;
                c_stat.PVB_C(i, j, indx) = alpha;

                //udate Vt
                c_stat.V_C<OT>(i, j, indx) = Output<OT>::make(curVal.x, curVal.y, curVal.z);
            }
            else
            {
                //update
                c_stat.PV_C(i, j, indx) += alpha;

                if (!foreground(i, j))
                {
                    c_stat.PVB_C(i, j, indx) += alpha;
                }
            }

            //re-sort Ct table by Pv
            const float PV_C_indx = c_stat.PV_C(i, j, indx);
            const float PVB_C_indx = c_stat.PVB_C(i, j, indx);
            OT V_C_indx = c_stat.V_C<OT>(i, j, indx);
            for (int k = 0; k < indx; ++k)
            {
                if (c_stat.PV_C(i, j, k) <= PV_C_indx)
                {
                    //shift elements
                    float Pv_tmp1;
                    float Pv_tmp2 = PV_C_indx;

                    float Pvb_tmp1;
                    float Pvb_tmp2 = PVB_C_indx;

                    OT v_tmp1;
                    OT v_tmp2 = V_C_indx;

                    for (int l = k; l <= indx; ++l)
                    {
                        Pv_tmp1 = c_stat.PV_C(i, j, l);
                        c_stat.PV_C(i, j, l) = Pv_tmp2;
                        Pv_tmp2 = Pv_tmp1;

                        Pvb_tmp1 = c_stat.PVB_C(i, j, l);
                        c_stat.PVB_C(i, j, l) = Pvb_tmp2;
                        Pvb_tmp2 = Pvb_tmp1;

                        v_tmp1 = c_stat.V_C<OT>(i, j, l);
                        c_stat.V_C<OT>(i, j, l) = v_tmp2;
                        v_tmp2 = v_tmp1;
                    }

                    break;
                }
            }

            // Check "once-off" changes:
            float sum1 = 0.0f;
            float sum2 = 0.0f;
            for (int k = 0; k < N1c; ++k)
            {
                const float PV_C = c_stat.PV_C(i, j, k);
                if (!PV_C)
                    break;

                sum1 += PV_C;
                sum2 += c_stat.PVB_C(i, j, k);
            }

            if (sum1 > T)
                c_stat.is_trained_st_model(i, j) = 1;

            float diff = sum1 - Pbc * sum2;

            // Update stat table:
            if (diff > T)
            {
                //new BG features are discovered
                for (int k = 0; k < N1c; ++k)
                {
                    const float PV_C = c_stat.PV_C(i, j, k);
                    if (!PV_C)
                        break;

                    c_stat.PVB_C(i, j, k) = (PV_C - Pbc * c_stat.PVB_C(i, j, k)) / (1.0f - Pbc);
                }

                c_stat.Pbc(i, j) = 1.0f - Pbc;
            }
            else
            {
                c_stat.Pbc(i, j) = Pbc;
            }
        } // if !(change detection) at pixel (i,j)

        // Update the reference BG image:
        if (!foreground(i, j))
        {
            CT curVal = curFrame(i, j);

            if (!Ftd(i, j) && !Fbd(i, j))
            {
                // Apply IIR filter:
                OT oldVal = background(i, j);

                int3 newVal = make_int3(
                    __float2int_rn(oldVal.x * (1.0f - alpha1) + curVal.x * alpha1),
                    __float2int_rn(oldVal.y * (1.0f - alpha1) + curVal.y * alpha1),
                    __float2int_rn(oldVal.z * (1.0f - alpha1) + curVal.z * alpha1)
                );

                background(i, j) = Output<OT>::make(
                    static_cast<uchar>(newVal.x),
                    static_cast<uchar>(newVal.y),
                    static_cast<uchar>(newVal.z)
                );
            }
            else
            {
                background(i, j) = Output<OT>::make(curVal.x, curVal.y, curVal.z);
            }
        }
    }

    template <typename PT, typename CT, typename OT>
    struct UpdateBackgroundModel
    {
        static void call(PtrStepSz<PT> prevFrame, PtrStepSz<CT> curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSz<OT> background,
                         int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T,
                         hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(prevFrame.cols, block.x), divUp(prevFrame.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(updateBackgroundModel<PT), CT, OT, PtrStep<PT>, PtrStep<CT>, PtrStepb, PtrStepb>, hipFuncCachePreferL1) );

            updateBackgroundModel<PT, CT, OT, PtrStep<PT>, PtrStep<CT>, PtrStepb, PtrStepb><<<grid, block, 0, stream>>>(
                prevFrame.cols, prevFrame.rows,
                prevFrame, curFrame,
                Ftd, Fbd, foreground, background,
                deltaC, deltaCC, alpha1, alpha2, alpha3, N1c, N1cc, N2c, N2cc, T);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    };

    template <typename PT, typename CT, typename OT>
    void updateBackgroundModel_gpu(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background,
                                   int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T,
                                   hipStream_t stream)
    {
        UpdateBackgroundModel<PT, CT, OT>::call(PtrStepSz<PT>(prevFrame), PtrStepSz<CT>(curFrame), Ftd, Fbd, foreground, PtrStepSz<OT>(background),
                                                deltaC, deltaCC, alpha1, alpha2, alpha3, N1c, N1cc, N2c, N2cc, T, stream);
    }

    template void updateBackgroundModel_gpu<uchar3, uchar3, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar3, uchar3, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar3, uchar4, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar3, uchar4, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar4, uchar3, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar4, uchar3, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar4, uchar4, uchar3>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
    template void updateBackgroundModel_gpu<uchar4, uchar4, uchar4>(PtrStepSzb prevFrame, PtrStepSzb curFrame, PtrStepSzb Ftd, PtrStepSzb Fbd, PtrStepSzb foreground, PtrStepSzb background, int deltaC, int deltaCC, float alpha1, float alpha2, float alpha3, int N1c, int N1cc, int N2c, int N2cc, float T, hipStream_t stream);
}

#endif /* CUDA_DISABLER */
