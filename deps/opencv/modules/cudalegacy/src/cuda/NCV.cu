#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <iostream>
#include <vector>

#include "opencv2/cudalegacy/NCV.hpp"

//===================================================================
//
// Operations with rectangles
//
//===================================================================


const Ncv32u NUMTHREADS_DRAWRECTS = 32;
const Ncv32u NUMTHREADS_DRAWRECTS_LOG2 = 5;


template <class T>
__global__ void drawRects(T *d_dst,
                          Ncv32u dstStride,
                          Ncv32u dstWidth,
                          Ncv32u dstHeight,
                          NcvRect32u *d_rects,
                          Ncv32u numRects,
                          T color)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    if (blockId > numRects * 4)
    {
        return;
    }

    NcvRect32u curRect = d_rects[blockId >> 2];
    NcvBool bVertical = blockId & 0x1;
    NcvBool bTopLeft = blockId & 0x2;

    Ncv32u pt0x, pt0y;
    if (bVertical)
    {
        Ncv32u numChunks = (curRect.height + NUMTHREADS_DRAWRECTS - 1) >> NUMTHREADS_DRAWRECTS_LOG2;

        pt0x = bTopLeft ? curRect.x : curRect.x + curRect.width - 1;
        pt0y = curRect.y;

        if (pt0x < dstWidth)
        {
            for (Ncv32u chunkId = 0; chunkId < numChunks; chunkId++)
            {
                Ncv32u ptY = pt0y + chunkId * NUMTHREADS_DRAWRECTS + threadIdx.x;
                if (ptY < pt0y + curRect.height && ptY < dstHeight)
                {
                    d_dst[ptY * dstStride + pt0x] = color;
                }
            }
        }
    }
    else
    {
        Ncv32u numChunks = (curRect.width + NUMTHREADS_DRAWRECTS - 1) >> NUMTHREADS_DRAWRECTS_LOG2;

        pt0x = curRect.x;
        pt0y = bTopLeft ? curRect.y : curRect.y + curRect.height - 1;

        if (pt0y < dstHeight)
        {
            for (Ncv32u chunkId = 0; chunkId < numChunks; chunkId++)
            {
                Ncv32u ptX = pt0x + chunkId * NUMTHREADS_DRAWRECTS + threadIdx.x;
                if (ptX < pt0x + curRect.width && ptX < dstWidth)
                {
                    d_dst[pt0y * dstStride + ptX] = color;
                }
            }
        }
    }
}


template <class T>
static NCVStatus drawRectsWrapperDevice(T *d_dst,
                                        Ncv32u dstStride,
                                        Ncv32u dstWidth,
                                        Ncv32u dstHeight,
                                        NcvRect32u *d_rects,
                                        Ncv32u numRects,
                                        T color,
                                        hipStream_t cuStream)
{
    (void)cuStream;
    ncvAssertReturn(d_dst != NULL && d_rects != NULL, NCV_NULL_PTR);
    ncvAssertReturn(dstWidth > 0 && dstHeight > 0, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(dstStride >= dstWidth, NCV_INVALID_STEP);
    ncvAssertReturn(numRects <= dstWidth * dstHeight, NCV_DIMENSIONS_INVALID);

    if (numRects == 0)
    {
        return NCV_SUCCESS;
    }

    dim3 grid(numRects * 4);
    dim3 block(NUMTHREADS_DRAWRECTS);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }

    drawRects<T><<<grid, block>>>(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color);

    ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);

    return NCV_SUCCESS;
}


NCVStatus ncvDrawRects_8u_device(Ncv8u *d_dst,
                                 Ncv32u dstStride,
                                 Ncv32u dstWidth,
                                 Ncv32u dstHeight,
                                 NcvRect32u *d_rects,
                                 Ncv32u numRects,
                                 Ncv8u color,
                                 hipStream_t cuStream)
{
    return drawRectsWrapperDevice(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color, cuStream);
}


NCVStatus ncvDrawRects_32u_device(Ncv32u *d_dst,
                                  Ncv32u dstStride,
                                  Ncv32u dstWidth,
                                  Ncv32u dstHeight,
                                  NcvRect32u *d_rects,
                                  Ncv32u numRects,
                                  Ncv32u color,
                                  hipStream_t cuStream)
{
    return drawRectsWrapperDevice(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color, cuStream);
}
