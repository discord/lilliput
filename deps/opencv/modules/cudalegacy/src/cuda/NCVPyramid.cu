#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "opencv2/core/cuda/common.hpp"

#include "opencv2/cudalegacy/NCV.hpp"
#include "opencv2/cudalegacy/NCVPyramid.hpp"

#include "NCVAlg.hpp"
#include "NCVPixelOperations.hpp"

template<typename T, Ncv32u CN> struct __average4_CN {static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11);};

template<typename T> struct __average4_CN<T, 1> {
static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11)
{
    T out;
    out.x = ((Ncv32s)p00.x + p01.x + p10.x + p11.x + 2) / 4;
    return out;
}};

template<> struct __average4_CN<float1, 1> {
static __host__ __device__ float1 _average4_CN(const float1 &p00, const float1 &p01, const float1 &p10, const float1 &p11)
{
    float1 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    return out;
}};

template<> struct __average4_CN<double1, 1> {
static __host__ __device__ double1 _average4_CN(const double1 &p00, const double1 &p01, const double1 &p10, const double1 &p11)
{
    double1 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    return out;
}};

template<typename T> struct __average4_CN<T, 3> {
static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11)
{
    T out;
    out.x = ((Ncv32s)p00.x + p01.x + p10.x + p11.x + 2) / 4;
    out.y = ((Ncv32s)p00.y + p01.y + p10.y + p11.y + 2) / 4;
    out.z = ((Ncv32s)p00.z + p01.z + p10.z + p11.z + 2) / 4;
    return out;
}};

template<> struct __average4_CN<float3, 3> {
static __host__ __device__ float3 _average4_CN(const float3 &p00, const float3 &p01, const float3 &p10, const float3 &p11)
{
    float3 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    return out;
}};

template<> struct __average4_CN<double3, 3> {
static __host__ __device__ double3 _average4_CN(const double3 &p00, const double3 &p01, const double3 &p10, const double3 &p11)
{
    double3 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    return out;
}};

template<typename T> struct __average4_CN<T, 4> {
static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11)
{
    T out;
    out.x = ((Ncv32s)p00.x + p01.x + p10.x + p11.x + 2) / 4;
    out.y = ((Ncv32s)p00.y + p01.y + p10.y + p11.y + 2) / 4;
    out.z = ((Ncv32s)p00.z + p01.z + p10.z + p11.z + 2) / 4;
    out.w = ((Ncv32s)p00.w + p01.w + p10.w + p11.w + 2) / 4;
    return out;
}};

template<> struct __average4_CN<float4, 4> {
static __host__ __device__ float4 _average4_CN(const float4 &p00, const float4 &p01, const float4 &p10, const float4 &p11)
{
    float4 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    out.w = (p00.w + p01.w + p10.w + p11.w) / 4;
    return out;
}};

template<> struct __average4_CN<double4, 4> {
static __host__ __device__ double4 _average4_CN(const double4 &p00, const double4 &p01, const double4 &p10, const double4 &p11)
{
    double4 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    out.w = (p00.w + p01.w + p10.w + p11.w) / 4;
    return out;
}};

template<typename T> static __host__ __device__ T _average4(const T &p00, const T &p01, const T &p10, const T &p11)
{
    return __average4_CN<T, NC(T)>::_average4_CN(p00, p01, p10, p11);
}


template<typename Tin, typename Tout, Ncv32u CN> struct __lerp_CN {static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d);};

template<typename Tin, typename Tout> struct __lerp_CN<Tin, Tout, 1> {
static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d)
{
    typedef typename TConvVec2Base<Tout>::TBase TB;
    return _pixMake(TB(b.x * d + a.x * (1 - d)));
}};

template<typename Tin, typename Tout> struct __lerp_CN<Tin, Tout, 3> {
static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d)
{
    typedef typename TConvVec2Base<Tout>::TBase TB;
    return _pixMake(TB(b.x * d + a.x * (1 - d)),
                    TB(b.y * d + a.y * (1 - d)),
                    TB(b.z * d + a.z * (1 - d)));
}};

template<typename Tin, typename Tout> struct __lerp_CN<Tin, Tout, 4> {
static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d)
{
    typedef typename TConvVec2Base<Tout>::TBase TB;
    return _pixMake(TB(b.x * d + a.x * (1 - d)),
                    TB(b.y * d + a.y * (1 - d)),
                    TB(b.z * d + a.z * (1 - d)),
                    TB(b.w * d + a.w * (1 - d)));
}};

template<typename Tin, typename Tout> static __host__ __device__ Tout _lerp(const Tin &a, const Tin &b, Ncv32f d)
{
    return __lerp_CN<Tin, Tout, NC(Tin)>::_lerp_CN(a, b, d);
}


template<typename T>
__global__ void kernelDownsampleX2(T *d_src,
                                   Ncv32u srcPitch,
                                   T *d_dst,
                                   Ncv32u dstPitch,
                                   NcvSize32u dstRoi)
{
    Ncv32u i = blockIdx.y * blockDim.y + threadIdx.y;
    Ncv32u j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < dstRoi.height && j < dstRoi.width)
    {
        T *d_src_line1 = (T *)((Ncv8u *)d_src + (2 * i + 0) * srcPitch);
        T *d_src_line2 = (T *)((Ncv8u *)d_src + (2 * i + 1) * srcPitch);
        T *d_dst_line = (T *)((Ncv8u *)d_dst + i * dstPitch);

        T p00 = d_src_line1[2*j+0];
        T p01 = d_src_line1[2*j+1];
        T p10 = d_src_line2[2*j+0];
        T p11 = d_src_line2[2*j+1];

        d_dst_line[j] = _average4(p00, p01, p10, p11);
    }
}

namespace cv { namespace cuda { namespace device
{
    namespace pyramid
    {
        template <typename T> void kernelDownsampleX2_gpu(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
        {
            dim3 bDim(16, 8);
            dim3 gDim(divUp(src.cols, bDim.x), divUp(src.rows, bDim.y));

            kernelDownsampleX2<<<gDim, bDim, 0, stream>>>((T*)src.data, static_cast<Ncv32u>(src.step),
                (T*)dst.data, static_cast<Ncv32u>(dst.step), NcvSize32u(dst.cols, dst.rows));

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void downsampleX2(PtrStepSzb src, PtrStepSzb dst, int depth, int cn, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

            static const func_t funcs[6][4] =
            {
                {kernelDownsampleX2_gpu<uchar1>       , 0 /*kernelDownsampleX2_gpu<uchar2>*/ , kernelDownsampleX2_gpu<uchar3>      , kernelDownsampleX2_gpu<uchar4>      },
                {0 /*kernelDownsampleX2_gpu<char1>*/  , 0 /*kernelDownsampleX2_gpu<char2>*/  , 0 /*kernelDownsampleX2_gpu<char3>*/ , 0 /*kernelDownsampleX2_gpu<char4>*/ },
                {kernelDownsampleX2_gpu<ushort1>      , 0 /*kernelDownsampleX2_gpu<ushort2>*/, kernelDownsampleX2_gpu<ushort3>     , kernelDownsampleX2_gpu<ushort4>     },
                {0 /*kernelDownsampleX2_gpu<short1>*/ , 0 /*kernelDownsampleX2_gpu<short2>*/ , 0 /*kernelDownsampleX2_gpu<short3>*/, 0 /*kernelDownsampleX2_gpu<short4>*/},
                {0 /*kernelDownsampleX2_gpu<int1>*/   , 0 /*kernelDownsampleX2_gpu<int2>*/   , 0 /*kernelDownsampleX2_gpu<int3>*/  , 0 /*kernelDownsampleX2_gpu<int4>*/  },
                {kernelDownsampleX2_gpu<float1>       , 0 /*kernelDownsampleX2_gpu<float2>*/ , kernelDownsampleX2_gpu<float3>      , kernelDownsampleX2_gpu<float4>      }
            };

            const func_t func = funcs[depth][cn - 1];
            CV_Assert(func != 0);

            func(src, dst, stream);
        }
    }
}}}




template<typename T>
__global__ void kernelInterpolateFrom1(T *d_srcTop,
                                       Ncv32u srcTopPitch,
                                       NcvSize32u szTopRoi,
                                       T *d_dst,
                                       Ncv32u dstPitch,
                                       NcvSize32u dstRoi)
{
    Ncv32u i = blockIdx.y * blockDim.y + threadIdx.y;
    Ncv32u j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < dstRoi.height && j < dstRoi.width)
    {
        Ncv32f ptTopX = 1.0f * (szTopRoi.width - 1) * j / (dstRoi.width - 1);
        Ncv32f ptTopY = 1.0f * (szTopRoi.height - 1) * i / (dstRoi.height - 1);
        Ncv32u xl = (Ncv32u)ptTopX;
        Ncv32u xh = xl+1;
        Ncv32f dx = ptTopX - xl;
        Ncv32u yl = (Ncv32u)ptTopY;
        Ncv32u yh = yl+1;
        Ncv32f dy = ptTopY - yl;

        T *d_src_line1 = (T *)((Ncv8u *)d_srcTop + yl * srcTopPitch);
        T *d_src_line2 = (T *)((Ncv8u *)d_srcTop + yh * srcTopPitch);
        T *d_dst_line = (T *)((Ncv8u *)d_dst + i * dstPitch);

        T p00, p01, p10, p11;
        p00 = d_src_line1[xl];
        p01 = xh < szTopRoi.width ? d_src_line1[xh] : p00;
        p10 = yh < szTopRoi.height ? d_src_line2[xl] : p00;
        p11 = (xh < szTopRoi.width && yh < szTopRoi.height) ? d_src_line2[xh] : p00;
        typedef typename TConvBase2Vec<Ncv32f, NC(T)>::TVec TVFlt;
        TVFlt m_00_01 = _lerp<T, TVFlt>(p00, p01, dx);
        TVFlt m_10_11 = _lerp<T, TVFlt>(p10, p11, dx);
        TVFlt mixture = _lerp<TVFlt, TVFlt>(m_00_01, m_10_11, dy);
        T outPix = _pixDemoteClampZ<TVFlt, T>(mixture);

        d_dst_line[j] = outPix;
    }
}
namespace cv { namespace cuda { namespace device
{
    namespace pyramid
    {
        template <typename T> void kernelInterpolateFrom1_gpu(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
        {
            dim3 bDim(16, 8);
            dim3 gDim(divUp(dst.cols, bDim.x), divUp(dst.rows, bDim.y));

            kernelInterpolateFrom1<<<gDim, bDim, 0, stream>>>((T*) src.data, static_cast<Ncv32u>(src.step), NcvSize32u(src.cols, src.rows),
                (T*) dst.data, static_cast<Ncv32u>(dst.step), NcvSize32u(dst.cols, dst.rows));

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void interpolateFrom1(PtrStepSzb src, PtrStepSzb dst, int depth, int cn, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

            static const func_t funcs[6][4] =
            {
                {kernelInterpolateFrom1_gpu<uchar1>      , 0 /*kernelInterpolateFrom1_gpu<uchar2>*/ , kernelInterpolateFrom1_gpu<uchar3>      , kernelInterpolateFrom1_gpu<uchar4>      },
                {0 /*kernelInterpolateFrom1_gpu<char1>*/ , 0 /*kernelInterpolateFrom1_gpu<char2>*/  , 0 /*kernelInterpolateFrom1_gpu<char3>*/ , 0 /*kernelInterpolateFrom1_gpu<char4>*/ },
                {kernelInterpolateFrom1_gpu<ushort1>     , 0 /*kernelInterpolateFrom1_gpu<ushort2>*/, kernelInterpolateFrom1_gpu<ushort3>     , kernelInterpolateFrom1_gpu<ushort4>     },
                {0 /*kernelInterpolateFrom1_gpu<short1>*/, 0 /*kernelInterpolateFrom1_gpu<short2>*/ , 0 /*kernelInterpolateFrom1_gpu<short3>*/, 0 /*kernelInterpolateFrom1_gpu<short4>*/},
                {0 /*kernelInterpolateFrom1_gpu<int1>*/  , 0 /*kernelInterpolateFrom1_gpu<int2>*/   , 0 /*kernelInterpolateFrom1_gpu<int3>*/  , 0 /*kernelInterpolateFrom1_gpu<int4>*/  },
                {kernelInterpolateFrom1_gpu<float1>      , 0 /*kernelInterpolateFrom1_gpu<float2>*/ , kernelInterpolateFrom1_gpu<float3>      , kernelInterpolateFrom1_gpu<float4>      }
            };

            const func_t func = funcs[depth][cn - 1];
            CV_Assert(func != 0);

            func(src, dst, stream);
        }
    }
}}}


#if 0 //def _WIN32

template<typename T>
static T _interpLinear(const T &a, const T &b, Ncv32f d)
{
    typedef typename TConvBase2Vec<Ncv32f, NC(T)>::TVec TVFlt;
    TVFlt tmp = _lerp<T, TVFlt>(a, b, d);
    return _pixDemoteClampZ<TVFlt, T>(tmp);
}


template<typename T>
static T _interpBilinear(const NCVMatrix<T> &refLayer, Ncv32f x, Ncv32f y)
{
    Ncv32u xl = (Ncv32u)x;
    Ncv32u xh = xl+1;
    Ncv32f dx = x - xl;
    Ncv32u yl = (Ncv32u)y;
    Ncv32u yh = yl+1;
    Ncv32f dy = y - yl;
    T p00, p01, p10, p11;
    p00 = refLayer.at(xl, yl);
    p01 = xh < refLayer.width() ? refLayer.at(xh, yl) : p00;
    p10 = yh < refLayer.height() ? refLayer.at(xl, yh) : p00;
    p11 = (xh < refLayer.width() && yh < refLayer.height()) ? refLayer.at(xh, yh) : p00;
    typedef typename TConvBase2Vec<Ncv32f, NC(T)>::TVec TVFlt;
    TVFlt m_00_01 = _lerp<T, TVFlt>(p00, p01, dx);
    TVFlt m_10_11 = _lerp<T, TVFlt>(p10, p11, dx);
    TVFlt mixture = _lerp<TVFlt, TVFlt>(m_00_01, m_10_11, dy);
    return _pixDemoteClampZ<TVFlt, T>(mixture);
}

template <class T>
NCVImagePyramid<T>::NCVImagePyramid(const NCVMatrix<T> &img,
                                    Ncv8u numLayers,
                                    INCVMemAllocator &alloc,
                                    hipStream_t cuStream)
{
    this->_isInitialized = false;
    ncvAssertPrintReturn(img.memType() == alloc.memType(), "NCVImagePyramid::ctor error", );

    this->layer0 = &img;
    NcvSize32u szLastLayer(img.width(), img.height());
    this->nLayers = 1;

    NCV_SET_SKIP_COND(alloc.isCounting());
    NcvBool bDeviceCode = alloc.memType() == NCVMemoryTypeDevice;

    if (numLayers == 0)
    {
        numLayers = 255; //it will cut-off when any of the dimensions goes 1
    }

#ifdef SELF_CHECK_GPU
    NCVMemNativeAllocator allocCPU(NCVMemoryTypeHostPinned, 512);
#endif

    for (Ncv32u i=0; i<(Ncv32u)numLayers-1; i++)
    {
        NcvSize32u szCurLayer(szLastLayer.width / 2, szLastLayer.height / 2);
        if (szCurLayer.width == 0 || szCurLayer.height == 0)
        {
            break;
        }

        this->pyramid.push_back(new NCVMatrixAlloc<T>(alloc, szCurLayer.width, szCurLayer.height));
        ncvAssertPrintReturn(((NCVMatrixAlloc<T> *)(this->pyramid[i]))->isMemAllocated(), "NCVImagePyramid::ctor error", );
        this->nLayers++;

        //fill in the layer
        NCV_SKIP_COND_BEGIN

        const NCVMatrix<T> *prevLayer = i == 0 ? this->layer0 : this->pyramid[i-1];
        NCVMatrix<T> *curLayer = this->pyramid[i];

        if (bDeviceCode)
        {
            dim3 bDim(16, 8);
            dim3 gDim(divUp(szCurLayer.width, bDim.x), divUp(szCurLayer.height, bDim.y));
            kernelDownsampleX2<<<gDim, bDim, 0, cuStream>>>(prevLayer->ptr(),
                                                            prevLayer->pitch(),
                                                            curLayer->ptr(),
                                                            curLayer->pitch(),
                                                            szCurLayer);
            ncvAssertPrintReturn(hipSuccess == hipGetLastError(), "NCVImagePyramid::ctor error", );

#ifdef SELF_CHECK_GPU
            NCVMatrixAlloc<T> h_prevLayer(allocCPU, prevLayer->width(), prevLayer->height());
            ncvAssertPrintReturn(h_prevLayer.isMemAllocated(), "Validation failure in NCVImagePyramid::ctor", );
            NCVMatrixAlloc<T> h_curLayer(allocCPU, curLayer->width(), curLayer->height());
            ncvAssertPrintReturn(h_curLayer.isMemAllocated(), "Validation failure in NCVImagePyramid::ctor", );
            ncvAssertPrintReturn(NCV_SUCCESS == prevLayer->copy2D(h_prevLayer, prevLayer->size(), cuStream), "Validation failure in NCVImagePyramid::ctor", );
            ncvAssertPrintReturn(NCV_SUCCESS == curLayer->copy2D(h_curLayer, curLayer->size(), cuStream), "Validation failure in NCVImagePyramid::ctor", );
            ncvAssertPrintReturn(hipSuccess == hipStreamSynchronize(cuStream), "Validation failure in NCVImagePyramid::ctor", );
            for (Ncv32u i=0; i<szCurLayer.height; i++)
            {
                for (Ncv32u j=0; j<szCurLayer.width; j++)
                {
                    T p00 = h_prevLayer.at(2*j+0, 2*i+0);
                    T p01 = h_prevLayer.at(2*j+1, 2*i+0);
                    T p10 = h_prevLayer.at(2*j+0, 2*i+1);
                    T p11 = h_prevLayer.at(2*j+1, 2*i+1);
                    T outGold = _average4(p00, p01, p10, p11);
                    T outGPU = h_curLayer.at(j, i);
                    ncvAssertPrintReturn(0 == memcmp(&outGold, &outGPU, sizeof(T)), "Validation failure in NCVImagePyramid::ctor with kernelDownsampleX2", );
                }
            }
#endif
        }
        else
        {
            for (Ncv32u i=0; i<szCurLayer.height; i++)
            {
                for (Ncv32u j=0; j<szCurLayer.width; j++)
                {
                    T p00 = prevLayer->at(2*j+0, 2*i+0);
                    T p01 = prevLayer->at(2*j+1, 2*i+0);
                    T p10 = prevLayer->at(2*j+0, 2*i+1);
                    T p11 = prevLayer->at(2*j+1, 2*i+1);
                    curLayer->at(j, i) = _average4(p00, p01, p10, p11);
                }
            }
        }

        NCV_SKIP_COND_END

        szLastLayer = szCurLayer;
    }

    this->_isInitialized = true;
}


template <class T>
NCVImagePyramid<T>::~NCVImagePyramid()
{
}


template <class T>
NcvBool NCVImagePyramid<T>::isInitialized() const
{
    return this->_isInitialized;
}


template <class T>
NCVStatus NCVImagePyramid<T>::getLayer(NCVMatrix<T> &outImg,
                                       NcvSize32u outRoi,
                                       NcvBool bTrilinear,
                                       hipStream_t cuStream) const
{
    ncvAssertReturn(this->isInitialized(), NCV_UNKNOWN_ERROR);
    ncvAssertReturn(outImg.memType() == this->layer0->memType(), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(outRoi.width <= this->layer0->width() && outRoi.height <= this->layer0->height() &&
                    outRoi.width > 0 && outRoi.height > 0, NCV_DIMENSIONS_INVALID);

    if (outRoi.width == this->layer0->width() && outRoi.height == this->layer0->height())
    {
        ncvAssertReturnNcvStat(this->layer0->copy2D(outImg, NcvSize32u(this->layer0->width(), this->layer0->height()), cuStream));
        return NCV_SUCCESS;
    }

    Ncv32f lastScale = 1.0f;
    Ncv32f curScale;
    const NCVMatrix<T> *lastLayer = this->layer0;
    const NCVMatrix<T> *curLayer = NULL;
    NcvBool bUse2Refs = false;

    for (Ncv32u i=0; i<this->nLayers-1; i++)
    {
        curScale = lastScale * 0.5f;
        curLayer = this->pyramid[i];

        if (outRoi.width == curLayer->width() && outRoi.height == curLayer->height())
        {
            ncvAssertReturnNcvStat(this->pyramid[i]->copy2D(outImg, NcvSize32u(this->pyramid[i]->width(), this->pyramid[i]->height()), cuStream));
            return NCV_SUCCESS;
        }

        if (outRoi.width >= curLayer->width() && outRoi.height >= curLayer->height())
        {
            if (outRoi.width < lastLayer->width() && outRoi.height < lastLayer->height())
            {
                bUse2Refs = true;
            }
            break;
        }

        lastScale = curScale;
        lastLayer = curLayer;
    }

    bUse2Refs = bUse2Refs && bTrilinear;

    NCV_SET_SKIP_COND(outImg.memType() == NCVMemoryTypeNone);
    NcvBool bDeviceCode = this->layer0->memType() == NCVMemoryTypeDevice;

#ifdef SELF_CHECK_GPU
    NCVMemNativeAllocator allocCPU(NCVMemoryTypeHostPinned, 512);
#endif

    NCV_SKIP_COND_BEGIN

    if (bDeviceCode)
    {
        ncvAssertReturn(bUse2Refs == false, NCV_NOT_IMPLEMENTED);

        dim3 bDim(16, 8);
        dim3 gDim(divUp(outRoi.width, bDim.x), divUp(outRoi.height, bDim.y));
        kernelInterpolateFrom1<<<gDim, bDim, 0, cuStream>>>(lastLayer->ptr(),
                                                            lastLayer->pitch(),
                                                            lastLayer->size(),
                                                            outImg.ptr(),
                                                            outImg.pitch(),
                                                            outRoi);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

#ifdef SELF_CHECK_GPU
        ncvSafeMatAlloc(h_lastLayer, T, allocCPU, lastLayer->width(), lastLayer->height(), NCV_ALLOCATOR_BAD_ALLOC);
        ncvSafeMatAlloc(h_outImg, T, allocCPU, outImg.width(), outImg.height(), NCV_ALLOCATOR_BAD_ALLOC);
        ncvAssertReturnNcvStat(lastLayer->copy2D(h_lastLayer, lastLayer->size(), cuStream));
        ncvAssertReturnNcvStat(outImg.copy2D(h_outImg, outRoi, cuStream));
        ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);

        for (Ncv32u i=0; i<outRoi.height; i++)
        {
            for (Ncv32u j=0; j<outRoi.width; j++)
            {
                NcvSize32u szTopLayer(lastLayer->width(), lastLayer->height());
                Ncv32f ptTopX = 1.0f * (szTopLayer.width - 1) * j / (outRoi.width - 1);
                Ncv32f ptTopY = 1.0f * (szTopLayer.height - 1) * i / (outRoi.height - 1);
                T outGold = _interpBilinear(h_lastLayer, ptTopX, ptTopY);
                ncvAssertPrintReturn(0 == memcmp(&outGold, &h_outImg.at(j,i), sizeof(T)), "Validation failure in NCVImagePyramid::ctor with kernelInterpolateFrom1", NCV_UNKNOWN_ERROR);
            }
        }
#endif
    }
    else
    {
        for (Ncv32u i=0; i<outRoi.height; i++)
        {
            for (Ncv32u j=0; j<outRoi.width; j++)
            {
                //top layer pixel (always exists)
                NcvSize32u szTopLayer(lastLayer->width(), lastLayer->height());
                Ncv32f ptTopX = 1.0f * (szTopLayer.width - 1) * j / (outRoi.width - 1);
                Ncv32f ptTopY = 1.0f * (szTopLayer.height - 1) * i / (outRoi.height - 1);
                T topPix = _interpBilinear(*lastLayer, ptTopX, ptTopY);
                T trilinearPix = topPix;

                if (bUse2Refs)
                {
                    //bottom layer pixel (exists only if the requested scale is greater than the smallest layer scale)
                    NcvSize32u szBottomLayer(curLayer->width(), curLayer->height());
                    Ncv32f ptBottomX = 1.0f * (szBottomLayer.width - 1) * j / (outRoi.width - 1);
                    Ncv32f ptBottomY = 1.0f * (szBottomLayer.height - 1) * i / (outRoi.height - 1);
                    T bottomPix = _interpBilinear(*curLayer, ptBottomX, ptBottomY);

                    Ncv32f scale = (1.0f * outRoi.width / layer0->width() + 1.0f * outRoi.height / layer0->height()) / 2;
                    Ncv32f dl = (scale - curScale) / (lastScale - curScale);
                    dl = CLAMP(dl, 0.0f, 1.0f);
                    trilinearPix = _interpLinear(bottomPix, topPix, dl);
                }

                outImg.at(j, i) = trilinearPix;
            }
        }
    }

    NCV_SKIP_COND_END

    return NCV_SUCCESS;
}


template class NCVImagePyramid<uchar1>;
template class NCVImagePyramid<uchar3>;
template class NCVImagePyramid<uchar4>;
template class NCVImagePyramid<ushort1>;
template class NCVImagePyramid<ushort3>;
template class NCVImagePyramid<ushort4>;
template class NCVImagePyramid<uint1>;
template class NCVImagePyramid<uint3>;
template class NCVImagePyramid<uint4>;
template class NCVImagePyramid<float1>;
template class NCVImagePyramid<float3>;
template class NCVImagePyramid<float4>;

#endif //_WIN32
