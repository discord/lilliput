#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"

namespace cv { namespace cuda { namespace device
{
    #define SOLVE_PNP_RANSAC_MAX_NUM_ITERS 200

    namespace transform_points
    {
        __constant__ float3 crot0;
        __constant__ float3 crot1;
        __constant__ float3 crot2;
        __constant__ float3 ctransl;

        struct TransformOp : unary_function<float3, float3>
        {
            __device__ __forceinline__ float3 operator()(const float3& p) const
            {
                return make_float3(
                        crot0.x * p.x + crot0.y * p.y + crot0.z * p.z + ctransl.x,
                        crot1.x * p.x + crot1.y * p.y + crot1.z * p.z + ctransl.y,
                        crot2.x * p.x + crot2.y * p.y + crot2.z * p.z + ctransl.z);
            }
            __host__ __device__ __forceinline__ TransformOp() {}
            __host__ __device__ __forceinline__ TransformOp(const TransformOp&) {}
        };

        void call(const PtrStepSz<float3> src, const float* rot,
                  const float* transl, PtrStepSz<float3> dst,
                  hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot0), rot, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot1), rot + 3, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot2), rot + 6, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctransl), transl, sizeof(float) * 3));
            cv::cuda::device::transform(src, dst, TransformOp(), WithOutMask(), stream);
        }
    } // namespace transform_points

    namespace project_points
    {
        __constant__ float3 crot0;
        __constant__ float3 crot1;
        __constant__ float3 crot2;
        __constant__ float3 ctransl;
        __constant__ float3 cproj0;
        __constant__ float3 cproj1;

        struct ProjectOp : unary_function<float3, float3>
        {
            __device__ __forceinline__ float2 operator()(const float3& p) const
            {
                // Rotate and translate in 3D
                float3 t = make_float3(
                        crot0.x * p.x + crot0.y * p.y + crot0.z * p.z + ctransl.x,
                        crot1.x * p.x + crot1.y * p.y + crot1.z * p.z + ctransl.y,
                        crot2.x * p.x + crot2.y * p.y + crot2.z * p.z + ctransl.z);
                // Project on 2D plane
                return make_float2(
                        (cproj0.x * t.x + cproj0.y * t.y) / t.z + cproj0.z,
                        (cproj1.x * t.x + cproj1.y * t.y) / t.z + cproj1.z);
            }
            __host__ __device__ __forceinline__ ProjectOp() {}
            __host__ __device__ __forceinline__ ProjectOp(const ProjectOp&) {}
        };

        void call(const PtrStepSz<float3> src, const float* rot,
                  const float* transl, const float* proj, PtrStepSz<float2> dst,
                  hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot0), rot, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot1), rot + 3, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot2), rot + 6, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctransl), transl, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cproj0), proj, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cproj1), proj + 3, sizeof(float) * 3));
            cv::cuda::device::transform(src, dst, ProjectOp(), WithOutMask(), stream);
        }
    } // namespace project_points

    namespace solve_pnp_ransac
    {
        __constant__ float3 crot_matrices[SOLVE_PNP_RANSAC_MAX_NUM_ITERS * 3];
        __constant__ float3 ctransl_vectors[SOLVE_PNP_RANSAC_MAX_NUM_ITERS];

        int maxNumIters()
        {
            return SOLVE_PNP_RANSAC_MAX_NUM_ITERS;
        }

        __device__ __forceinline__ float sqr(float x)
        {
            return x * x;
        }

        template <int BLOCK_SIZE>
        __global__ void computeHypothesisScoresKernel(
                const int num_points, const float3* object, const float2* image,
                const float dist_threshold, int* g_num_inliers)
        {
            const float3* const &rot_mat = crot_matrices + blockIdx.x * 3;
            const float3 &transl_vec = ctransl_vectors[blockIdx.x];
            int num_inliers = 0;

            for (int i = threadIdx.x; i < num_points; i += blockDim.x)
            {
                float3 p = object[i];
                p = make_float3(
                        rot_mat[0].x * p.x + rot_mat[0].y * p.y + rot_mat[0].z * p.z + transl_vec.x,
                        rot_mat[1].x * p.x + rot_mat[1].y * p.y + rot_mat[1].z * p.z + transl_vec.y,
                        rot_mat[2].x * p.x + rot_mat[2].y * p.y + rot_mat[2].z * p.z + transl_vec.z);
                p.x /= p.z;
                p.y /= p.z;
                float2 image_p = image[i];
                if (sqr(p.x - image_p.x) + sqr(p.y - image_p.y) < dist_threshold)
                    ++num_inliers;
            }

            __shared__ int s_num_inliers[BLOCK_SIZE];
            reduce<BLOCK_SIZE>(s_num_inliers, num_inliers, threadIdx.x, plus<int>());

            if (threadIdx.x == 0)
                g_num_inliers[blockIdx.x] = num_inliers;
        }

        void computeHypothesisScores(
                const int num_hypotheses, const int num_points, const float* rot_matrices,
                const float3* transl_vectors, const float3* object, const float2* image,
                const float dist_threshold, int* hypothesis_scores)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot_matrices), rot_matrices, num_hypotheses * 3 * sizeof(float3)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctransl_vectors), transl_vectors, num_hypotheses * sizeof(float3)));

            dim3 threads(256);
            dim3 grid(num_hypotheses);

            computeHypothesisScoresKernel<256><<<grid, threads>>>(
                    num_points, object, image, dist_threshold, hypothesis_scores);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }
    } // namespace solvepnp_ransac
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
