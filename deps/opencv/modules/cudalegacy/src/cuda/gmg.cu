#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/limits.hpp"

namespace cv { namespace cuda { namespace device {
    namespace gmg
    {
        __constant__ int   c_width;
        __constant__ int   c_height;
        __constant__ float c_minVal;
        __constant__ float c_maxVal;
        __constant__ int   c_quantizationLevels;
        __constant__ float c_backgroundPrior;
        __constant__ float c_decisionThreshold;
        __constant__ int   c_maxFeatures;
        __constant__ int   c_numInitializationFrames;

        void loadConstants(int width, int height, float minVal, float maxVal, int quantizationLevels, float backgroundPrior,
                           float decisionThreshold, int maxFeatures, int numInitializationFrames)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_width), &width, sizeof(width)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_height), &height, sizeof(height)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_minVal), &minVal, sizeof(minVal)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_maxVal), &maxVal, sizeof(maxVal)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_quantizationLevels), &quantizationLevels, sizeof(quantizationLevels)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_backgroundPrior), &backgroundPrior, sizeof(backgroundPrior)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_decisionThreshold), &decisionThreshold, sizeof(decisionThreshold)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_maxFeatures), &maxFeatures, sizeof(maxFeatures)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_numInitializationFrames), &numInitializationFrames, sizeof(numInitializationFrames)) );
        }

        __device__ float findFeature(const int color, const PtrStepi& colors, const PtrStepf& weights, const int x, const int y, const int nfeatures)
        {
            for (int i = 0, fy = y; i < nfeatures; ++i, fy += c_height)
            {
                if (color == colors(fy, x))
                    return weights(fy, x);
            }

            // not in histogram, so return 0.
            return 0.0f;
        }

        __device__ void normalizeHistogram(PtrStepf weights, const int x, const int y, const int nfeatures)
        {
            float total = 0.0f;
            for (int i = 0, fy = y; i < nfeatures; ++i, fy += c_height)
                total += weights(fy, x);

            if (total != 0.0f)
            {
                for (int i = 0, fy = y; i < nfeatures; ++i, fy += c_height)
                    weights(fy, x) /= total;
            }
        }

        __device__ bool insertFeature(const int color, const float weight, PtrStepi colors, PtrStepf weights, const int x, const int y, int& nfeatures)
        {
            for (int i = 0, fy = y; i < nfeatures; ++i, fy += c_height)
            {
                if (color == colors(fy, x))
                {
                    // feature in histogram

                    weights(fy, x) += weight;

                    return false;
                }
            }

            if (nfeatures == c_maxFeatures)
            {
                // discard oldest feature

                int idx = -1;
                float minVal = numeric_limits<float>::max();
                for (int i = 0, fy = y; i < nfeatures; ++i, fy += c_height)
                {
                    const float w = weights(fy, x);
                    if (w < minVal)
                    {
                        minVal = w;
                        idx = fy;
                    }
                }

                colors(idx, x) = color;
                weights(idx, x) = weight;

                return false;
            }

            colors(nfeatures * c_height + y, x) = color;
            weights(nfeatures * c_height + y, x) = weight;

            ++nfeatures;

            return true;
        }

        namespace detail
        {
            template <int cn> struct Quantization
            {
                template <typename T>
                __device__ static int apply(const T& val)
                {
                    int res = 0;
                    res |= static_cast<int>((val.x - c_minVal) * c_quantizationLevels / (c_maxVal - c_minVal));
                    res |= static_cast<int>((val.y - c_minVal) * c_quantizationLevels / (c_maxVal - c_minVal)) << 8;
                    res |= static_cast<int>((val.z - c_minVal) * c_quantizationLevels / (c_maxVal - c_minVal)) << 16;
                    return res;
                }
            };

            template <> struct Quantization<1>
            {
                template <typename T>
                __device__ static int apply(T val)
                {
                    return static_cast<int>((val - c_minVal) * c_quantizationLevels / (c_maxVal - c_minVal));
                }
            };
        }

        template <typename T> struct Quantization : detail::Quantization<VecTraits<T>::cn> {};

        template <typename SrcT>
        __global__ void update(const PtrStep<SrcT> frame, PtrStepb fgmask, PtrStepi colors_, PtrStepf weights_, PtrStepi nfeatures_,
                               const int frameNum, const float learningRate, const bool updateBackgroundModel)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= c_width || y >= c_height)
                return;

            const SrcT pix = frame(y, x);
            const int newFeatureColor = Quantization<SrcT>::apply(pix);

            int nfeatures = nfeatures_(y, x);

            if (frameNum >= c_numInitializationFrames)
            {
                // typical operation

                const float weight = findFeature(newFeatureColor, colors_, weights_, x, y, nfeatures);

                // see Godbehere, Matsukawa, Goldberg (2012) for reasoning behind this implementation of Bayes rule
                const float posterior = (weight * c_backgroundPrior) / (weight * c_backgroundPrior + (1.0f - weight) * (1.0f - c_backgroundPrior));

                const bool isForeground = ((1.0f - posterior) > c_decisionThreshold);
                fgmask(y, x) = (uchar)(-isForeground);

                // update histogram.

                if (updateBackgroundModel)
                {
                    for (int i = 0, fy = y; i < nfeatures; ++i, fy += c_height)
                        weights_(fy, x) *= 1.0f - learningRate;

                    bool inserted = insertFeature(newFeatureColor, learningRate, colors_, weights_, x, y, nfeatures);

                    if (inserted)
                    {
                        normalizeHistogram(weights_, x, y, nfeatures);
                        nfeatures_(y, x) = nfeatures;
                    }
                }
            }
            else if (updateBackgroundModel)
            {
                // training-mode update

                insertFeature(newFeatureColor, 1.0f, colors_, weights_, x, y, nfeatures);

                if (frameNum == c_numInitializationFrames - 1)
                    normalizeHistogram(weights_, x, y, nfeatures);
            }
        }

        template <typename SrcT>
        void update_gpu(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures,
                        int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream)
        {
            const dim3 block(32, 8);
            const dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(update<SrcT>), hipFuncCachePreferL1) );

            update<SrcT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask, colors, weights, nfeatures, frameNum, learningRate, updateBackgroundModel);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void update_gpu<uchar  >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
        template void update_gpu<uchar3 >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
        template void update_gpu<uchar4 >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);

        template void update_gpu<ushort >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
        template void update_gpu<ushort3>(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
        template void update_gpu<ushort4>(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);

        template void update_gpu<float  >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
        template void update_gpu<float3 >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
        template void update_gpu<float4 >(PtrStepSzb frame, PtrStepb fgmask, PtrStepSzi colors, PtrStepf weights, PtrStepi nfeatures, int frameNum, float learningRate, bool updateBackgroundModel, hipStream_t stream);
    }
}}}


#endif /* CUDA_DISABLER */
