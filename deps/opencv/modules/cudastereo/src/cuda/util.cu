#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"

namespace cv { namespace cuda { namespace device
{
    /////////////////////////////////// reprojectImageTo3D ///////////////////////////////////////////////

    __constant__ float cq[16];

    template <typename T, typename D>
    __global__ void reprojectImageTo3D(const PtrStepSz<T> disp, PtrStep<D> xyz)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y >= disp.rows || x >= disp.cols)
            return;

        const float qx = x * cq[ 0] + y * cq[ 1] + cq[ 3];
        const float qy = x * cq[ 4] + y * cq[ 5] + cq[ 7];
        const float qz = x * cq[ 8] + y * cq[ 9] + cq[11];
        const float qw = x * cq[12] + y * cq[13] + cq[15];

        const T d = disp(y, x);

        const float iW = 1.f / (qw + cq[14] * d);

        D v = VecTraits<D>::all(1.0f);
        v.x = (qx + cq[2] * d) * iW;
        v.y = (qy + cq[6] * d) * iW;
        v.z = (qz + cq[10] * d) * iW;

        xyz(y, x) = v;
    }

    template <typename T, typename D>
    void reprojectImageTo3D_gpu(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream)
    {
        dim3 block(32, 8);
        dim3 grid(divUp(disp.cols, block.x), divUp(disp.rows, block.y));

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)) );

        reprojectImageTo3D<T, D><<<grid, block, 0, stream>>>((PtrStepSz<T>)disp, (PtrStepSz<D>)xyz);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void reprojectImageTo3D_gpu<uchar, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<uchar, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<short, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<short, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<int, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<int, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<float, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
    template void reprojectImageTo3D_gpu<float, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);

    /////////////////////////////////// drawColorDisp ///////////////////////////////////////////////

    template <typename T>
    __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
    {
        unsigned int H = ((ndisp-d) * 240)/ndisp;

        unsigned int hi = (H/60) % 6;
        float f = H/60.f - H/60;
        float p = V * (1 - S);
        float q = V * (1 - f * S);
        float t = V * (1 - (1 - f) * S);

        float3 res;

        if (hi == 0) //R = V,	G = t,	B = p
        {
            res.x = p;
            res.y = t;
            res.z = V;
        }

        if (hi == 1) // R = q,	G = V,	B = p
        {
            res.x = p;
            res.y = V;
            res.z = q;
        }

        if (hi == 2) // R = p,	G = V,	B = t
        {
            res.x = t;
            res.y = V;
            res.z = p;
        }

        if (hi == 3) // R = p,	G = q,	B = V
        {
            res.x = V;
            res.y = q;
            res.z = p;
        }

        if (hi == 4) // R = t,	G = p,	B = V
        {
            res.x = V;
            res.y = p;
            res.z = t;
        }

        if (hi == 5) // R = V,	G = p,	B = q
        {
            res.x = q;
            res.y = p;
            res.z = V;
        }
        const unsigned int b = (unsigned int)(::max(0.f, ::min(res.x, 1.f)) * 255.f);
        const unsigned int g = (unsigned int)(::max(0.f, ::min(res.y, 1.f)) * 255.f);
        const unsigned int r = (unsigned int)(::max(0.f, ::min(res.z, 1.f)) * 255.f);
        const unsigned int a = 255U;

        return (a << 24) + (r << 16) + (g << 8) + b;
    }

    __global__ void drawColorDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height)
        {
            uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

            uint4 res;
            res.x = cvtPixel(d4.x, ndisp);
            res.y = cvtPixel(d4.y, ndisp);
            res.z = cvtPixel(d4.z, ndisp);
            res.w = cvtPixel(d4.w, ndisp);

            uint4* line = (uint4*)(out_image + y * out_step);
            line[x >> 2] = res;
        }
    }

    __global__ void drawColorDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height)
        {
            short2 d2 = *(short2*)(disp + y * disp_step + x);

            uint2 res;
            res.x = cvtPixel(d2.x, ndisp);
            res.y = cvtPixel(d2.y, ndisp);

            uint2* line = (uint2*)(out_image + y * out_step);
            line[x >> 1] = res;
        }
    }

    __global__ void drawColorDisp(int* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height)
        {
            uint *line = (uint*)(out_image + y * out_step);
            line[x] = cvtPixel(disp[y*disp_step + x], ndisp);
        }
    }

    __global__ void drawColorDisp(float* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height)
        {
            uint *line = (uint*)(out_image + y * out_step);
            line[x] = cvtPixel(disp[y*disp_step + x], ndisp);
        }
    }

    void drawColorDisp_gpu(const PtrStepSzb& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 2);
        grid.y = divUp(src.rows, threads.y);

        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step, dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void drawColorDisp_gpu(const PtrStepSz<short>& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 1);
        grid.y = divUp(src.rows, threads.y);

        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(short), dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void drawColorDisp_gpu(const PtrStepSz<int>& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(int), dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void drawColorDisp_gpu(const PtrStepSz<float>& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(float), dst.data, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
