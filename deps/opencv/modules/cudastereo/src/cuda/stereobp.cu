#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/limits.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace stereobp
    {
        ///////////////////////////////////////////////////////////////
        /////////////////////// load constants ////////////////////////
        ///////////////////////////////////////////////////////////////

        __constant__ int   cndisp;
        __constant__ float cmax_data_term;
        __constant__ float cdata_weight;
        __constant__ float cmax_disc_term;
        __constant__ float cdisc_single_jump;

        void load_constants(int ndisp, float max_data_term, float data_weight, float max_disc_term, float disc_single_jump)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cndisp),            &ndisp,            sizeof(int  )) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_data_term),    &max_data_term,    sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdata_weight),      &data_weight,      sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_disc_term),    &max_disc_term,    sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisc_single_jump), &disc_single_jump, sizeof(float)) );
        }

        ///////////////////////////////////////////////////////////////
        ////////////////////////// comp data //////////////////////////
        ///////////////////////////////////////////////////////////////

        template <int cn> struct PixDiff;
        template <> struct PixDiff<1>
        {
            __device__ __forceinline__ PixDiff(const uchar* ls)
            {
                l = *ls;
            }
            __device__ __forceinline__ float operator()(const uchar* rs) const
            {
                return ::abs((int)l - *rs);
            }
            uchar l;
        };
        template <> struct PixDiff<3>
        {
            __device__ __forceinline__ PixDiff(const uchar* ls)
            {
                l = *((uchar3*)ls);
            }
            __device__ __forceinline__ float operator()(const uchar* rs) const
            {
                const float tr = 0.299f;
                const float tg = 0.587f;
                const float tb = 0.114f;

                float val  = tb * ::abs((int)l.x - rs[0]);
                      val += tg * ::abs((int)l.y - rs[1]);
                      val += tr * ::abs((int)l.z - rs[2]);

                return val;
            }
            uchar3 l;
        };
        template <> struct PixDiff<4>
        {
            __device__ __forceinline__ PixDiff(const uchar* ls)
            {
                l = *((uchar4*)ls);
            }
            __device__ __forceinline__ float operator()(const uchar* rs) const
            {
                const float tr = 0.299f;
                const float tg = 0.587f;
                const float tb = 0.114f;

                uchar4 r = *((uchar4*)rs);

                float val  = tb * ::abs((int)l.x - r.x);
                      val += tg * ::abs((int)l.y - r.y);
                      val += tr * ::abs((int)l.z - r.z);

                return val;
            }
            uchar4 l;
        };

        template <int cn, typename D>
        __global__ void comp_data(const PtrStepSzb left, const PtrStepb right, PtrStep<D> data)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y > 0 && y < left.rows - 1 && x > 0 && x < left.cols - 1)
            {
                const uchar* ls = left.ptr(y) + x * cn;
                const PixDiff<cn> pixDiff(ls);
                const uchar* rs = right.ptr(y) + x * cn;

                D* ds = data.ptr(y) + x;
                const size_t disp_step = data.step * left.rows / sizeof(D);

                for (int disp = 0; disp < cndisp; disp++)
                {
                    if (x - disp >= 1)
                    {
                        float val = pixDiff(rs - disp * cn);

                        ds[disp * disp_step] = saturate_cast<D>(fmin(cdata_weight * val, cdata_weight * cmax_data_term));
                    }
                    else
                    {
                        ds[disp * disp_step] = saturate_cast<D>(cdata_weight * cmax_data_term);
                    }
                }
            }
        }

        template<typename T, typename D>
        void comp_data_gpu(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream);

        template <> void comp_data_gpu<uchar, short>(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<1, short><<<grid, threads, 0, stream>>>(left, right, (PtrStepSz<short>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <> void comp_data_gpu<uchar, float>(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<1, float><<<grid, threads, 0, stream>>>(left, right, (PtrStepSz<float>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <> void comp_data_gpu<uchar3, short>(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<3, short><<<grid, threads, 0, stream>>>(left, right, (PtrStepSz<short>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <> void comp_data_gpu<uchar3, float>(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<3, float><<<grid, threads, 0, stream>>>(left, right, (PtrStepSz<float>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <> void comp_data_gpu<uchar4, short>(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<4, short><<<grid, threads, 0, stream>>>(left, right, (PtrStepSz<short>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
        template <> void comp_data_gpu<uchar4, float>(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(left.cols, threads.x);
            grid.y = divUp(left.rows, threads.y);

            comp_data<4, float><<<grid, threads, 0, stream>>>(left, right, (PtrStepSz<float>)data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////
        //////////////////////// data step down ///////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __global__ void data_step_down(int dst_cols, int dst_rows, int src_rows, const PtrStep<T> src, PtrStep<T> dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst_cols && y < dst_rows)
            {
                for (int d = 0; d < cndisp; ++d)
                {
                    float dst_reg  = src.ptr(d * src_rows + (2*y+0))[(2*x+0)];
                          dst_reg += src.ptr(d * src_rows + (2*y+1))[(2*x+0)];
                          dst_reg += src.ptr(d * src_rows + (2*y+0))[(2*x+1)];
                          dst_reg += src.ptr(d * src_rows + (2*y+1))[(2*x+1)];

                    dst.ptr(d * dst_rows + y)[x] = saturate_cast<T>(dst_reg);
                }
            }
        }

        template<typename T>
        void data_step_down_gpu(int dst_cols, int dst_rows, int src_rows, const PtrStepSzb& src, const PtrStepSzb& dst, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(dst_cols, threads.x);
            grid.y = divUp(dst_rows, threads.y);

            data_step_down<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (PtrStepSz<T>)src, (PtrStepSz<T>)dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void data_step_down_gpu<short>(int dst_cols, int dst_rows, int src_rows, const PtrStepSzb& src, const PtrStepSzb& dst, hipStream_t stream);
        template void data_step_down_gpu<float>(int dst_cols, int dst_rows, int src_rows, const PtrStepSzb& src, const PtrStepSzb& dst, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        /////////////////// level up messages  ////////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __global__ void level_up_message(int dst_cols, int dst_rows, int src_rows, const PtrStep<T> src, PtrStep<T> dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst_cols && y < dst_rows)
            {
                const size_t dst_disp_step = dst.step * dst_rows / sizeof(T);
                const size_t src_disp_step = src.step * src_rows / sizeof(T);

                T*       dstr = dst.ptr(y  ) + x;
                const T* srcr = src.ptr(y/2) + x/2;

                for (int d = 0; d < cndisp; ++d)
                    dstr[d * dst_disp_step] = srcr[d * src_disp_step];
            }
        }

        template <typename T>
        void level_up_messages_gpu(int dst_idx, int dst_cols, int dst_rows, int src_rows, PtrStepSzb* mus, PtrStepSzb* mds, PtrStepSzb* mls, PtrStepSzb* mrs, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(dst_cols, threads.x);
            grid.y = divUp(dst_rows, threads.y);

            int src_idx = (dst_idx + 1) & 1;

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (PtrStepSz<T>)mus[src_idx], (PtrStepSz<T>)mus[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (PtrStepSz<T>)mds[src_idx], (PtrStepSz<T>)mds[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (PtrStepSz<T>)mls[src_idx], (PtrStepSz<T>)mls[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (PtrStepSz<T>)mrs[src_idx], (PtrStepSz<T>)mrs[dst_idx]);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void level_up_messages_gpu<short>(int dst_idx, int dst_cols, int dst_rows, int src_rows, PtrStepSzb* mus, PtrStepSzb* mds, PtrStepSzb* mls, PtrStepSzb* mrs, hipStream_t stream);
        template void level_up_messages_gpu<float>(int dst_idx, int dst_cols, int dst_rows, int src_rows, PtrStepSzb* mus, PtrStepSzb* mds, PtrStepSzb* mls, PtrStepSzb* mrs, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        ////////////////////  calc all iterations /////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __device__ void calc_min_linear_penalty(T* dst, size_t step)
        {
            float prev = dst[0];
            float cur;
            for (int disp = 1; disp < cndisp; ++disp)
            {
                prev += cdisc_single_jump;
                cur = dst[step * disp];
                if (prev < cur)
                {
                    cur = prev;
                    dst[step * disp] = saturate_cast<T>(prev);
                }
                prev = cur;
            }

            prev = dst[(cndisp - 1) * step];
            for (int disp = cndisp - 2; disp >= 0; disp--)
            {
                prev += cdisc_single_jump;
                cur = dst[step * disp];
                if (prev < cur)
                {
                    cur = prev;
                    dst[step * disp] = saturate_cast<T>(prev);
                }
                prev = cur;
            }
        }

        template <typename T>
        __device__ void message(const T* msg1, const T* msg2, const T* msg3, const T* data, T* dst, size_t msg_disp_step, size_t data_disp_step)
        {
            float minimum = device::numeric_limits<float>::max();

            for(int i = 0; i < cndisp; ++i)
            {
                float dst_reg  = msg1[msg_disp_step * i];
                      dst_reg += msg2[msg_disp_step * i];
                      dst_reg += msg3[msg_disp_step * i];
                      dst_reg += data[data_disp_step * i];

                if (dst_reg < minimum)
                    minimum = dst_reg;

                dst[msg_disp_step * i] = saturate_cast<T>(dst_reg);
            }

            calc_min_linear_penalty(dst, msg_disp_step);

            minimum += cmax_disc_term;

            float sum = 0;
            for(int i = 0; i < cndisp; ++i)
            {
                float dst_reg = dst[msg_disp_step * i];
                if (dst_reg > minimum)
                {
                    dst_reg = minimum;
                    dst[msg_disp_step * i] = saturate_cast<T>(minimum);
                }
                sum += dst_reg;
            }
            sum /= cndisp;

            for(int i = 0; i < cndisp; ++i)
                dst[msg_disp_step * i] -= sum;
        }

        template <typename T>
        __global__ void one_iteration(int t, int elem_step, T* u, T* d, T* l, T* r, const PtrStep<T> data, int cols, int rows)
        {
            const int y = blockIdx.y * blockDim.y + threadIdx.y;
            const int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + t) & 1);

            if ((y > 0) && (y < rows - 1) && (x > 0) && (x < cols - 1))
            {
                T* us = u + y * elem_step + x;
                T* ds = d + y * elem_step + x;
                T* ls = l + y * elem_step + x;
                T* rs = r + y * elem_step + x;
                const T* dt = data.ptr(y) + x;

                size_t msg_disp_step = elem_step * rows;
                size_t data_disp_step = data.step * rows / sizeof(T);

                message(us + elem_step, ls         + 1, rs - 1, dt, us, msg_disp_step, data_disp_step);
                message(ds - elem_step, ls         + 1, rs - 1, dt, ds, msg_disp_step, data_disp_step);
                message(us + elem_step, ds - elem_step, rs - 1, dt, rs, msg_disp_step, data_disp_step);
                message(us + elem_step, ds - elem_step, ls + 1, dt, ls, msg_disp_step, data_disp_step);
            }
        }

        template <typename T>
        void calc_all_iterations_gpu(int cols, int rows, int iters, const PtrStepSzb& u, const PtrStepSzb& d,
            const PtrStepSzb& l, const PtrStepSzb& r, const PtrStepSzb& data, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(cols, threads.x << 1);
            grid.y = divUp(rows, threads.y);

            int elem_step = (int)(u.step / sizeof(T));

            for(int t = 0; t < iters; ++t)
            {
                one_iteration<T><<<grid, threads, 0, stream>>>(t, elem_step, (T*)u.data, (T*)d.data, (T*)l.data, (T*)r.data, (PtrStepSz<T>)data, cols, rows);
                cudaSafeCall( hipGetLastError() );

                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        }

        template void calc_all_iterations_gpu<short>(int cols, int rows, int iters, const PtrStepSzb& u, const PtrStepSzb& d, const PtrStepSzb& l, const PtrStepSzb& r, const PtrStepSzb& data, hipStream_t stream);
        template void calc_all_iterations_gpu<float>(int cols, int rows, int iters, const PtrStepSzb& u, const PtrStepSzb& d, const PtrStepSzb& l, const PtrStepSzb& r, const PtrStepSzb& data, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        /////////////////////////// output ////////////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __global__ void output(const int elem_step, const T* u, const T* d, const T* l, const T* r, const T* data,
            PtrStepSz<short> disp)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y > 0 && y < disp.rows - 1 && x > 0 && x < disp.cols - 1)
            {
                const T* us = u + (y + 1) * elem_step + x;
                const T* ds = d + (y - 1) * elem_step + x;
                const T* ls = l + y * elem_step + (x + 1);
                const T* rs = r + y * elem_step+ (x - 1);
                const T* dt = data + y * elem_step + x;

                size_t disp_step = disp.rows * elem_step;

                int best = 0;
                float best_val = numeric_limits<float>::max();
                for (int d = 0; d < cndisp; ++d)
                {
                    float val  = us[d * disp_step];
                          val += ds[d * disp_step];
                          val += ls[d * disp_step];
                          val += rs[d * disp_step];
                          val += dt[d * disp_step];

                    if (val < best_val)
                    {
                        best_val = val;
                        best = d;
                    }
                }

                disp.ptr(y)[x] = saturate_cast<short>(best);
            }
        }

        template <typename T>
        void output_gpu(const PtrStepSzb& u, const PtrStepSzb& d, const PtrStepSzb& l, const PtrStepSzb& r, const PtrStepSzb& data,
            const PtrStepSz<short>& disp, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(disp.cols, threads.x);
            grid.y = divUp(disp.rows, threads.y);

            int elem_step = static_cast<int>(u.step/sizeof(T));

            output<T><<<grid, threads, 0, stream>>>(elem_step, (const T*)u.data, (const T*)d.data, (const T*)l.data, (const T*)r.data, (const T*)data.data, disp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void output_gpu<short>(const PtrStepSzb& u, const PtrStepSzb& d, const PtrStepSzb& l, const PtrStepSzb& r, const PtrStepSzb& data, const PtrStepSz<short>& disp, hipStream_t stream);
        template void output_gpu<float>(const PtrStepSzb& u, const PtrStepSzb& d, const PtrStepSzb& l, const PtrStepSzb& r, const PtrStepSzb& data, const PtrStepSz<short>& disp, hipStream_t stream);
    } // namespace stereobp
}}} // namespace cv { namespace cuda { namespace cudev

#endif /* CUDA_DISABLER */
