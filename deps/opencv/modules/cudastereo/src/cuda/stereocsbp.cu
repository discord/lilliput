#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"

#include "cuda/stereocsbp.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace stereocsbp
    {
        ///////////////////////////////////////////////////////////////
        /////////////////////// init data cost ////////////////////////
        ///////////////////////////////////////////////////////////////

        template <int channels> static float __device__ pixeldiff(const uchar* left, const uchar* right, float max_data_term);
        template<> __device__ __forceinline__ float pixeldiff<1>(const uchar* left, const uchar* right, float max_data_term)
        {
            return fminf( ::abs((int)*left - *right), max_data_term);
        }
        template<> __device__ __forceinline__ float pixeldiff<3>(const uchar* left, const uchar* right, float max_data_term)
        {
            float tb = 0.114f * ::abs((int)left[0] - right[0]);
            float tg = 0.587f * ::abs((int)left[1] - right[1]);
            float tr = 0.299f * ::abs((int)left[2] - right[2]);

            return fminf(tr + tg + tb, max_data_term);
        }
        template<> __device__ __forceinline__ float pixeldiff<4>(const uchar* left, const uchar* right, float max_data_term)
        {
            uchar4 l = *((const uchar4*)left);
            uchar4 r = *((const uchar4*)right);

            float tb = 0.114f * ::abs((int)l.x - r.x);
            float tg = 0.587f * ::abs((int)l.y - r.y);
            float tr = 0.299f * ::abs((int)l.z - r.z);

            return fminf(tr + tg + tb, max_data_term);
        }

        template <typename T>
        __global__ void get_first_k_initial_global(uchar *ctemp, T* data_cost_selected_, T *selected_disp_pyr, int h, int w, int nr_plane, int ndisp,
            size_t msg_step, size_t disp_step)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                T* selected_disparity = selected_disp_pyr + y * msg_step + x;
                T* data_cost_selected = data_cost_selected_ + y * msg_step + x;
                T* data_cost = (T*)ctemp + y * msg_step + x;

                for(int i = 0; i < nr_plane; i++)
                {
                    T minimum = device::numeric_limits<T>::max();
                    int id = 0;
                    for(int d = 0; d < ndisp; d++)
                    {
                        T cur = data_cost[d * disp_step];
                        if(cur < minimum)
                        {
                            minimum = cur;
                            id = d;
                        }
                    }

                    data_cost_selected[i  * disp_step] = minimum;
                    selected_disparity[i  * disp_step] = id;
                    data_cost         [id * disp_step] = numeric_limits<T>::max();
                }
            }
        }


        template <typename T>
        __global__ void get_first_k_initial_local(uchar *ctemp, T* data_cost_selected_, T* selected_disp_pyr, int h, int w, int nr_plane, int ndisp,
            size_t msg_step, size_t disp_step)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                T* selected_disparity = selected_disp_pyr + y * msg_step + x;
                T* data_cost_selected = data_cost_selected_ + y * msg_step + x;
                T* data_cost = (T*)ctemp + y * msg_step + x;

                int nr_local_minimum = 0;

                T prev = data_cost[0 * disp_step];
                T cur  = data_cost[1 * disp_step];
                T next = data_cost[2 * disp_step];

                for (int d = 1; d < ndisp - 1 && nr_local_minimum < nr_plane; d++)
                {
                    if (cur < prev && cur < next)
                    {
                        data_cost_selected[nr_local_minimum * disp_step] = cur;
                        selected_disparity[nr_local_minimum * disp_step] = d;

                        data_cost[d * disp_step] = numeric_limits<T>::max();

                        nr_local_minimum++;
                    }
                    prev = cur;
                    cur = next;
                    next = data_cost[(d + 1) * disp_step];
                }

                for (int i = nr_local_minimum; i < nr_plane; i++)
                {
                    T minimum = numeric_limits<T>::max();
                    int id = 0;

                    for (int d = 0; d < ndisp; d++)
                    {
                        cur = data_cost[d * disp_step];
                        if (cur < minimum)
                        {
                            minimum = cur;
                            id = d;
                        }
                    }
                    data_cost_selected[i * disp_step] = minimum;
                    selected_disparity[i * disp_step] = id;

                    data_cost[id * disp_step] = numeric_limits<T>::max();
                }
            }
        }

        template <typename T, int channels>
        __global__ void init_data_cost(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step,
                                      int h, int w, int level, int ndisp, float data_weight, float max_data_term,
                                      int min_disp, size_t msg_step, size_t disp_step)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                int y0 = y << level;
                int yt = (y + 1) << level;

                int x0 = x << level;
                int xt = (x + 1) << level;

                T* data_cost = (T*)ctemp + y * msg_step + x;

                for(int d = 0; d < ndisp; ++d)
                {
                    float val = 0.0f;
                    for(int yi = y0; yi < yt; yi++)
                    {
                        for(int xi = x0; xi < xt; xi++)
                        {
                            int xr = xi - d;
                            if(d < min_disp || xr < 0)
                                val += data_weight * max_data_term;
                            else
                            {
                                const uchar* lle = cleft + yi * cimg_step + xi * channels;
                                const uchar* lri = cright + yi * cimg_step + xr * channels;

                                val += data_weight * pixeldiff<channels>(lle, lri, max_data_term);
                            }
                        }
                    }
                    data_cost[disp_step * d] = saturate_cast<T>(val);
                }
            }
        }

        template <typename T, int winsz, int channels>
        __global__ void init_data_cost_reduce(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step,
                                              int level, int rows, int cols, int h, int ndisp, float data_weight, float max_data_term,
                                              int min_disp, size_t msg_step, size_t disp_step)
        {
            int x_out = blockIdx.x;
            int y_out = blockIdx.y % h;
            int d = (blockIdx.y / h) * blockDim.z + threadIdx.z;

            int tid = threadIdx.x;

            if (d < ndisp)
            {
                int x0 = x_out << level;
                int y0 = y_out << level;

                int len = ::min(y0 + winsz, rows) - y0;

                float val = 0.0f;
                if (x0 + tid < cols)
                {
                    if (x0 + tid - d < 0 || d < min_disp)
                        val = data_weight * max_data_term * len;
                    else
                    {
                        const uchar* lle =  cleft + y0 * cimg_step + channels * (x0 + tid    );
                        const uchar* lri = cright + y0 * cimg_step + channels * (x0 + tid - d);

                        for(int y = 0; y < len; ++y)
                        {
                            val += data_weight * pixeldiff<channels>(lle, lri, max_data_term);

                            lle += cimg_step;
                            lri += cimg_step;
                        }
                    }
                }

                extern __shared__ float smem[];

                reduce<winsz>(smem + winsz * threadIdx.z, val, tid, plus<float>());

                T* data_cost = (T*)ctemp + y_out * msg_step + x_out;

                if (tid == 0)
                    data_cost[disp_step * d] = saturate_cast<T>(val);
            }
        }


        template <typename T>
        void init_data_cost_caller_(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step, int /*rows*/, int /*cols*/, int h, int w, int level, int ndisp, int channels, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            switch (channels)
            {
            case 1: init_data_cost<T, 1><<<grid, threads, 0, stream>>>(cleft, cright, ctemp, cimg_step, h, w, level, ndisp, data_weight, max_data_term, min_disp, msg_step, disp_step); break;
            case 3: init_data_cost<T, 3><<<grid, threads, 0, stream>>>(cleft, cright, ctemp, cimg_step, h, w, level, ndisp, data_weight, max_data_term, min_disp, msg_step, disp_step); break;
            case 4: init_data_cost<T, 4><<<grid, threads, 0, stream>>>(cleft, cright, ctemp, cimg_step, h, w, level, ndisp, data_weight, max_data_term, min_disp, msg_step, disp_step); break;
            default: CV_Error(cv::Error::BadNumChannels, "Unsupported channels count");
            }
        }

        template <typename T, int winsz>
        void init_data_cost_reduce_caller_(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step, int rows, int cols, int h, int w, int level, int ndisp, int channels, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step, hipStream_t stream)
        {
            const int threadsNum = 256;
            const size_t smem_size = threadsNum * sizeof(float);

            dim3 threads(winsz, 1, threadsNum / winsz);
            dim3 grid(w, h, 1);
            grid.y *= divUp(ndisp, threads.z);

            switch (channels)
            {
            case 1: init_data_cost_reduce<T, winsz, 1><<<grid, threads, smem_size, stream>>>(cleft, cright, ctemp, cimg_step, level, rows, cols, h, ndisp, data_weight, max_data_term, min_disp, msg_step, disp_step); break;
            case 3: init_data_cost_reduce<T, winsz, 3><<<grid, threads, smem_size, stream>>>(cleft, cright, ctemp, cimg_step, level, rows, cols, h, ndisp, data_weight, max_data_term, min_disp, msg_step, disp_step); break;
            case 4: init_data_cost_reduce<T, winsz, 4><<<grid, threads, smem_size, stream>>>(cleft, cright, ctemp, cimg_step, level, rows, cols, h, ndisp, data_weight, max_data_term, min_disp, msg_step, disp_step); break;
            default: CV_Error(cv::Error::BadNumChannels, "Unsupported channels count");
            }
        }

        template<class T>
        void init_data_cost(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step, int rows, int cols, T* disp_selected_pyr, T* data_cost_selected, size_t msg_step,
                    int h, int w, int level, int nr_plane, int ndisp, int channels, float data_weight, float max_data_term, int min_disp, bool use_local_init_data_cost, hipStream_t stream)
        {

            typedef void (*InitDataCostCaller)(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step, int cols, int rows, int w, int h, int level, int ndisp, int channels, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step, hipStream_t stream);

            static const InitDataCostCaller init_data_cost_callers[] =
            {
                init_data_cost_caller_<T>, init_data_cost_caller_<T>, init_data_cost_reduce_caller_<T, 4>,
                init_data_cost_reduce_caller_<T, 8>, init_data_cost_reduce_caller_<T, 16>, init_data_cost_reduce_caller_<T, 32>,
                init_data_cost_reduce_caller_<T, 64>, init_data_cost_reduce_caller_<T, 128>, init_data_cost_reduce_caller_<T, 256>
            };

            size_t disp_step = msg_step * h;

            init_data_cost_callers[level](cleft, cright, ctemp, cimg_step, rows, cols, h, w, level, ndisp, channels, data_weight, max_data_term, min_disp, msg_step, disp_step, stream);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            if (use_local_init_data_cost == true)
                get_first_k_initial_local<<<grid, threads, 0, stream>>> (ctemp, data_cost_selected, disp_selected_pyr, h, w, nr_plane, ndisp, msg_step, disp_step);
            else
                get_first_k_initial_global<<<grid, threads, 0, stream>>>(ctemp, data_cost_selected, disp_selected_pyr, h, w, nr_plane, ndisp, msg_step, disp_step);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void init_data_cost<short>(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step, int rows, int cols, short* disp_selected_pyr, short* data_cost_selected, size_t msg_step,
                    int h, int w, int level, int nr_plane, int ndisp, int channels, float data_weight, float max_data_term, int min_disp, bool use_local_init_data_cost, hipStream_t stream);

        template void init_data_cost<float>(const uchar *cleft, const uchar *cright, uchar *ctemp, size_t cimg_step, int rows, int cols, float* disp_selected_pyr, float* data_cost_selected, size_t msg_step,
                    int h, int w, int level, int nr_plane, int ndisp, int channels, float data_weight, float max_data_term, int min_disp, bool use_local_init_data_cost, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        ////////////////////// compute data cost //////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T, int channels>
        __global__ void compute_data_cost(const uchar *cleft, const uchar *cright, size_t cimg_step, const T* selected_disp_pyr, T* data_cost_, int h, int w, int level, int nr_plane, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step1, size_t disp_step2)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                int y0 = y << level;
                int yt = (y + 1) << level;

                int x0 = x << level;
                int xt = (x + 1) << level;

                const T* selected_disparity = selected_disp_pyr + y/2 * msg_step + x/2;
                T* data_cost = data_cost_ + y * msg_step + x;

                for(int d = 0; d < nr_plane; d++)
                {
                    float val = 0.0f;
                    for(int yi = y0; yi < yt; yi++)
                    {
                        for(int xi = x0; xi < xt; xi++)
                        {
                            int sel_disp = selected_disparity[d * disp_step2];
                            int xr = xi - sel_disp;

                            if (xr < 0 || sel_disp < min_disp)
                                val += data_weight * max_data_term;
                            else
                            {
                                const uchar* left_x = cleft + yi * cimg_step + xi * channels;
                                const uchar* right_x = cright + yi * cimg_step + xr * channels;

                                val += data_weight * pixeldiff<channels>(left_x, right_x, max_data_term);
                            }
                        }
                    }
                    data_cost[disp_step1 * d] = saturate_cast<T>(val);
                }
            }
        }

        template <typename T, int winsz, int channels>
        __global__ void compute_data_cost_reduce(const uchar *cleft, const uchar *cright, size_t cimg_step, const T* selected_disp_pyr, T* data_cost_, int level, int rows, int cols, int h, int nr_plane, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step1, size_t disp_step2)
        {
            int x_out = blockIdx.x;
            int y_out = blockIdx.y % h;
            int d = (blockIdx.y / h) * blockDim.z + threadIdx.z;

            int tid = threadIdx.x;

            const T* selected_disparity = selected_disp_pyr + y_out/2 * msg_step + x_out/2;
            T* data_cost = data_cost_ + y_out * msg_step + x_out;

            if (d < nr_plane)
            {
                int sel_disp = selected_disparity[d * disp_step2];

                int x0 = x_out << level;
                int y0 = y_out << level;

                int len = ::min(y0 + winsz, rows) - y0;

                float val = 0.0f;
                if (x0 + tid < cols)
                {
                    if (x0 + tid - sel_disp < 0 || sel_disp < min_disp)
                        val = data_weight * max_data_term * len;
                    else
                    {
                        const uchar* lle =  cleft + y0 * cimg_step + channels * (x0 + tid    );
                        const uchar* lri = cright + y0 * cimg_step + channels * (x0 + tid - sel_disp);

                        for(int y = 0; y < len; ++y)
                        {
                            val += data_weight * pixeldiff<channels>(lle, lri, max_data_term);

                            lle += cimg_step;
                            lri += cimg_step;
                        }
                    }
                }

                extern __shared__ float smem[];

                reduce<winsz>(smem + winsz * threadIdx.z, val, tid, plus<float>());

                if (tid == 0)
                    data_cost[disp_step1 * d] = saturate_cast<T>(val);
            }
        }

        template <typename T>
        void compute_data_cost_caller_(const uchar *cleft, const uchar *cright, size_t cimg_step, const T* disp_selected_pyr, T* data_cost, int /*rows*/, int /*cols*/,
                                      int h, int w, int level, int nr_plane, int channels, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step1, size_t disp_step2, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            switch(channels)
            {
            case 1: compute_data_cost<T, 1><<<grid, threads, 0, stream>>>(cleft, cright, cimg_step, disp_selected_pyr, data_cost, h, w, level, nr_plane, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2); break;
            case 3: compute_data_cost<T, 3><<<grid, threads, 0, stream>>>(cleft, cright, cimg_step, disp_selected_pyr, data_cost, h, w, level, nr_plane, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2); break;
            case 4: compute_data_cost<T, 4><<<grid, threads, 0, stream>>>(cleft, cright, cimg_step, disp_selected_pyr, data_cost, h, w, level, nr_plane, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2); break;
            default: CV_Error(cv::Error::BadNumChannels, "Unsupported channels count");
            }
        }

        template <typename T, int winsz>
        void compute_data_cost_reduce_caller_(const uchar *cleft, const uchar *cright, size_t cimg_step, const T* disp_selected_pyr, T* data_cost, int rows, int cols,
                                      int h, int w, int level, int nr_plane, int channels, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step1, size_t disp_step2, hipStream_t stream)
        {
            const int threadsNum = 256;
            const size_t smem_size = threadsNum * sizeof(float);

            dim3 threads(winsz, 1, threadsNum / winsz);
            dim3 grid(w, h, 1);
            grid.y *= divUp(nr_plane, threads.z);

            switch (channels)
            {
            case 1: compute_data_cost_reduce<T, winsz, 1><<<grid, threads, smem_size, stream>>>(cleft, cright, cimg_step, disp_selected_pyr, data_cost, level, rows, cols, h, nr_plane, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2); break;
            case 3: compute_data_cost_reduce<T, winsz, 3><<<grid, threads, smem_size, stream>>>(cleft, cright, cimg_step, disp_selected_pyr, data_cost, level, rows, cols, h, nr_plane, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2); break;
            case 4: compute_data_cost_reduce<T, winsz, 4><<<grid, threads, smem_size, stream>>>(cleft, cright, cimg_step, disp_selected_pyr, data_cost, level, rows, cols, h, nr_plane, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2); break;
            default: CV_Error(cv::Error::BadNumChannels, "Unsupported channels count");
            }
        }

        template<class T>
        void compute_data_cost(const uchar *cleft, const uchar *cright, size_t cimg_step, const T* disp_selected_pyr, T* data_cost, size_t msg_step,
                               int rows, int cols, int h, int w, int h2, int level, int nr_plane, int channels, float data_weight, float max_data_term,
                               int min_disp, hipStream_t stream)
        {
            typedef void (*ComputeDataCostCaller)(const uchar *cleft, const uchar *cright, size_t cimg_step, const T* disp_selected_pyr, T* data_cost, int rows, int cols,
                int h, int w, int level, int nr_plane, int channels, float data_weight, float max_data_term, int min_disp, size_t msg_step, size_t disp_step1, size_t disp_step2, hipStream_t stream);

            static const ComputeDataCostCaller callers[] =
            {
                compute_data_cost_caller_<T>, compute_data_cost_caller_<T>, compute_data_cost_reduce_caller_<T, 4>,
                compute_data_cost_reduce_caller_<T, 8>, compute_data_cost_reduce_caller_<T, 16>, compute_data_cost_reduce_caller_<T, 32>,
                compute_data_cost_reduce_caller_<T, 64>, compute_data_cost_reduce_caller_<T, 128>, compute_data_cost_reduce_caller_<T, 256>
            };

            size_t disp_step1 = msg_step * h;
            size_t disp_step2 = msg_step * h2;

            callers[level](cleft, cright, cimg_step, disp_selected_pyr, data_cost, rows, cols, h, w, level, nr_plane, channels, data_weight, max_data_term, min_disp, msg_step, disp_step1, disp_step2, stream);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void compute_data_cost(const uchar *cleft, const uchar *cright, size_t cimg_step, const short* disp_selected_pyr, short* data_cost, size_t msg_step,
                               int rows, int cols, int h, int w, int h2, int level, int nr_plane, int channels, float data_weight, float max_data_term, int min_disp, hipStream_t stream);

        template void compute_data_cost(const uchar *cleft, const uchar *cright, size_t cimg_step, const float* disp_selected_pyr, float* data_cost, size_t msg_step,
                               int rows, int cols, int h, int w, int h2, int level, int nr_plane, int channels, float data_weight, float max_data_term, int min_disp, hipStream_t stream);


        ///////////////////////////////////////////////////////////////
        //////////////////////// init message /////////////////////////
        ///////////////////////////////////////////////////////////////


         template <typename T>
        __device__ void get_first_k_element_increase(T* u_new, T* d_new, T* l_new, T* r_new,
                                                     const T* u_cur, const T* d_cur, const T* l_cur, const T* r_cur,
                                                     T* data_cost_selected, T* disparity_selected_new, T* data_cost_new,
                                                     const T* data_cost_cur, const T* disparity_selected_cur,
                                                     int nr_plane, int nr_plane2, size_t disp_step1, size_t disp_step2)
        {
            for(int i = 0; i < nr_plane; i++)
            {
                T minimum = numeric_limits<T>::max();
                int id = 0;
                for(int j = 0; j < nr_plane2; j++)
                {
                    T cur = data_cost_new[j * disp_step1];
                    if(cur < minimum)
                    {
                        minimum = cur;
                        id = j;
                    }
                }

                data_cost_selected[i * disp_step1] = data_cost_cur[id * disp_step1];
                disparity_selected_new[i * disp_step1] = disparity_selected_cur[id * disp_step2];

                u_new[i * disp_step1] = u_cur[id * disp_step2];
                d_new[i * disp_step1] = d_cur[id * disp_step2];
                l_new[i * disp_step1] = l_cur[id * disp_step2];
                r_new[i * disp_step1] = r_cur[id * disp_step2];

                data_cost_new[id * disp_step1] = numeric_limits<T>::max();
            }
        }

        template <typename T>
        __global__ void init_message(uchar *ctemp, T* u_new_, T* d_new_, T* l_new_, T* r_new_,
                                     const T* u_cur_, const T* d_cur_, const T* l_cur_, const T* r_cur_,
                                     T* selected_disp_pyr_new, const T* selected_disp_pyr_cur,
                                     T* data_cost_selected_, const T* data_cost_,
                                     int h, int w, int nr_plane, int h2, int w2, int nr_plane2,
                                     size_t msg_step, size_t disp_step1, size_t disp_step2)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                const T* u_cur = u_cur_ + ::min(h2-1, y/2 + 1) * msg_step + x/2;
                const T* d_cur = d_cur_ + ::max(0, y/2 - 1)    * msg_step + x/2;
                const T* l_cur = l_cur_ + (y/2)                * msg_step + ::min(w2-1, x/2 + 1);
                const T* r_cur = r_cur_ + (y/2)                * msg_step + ::max(0, x/2 - 1);

                T* data_cost_new = (T*)ctemp + y * msg_step + x;

                const T* disparity_selected_cur = selected_disp_pyr_cur + y/2 * msg_step + x/2;
                const T* data_cost = data_cost_ + y * msg_step + x;

                for(int d = 0; d < nr_plane2; d++)
                {
                    int idx2 = d * disp_step2;

                    T val  = data_cost[d * disp_step1] + u_cur[idx2] + d_cur[idx2] + l_cur[idx2] + r_cur[idx2];
                    data_cost_new[d * disp_step1] = val;
                }

                T* data_cost_selected = data_cost_selected_ + y * msg_step + x;
                T* disparity_selected_new = selected_disp_pyr_new + y * msg_step + x;

                T* u_new = u_new_ + y * msg_step + x;
                T* d_new = d_new_ + y * msg_step + x;
                T* l_new = l_new_ + y * msg_step + x;
                T* r_new = r_new_ + y * msg_step + x;

                u_cur = u_cur_ + y/2 * msg_step + x/2;
                d_cur = d_cur_ + y/2 * msg_step + x/2;
                l_cur = l_cur_ + y/2 * msg_step + x/2;
                r_cur = r_cur_ + y/2 * msg_step + x/2;

                get_first_k_element_increase(u_new, d_new, l_new, r_new, u_cur, d_cur, l_cur, r_cur,
                                             data_cost_selected, disparity_selected_new, data_cost_new,
                                             data_cost, disparity_selected_cur, nr_plane, nr_plane2,
                                             disp_step1, disp_step2);
            }
        }


        template<class T>
        void init_message(uchar *ctemp, T* u_new, T* d_new, T* l_new, T* r_new,
                          const T* u_cur, const T* d_cur, const T* l_cur, const T* r_cur,
                          T* selected_disp_pyr_new, const T* selected_disp_pyr_cur,
                          T* data_cost_selected, const T* data_cost, size_t msg_step,
                          int h, int w, int nr_plane, int h2, int w2, int nr_plane2, hipStream_t stream)
        {

            size_t disp_step1 = msg_step * h;
            size_t disp_step2 = msg_step * h2;

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            init_message<<<grid, threads, 0, stream>>>(ctemp, u_new, d_new, l_new, r_new,
                                                       u_cur, d_cur, l_cur, r_cur,
                                                       selected_disp_pyr_new, selected_disp_pyr_cur,
                                                       data_cost_selected, data_cost,
                                                       h, w, nr_plane, h2, w2, nr_plane2,
                                                       msg_step, disp_step1, disp_step2);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }


        template void init_message(uchar *ctemp, short* u_new, short* d_new, short* l_new, short* r_new,
                          const short* u_cur, const short* d_cur, const short* l_cur, const short* r_cur,
                          short* selected_disp_pyr_new, const short* selected_disp_pyr_cur,
                          short* data_cost_selected, const short* data_cost, size_t msg_step,
                          int h, int w, int nr_plane, int h2, int w2, int nr_plane2, hipStream_t stream);

        template void init_message(uchar *ctemp, float* u_new, float* d_new, float* l_new, float* r_new,
                          const float* u_cur, const float* d_cur, const float* l_cur, const float* r_cur,
                          float* selected_disp_pyr_new, const float* selected_disp_pyr_cur,
                          float* data_cost_selected, const float* data_cost, size_t msg_step,
                          int h, int w, int nr_plane, int h2, int w2, int nr_plane2, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        ////////////////////  calc all iterations /////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __device__ void message_per_pixel(const T* data, T* msg_dst, const T* msg1, const T* msg2, const T* msg3,
                                          const T* dst_disp, const T* src_disp, int nr_plane, int max_disc_term, float disc_single_jump, volatile T* temp,
                                          size_t disp_step)
        {
            T minimum = numeric_limits<T>::max();

            for(int d = 0; d < nr_plane; d++)
            {
                int idx = d * disp_step;
                T val  = data[idx] + msg1[idx] + msg2[idx] + msg3[idx];

                if(val < minimum)
                    minimum = val;

                msg_dst[idx] = val;
            }

            float sum = 0;
            for(int d = 0; d < nr_plane; d++)
            {
                float cost_min = minimum + max_disc_term;
                T src_disp_reg = src_disp[d * disp_step];

                for(int d2 = 0; d2 < nr_plane; d2++)
                    cost_min = fmin(cost_min, msg_dst[d2 * disp_step] + disc_single_jump * ::abs(dst_disp[d2 * disp_step] - src_disp_reg));

                temp[d * disp_step] = saturate_cast<T>(cost_min);
                sum += cost_min;
            }
            sum /= nr_plane;

            for(int d = 0; d < nr_plane; d++)
                msg_dst[d * disp_step] = saturate_cast<T>(temp[d * disp_step] - sum);
        }

        template <typename T>
        __global__ void compute_message(uchar *ctemp, T* u_, T* d_, T* l_, T* r_, const T* data_cost_selected, const T* selected_disp_pyr_cur, int h, int w, int nr_plane, int i, int max_disc_term, float disc_single_jump, size_t msg_step, size_t disp_step)
        {
            int y = blockIdx.y * blockDim.y + threadIdx.y;
            int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + i) & 1);

            if (y > 0 && y < h - 1 && x > 0 && x < w - 1)
            {
                const T* data = data_cost_selected + y * msg_step + x;

                T* u = u_ + y * msg_step + x;
                T* d = d_ + y * msg_step + x;
                T* l = l_ + y * msg_step + x;
                T* r = r_ + y * msg_step + x;

                const T* disp = selected_disp_pyr_cur + y * msg_step + x;

                T* temp = (T*)ctemp + y * msg_step + x;

                message_per_pixel(data, u, r - 1, u + msg_step, l + 1, disp, disp - msg_step, nr_plane, max_disc_term, disc_single_jump, temp, disp_step);
                message_per_pixel(data, d, d - msg_step, r - 1, l + 1, disp, disp + msg_step, nr_plane, max_disc_term, disc_single_jump, temp, disp_step);
                message_per_pixel(data, l, u + msg_step, d - msg_step, l + 1, disp, disp - 1, nr_plane, max_disc_term, disc_single_jump, temp, disp_step);
                message_per_pixel(data, r, u + msg_step, d - msg_step, r - 1, disp, disp + 1, nr_plane, max_disc_term, disc_single_jump, temp, disp_step);
            }
        }


        template<class T>
        void calc_all_iterations(uchar *ctemp, T* u, T* d, T* l, T* r, const T* data_cost_selected,
            const T* selected_disp_pyr_cur, size_t msg_step, int h, int w, int nr_plane, int iters, int max_disc_term, float disc_single_jump, hipStream_t stream)
        {
            size_t disp_step = msg_step * h;

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x << 1);
            grid.y = divUp(h, threads.y);

            for(int t = 0; t < iters; ++t)
            {
                compute_message<<<grid, threads, 0, stream>>>(ctemp, u, d, l, r, data_cost_selected, selected_disp_pyr_cur, h, w, nr_plane, t & 1, max_disc_term, disc_single_jump, msg_step, disp_step);
                cudaSafeCall( hipGetLastError() );
            }
            if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
        };

        template void calc_all_iterations(uchar *ctemp, short* u, short* d, short* l, short* r, const short* data_cost_selected, const short* selected_disp_pyr_cur, size_t msg_step,
            int h, int w, int nr_plane, int iters, int max_disc_term, float disc_single_jump, hipStream_t stream);

        template void calc_all_iterations(uchar *ctemp, float* u, float* d, float* l, float* r, const float* data_cost_selected, const float* selected_disp_pyr_cur, size_t msg_step,
            int h, int w, int nr_plane, int iters, int max_disc_term, float disc_single_jump, hipStream_t stream);


        ///////////////////////////////////////////////////////////////
        /////////////////////////// output ////////////////////////////
        ///////////////////////////////////////////////////////////////


        template <typename T>
        __global__ void compute_disp(const T* u_, const T* d_, const T* l_, const T* r_,
                                     const T* data_cost_selected, const T* disp_selected_pyr,
                                     PtrStepSz<short> disp, int nr_plane, size_t msg_step, size_t disp_step)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y > 0 && y < disp.rows - 1 && x > 0 && x < disp.cols - 1)
            {
                const T* data = data_cost_selected + y * msg_step + x;
                const T* disp_selected = disp_selected_pyr + y * msg_step + x;

                const T* u = u_ + (y+1) * msg_step + (x+0);
                const T* d = d_ + (y-1) * msg_step + (x+0);
                const T* l = l_ + (y+0) * msg_step + (x+1);
                const T* r = r_ + (y+0) * msg_step + (x-1);

                int best = 0;
                T best_val = numeric_limits<T>::max();
                for (int i = 0; i < nr_plane; ++i)
                {
                    int idx = i * disp_step;
                    T val = data[idx]+ u[idx] + d[idx] + l[idx] + r[idx];

                    if (val < best_val)
                    {
                        best_val = val;
                        best = saturate_cast<short>(disp_selected[idx]);
                    }
                }
                disp(y, x) = best;
            }
        }

        template<class T>
        void compute_disp(const T* u, const T* d, const T* l, const T* r, const T* data_cost_selected, const T* disp_selected, size_t msg_step,
            const PtrStepSz<short>& disp, int nr_plane, hipStream_t stream)
        {
            size_t disp_step = disp.rows * msg_step;

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(disp.cols, threads.x);
            grid.y = divUp(disp.rows, threads.y);

            compute_disp<<<grid, threads, 0, stream>>>(u, d, l, r, data_cost_selected, disp_selected, disp, nr_plane, msg_step, disp_step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void compute_disp(const short* u, const short* d, const short* l, const short* r, const short* data_cost_selected, const short* disp_selected, size_t msg_step,
            const PtrStepSz<short>& disp, int nr_plane, hipStream_t stream);

        template void compute_disp(const float* u, const float* d, const float* l, const float* r, const float* data_cost_selected, const float* disp_selected, size_t msg_step,
            const PtrStepSz<short>& disp, int nr_plane, hipStream_t stream);
    } // namespace stereocsbp
}}} // namespace cv { namespace cuda { namespace cudev {

#endif /* CUDA_DISABLER */
