#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace stereobm
    {
        //////////////////////////////////////////////////////////////////////////////////////////////////
        /////////////////////////////////////// Stereo BM ////////////////////////////////////////////////
        //////////////////////////////////////////////////////////////////////////////////////////////////

        #define ROWSperTHREAD 21     // the number of rows a thread will process

        #define BLOCK_W 128          // the thread block width (464)
        #define N_DISPARITIES 8

        #define STEREO_MIND 0                    // The minimum d range to check
        #define STEREO_DISP_STEP N_DISPARITIES   // the d step, must be <= 1 to avoid aliasing

        __constant__ unsigned int* cminSSDImage;
        __constant__ size_t cminSSD_step;
        __constant__ int cwidth;
        __constant__ int cheight;

        __device__ __forceinline__ int SQ(int a)
        {
            return a * a;
        }

        template<int RADIUS>
        __device__ unsigned int CalcSSD(volatile unsigned int *col_ssd_cache, volatile unsigned int *col_ssd)
        {
            unsigned int cache = 0;
            unsigned int cache2 = 0;

            for(int i = 1; i <= RADIUS; i++)
                cache += col_ssd[i];

            col_ssd_cache[0] = cache;

            __syncthreads();

            if (threadIdx.x < BLOCK_W - RADIUS)
                cache2 = col_ssd_cache[RADIUS];
            else
                for(int i = RADIUS + 1; i < (2 * RADIUS + 1); i++)
                    cache2 += col_ssd[i];

            return col_ssd[0] + cache + cache2;
        }

        template<int RADIUS>
        __device__ uint2 MinSSD(volatile unsigned int *col_ssd_cache, volatile unsigned int *col_ssd)
        {
            unsigned int ssd[N_DISPARITIES];

            //See above:  #define COL_SSD_SIZE (BLOCK_W + 2 * RADIUS)
            ssd[0] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 0 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[1] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 1 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[2] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 2 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[3] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 3 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[4] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 4 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[5] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 5 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[6] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 6 * (BLOCK_W + 2 * RADIUS));
            __syncthreads();
            ssd[7] = CalcSSD<RADIUS>(col_ssd_cache, col_ssd + 7 * (BLOCK_W + 2 * RADIUS));

            int mssd = ::min(::min(::min(ssd[0], ssd[1]), ::min(ssd[4], ssd[5])), ::min(::min(ssd[2], ssd[3]), ::min(ssd[6], ssd[7])));

            int bestIdx = 0;
            for (int i = 0; i < N_DISPARITIES; i++)
            {
                if (mssd == ssd[i])
                    bestIdx = i;
            }

            return make_uint2(mssd, bestIdx);
        }

        template<int RADIUS>
        __device__ void StepDown(int idx1, int idx2, unsigned char* imageL, unsigned char* imageR, int d, volatile unsigned int *col_ssd)
        {
            unsigned char leftPixel1;
            unsigned char leftPixel2;
            unsigned char rightPixel1[8];
            unsigned char rightPixel2[8];
            unsigned int diff1, diff2;

            leftPixel1 = imageL[idx1];
            leftPixel2 = imageL[idx2];

            idx1 = idx1 - d;
            idx2 = idx2 - d;

            rightPixel1[7] = imageR[idx1 - 7];
            rightPixel1[0] = imageR[idx1 - 0];
            rightPixel1[1] = imageR[idx1 - 1];
            rightPixel1[2] = imageR[idx1 - 2];
            rightPixel1[3] = imageR[idx1 - 3];
            rightPixel1[4] = imageR[idx1 - 4];
            rightPixel1[5] = imageR[idx1 - 5];
            rightPixel1[6] = imageR[idx1 - 6];

            rightPixel2[7] = imageR[idx2 - 7];
            rightPixel2[0] = imageR[idx2 - 0];
            rightPixel2[1] = imageR[idx2 - 1];
            rightPixel2[2] = imageR[idx2 - 2];
            rightPixel2[3] = imageR[idx2 - 3];
            rightPixel2[4] = imageR[idx2 - 4];
            rightPixel2[5] = imageR[idx2 - 5];
            rightPixel2[6] = imageR[idx2 - 6];

            //See above:  #define COL_SSD_SIZE (BLOCK_W + 2 * RADIUS)
            diff1 = leftPixel1 - rightPixel1[0];
            diff2 = leftPixel2 - rightPixel2[0];
            col_ssd[0 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[1];
            diff2 = leftPixel2 - rightPixel2[1];
            col_ssd[1 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[2];
            diff2 = leftPixel2 - rightPixel2[2];
            col_ssd[2 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[3];
            diff2 = leftPixel2 - rightPixel2[3];
            col_ssd[3 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[4];
            diff2 = leftPixel2 - rightPixel2[4];
            col_ssd[4 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[5];
            diff2 = leftPixel2 - rightPixel2[5];
            col_ssd[5 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[6];
            diff2 = leftPixel2 - rightPixel2[6];
            col_ssd[6 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);

            diff1 = leftPixel1 - rightPixel1[7];
            diff2 = leftPixel2 - rightPixel2[7];
            col_ssd[7 * (BLOCK_W + 2 * RADIUS)] += SQ(diff2) - SQ(diff1);
        }

        template<int RADIUS>
        __device__ void InitColSSD(int x_tex, int y_tex, int im_pitch, unsigned char* imageL, unsigned char* imageR, int d, volatile unsigned int *col_ssd)
        {
            unsigned char leftPixel1;
            int idx;
            unsigned int diffa[] = {0, 0, 0, 0, 0, 0, 0, 0};

            for(int i = 0; i < (2 * RADIUS + 1); i++)
            {
                idx = y_tex * im_pitch + x_tex;
                leftPixel1 = imageL[idx];
                idx = idx - d;

                diffa[0] += SQ(leftPixel1 - imageR[idx - 0]);
                diffa[1] += SQ(leftPixel1 - imageR[idx - 1]);
                diffa[2] += SQ(leftPixel1 - imageR[idx - 2]);
                diffa[3] += SQ(leftPixel1 - imageR[idx - 3]);
                diffa[4] += SQ(leftPixel1 - imageR[idx - 4]);
                diffa[5] += SQ(leftPixel1 - imageR[idx - 5]);
                diffa[6] += SQ(leftPixel1 - imageR[idx - 6]);
                diffa[7] += SQ(leftPixel1 - imageR[idx - 7]);

                y_tex += 1;
            }
            //See above:  #define COL_SSD_SIZE (BLOCK_W + 2 * RADIUS)
            col_ssd[0 * (BLOCK_W + 2 * RADIUS)] = diffa[0];
            col_ssd[1 * (BLOCK_W + 2 * RADIUS)] = diffa[1];
            col_ssd[2 * (BLOCK_W + 2 * RADIUS)] = diffa[2];
            col_ssd[3 * (BLOCK_W + 2 * RADIUS)] = diffa[3];
            col_ssd[4 * (BLOCK_W + 2 * RADIUS)] = diffa[4];
            col_ssd[5 * (BLOCK_W + 2 * RADIUS)] = diffa[5];
            col_ssd[6 * (BLOCK_W + 2 * RADIUS)] = diffa[6];
            col_ssd[7 * (BLOCK_W + 2 * RADIUS)] = diffa[7];
        }

        template<int RADIUS>
        __global__ void stereoKernel(unsigned char *left, unsigned char *right, size_t img_step, PtrStepb disp, int maxdisp)
        {
            extern __shared__ unsigned int col_ssd_cache[];
            volatile unsigned int *col_ssd = col_ssd_cache + BLOCK_W + threadIdx.x;
            volatile unsigned int *col_ssd_extra = threadIdx.x < (2 * RADIUS) ? col_ssd + BLOCK_W : 0;  //#define N_DIRTY_PIXELS (2 * RADIUS)

            //#define X (blockIdx.x * BLOCK_W + threadIdx.x + STEREO_MAXD)
            int X = (blockIdx.x * BLOCK_W + threadIdx.x + maxdisp + RADIUS);
            //#define Y (__mul24(blockIdx.y, ROWSperTHREAD) + RADIUS)
            #define Y (blockIdx.y * ROWSperTHREAD + RADIUS)
            //int Y = blockIdx.y * ROWSperTHREAD + RADIUS;

            unsigned int* minSSDImage = cminSSDImage + X + Y * cminSSD_step;
            unsigned char* disparImage = disp.data + X + Y * disp.step;
         /*   if (X < cwidth)
            {
                unsigned int *minSSDImage_end = minSSDImage + min(ROWSperTHREAD, cheight - Y) * minssd_step;
                for(uint *ptr = minSSDImage; ptr != minSSDImage_end; ptr += minssd_step )
                    *ptr = 0xFFFFFFFF;
            }*/
            int end_row = ::min(ROWSperTHREAD, cheight - Y - RADIUS);
            int y_tex;
            int x_tex = X - RADIUS;

            if (x_tex >= cwidth)
                return;

            for(int d = STEREO_MIND; d < maxdisp; d += STEREO_DISP_STEP)
            {
                y_tex = Y - RADIUS;

                InitColSSD<RADIUS>(x_tex, y_tex, img_step, left, right, d, col_ssd);

                if (col_ssd_extra > 0)
                    if (x_tex + BLOCK_W < cwidth)
                        InitColSSD<RADIUS>(x_tex + BLOCK_W, y_tex, img_step, left, right, d, col_ssd_extra);

                __syncthreads(); //before MinSSD function

                if (X < cwidth - RADIUS && Y < cheight - RADIUS)
                {
                    uint2 minSSD = MinSSD<RADIUS>(col_ssd_cache + threadIdx.x, col_ssd);
                    if (minSSD.x < minSSDImage[0])
                    {
                        disparImage[0] = (unsigned char)(d + minSSD.y);
                        minSSDImage[0] = minSSD.x;
                    }
                }

                for(int row = 1; row < end_row; row++)
                {
                    int idx1 = y_tex * img_step + x_tex;
                    int idx2 = (y_tex + (2 * RADIUS + 1)) * img_step + x_tex;

                    __syncthreads();

                    StepDown<RADIUS>(idx1, idx2, left, right, d, col_ssd);

                    if (col_ssd_extra)
                        if (x_tex + BLOCK_W < cwidth)
                            StepDown<RADIUS>(idx1, idx2, left + BLOCK_W, right + BLOCK_W, d, col_ssd_extra);

                    y_tex += 1;

                    __syncthreads(); //before MinSSD function

                    if (X < cwidth - RADIUS && row < cheight - RADIUS - Y)
                    {
                        int idx = row * cminSSD_step;
                        uint2 minSSD = MinSSD<RADIUS>(col_ssd_cache + threadIdx.x, col_ssd);
                        if (minSSD.x < minSSDImage[idx])
                        {
                            disparImage[disp.step * row] = (unsigned char)(d + minSSD.y);
                            minSSDImage[idx] = minSSD.x;
                        }
                    }
                } // for row loop
            } // for d loop
        }


        template<int RADIUS> void kernel_caller(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& disp, int maxdisp, hipStream_t & stream)
        {
            dim3 grid(1,1,1);
            dim3 threads(BLOCK_W, 1, 1);

            grid.x = divUp(left.cols - maxdisp - 2 * RADIUS, BLOCK_W);
            grid.y = divUp(left.rows - 2 * RADIUS, ROWSperTHREAD);

            //See above:  #define COL_SSD_SIZE (BLOCK_W + 2 * RADIUS)
            size_t smem_size = (BLOCK_W + N_DISPARITIES * (BLOCK_W + 2 * RADIUS)) * sizeof(unsigned int);

            stereoKernel<RADIUS><<<grid, threads, smem_size, stream>>>(left.data, right.data, left.step, disp, maxdisp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        };

        typedef void (*kernel_caller_t)(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& disp, int maxdisp, hipStream_t & stream);

        const static kernel_caller_t callers[] =
        {
            0,
            kernel_caller< 1>, kernel_caller< 2>, kernel_caller< 3>, kernel_caller< 4>, kernel_caller< 5>,
            kernel_caller< 6>, kernel_caller< 7>, kernel_caller< 8>, kernel_caller< 9>, kernel_caller<10>,
            kernel_caller<11>, kernel_caller<12>, kernel_caller<13>, kernel_caller<14>, kernel_caller<15>,
            kernel_caller<16>, kernel_caller<17>, kernel_caller<18>, kernel_caller<19>, kernel_caller<20>,
            kernel_caller<21>, kernel_caller<22>, kernel_caller<23>, kernel_caller<24>, kernel_caller<25>

            //0,0,0, 0,0,0, 0,0,kernel_caller<9>
        };
        const int calles_num = sizeof(callers)/sizeof(callers[0]);

        void stereoBM_CUDA(const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& disp, int maxdisp, int winsz, const PtrStepSz<unsigned int>& minSSD_buf, hipStream_t& stream)
        {
            int winsz2 = winsz >> 1;

            if (winsz2 == 0 || winsz2 >= calles_num)
                CV_Error(cv::Error::StsBadArg, "Unsupported window size");

            //cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(&stereoKernel), hipFuncCachePreferL1) );
            //cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(&stereoKernel), hipFuncCachePreferShared) );

            cudaSafeCall( hipMemset2D(disp.data, disp.step, 0, disp.cols, disp.rows) );
            cudaSafeCall( hipMemset2D(minSSD_buf.data, minSSD_buf.step, 0xFF, minSSD_buf.cols * minSSD_buf.elemSize(), disp.rows) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cwidth), &left.cols, sizeof(left.cols) ) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cheight), &left.rows, sizeof(left.rows) ) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cminSSDImage), &minSSD_buf.data, sizeof(minSSD_buf.data) ) );

            size_t minssd_step = minSSD_buf.step/minSSD_buf.elemSize();
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cminSSD_step),  &minssd_step, sizeof(minssd_step) ) );

            callers[winsz2](left, right, disp, maxdisp, stream);
        }

        //////////////////////////////////////////////////////////////////////////////////////////////////
        /////////////////////////////////////// Sobel Prefiler ///////////////////////////////////////////
        //////////////////////////////////////////////////////////////////////////////////////////////////

        texture<unsigned char, 2, hipReadModeElementType> texForSobel;

        __global__ void prefilter_kernel(PtrStepSzb output, int prefilterCap)
        {
            int x = blockDim.x * blockIdx.x + threadIdx.x;
            int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < output.cols && y < output.rows)
            {
                int conv = (int)tex2D(texForSobel, x - 1, y - 1) * (-1) + (int)tex2D(texForSobel, x + 1, y - 1) * (1) +
                           (int)tex2D(texForSobel, x - 1, y    ) * (-2) + (int)tex2D(texForSobel, x + 1, y    ) * (2) +
                           (int)tex2D(texForSobel, x - 1, y + 1) * (-1) + (int)tex2D(texForSobel, x + 1, y + 1) * (1);


                conv = ::min(::min(::max(-prefilterCap, conv), prefilterCap) + prefilterCap, 255);
                output.ptr(y)[x] = conv & 0xFF;
            }
        }

        void prefilter_xsobel(const PtrStepSzb& input, const PtrStepSzb& output, int prefilterCap, hipStream_t & stream)
        {
            hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
            cudaSafeCall( hipBindTexture2D( 0, texForSobel, input.data, desc, input.cols, input.rows, input.step ) );

            dim3 threads(16, 16, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(input.cols, threads.x);
            grid.y = divUp(input.rows, threads.y);

            prefilter_kernel<<<grid, threads, 0, stream>>>(output, prefilterCap);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            cudaSafeCall( hipUnbindTexture (texForSobel ) );
        }


        //////////////////////////////////////////////////////////////////////////////////////////////////
        /////////////////////////////////// Textureness filtering ////////////////////////////////////////
        //////////////////////////////////////////////////////////////////////////////////////////////////

        texture<unsigned char, 2, hipReadModeNormalizedFloat> texForTF;

        __device__ __forceinline__ float sobel(int x, int y)
        {
            float conv = tex2D(texForTF, x - 1, y - 1) * (-1) + tex2D(texForTF, x + 1, y - 1) * (1) +
                         tex2D(texForTF, x - 1, y    ) * (-2) + tex2D(texForTF, x + 1, y    ) * (2) +
                         tex2D(texForTF, x - 1, y + 1) * (-1) + tex2D(texForTF, x + 1, y + 1) * (1);
            return fabs(conv);
        }

        __device__ float CalcSums(float *cols, float *cols_cache, int winsz)
        {
            float cache = 0;
            float cache2 = 0;
            int winsz2 = winsz/2;

            for(int i = 1; i <= winsz2; i++)
                cache += cols[i];

            cols_cache[0] = cache;

            __syncthreads();

            if (threadIdx.x < blockDim.x - winsz2)
                cache2 = cols_cache[winsz2];
            else
                for(int i = winsz2 + 1; i < winsz; i++)
                    cache2 += cols[i];

            return cols[0] + cache + cache2;
        }

        #define RpT (2 * ROWSperTHREAD)  // got experimentally

        __global__ void textureness_kernel(PtrStepSzb disp, int winsz, float threshold)
        {
            int winsz2 = winsz/2;
            int n_dirty_pixels = (winsz2) * 2;

            extern __shared__ float cols_cache[];
            float *cols = cols_cache + blockDim.x + threadIdx.x;
            float *cols_extra = threadIdx.x < n_dirty_pixels ? cols + blockDim.x : 0;

            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int beg_row = blockIdx.y * RpT;
            int end_row = ::min(beg_row + RpT, disp.rows);

            if (x < disp.cols)
            {
                int y = beg_row;

                float sum = 0;
                float sum_extra = 0;

                for(int i = y - winsz2; i <= y + winsz2; ++i)
                {
                    sum += sobel(x - winsz2, i);
                    if (cols_extra)
                        sum_extra += sobel(x + blockDim.x - winsz2, i);
                }
                *cols = sum;
                if (cols_extra)
                    *cols_extra = sum_extra;

                __syncthreads();

                float sum_win = CalcSums(cols, cols_cache + threadIdx.x, winsz) * 255;
                if (sum_win < threshold)
                    disp.data[y * disp.step + x] = 0;

                __syncthreads();

                for(int y = beg_row + 1; y < end_row; ++y)
                {
                    sum = sum - sobel(x - winsz2, y - winsz2 - 1) + sobel(x - winsz2, y + winsz2);
                    *cols = sum;

                    if (cols_extra)
                    {
                        sum_extra = sum_extra - sobel(x + blockDim.x - winsz2, y - winsz2 - 1) + sobel(x + blockDim.x - winsz2, y + winsz2);
                        *cols_extra = sum_extra;
                    }

                    __syncthreads();
                    float sum_win = CalcSums(cols, cols_cache + threadIdx.x, winsz) * 255;
                    if (sum_win < threshold)
                        disp.data[y * disp.step + x] = 0;

                    __syncthreads();
                }
            }
        }

        void postfilter_textureness(const PtrStepSzb& input, int winsz, float avgTexturenessThreshold, const PtrStepSzb& disp, hipStream_t & stream)
        {
            avgTexturenessThreshold *= winsz * winsz;

            texForTF.filterMode     = hipFilterModeLinear;
            texForTF.addressMode[0] = hipAddressModeWrap;
            texForTF.addressMode[1] = hipAddressModeWrap;

            hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
            cudaSafeCall( hipBindTexture2D( 0, texForTF, input.data, desc, input.cols, input.rows, input.step ) );

            dim3 threads(128, 1, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(input.cols, threads.x);
            grid.y = divUp(input.rows, RpT);

            size_t smem_size = (threads.x + threads.x + (winsz/2) * 2 ) * sizeof(float);
            textureness_kernel<<<grid, threads, smem_size, stream>>>(disp, winsz, avgTexturenessThreshold);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            cudaSafeCall( hipUnbindTexture (texForTF) );
        }
    } // namespace stereobm
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
