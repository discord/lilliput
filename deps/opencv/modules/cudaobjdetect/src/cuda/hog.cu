#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/warp_shuffle.hpp"

namespace cv { namespace cuda { namespace device
{

    namespace hog
    {
        __constant__ int cnbins;
        __constant__ int cblock_stride_x;
        __constant__ int cblock_stride_y;
        __constant__ int cnblocks_win_x;
        __constant__ int cnblocks_win_y;
        __constant__ int cncells_block_x;
        __constant__ int cncells_block_y;
        __constant__ int cblock_hist_size;
        __constant__ int cblock_hist_size_2up;
        __constant__ int cdescr_size;
        __constant__ int cdescr_width;


        /* Returns the nearest upper power of two, works only for
        the typical GPU thread count (pert block) values */
        int power_2up(unsigned int n)
        {
            if (n <= 1) return 1;
            else if (n <= 2) return 2;
            else if (n <= 4) return 4;
            else if (n <= 8) return 8;
            else if (n <= 16) return 16;
            else if (n <= 32) return 32;
            else if (n <= 64) return 64;
            else if (n <= 128) return 128;
            else if (n <= 256) return 256;
            else if (n <= 512) return 512;
            else if (n <= 1024) return 1024;
            return -1; // Input is too big
        }

        /* Returns the max size for nblocks */
        int max_nblocks(int nthreads, int ncells_block = 1)
        {
            int threads = nthreads * ncells_block;
            if(threads * 4 <= 256)
                return 4;
            else if(threads * 3 <= 256)
                return 3;
            else if(threads * 2 <= 256)
                return 2;
            else
                return 1;
        }


        void set_up_constants(int nbins, int block_stride_x, int block_stride_y,
                              int nblocks_win_x, int nblocks_win_y, int ncells_block_x, int ncells_block_y)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cnbins), &nbins, sizeof(nbins)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cblock_stride_x), &block_stride_x, sizeof(block_stride_x)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cblock_stride_y), &block_stride_y, sizeof(block_stride_y)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cnblocks_win_x), &nblocks_win_x, sizeof(nblocks_win_x)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cnblocks_win_y), &nblocks_win_y, sizeof(nblocks_win_y)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cncells_block_x), &ncells_block_x, sizeof(ncells_block_x)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cncells_block_y), &ncells_block_y, sizeof(ncells_block_y)) );

            int block_hist_size = nbins * ncells_block_x * ncells_block_y;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cblock_hist_size), &block_hist_size, sizeof(block_hist_size)) );

            int block_hist_size_2up = power_2up(block_hist_size);
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cblock_hist_size_2up), &block_hist_size_2up, sizeof(block_hist_size_2up)) );

            int descr_width = nblocks_win_x * block_hist_size;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdescr_width), &descr_width, sizeof(descr_width)) );

            int descr_size = descr_width * nblocks_win_y;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdescr_size), &descr_size, sizeof(descr_size)) );
        }


        //----------------------------------------------------------------------------
        // Histogram computation
        //
        // CUDA kernel to compute the histograms
        template <int nblocks> // Number of histogram blocks processed by single GPU thread block
        __global__ void compute_hists_kernel_many_blocks(const int img_block_width, const PtrStepf grad,
                                                         const PtrStepb qangle, float scale, float* block_hists,
                                                         int cell_size, int patch_size, int block_patch_size,
                                                         int threads_cell, int threads_block, int half_cell_size)
        {
            const int block_x = threadIdx.z;
            const int cell_x = threadIdx.x / threads_cell;
            const int cell_y = threadIdx.y;
            const int cell_thread_x = threadIdx.x & (threads_cell - 1);

            if (blockIdx.x * blockDim.z + block_x >= img_block_width)
                return;

            extern __shared__ float smem[];
            float* hists = smem;
            float* final_hist = smem + cnbins * block_patch_size * nblocks;

            // patch_size means that patch_size pixels affect on block's cell
            if (cell_thread_x < patch_size)
            {
                const int offset_x = (blockIdx.x * blockDim.z + block_x) * cblock_stride_x +
                                     half_cell_size * cell_x + cell_thread_x;
                const int offset_y = blockIdx.y * cblock_stride_y + half_cell_size * cell_y;

                const float* grad_ptr = grad.ptr(offset_y) + offset_x * 2;
                const unsigned char* qangle_ptr = qangle.ptr(offset_y) + offset_x * 2;


                float* hist = hists + patch_size * (cell_y * blockDim.z * cncells_block_y +
                                            cell_x + block_x * cncells_block_x) +
                                           cell_thread_x;
                for (int bin_id = 0; bin_id < cnbins; ++bin_id)
                    hist[bin_id * block_patch_size * nblocks] = 0.f;

                //(dist_x, dist_y) : distance between current pixel in patch and cell's center
                const int dist_x = -half_cell_size + (int)cell_thread_x - half_cell_size * cell_x;

                const int dist_y_begin = -half_cell_size - half_cell_size * (int)threadIdx.y;
                for (int dist_y = dist_y_begin; dist_y < dist_y_begin + patch_size; ++dist_y)
                {
                    float2 vote = *(const float2*)grad_ptr;
                    uchar2 bin = *(const uchar2*)qangle_ptr;

                    grad_ptr += grad.step/sizeof(float);
                    qangle_ptr += qangle.step;

                    //(dist_center_x, dist_center_y) : distance between current pixel in patch and block's center
                    int dist_center_y = dist_y - half_cell_size * (1 - 2 * cell_y);
                    int dist_center_x = dist_x - half_cell_size * (1 - 2 * cell_x);

                    float gaussian = ::expf(-(dist_center_y * dist_center_y +
                                              dist_center_x * dist_center_x) * scale);

                    float interp_weight = ((float)cell_size - ::fabs(dist_y + 0.5f)) *
                                          ((float)cell_size - ::fabs(dist_x + 0.5f)) / (float)threads_block;

                    hist[bin.x * block_patch_size * nblocks] += gaussian * interp_weight * vote.x;
                    hist[bin.y * block_patch_size * nblocks] += gaussian * interp_weight * vote.y;
                }

                //reduction of the histograms
                volatile float* hist_ = hist;
                for (int bin_id = 0; bin_id < cnbins; ++bin_id, hist_ += block_patch_size * nblocks)
                {
                    if (cell_thread_x < patch_size/2) hist_[0] += hist_[patch_size/2];
                    if (cell_thread_x < patch_size/4 && (!((patch_size/4) < 3 && cell_thread_x == 0)))
                            hist_[0] += hist_[patch_size/4];
                    if (cell_thread_x == 0)
                        final_hist[((cell_x + block_x * cncells_block_x) * cncells_block_y + cell_y) * cnbins + bin_id]
                            = hist_[0] + hist_[1] + hist_[2];
                }
            }

            __syncthreads();

            float* block_hist = block_hists + (blockIdx.y * img_block_width +
                                               blockIdx.x * blockDim.z + block_x) *
                                              cblock_hist_size;

            //copying from final_hist to block_hist
            int tid;
            if(threads_cell < cnbins)
            {
                tid = (cell_y * cncells_block_y + cell_x) * cnbins + cell_thread_x;
            } else
            {
                tid = (cell_y * cncells_block_y + cell_x) * threads_cell + cell_thread_x;
            }
            if (tid < cblock_hist_size)
            {
                block_hist[tid] = final_hist[block_x * cblock_hist_size + tid];
                if(threads_cell < cnbins && cell_thread_x == (threads_cell-1))
                {
                    for(int i=1;i<=(cnbins - threads_cell);++i)
                    {
                        block_hist[tid + i] = final_hist[block_x * cblock_hist_size + tid + i];
                    }
                }
            }
        }

        //declaration of variables and invoke the kernel with the calculated number of blocks
        void compute_hists(int nbins, int block_stride_x, int block_stride_y,
                           int height, int width, const PtrStepSzf& grad,
                           const PtrStepSzb& qangle, float sigma, float* block_hists,
                           int cell_size_x, int cell_size_y, int ncells_block_x, int ncells_block_y)
        {
            const int ncells_block = ncells_block_x * ncells_block_y;
            const int patch_side = cell_size_x / 4;
            const int patch_size = cell_size_x + (patch_side * 2);
            const int block_patch_size = ncells_block * patch_size;
            const int threads_cell = power_2up(patch_size);
            const int threads_block = ncells_block * threads_cell;
            const int half_cell_size = cell_size_x / 2;

            int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) /
                                  block_stride_x;
            int img_block_height = (height - ncells_block_y * cell_size_y + block_stride_y) /
                                   block_stride_y;

            const int nblocks = max_nblocks(threads_cell, ncells_block);
            dim3 grid(divUp(img_block_width, nblocks), img_block_height);
            dim3 threads(threads_cell * ncells_block_x, ncells_block_y, nblocks);

            // Precompute gaussian spatial window parameter
            float scale = 1.f / (2.f * sigma * sigma);

            int hists_size = (nbins * ncells_block * patch_size * nblocks) * sizeof(float);
            int final_hists_size = (nbins * ncells_block * nblocks) * sizeof(float);
            int smem = hists_size + final_hists_size;
            if (nblocks == 4)
                compute_hists_kernel_many_blocks<4><<<grid, threads, smem>>>(
                    img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
            else if (nblocks == 3)
                compute_hists_kernel_many_blocks<3><<<grid, threads, smem>>>(
                    img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
            else if (nblocks == 2)
                compute_hists_kernel_many_blocks<2><<<grid, threads, smem>>>(
                    img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
            else
                compute_hists_kernel_many_blocks<1><<<grid, threads, smem>>>(
                    img_block_width, grad, qangle, scale, block_hists, cell_size_x, patch_size, block_patch_size, threads_cell, threads_block, half_cell_size);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }


        //-------------------------------------------------------------
        //  Normalization of histograms via L2Hys_norm
        //


        template<int size>
        __device__ float reduce_smem(float* smem, float val)
        {
            unsigned int tid = threadIdx.x;
            float sum = val;

            reduce<size>(smem, sum, tid, plus<float>());

            if (size == 32)
            {
            #if __CUDA_ARCH__ >= 300
                return shfl(sum, 0);
            #else
                return smem[0];
            #endif
            }
            else
            {
            #if __CUDA_ARCH__ >= 300
                if (threadIdx.x == 0)
                    smem[0] = sum;
            #endif

                __syncthreads();

                return smem[0];
            }
        }


        template <int nthreads, // Number of threads which process one block historgam
                  int nblocks> // Number of block hisograms processed by one GPU thread block
        __global__ void normalize_hists_kernel_many_blocks(const int block_hist_size,
                                                           const int img_block_width,
                                                           float* block_hists, float threshold)
        {
            if (blockIdx.x * blockDim.z + threadIdx.z >= img_block_width)
                return;

            float* hist = block_hists + (blockIdx.y * img_block_width +
                                         blockIdx.x * blockDim.z + threadIdx.z) *
                                        block_hist_size + threadIdx.x;

            __shared__ float sh_squares[nthreads * nblocks];
            float* squares = sh_squares + threadIdx.z * nthreads;

            float elem = 0.f;
            if (threadIdx.x < block_hist_size)
                elem = hist[0];

            __syncthreads(); // prevent race condition (redundant?)
            float sum = reduce_smem<nthreads>(squares, elem * elem);

            float scale = 1.0f / (::sqrtf(sum) + 0.1f * block_hist_size);
            elem = ::min(elem * scale, threshold);

            __syncthreads(); // prevent race condition
            sum = reduce_smem<nthreads>(squares, elem * elem);

            scale = 1.0f / (::sqrtf(sum) + 1e-3f);

            if (threadIdx.x < block_hist_size)
                hist[0] = elem * scale;
        }


        void normalize_hists(int nbins, int block_stride_x, int block_stride_y,
                             int height, int width, float* block_hists, float threshold, int cell_size_x, int cell_size_y, int ncells_block_x, int ncells_block_y)
        {
            const int nblocks = 1;

            int block_hist_size = nbins * ncells_block_x * ncells_block_y;
            int nthreads = power_2up(block_hist_size);
            dim3 threads(nthreads, 1, nblocks);

            int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
            int img_block_height = (height - ncells_block_y * cell_size_y + block_stride_y) / block_stride_y;
            dim3 grid(divUp(img_block_width, nblocks), img_block_height);

            if (nthreads == 32)
                normalize_hists_kernel_many_blocks<32, nblocks><<<grid, threads>>>(block_hist_size, img_block_width, block_hists, threshold);
            else if (nthreads == 64)
                normalize_hists_kernel_many_blocks<64, nblocks><<<grid, threads>>>(block_hist_size, img_block_width, block_hists, threshold);
            else if (nthreads == 128)
                normalize_hists_kernel_many_blocks<128, nblocks><<<grid, threads>>>(block_hist_size, img_block_width, block_hists, threshold);
            else if (nthreads == 256)
                normalize_hists_kernel_many_blocks<256, nblocks><<<grid, threads>>>(block_hist_size, img_block_width, block_hists, threshold);
            else if (nthreads == 512)
                normalize_hists_kernel_many_blocks<512, nblocks><<<grid, threads>>>(block_hist_size, img_block_width, block_hists, threshold);
            else
                CV_Error(cv::Error::StsBadArg, "normalize_hists: histogram's size is too big, try to decrease number of bins");

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }


        //---------------------------------------------------------------------
        //  Linear SVM based classification
        //

       // return confidence values not just positive location
       template <int nthreads, // Number of threads per one histogram block
                 int nblocks>  // Number of histogram block processed by single GPU thread block
       __global__ void compute_confidence_hists_kernel_many_blocks(const int img_win_width, const int img_block_width,
                                                                                                           const int win_block_stride_x, const int win_block_stride_y,
                                                                                                           const float* block_hists, const float* coefs,
                                                                                                           float free_coef, float threshold, float* confidences)
       {
           const int win_x = threadIdx.z;
           if (blockIdx.x * blockDim.z + win_x >= img_win_width)
                   return;

           const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
                                                                                blockIdx.x * win_block_stride_x * blockDim.z + win_x) *
                                                                               cblock_hist_size;

           float product = 0.f;
           for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
           {
                   int offset_y = i / cdescr_width;
                   int offset_x = i - offset_y * cdescr_width;
                   product += coefs[i] * hist[offset_y * img_block_width * cblock_hist_size + offset_x];
           }

           __shared__ float products[nthreads * nblocks];

           const int tid = threadIdx.z * nthreads + threadIdx.x;

           reduce<nthreads>(products, product, tid, plus<float>());

           if (threadIdx.x == 0)
               confidences[blockIdx.y * img_win_width + blockIdx.x * blockDim.z + win_x] = product + free_coef;

       }

       void compute_confidence_hists(int win_height, int win_width, int block_stride_y, int block_stride_x,
                                               int win_stride_y, int win_stride_x, int height, int width, float* block_hists,
                                               float* coefs, float free_coef, float threshold, int cell_size_x, int ncells_block_x, float *confidences)
       {
           const int nthreads = 256;
           const int nblocks = 1;

           int win_block_stride_x = win_stride_x / block_stride_x;
           int win_block_stride_y = win_stride_y / block_stride_y;
           int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
           int img_win_height = (height - win_height + win_stride_y) / win_stride_y;

           dim3 threads(nthreads, 1, nblocks);
           dim3 grid(divUp(img_win_width, nblocks), img_win_height);

           cudaSafeCall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_confidence_hists_kernel_many_blocks<nthreads), nblocks>,
                                                                                   hipFuncCachePreferL1));

           int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) /
                                                       block_stride_x;
           compute_confidence_hists_kernel_many_blocks<nthreads, nblocks><<<grid, threads>>>(
                   img_win_width, img_block_width, win_block_stride_x, win_block_stride_y,
                   block_hists, coefs, free_coef, threshold, confidences);
           cudaSafeCall(hipDeviceSynchronize());
       }



        template <int nthreads, // Number of threads per one histogram block
                  int nblocks>  // Number of histogram block processed by single GPU thread block
        __global__ void classify_hists_kernel_many_blocks(const int img_win_width, const int img_block_width,
                                                          const int win_block_stride_x, const int win_block_stride_y,
                                                          const float* block_hists, const float* coefs,
                                                          float free_coef, float threshold, unsigned char* labels)
        {
            const int win_x = threadIdx.z;
            if (blockIdx.x * blockDim.z + win_x >= img_win_width)
                return;

            const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
                                               blockIdx.x * win_block_stride_x * blockDim.z + win_x) *
                                              cblock_hist_size;

            float product = 0.f;
            for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
            {
                int offset_y = i / cdescr_width;
                int offset_x = i - offset_y * cdescr_width;
                product += coefs[i] * hist[offset_y * img_block_width * cblock_hist_size + offset_x];
            }

            __shared__ float products[nthreads * nblocks];

            const int tid = threadIdx.z * nthreads + threadIdx.x;

            reduce<nthreads>(products, product, tid, plus<float>());

            if (threadIdx.x == 0)
                labels[blockIdx.y * img_win_width + blockIdx.x * blockDim.z + win_x] = (product + free_coef >= threshold);
        }


        void classify_hists(int win_height, int win_width, int block_stride_y, int block_stride_x,
                            int win_stride_y, int win_stride_x, int height, int width, float* block_hists,
                            float* coefs, float free_coef, float threshold, int cell_size_x, int ncells_block_x, unsigned char* labels)
        {
            const int nthreads = 256;
            const int nblocks = 1;

            int win_block_stride_x = win_stride_x / block_stride_x;
            int win_block_stride_y = win_stride_y / block_stride_y;
            int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
            int img_win_height = (height - win_height + win_stride_y) / win_stride_y;

            dim3 threads(nthreads, 1, nblocks);
            dim3 grid(divUp(img_win_width, nblocks), img_win_height);

            cudaSafeCall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(classify_hists_kernel_many_blocks<nthreads), nblocks>, hipFuncCachePreferL1));

            int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
            classify_hists_kernel_many_blocks<nthreads, nblocks><<<grid, threads>>>(
                img_win_width, img_block_width, win_block_stride_x, win_block_stride_y,
                block_hists, coefs, free_coef, threshold, labels);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        //----------------------------------------------------------------------------
        // Extract descriptors


        template <int nthreads>
        __global__ void extract_descrs_by_rows_kernel(const int img_block_width, const int win_block_stride_x, const int win_block_stride_y,
                                                      const float* block_hists, PtrStepf descriptors)
        {
            // Get left top corner of the window in src
            const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
                                               blockIdx.x * win_block_stride_x) * cblock_hist_size;

            // Get left top corner of the window in dst
            float* descriptor = descriptors.ptr(blockIdx.y * gridDim.x + blockIdx.x);

            // Copy elements from src to dst
            for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
            {
                int offset_y = i / cdescr_width;
                int offset_x = i - offset_y * cdescr_width;
                descriptor[i] = hist[offset_y * img_block_width * cblock_hist_size + offset_x];
            }
        }


        void extract_descrs_by_rows(int win_height, int win_width, int block_stride_y, int block_stride_x, int win_stride_y, int win_stride_x,
                                    int height, int width, float* block_hists, int cell_size_x, int ncells_block_x, PtrStepSzf descriptors)
        {
            const int nthreads = 256;

            int win_block_stride_x = win_stride_x / block_stride_x;
            int win_block_stride_y = win_stride_y / block_stride_y;
            int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
            int img_win_height = (height - win_height + win_stride_y) / win_stride_y;
            dim3 threads(nthreads, 1);
            dim3 grid(img_win_width, img_win_height);

            int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
            extract_descrs_by_rows_kernel<nthreads><<<grid, threads>>>(
                img_block_width, win_block_stride_x, win_block_stride_y, block_hists, descriptors);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }


        template <int nthreads>
        __global__ void extract_descrs_by_cols_kernel(const int img_block_width, const int win_block_stride_x,
                                                      const int win_block_stride_y, const float* block_hists,
                                                      PtrStepf descriptors)
        {
            // Get left top corner of the window in src
            const float* hist = block_hists + (blockIdx.y * win_block_stride_y * img_block_width +
                                               blockIdx.x * win_block_stride_x) * cblock_hist_size;

            // Get left top corner of the window in dst
            float* descriptor = descriptors.ptr(blockIdx.y * gridDim.x + blockIdx.x);

            // Copy elements from src to dst
            for (int i = threadIdx.x; i < cdescr_size; i += nthreads)
            {
                int block_idx = i / cblock_hist_size;
                int idx_in_block = i - block_idx * cblock_hist_size;

                int y = block_idx / cnblocks_win_x;
                int x = block_idx - y * cnblocks_win_x;

                descriptor[(x * cnblocks_win_y + y) * cblock_hist_size + idx_in_block]
                    = hist[(y * img_block_width  + x) * cblock_hist_size + idx_in_block];
            }
        }


        void extract_descrs_by_cols(int win_height, int win_width, int block_stride_y, int block_stride_x,
                                    int win_stride_y, int win_stride_x, int height, int width, float* block_hists, int cell_size_x, int ncells_block_x,
                                    PtrStepSzf descriptors)
        {
            const int nthreads = 256;

            int win_block_stride_x = win_stride_x / block_stride_x;
            int win_block_stride_y = win_stride_y / block_stride_y;
            int img_win_width = (width - win_width + win_stride_x) / win_stride_x;
            int img_win_height = (height - win_height + win_stride_y) / win_stride_y;
            dim3 threads(nthreads, 1);
            dim3 grid(img_win_width, img_win_height);

            int img_block_width = (width - ncells_block_x * cell_size_x + block_stride_x) / block_stride_x;
            extract_descrs_by_cols_kernel<nthreads><<<grid, threads>>>(
                img_block_width, win_block_stride_x, win_block_stride_y, block_hists, descriptors);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        //----------------------------------------------------------------------------
        // Gradients computation


        template <int nthreads, int correct_gamma>
        __global__ void compute_gradients_8UC4_kernel(int height, int width, const PtrStepb img,
                                                      float angle_scale, PtrStepf grad, PtrStepb qangle)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;

            const uchar4* row = (const uchar4*)img.ptr(blockIdx.y);

            __shared__ float sh_row[(nthreads + 2) * 3];

            uchar4 val;
            if (x < width)
                val = row[x];
            else
                val = row[width - 2];

            sh_row[threadIdx.x + 1] = val.x;
            sh_row[threadIdx.x + 1 + (nthreads + 2)] = val.y;
            sh_row[threadIdx.x + 1 + 2 * (nthreads + 2)] = val.z;

            if (threadIdx.x == 0)
            {
                val = row[::max(x - 1, 1)];
                sh_row[0] = val.x;
                sh_row[(nthreads + 2)] = val.y;
                sh_row[2 * (nthreads + 2)] = val.z;
            }

            if (threadIdx.x == blockDim.x - 1)
            {
                val = row[::min(x + 1, width - 2)];
                sh_row[blockDim.x + 1] = val.x;
                sh_row[blockDim.x + 1 + (nthreads + 2)] = val.y;
                sh_row[blockDim.x + 1 + 2 * (nthreads + 2)] = val.z;
            }

            __syncthreads();
            if (x < width)
            {
                float3 a, b;

                b.x = sh_row[threadIdx.x + 2];
                b.y = sh_row[threadIdx.x + 2 + (nthreads + 2)];
                b.z = sh_row[threadIdx.x + 2 + 2 * (nthreads + 2)];
                a.x = sh_row[threadIdx.x];
                a.y = sh_row[threadIdx.x + (nthreads + 2)];
                a.z = sh_row[threadIdx.x + 2 * (nthreads + 2)];

                float3 dx;
                if (correct_gamma)
                    dx = make_float3(::sqrtf(b.x) - ::sqrtf(a.x), ::sqrtf(b.y) - ::sqrtf(a.y), ::sqrtf(b.z) - ::sqrtf(a.z));
                else
                    dx = make_float3(b.x - a.x, b.y - a.y, b.z - a.z);

                float3 dy = make_float3(0.f, 0.f, 0.f);

                if (blockIdx.y > 0 && blockIdx.y < height - 1)
                {
                    val = ((const uchar4*)img.ptr(blockIdx.y - 1))[x];
                    a = make_float3(val.x, val.y, val.z);

                    val = ((const uchar4*)img.ptr(blockIdx.y + 1))[x];
                    b = make_float3(val.x, val.y, val.z);

                    if (correct_gamma)
                        dy = make_float3(::sqrtf(b.x) - ::sqrtf(a.x), ::sqrtf(b.y) - ::sqrtf(a.y), ::sqrtf(b.z) - ::sqrtf(a.z));
                    else
                        dy = make_float3(b.x - a.x, b.y - a.y, b.z - a.z);
                }

                float best_dx = dx.x;
                float best_dy = dy.x;

                float mag0 = dx.x * dx.x + dy.x * dy.x;
                float mag1 = dx.y * dx.y + dy.y * dy.y;
                if (mag0 < mag1)
                {
                    best_dx = dx.y;
                    best_dy = dy.y;
                    mag0 = mag1;
                }

                mag1 = dx.z * dx.z + dy.z * dy.z;
                if (mag0 < mag1)
                {
                    best_dx = dx.z;
                    best_dy = dy.z;
                    mag0 = mag1;
                }

                mag0 = ::sqrtf(mag0);

                float ang = (::atan2f(best_dy, best_dx) + CV_PI_F) * angle_scale - 0.5f;
                int hidx = (int)::floorf(ang);
                ang -= hidx;
                hidx = (hidx + cnbins) % cnbins;

                ((uchar2*)qangle.ptr(blockIdx.y))[x] = make_uchar2(hidx, (hidx + 1) % cnbins);
                ((float2*)grad.ptr(blockIdx.y))[x] = make_float2(mag0 * (1.f - ang), mag0 * ang);
            }
        }


        void compute_gradients_8UC4(int nbins, int height, int width, const PtrStepSzb& img,
                                    float angle_scale, PtrStepSzf grad, PtrStepSzb qangle, bool correct_gamma)
        {
            (void)nbins;
            const int nthreads = 256;

            dim3 bdim(nthreads, 1);
            dim3 gdim(divUp(width, bdim.x), divUp(height, bdim.y));

            if (correct_gamma)
                compute_gradients_8UC4_kernel<nthreads, 1><<<gdim, bdim>>>(height, width, img, angle_scale, grad, qangle);
            else
                compute_gradients_8UC4_kernel<nthreads, 0><<<gdim, bdim>>>(height, width, img, angle_scale, grad, qangle);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int nthreads, int correct_gamma>
        __global__ void compute_gradients_8UC1_kernel(int height, int width, const PtrStepb img,
                                                      float angle_scale, PtrStepf grad, PtrStepb qangle)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;

            const unsigned char* row = (const unsigned char*)img.ptr(blockIdx.y);

            __shared__ float sh_row[nthreads + 2];

            if (x < width)
                sh_row[threadIdx.x + 1] = row[x];
            else
                sh_row[threadIdx.x + 1] = row[width - 2];

            if (threadIdx.x == 0)
                sh_row[0] = row[::max(x - 1, 1)];

            if (threadIdx.x == blockDim.x - 1)
                sh_row[blockDim.x + 1] = row[::min(x + 1, width - 2)];

            __syncthreads();
            if (x < width)
            {
                float dx;

                if (correct_gamma)
                    dx = ::sqrtf(sh_row[threadIdx.x + 2]) - ::sqrtf(sh_row[threadIdx.x]);
                else
                    dx = sh_row[threadIdx.x + 2] - sh_row[threadIdx.x];

                float dy = 0.f;
                if (blockIdx.y > 0 && blockIdx.y < height - 1)
                {
                    float a = ((const unsigned char*)img.ptr(blockIdx.y + 1))[x];
                    float b = ((const unsigned char*)img.ptr(blockIdx.y - 1))[x];
                    if (correct_gamma)
                        dy = ::sqrtf(a) - ::sqrtf(b);
                    else
                        dy = a - b;
                }
                float mag = ::sqrtf(dx * dx + dy * dy);

                float ang = (::atan2f(dy, dx) + CV_PI_F) * angle_scale - 0.5f;
                int hidx = (int)::floorf(ang);
                ang -= hidx;
                hidx = (hidx + cnbins) % cnbins;

                ((uchar2*)qangle.ptr(blockIdx.y))[x] = make_uchar2(hidx, (hidx + 1) % cnbins);
                ((float2*)  grad.ptr(blockIdx.y))[x] = make_float2(mag * (1.f - ang), mag * ang);
            }
        }


        void compute_gradients_8UC1(int nbins, int height, int width, const PtrStepSzb& img,
                                    float angle_scale, PtrStepSzf grad, PtrStepSzb qangle, bool correct_gamma)
        {
            (void)nbins;
            const int nthreads = 256;

            dim3 bdim(nthreads, 1);
            dim3 gdim(divUp(width, bdim.x), divUp(height, bdim.y));

            if (correct_gamma)
                compute_gradients_8UC1_kernel<nthreads, 1><<<gdim, bdim>>>(height, width, img, angle_scale, grad, qangle);
            else
                compute_gradients_8UC1_kernel<nthreads, 0><<<gdim, bdim>>>(height, width, img, angle_scale, grad, qangle);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }



        //-------------------------------------------------------------------
        // Resize

        texture<uchar4, 2, hipReadModeNormalizedFloat> resize8UC4_tex;
        texture<uchar,  2, hipReadModeNormalizedFloat> resize8UC1_tex;

        __global__ void resize_for_hog_kernel(float sx, float sy, PtrStepSz<uchar> dst, int colOfs)
        {
            unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
                dst.ptr(y)[x] = tex2D(resize8UC1_tex, x * sx + colOfs, y * sy) * 255;
        }

        __global__ void resize_for_hog_kernel(float sx, float sy, PtrStepSz<uchar4> dst, int colOfs)
        {
            unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
            unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float4 val = tex2D(resize8UC4_tex, x * sx + colOfs, y * sy);
                dst.ptr(y)[x] = make_uchar4(val.x * 255, val.y * 255, val.z * 255, val.w * 255);
            }
        }

        template<class T, class TEX>
        static void resize_for_hog(const PtrStepSzb& src, PtrStepSzb dst, TEX& tex)
        {
            tex.filterMode = hipFilterModeLinear;

            size_t texOfs = 0;
            int colOfs = 0;

            hipChannelFormatDesc desc = hipCreateChannelDesc<T>();
            cudaSafeCall( hipBindTexture2D(&texOfs, tex, src.data, desc, src.cols, src.rows, src.step) );

            if (texOfs != 0)
            {
                colOfs = static_cast<int>( texOfs/sizeof(T) );
                cudaSafeCall( hipUnbindTexture(tex) );
                cudaSafeCall( hipBindTexture2D(&texOfs, tex, src.data, desc, src.cols, src.rows, src.step) );
            }

            dim3 threads(32, 8);
            dim3 grid(divUp(dst.cols, threads.x), divUp(dst.rows, threads.y));

            float sx = static_cast<float>(src.cols) / dst.cols;
            float sy = static_cast<float>(src.rows) / dst.rows;

            resize_for_hog_kernel<<<grid, threads>>>(sx, sy, (PtrStepSz<T>)dst, colOfs);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            cudaSafeCall( hipUnbindTexture(tex) );
        }

        void resize_8UC1(const PtrStepSzb& src, PtrStepSzb dst) { resize_for_hog<uchar> (src, dst, resize8UC1_tex); }
        void resize_8UC4(const PtrStepSzb& src, PtrStepSzb dst) { resize_for_hog<uchar4>(src, dst, resize8UC4_tex); }
    } // namespace hog
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
