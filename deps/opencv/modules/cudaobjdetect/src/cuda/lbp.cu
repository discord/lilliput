#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "lbp.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace lbp
    {
        struct LBP
        {
            __host__ __device__ __forceinline__ LBP() {}

            __device__ __forceinline__ int operator() (const int* integral, int ty, int fh, int fw, int& shift) const
            {
                int anchors[9];

                anchors[0]  = integral[ty];
                anchors[1]  = integral[ty + fw];
                anchors[0] -= anchors[1];
                anchors[2]  = integral[ty + fw * 2];
                anchors[1] -= anchors[2];
                anchors[2] -= integral[ty + fw * 3];

                ty += fh;
                anchors[3]  = integral[ty];
                anchors[4]  = integral[ty + fw];
                anchors[3] -= anchors[4];
                anchors[5]  = integral[ty + fw * 2];
                anchors[4] -= anchors[5];
                anchors[5] -= integral[ty + fw * 3];

                anchors[0] -= anchors[3];
                anchors[1] -= anchors[4];
                anchors[2] -= anchors[5];
                // 0 - 2 contains s0 - s2

                ty += fh;
                anchors[6]  = integral[ty];
                anchors[7]  = integral[ty + fw];
                anchors[6] -= anchors[7];
                anchors[8]  = integral[ty + fw * 2];
                anchors[7] -= anchors[8];
                anchors[8] -= integral[ty + fw * 3];

                anchors[3] -= anchors[6];
                anchors[4] -= anchors[7];
                anchors[5] -= anchors[8];
                // 3 - 5 contains s3 - s5

                anchors[0] -= anchors[4];
                anchors[1] -= anchors[4];
                anchors[2] -= anchors[4];
                anchors[3] -= anchors[4];
                anchors[5] -= anchors[4];

                int response = (~(anchors[0] >> 31)) & 4;
                response |= (~(anchors[1] >> 31)) & 2;;
                response |= (~(anchors[2] >> 31)) & 1;

                shift = (~(anchors[5] >> 31)) & 16;
                shift |= (~(anchors[3] >> 31)) & 1;

                ty += fh;
                anchors[0]  = integral[ty];
                anchors[1]  = integral[ty + fw];
                anchors[0] -= anchors[1];
                anchors[2]  = integral[ty + fw * 2];
                anchors[1] -= anchors[2];
                anchors[2] -= integral[ty + fw * 3];

                anchors[6] -= anchors[0];
                anchors[7] -= anchors[1];
                anchors[8] -= anchors[2];
                // 0 -2 contains s6 - s8

                anchors[6] -= anchors[4];
                anchors[7] -= anchors[4];
                anchors[8] -= anchors[4];

                shift |= (~(anchors[6] >> 31)) & 2;
                shift |= (~(anchors[7] >> 31)) & 4;
                shift |= (~(anchors[8] >> 31)) & 8;
                return response;
            }
        };

        template<typename Pr>
        __global__ void disjoin(int4* candidates, int4* objects, unsigned int n, int groupThreshold, float grouping_eps, unsigned int* nclasses)
        {
            unsigned int tid = threadIdx.x;
            extern __shared__ int sbuff[];

            int* labels = sbuff;
            int* rrects = sbuff + n;

            Pr predicate(grouping_eps);
            partition(candidates, n, labels, predicate);

            rrects[tid * 4 + 0] = 0;
            rrects[tid * 4 + 1] = 0;
            rrects[tid * 4 + 2] = 0;
            rrects[tid * 4 + 3] = 0;
            __syncthreads();

            int cls = labels[tid];
            Emulation::smem::atomicAdd((rrects + cls * 4 + 0), candidates[tid].x);
            Emulation::smem::atomicAdd((rrects + cls * 4 + 1), candidates[tid].y);
            Emulation::smem::atomicAdd((rrects + cls * 4 + 2), candidates[tid].z);
            Emulation::smem::atomicAdd((rrects + cls * 4 + 3), candidates[tid].w);

            __syncthreads();
            labels[tid] = 0;

            __syncthreads();
            Emulation::smem::atomicInc((unsigned int*)labels + cls, n);

            __syncthreads();
            *nclasses = 0;

            int active = labels[tid];
            if (active)
            {
                int* r1 = rrects + tid * 4;
                float s = 1.f / active;
                r1[0] = saturate_cast<int>(r1[0] * s);
                r1[1] = saturate_cast<int>(r1[1] * s);
                r1[2] = saturate_cast<int>(r1[2] * s);
                r1[3] = saturate_cast<int>(r1[3] * s);
            }
            __syncthreads();

            if (active && active >= groupThreshold)
            {
                int* r1 = rrects + tid * 4;
                int4 r_out = make_int4(r1[0], r1[1], r1[2], r1[3]);

                int aidx = Emulation::smem::atomicInc(nclasses, n);
                objects[aidx] = r_out;
            }
        }

        void connectedConmonents(PtrStepSz<int4> candidates, int ncandidates, PtrStepSz<int4> objects, int groupThreshold, float grouping_eps, unsigned int* nclasses)
        {
            if (!ncandidates) return;
            int block = ncandidates;
            int smem  = block * ( sizeof(int) + sizeof(int4) );
            disjoin<InSameComponint><<<1, block, smem>>>(candidates, objects, ncandidates, groupThreshold, grouping_eps, nclasses);
            cudaSafeCall( hipGetLastError() );
        }

        struct Cascade
        {
            __host__ __device__ __forceinline__ Cascade(const Stage* _stages, int _nstages, const ClNode* _nodes, const float* _leaves,
                const int* _subsets, const uchar4* _features, int _subsetSize)

            : stages(_stages), nstages(_nstages), nodes(_nodes), leaves(_leaves), subsets(_subsets), features(_features), subsetSize(_subsetSize){}

            __device__ __forceinline__ bool operator() (int y, int x, int* integral, const int pitch) const
            {
                int current_node = 0;
                int current_leave = 0;

                for (int s = 0; s < nstages; ++s)
                {
                    float sum = 0;
                    Stage stage = stages[s];
                    for (int t = 0; t < stage.ntrees; t++)
                    {
                        ClNode node = nodes[current_node];
                        uchar4 feature = features[node.featureIdx];

                        int shift;
                        int c = evaluator(integral, (y + feature.y) * pitch + x + feature.x, feature.w * pitch, feature.z, shift);
                        int idx =  (subsets[ current_node * subsetSize + c] & ( 1 << shift)) ? current_leave : current_leave + 1;
                        sum += leaves[idx];

                        current_node += 1;
                        current_leave += 2;
                    }

                    if (sum < stage.threshold)
                        return false;
                }

                return true;
            }

            const Stage*  stages;
            const int nstages;

            const ClNode* nodes;
            const float* leaves;
            const int* subsets;
            const uchar4* features;

            const int subsetSize;
            const LBP evaluator;
        };

        // stepShift, scale, width_k, sum_prev => y =  sum_prev + tid_k / width_k, x = tid_k - tid_k / width_k
        __global__ void lbp_cascade(const Cascade cascade, int frameW, int frameH, int windowW, int windowH, float scale, const float factor,
            const int total, int* integral, const int pitch, PtrStepSz<int4> objects, unsigned int* classified)
        {
            int ftid = blockIdx.x * blockDim.x + threadIdx.x;
            if (ftid >= total) return;

            int step = (scale <= 2.f);

            int windowsForLine = (__float2int_rn( __fdividef(frameW, scale)) - windowW) >> step;
            int stotal = windowsForLine * ( (__float2int_rn( __fdividef(frameH, scale)) - windowH) >> step);
            int wshift = 0;

            int scaleTid = ftid;

            while (scaleTid >= stotal)
            {
                scaleTid -= stotal;
                wshift += __float2int_rn(__fdividef(frameW, scale)) + 1;
                scale *= factor;
                step = (scale <= 2.f);
                windowsForLine = ( ((__float2int_rn(__fdividef(frameW, scale)) - windowW) >> step));
                stotal = windowsForLine * ( (__float2int_rn(__fdividef(frameH, scale)) - windowH) >> step);
            }

            int y = __fdividef(scaleTid, windowsForLine);
            int x = scaleTid - y * windowsForLine;

            x <<= step;
            y <<= step;

            if (cascade(y, x + wshift, integral, pitch))
            {
                if(x >= __float2int_rn(__fdividef(frameW, scale)) - windowW) return;

                int4 rect;
                rect.x = __float2int_rn(x * scale);
                rect.y = __float2int_rn(y * scale);
                rect.z = __float2int_rn(windowW * scale);
                rect.w = __float2int_rn(windowH * scale);

                int res = atomicInc(classified, (unsigned int)objects.cols);
                objects(0, res) = rect;
            }
        }

        void classifyPyramid(int frameW, int frameH, int windowW, int windowH, float initialScale, float factor, int workAmount,
            const PtrStepSzb& mstages, const int nstages, const PtrStepSzi& mnodes, const PtrStepSzf& mleaves, const PtrStepSzi& msubsets, const PtrStepSzb& mfeatures,
            const int subsetSize, PtrStepSz<int4> objects, unsigned int* classified, PtrStepSzi integral)
        {
            const int block = 128;
            int grid = divUp(workAmount, block);
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(lbp_cascade), hipFuncCachePreferL1);
            Cascade cascade((Stage*)mstages.ptr(), nstages, (ClNode*)mnodes.ptr(), mleaves.ptr(), msubsets.ptr(), (uchar4*)mfeatures.ptr(), subsetSize);
            lbp_cascade<<<grid, block>>>(cascade, frameW, frameH, windowW, windowH, initialScale, factor, workAmount, integral.ptr(), (int)integral.step / sizeof(int), objects, classified);
        }
    }
}}}

#endif /* CUDA_DISABLER */
