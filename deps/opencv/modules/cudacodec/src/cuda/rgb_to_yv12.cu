#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev/ptr2d/glob.hpp"

using namespace cv::cudev;

void RGB_to_YV12(const GpuMat& src, GpuMat& dst);

namespace
{
    __device__ __forceinline__ void rgb_to_y(const uchar b, const uchar g, const uchar r, uchar& y)
    {
        y = static_cast<uchar>(((int)(30 * r) + (int)(59 * g) + (int)(11 * b)) / 100);
    }

    __device__ __forceinline__ void rgb_to_yuv(const uchar b, const uchar g, const uchar r, uchar& y, uchar& u, uchar& v)
    {
        rgb_to_y(b, g, r, y);
        u = static_cast<uchar>(((int)(-17 * r) - (int)(33 * g) + (int)(50 * b) + 12800) / 100);
        v = static_cast<uchar>(((int)(50 * r) - (int)(42 * g) - (int)(8 * b) + 12800) / 100);
    }

    __global__ void Gray_to_YV12(const GlobPtrSz<uchar> src, GlobPtr<uchar> dst)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
        const int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

        if (x + 1 >= src.cols || y + 1 >= src.rows)
            return;

        // get pointers to the data
        const size_t planeSize = src.rows * dst.step;
        GlobPtr<uchar> y_plane = globPtr(dst.data, dst.step);
        GlobPtr<uchar> u_plane = globPtr(y_plane.data + planeSize, dst.step / 2);
        GlobPtr<uchar> v_plane = globPtr(u_plane.data + (planeSize / 4), dst.step / 2);

        uchar pix;
        uchar y_val, u_val, v_val;

        pix = src(y, x);
        rgb_to_y(pix, pix, pix, y_val);
        y_plane(y, x) = y_val;

        pix = src(y, x + 1);
        rgb_to_y(pix, pix, pix, y_val);
        y_plane(y, x + 1) = y_val;

        pix = src(y + 1, x);
        rgb_to_y(pix, pix, pix, y_val);
        y_plane(y + 1, x) = y_val;

        pix = src(y + 1, x + 1);
        rgb_to_yuv(pix, pix, pix, y_val, u_val, v_val);
        y_plane(y + 1, x + 1) = y_val;
        u_plane(y / 2, x / 2) = u_val;
        v_plane(y / 2, x / 2) = v_val;
    }

    template <typename T>
    __global__ void RGB_to_YV12(const GlobPtrSz<T> src, GlobPtr<uchar> dst)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
        const int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

        if (x + 1 >= src.cols || y + 1 >= src.rows)
            return;

        // get pointers to the data
        const size_t planeSize = src.rows * dst.step;
        GlobPtr<uchar> y_plane = globPtr(dst.data, dst.step);
        GlobPtr<uchar> u_plane = globPtr(y_plane.data + planeSize, dst.step / 2);
        GlobPtr<uchar> v_plane = globPtr(u_plane.data + (planeSize / 4), dst.step / 2);

        T pix;
        uchar y_val, u_val, v_val;

        pix = src(y, x);
        rgb_to_y(pix.z, pix.y, pix.x, y_val);
        y_plane(y, x) = y_val;

        pix = src(y, x + 1);
        rgb_to_y(pix.z, pix.y, pix.x, y_val);
        y_plane(y, x + 1) = y_val;

        pix = src(y + 1, x);
        rgb_to_y(pix.z, pix.y, pix.x, y_val);
        y_plane(y + 1, x) = y_val;

        pix = src(y + 1, x + 1);
        rgb_to_yuv(pix.z, pix.y, pix.x, y_val, u_val, v_val);
        y_plane(y + 1, x + 1) = y_val;
        u_plane(y / 2, x / 2) = u_val;
        v_plane(y / 2, x / 2) = v_val;
    }
}

void RGB_to_YV12(const GpuMat& src, GpuMat& dst)
{
    const dim3 block(32, 8);
    const dim3 grid(divUp(src.cols, block.x * 2), divUp(src.rows, block.y * 2));

    switch (src.channels())
    {
    case 1:
        Gray_to_YV12<<<grid, block>>>(globPtr<uchar>(src), globPtr<uchar>(dst));
        break;
    case 3:
        RGB_to_YV12<<<grid, block>>>(globPtr<uchar3>(src), globPtr<uchar>(dst));
        break;
    case 4:
        RGB_to_YV12<<<grid, block>>>(globPtr<uchar4>(src), globPtr<uchar>(dst));
        break;
    }

    CV_CUDEV_SAFE_CALL( hipGetLastError() );
    CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

#endif
