#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        template <typename T, typename B> __global__ void pyrDown(const PtrStepSz<T> src, PtrStep<T> dst, const B b, int dst_cols)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_t;

            __shared__ work_t smem[256 + 4];

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y;

            const int src_y = 2 * y;

            if (src_y >= 2 && src_y < src.rows - 2 && x >= 2 && x < src.cols - 2)
            {
                {
                    work_t sum;

                    sum =       0.0625f * src(src_y - 2, x);
                    sum = sum + 0.25f   * src(src_y - 1, x);
                    sum = sum + 0.375f  * src(src_y    , x);
                    sum = sum + 0.25f   * src(src_y + 1, x);
                    sum = sum + 0.0625f * src(src_y + 2, x);

                    smem[2 + threadIdx.x] = sum;
                }

                if (threadIdx.x < 2)
                {
                    const int left_x = x - 2;

                    work_t sum;

                    sum =       0.0625f * src(src_y - 2, left_x);
                    sum = sum + 0.25f   * src(src_y - 1, left_x);
                    sum = sum + 0.375f  * src(src_y    , left_x);
                    sum = sum + 0.25f   * src(src_y + 1, left_x);
                    sum = sum + 0.0625f * src(src_y + 2, left_x);

                    smem[threadIdx.x] = sum;
                }

                if (threadIdx.x > 253)
                {
                    const int right_x = x + 2;

                    work_t sum;

                    sum =       0.0625f * src(src_y - 2, right_x);
                    sum = sum + 0.25f   * src(src_y - 1, right_x);
                    sum = sum + 0.375f  * src(src_y    , right_x);
                    sum = sum + 0.25f   * src(src_y + 1, right_x);
                    sum = sum + 0.0625f * src(src_y + 2, right_x);

                    smem[4 + threadIdx.x] = sum;
                }
            }
            else
            {
                {
                    work_t sum;

                    sum =       0.0625f * src(b.idx_row_low (src_y - 2), b.idx_col_high(x));
                    sum = sum + 0.25f   * src(b.idx_row_low (src_y - 1), b.idx_col_high(x));
                    sum = sum + 0.375f  * src(src_y                    , b.idx_col_high(x));
                    sum = sum + 0.25f   * src(b.idx_row_high(src_y + 1), b.idx_col_high(x));
                    sum = sum + 0.0625f * src(b.idx_row_high(src_y + 2), b.idx_col_high(x));

                    smem[2 + threadIdx.x] = sum;
                }

                if (threadIdx.x < 2)
                {
                    const int left_x = x - 2;

                    work_t sum;

                    sum =       0.0625f * src(b.idx_row_low (src_y - 2), b.idx_col(left_x));
                    sum = sum + 0.25f   * src(b.idx_row_low (src_y - 1), b.idx_col(left_x));
                    sum = sum + 0.375f  * src(src_y                    , b.idx_col(left_x));
                    sum = sum + 0.25f   * src(b.idx_row_high(src_y + 1), b.idx_col(left_x));
                    sum = sum + 0.0625f * src(b.idx_row_high(src_y + 2), b.idx_col(left_x));

                    smem[threadIdx.x] = sum;
                }

                if (threadIdx.x > 253)
                {
                    const int right_x = x + 2;

                    work_t sum;

                    sum =       0.0625f * src(b.idx_row_low (src_y - 2), b.idx_col_high(right_x));
                    sum = sum + 0.25f   * src(b.idx_row_low (src_y - 1), b.idx_col_high(right_x));
                    sum = sum + 0.375f  * src(src_y                    , b.idx_col_high(right_x));
                    sum = sum + 0.25f   * src(b.idx_row_high(src_y + 1), b.idx_col_high(right_x));
                    sum = sum + 0.0625f * src(b.idx_row_high(src_y + 2), b.idx_col_high(right_x));

                    smem[4 + threadIdx.x] = sum;
                }
            }

            __syncthreads();

            if (threadIdx.x < 128)
            {
                const int tid2 = threadIdx.x * 2;

                work_t sum;

                sum =       0.0625f * smem[2 + tid2 - 2];
                sum = sum + 0.25f   * smem[2 + tid2 - 1];
                sum = sum + 0.375f  * smem[2 + tid2    ];
                sum = sum + 0.25f   * smem[2 + tid2 + 1];
                sum = sum + 0.0625f * smem[2 + tid2 + 2];

                const int dst_x = (blockIdx.x * blockDim.x + tid2) / 2;

                if (dst_x < dst_cols)
                    dst.ptr(y)[dst_x] = saturate_cast<T>(sum);
            }
        }

        template <typename T, template <typename> class B> void pyrDown_caller(PtrStepSz<T> src, PtrStepSz<T> dst, hipStream_t stream)
        {
            const dim3 block(256);
            const dim3 grid(divUp(src.cols, block.x), dst.rows);

            B<T> b(src.rows, src.cols);

            pyrDown<T><<<grid, block, 0, stream>>>(src, dst, b, dst.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <typename T> void pyrDown_gpu(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
        {
            pyrDown_caller<T, BrdReflect101>(static_cast< PtrStepSz<T> >(src), static_cast< PtrStepSz<T> >(dst), stream);
        }

        template void pyrDown_gpu<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<uchar2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<uchar3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<uchar4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        //template void pyrDown_gpu<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<char2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<char3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<char4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrDown_gpu<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<ushort2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<ushort3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<ushort4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrDown_gpu<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<short2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<short3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<short4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrDown_gpu<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<int2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<int3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<int4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrDown_gpu<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrDown_gpu<float2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<float3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrDown_gpu<float4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */