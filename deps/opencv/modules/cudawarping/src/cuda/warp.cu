#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/filters.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        __constant__ float c_warpMat[3 * 3];

        struct AffineTransform
        {
            static __device__ __forceinline__ float2 calcCoord(int x, int y)
            {
                const float xcoo = c_warpMat[0] * x + c_warpMat[1] * y + c_warpMat[2];
                const float ycoo = c_warpMat[3] * x + c_warpMat[4] * y + c_warpMat[5];

                return make_float2(xcoo, ycoo);
            }
        };

        struct PerspectiveTransform
        {
            static __device__ __forceinline__ float2 calcCoord(int x, int y)
            {
                const float coeff = 1.0f / (c_warpMat[6] * x + c_warpMat[7] * y + c_warpMat[8]);

                const float xcoo = coeff * (c_warpMat[0] * x + c_warpMat[1] * y + c_warpMat[2]);
                const float ycoo = coeff * (c_warpMat[3] * x + c_warpMat[4] * y + c_warpMat[5]);

                return make_float2(xcoo, ycoo);
            }
        };

        ///////////////////////////////////////////////////////////////////
        // Build Maps

        template <class Transform> __global__ void buildWarpMaps(PtrStepSzf xmap, PtrStepf ymap)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < xmap.cols && y < xmap.rows)
            {
                const float2 coord = Transform::calcCoord(x, y);

                xmap(y, x) = coord.x;
                ymap(y, x) = coord.y;
            }
        }

        template <class Transform> void buildWarpMaps_caller(PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(xmap.cols, block.x), divUp(xmap.rows, block.y));

            buildWarpMaps<Transform><<<grid, block, 0, stream>>>(xmap, ymap);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void buildWarpAffineMaps_gpu(float coeffs[2 * 3], PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_warpMat), coeffs, 2 * 3 * sizeof(float)) );

            buildWarpMaps_caller<AffineTransform>(xmap, ymap, stream);
        }

        void buildWarpPerspectiveMaps_gpu(float coeffs[3 * 3], PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_warpMat), coeffs, 3 * 3 * sizeof(float)) );

            buildWarpMaps_caller<PerspectiveTransform>(xmap, ymap, stream);
        }

        ///////////////////////////////////////////////////////////////////
        // Warp

        template <class Transform, class Ptr2D, typename T> __global__ void warp(const Ptr2D src, PtrStepSz<T> dst)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                const float2 coord = Transform::calcCoord(x, y);

                dst.ptr(y)[x] = saturate_cast<T>(src(coord.y, coord.x));
            }
        }

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> dst, const float* borderValue, hipStream_t stream, bool)
            {
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader< PtrStep<T>, B<work_type> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, B<work_type> > > filter_src(brdSrc);

                warp<Transform><<<grid, block, 0, stream>>>(filter_src, dst);
                cudaSafeCall( hipGetLastError() );
            }
        };

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherNonStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, bool)
            {
                (void)xoff;
                (void)yoff;
                (void)srcWhole;

                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader< PtrStep<T>, B<work_type> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, B<work_type> > > filter_src(brdSrc);

                warp<Transform><<<grid, block>>>(filter_src, dst);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        #define OPENCV_CUDA_IMPLEMENT_WARP_TEX(type) \
            texture< type , hipTextureType2D > tex_warp_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
            struct tex_warp_ ## type ## _reader \
            { \
                typedef type elem_type; \
                typedef int index_type; \
                int xoff, yoff; \
                tex_warp_ ## type ## _reader (int xoff_, int yoff_) : xoff(xoff_), yoff(yoff_) {} \
                __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
                { \
                    return tex2D(tex_warp_ ## type , x + xoff, y + yoff); \
                } \
            }; \
            template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, type> \
            { \
                static void call(PtrStepSz< type > src, PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSz< type > dst, const float* borderValue, bool cc20) \
                { \
                    typedef typename TypeVec<float, VecTraits< type >::cn>::vec_type work_type; \
                    dim3 block(32, cc20 ? 8 : 4); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_warp_ ## type , srcWhole); \
                    tex_warp_ ## type ##_reader texSrc(xoff, yoff); \
                    B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue)); \
                    BorderReader< tex_warp_ ## type ##_reader, B<work_type> > brdSrc(texSrc, brd); \
                    Filter< BorderReader< tex_warp_ ## type ##_reader, B<work_type> > > filter_src(brdSrc); \
                    warp<Transform><<<grid, block>>>(filter_src, dst); \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            }; \
            template <class Transform, template <typename> class Filter> struct WarpDispatcherNonStream<Transform, Filter, BrdReplicate, type> \
            { \
                static void call(PtrStepSz< type > src, PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSz< type > dst, const float*, bool) \
                { \
                    dim3 block(32, 8); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_warp_ ## type , srcWhole); \
                    tex_warp_ ## type ##_reader texSrc(xoff, yoff); \
                    if (srcWhole.cols == src.cols && srcWhole.rows == src.rows) \
                    { \
                        Filter< tex_warp_ ## type ##_reader > filter_src(texSrc); \
                        warp<Transform><<<grid, block>>>(filter_src, dst); \
                    } \
                    else \
                    { \
                        BrdReplicate<type> brd(src.rows, src.cols); \
                        BorderReader< tex_warp_ ## type ##_reader, BrdReplicate<type> > brdSrc(texSrc, brd); \
                        Filter< BorderReader< tex_warp_ ## type ##_reader, BrdReplicate<type> > > filter_src(brdSrc); \
                        warp<Transform><<<grid, block>>>(filter_src, dst); \
                    } \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            };

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(uchar)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(uchar2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(uchar4)

        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(schar)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(char2)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(char4)

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(ushort)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(ushort2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(ushort4)

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(short)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(short2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(short4)

        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(int)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(int2)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(int4)

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(float)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(float2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(float4)

        #undef OPENCV_CUDA_IMPLEMENT_WARP_TEX

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcher
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, hipStream_t stream, bool cc20)
            {
                if (stream == 0)
                    WarpDispatcherNonStream<Transform, Filter, B, T>::call(src, srcWhole, xoff, yoff, dst, borderValue, cc20);
                else
                    WarpDispatcherStream<Transform, Filter, B, T>::call(src, dst, borderValue, stream, cc20);
            }
        };

        template <class Transform, typename T>
        void warp_caller(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzb dst, int interpolation,
                         int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            typedef void (*func_t)(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, hipStream_t stream, bool cc20);

            static const func_t funcs[3][5] =
            {
                {
                    WarpDispatcher<Transform, PointFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReflect101, T>::call
                },
                {
                    WarpDispatcher<Transform, LinearFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReflect101, T>::call
                },
                {
                    WarpDispatcher<Transform, CubicFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReflect101, T>::call
                }
            };

            funcs[interpolation][borderMode](static_cast< PtrStepSz<T> >(src), static_cast< PtrStepSz<T> >(srcWhole), xoff, yoff,
                static_cast< PtrStepSz<T> >(dst), borderValue, stream, cc20);
        }

        template <typename T> void warpAffine_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation,
                                                  int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_warpMat), coeffs, 2 * 3 * sizeof(float)) );

            warp_caller<AffineTransform, T>(src, srcWhole, xoff, yoff, dst, interpolation, borderMode, borderValue, stream, cc20);
        }

        template void warpAffine_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<uchar2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpAffine_gpu<schar>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<char2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<char3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<char4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<ushort2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<short2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpAffine_gpu<int >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<int2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<int3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<int4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<float2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template <typename T> void warpPerspective_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation,
                                                  int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_warpMat), coeffs, 3 * 3 * sizeof(float)) );

            warp_caller<PerspectiveTransform, T>(src, srcWhole, xoff, yoff, dst, interpolation, borderMode, borderValue, stream, cc20);
        }

        template void warpPerspective_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<uchar2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpPerspective_gpu<schar>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<char2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<char3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<char4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<ushort2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<short2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpPerspective_gpu<int >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<int2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<int3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<int4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<float2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
