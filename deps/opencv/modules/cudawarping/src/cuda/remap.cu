#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/filters.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        template <typename Ptr2D, typename T> __global__ void remap(const Ptr2D src, const PtrStepf mapx, const PtrStepf mapy, PtrStepSz<T> dst)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                const float xcoo = mapx.ptr(y)[x];
                const float ycoo = mapy.ptr(y)[x];

                dst.ptr(y)[x] = saturate_cast<T>(src(ycoo, xcoo));
            }
        }

        template <template <typename> class Filter, template <typename> class B, typename T> struct RemapDispatcherStream
        {
            static void call(PtrStepSz<T> src, PtrStepSzf mapx, PtrStepSzf mapy, PtrStepSz<T> dst, const float* borderValue, hipStream_t stream, bool)
            {
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader< PtrStep<T>, B<work_type> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, B<work_type> > > filter_src(brdSrc);

                remap<<<grid, block, 0, stream>>>(filter_src, mapx, mapy, dst);
                cudaSafeCall( hipGetLastError() );
            }
        };

        template <template <typename> class Filter, template <typename> class B, typename T> struct RemapDispatcherNonStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSzf mapx, PtrStepSzf mapy, PtrStepSz<T> dst, const float* borderValue, bool)
            {
                (void)srcWhole;
                (void)xoff;
                (void)yoff;
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader< PtrStep<T>, B<work_type> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, B<work_type> > > filter_src(brdSrc);

                remap<<<grid, block>>>(filter_src, mapx, mapy, dst);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        #define OPENCV_CUDA_IMPLEMENT_REMAP_TEX(type) \
            texture< type , hipTextureType2D> tex_remap_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
            struct tex_remap_ ## type ## _reader \
            { \
                typedef type elem_type; \
                typedef int index_type; \
                int xoff, yoff; \
                tex_remap_ ## type ## _reader (int xoff_, int yoff_) : xoff(xoff_), yoff(yoff_) {} \
                __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
                { \
                    return tex2D(tex_remap_ ## type , x + xoff, y + yoff); \
                } \
            }; \
            template <template <typename> class Filter, template <typename> class B> struct RemapDispatcherNonStream<Filter, B, type> \
            { \
                static void call(PtrStepSz< type > src, PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSzf mapx, PtrStepSzf mapy, \
                    PtrStepSz< type > dst, const float* borderValue, bool cc20) \
                { \
                    typedef typename TypeVec<float, VecTraits< type >::cn>::vec_type work_type; \
                    dim3 block(32, cc20 ? 8 : 4); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_remap_ ## type , srcWhole); \
                    tex_remap_ ## type ##_reader texSrc(xoff, yoff); \
                    B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue)); \
                    BorderReader< tex_remap_ ## type ##_reader, B<work_type> > brdSrc(texSrc, brd); \
                    Filter< BorderReader< tex_remap_ ## type ##_reader, B<work_type> > > filter_src(brdSrc); \
                    remap<<<grid, block>>>(filter_src, mapx, mapy, dst); \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            }; \
            template <template <typename> class Filter> struct RemapDispatcherNonStream<Filter, BrdReplicate, type> \
            { \
                static void call(PtrStepSz< type > src, PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSzf mapx, PtrStepSzf mapy, \
                    PtrStepSz< type > dst, const float*, bool) \
                { \
                    dim3 block(32, 8); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_remap_ ## type , srcWhole); \
                    tex_remap_ ## type ##_reader texSrc(xoff, yoff); \
                    if (srcWhole.cols == src.cols && srcWhole.rows == src.rows) \
                    { \
                        Filter< tex_remap_ ## type ##_reader > filter_src(texSrc); \
                        remap<<<grid, block>>>(filter_src, mapx, mapy, dst); \
                    } \
                    else \
                    { \
                        BrdReplicate<type> brd(src.rows, src.cols); \
                        BorderReader< tex_remap_ ## type ##_reader, BrdReplicate<type> > brdSrc(texSrc, brd); \
                        Filter< BorderReader< tex_remap_ ## type ##_reader, BrdReplicate<type> > > filter_src(brdSrc); \
                        remap<<<grid, block>>>(filter_src, mapx, mapy, dst); \
                    } \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            };

        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(uchar)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(uchar2)
        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(uchar4)

        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(schar)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(char2)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(char4)

        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(ushort)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(ushort2)
        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(ushort4)

        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(short)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(short2)
        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(short4)

        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(int)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(int2)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(int4)

        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(float)
        //OPENCV_CUDA_IMPLEMENT_REMAP_TEX(float2)
        OPENCV_CUDA_IMPLEMENT_REMAP_TEX(float4)

        #undef OPENCV_CUDA_IMPLEMENT_REMAP_TEX

        template <template <typename> class Filter, template <typename> class B, typename T> struct RemapDispatcher
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSzf mapx, PtrStepSzf mapy,
                PtrStepSz<T> dst, const float* borderValue, hipStream_t stream, bool cc20)
            {
                if (stream == 0)
                    RemapDispatcherNonStream<Filter, B, T>::call(src, srcWhole, xoff, yoff, mapx, mapy, dst, borderValue, cc20);
                else
                    RemapDispatcherStream<Filter, B, T>::call(src, mapx, mapy, dst, borderValue, stream, cc20);
            }
        };

        template <typename T> void remap_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap,
            PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            typedef void (*caller_t)(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap,
                PtrStepSz<T> dst, const float* borderValue, hipStream_t stream, bool cc20);

            static const caller_t callers[3][5] =
            {
                {
                    RemapDispatcher<PointFilter, BrdConstant, T>::call,
                    RemapDispatcher<PointFilter, BrdReplicate, T>::call,
                    RemapDispatcher<PointFilter, BrdReflect, T>::call,
                    RemapDispatcher<PointFilter, BrdWrap, T>::call,
                    RemapDispatcher<PointFilter, BrdReflect101, T>::call
                },
                {
                    RemapDispatcher<LinearFilter, BrdConstant, T>::call,
                    RemapDispatcher<LinearFilter, BrdReplicate, T>::call,
                    RemapDispatcher<LinearFilter, BrdReflect, T>::call,
                    RemapDispatcher<LinearFilter, BrdWrap, T>::call,
                    RemapDispatcher<LinearFilter, BrdReflect101, T>::call
                },
                {
                    RemapDispatcher<CubicFilter, BrdConstant, T>::call,
                    RemapDispatcher<CubicFilter, BrdReplicate, T>::call,
                    RemapDispatcher<CubicFilter, BrdReflect, T>::call,
                    RemapDispatcher<CubicFilter, BrdWrap, T>::call,
                    RemapDispatcher<CubicFilter, BrdReflect101, T>::call
                }
            };

            callers[interpolation][borderMode](static_cast< PtrStepSz<T> >(src), static_cast< PtrStepSz<T> >(srcWhole), xoff, yoff, xmap, ymap,
                static_cast< PtrStepSz<T> >(dst), borderValue, stream, cc20);
        }

        template void remap_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<uchar2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void remap_gpu<schar>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<char2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<char3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<char4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void remap_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<ushort2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void remap_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<short2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void remap_gpu<int >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<int2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<int3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<int4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void remap_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void remap_gpu<float2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void remap_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzf xmap, PtrStepSzf ymap, PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
