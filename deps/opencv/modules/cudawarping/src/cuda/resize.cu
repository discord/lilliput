#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <cfloat>
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/filters.hpp"

namespace cv { namespace cuda { namespace device
{
    // kernels

    template <typename T> __global__ void resize_nearest(const PtrStep<T> src, PtrStepSz<T> dst, const float fy, const float fx)
    {
        const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
        const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

        if (dst_x < dst.cols && dst_y < dst.rows)
        {
            const float src_x = dst_x * fx;
            const float src_y = dst_y * fy;

            dst(dst_y, dst_x) = src(__float2int_rz(src_y), __float2int_rz(src_x));
        }
    }

    template <typename T> __global__ void resize_linear(const PtrStepSz<T> src, PtrStepSz<T> dst, const float fy, const float fx)
    {
        typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

        const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
        const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

        if (dst_x < dst.cols && dst_y < dst.rows)
        {
            const float src_x = dst_x * fx;
            const float src_y = dst_y * fy;

            work_type out = VecTraits<work_type>::all(0);

            const int x1 = __float2int_rd(src_x);
            const int y1 = __float2int_rd(src_y);
            const int x2 = x1 + 1;
            const int y2 = y1 + 1;
            const int x2_read = ::min(x2, src.cols - 1);
            const int y2_read = ::min(y2, src.rows - 1);

            T src_reg = src(y1, x1);
            out = out + src_reg * ((x2 - src_x) * (y2 - src_y));

            src_reg = src(y1, x2_read);
            out = out + src_reg * ((src_x - x1) * (y2 - src_y));

            src_reg = src(y2_read, x1);
            out = out + src_reg * ((x2 - src_x) * (src_y - y1));

            src_reg = src(y2_read, x2_read);
            out = out + src_reg * ((src_x - x1) * (src_y - y1));

            dst(dst_y, dst_x) = saturate_cast<T>(out);
        }
    }

    template <class Ptr2D, typename T> __global__ void resize(const Ptr2D src, PtrStepSz<T> dst, const float fy, const float fx)
    {
        const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
        const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

        if (dst_x < dst.cols && dst_y < dst.rows)
        {
            const float src_x = dst_x * fx;
            const float src_y = dst_y * fy;

            dst(dst_y, dst_x) = src(src_y, src_x);
        }
    }

    template <typename Ptr2D, typename T> __global__ void resize_area(const Ptr2D src, PtrStepSz<T> dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < dst.cols && y < dst.rows)
        {
            dst(y, x) = src(y, x);
        }
    }

    // textures

    template <typename T> struct TextureAccessor;

    #define OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(type) \
        texture<type, hipTextureType2D, hipReadModeElementType> tex_resize_##type (0, hipFilterModePoint, hipAddressModeClamp); \
        template <> struct TextureAccessor<type> \
        { \
            typedef type elem_type; \
            typedef int index_type; \
            int xoff; \
            int yoff; \
            __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
            { \
                return tex2D(tex_resize_##type, x + xoff, y + yoff); \
            } \
            __host__ static void bind(const PtrStepSz<type>& mat) \
            { \
                bindTexture(&tex_resize_##type, mat); \
            } \
        };

    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(uchar)
    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(uchar4)

    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(ushort)
    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(ushort4)

    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(short)
    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(short4)

    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(float)
    OPENCV_CUDA_IMPLEMENT_RESIZE_TEX(float4)

    #undef OPENCV_CUDA_IMPLEMENT_RESIZE_TEX

    template <typename T>
    TextureAccessor<T> texAccessor(const PtrStepSz<T>& mat, int yoff, int xoff)
    {
        TextureAccessor<T>::bind(mat);

        TextureAccessor<T> t;
        t.xoff = xoff;
        t.yoff = yoff;

        return t;
    }

    // callers for nearest interpolation

    template <typename T>
    void call_resize_nearest_glob(const PtrStepSz<T>& src, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        resize_nearest<<<grid, block, 0, stream>>>(src, dst, fy, fx);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T>
    void call_resize_nearest_tex(const PtrStepSz<T>& /*src*/, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        resize<<<grid, block>>>(texAccessor(srcWhole, yoff, xoff), dst, fy, fx);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

    // callers for linear interpolation

    template <typename T>
    void call_resize_linear_glob(const PtrStepSz<T>& src, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        resize_linear<<<grid, block, 0, stream>>>(src, dst, fy, fx);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T>
    void call_resize_linear_tex(const PtrStepSz<T>& src, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        if (srcWhole.data == src.data)
        {
            TextureAccessor<T> texSrc = texAccessor(src, 0, 0);
            LinearFilter< TextureAccessor<T> > filteredSrc(texSrc);

            resize<<<grid, block>>>(filteredSrc, dst, fy, fx);
        }
        else
        {
            TextureAccessor<T> texSrc = texAccessor(srcWhole, yoff, xoff);

            BrdReplicate<T> brd(src.rows, src.cols);
            BorderReader<TextureAccessor<T>, BrdReplicate<T> > brdSrc(texSrc, brd);
            LinearFilter< BorderReader<TextureAccessor<T>, BrdReplicate<T> > > filteredSrc(brdSrc);

            resize<<<grid, block>>>(filteredSrc, dst, fy, fx);
        }

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

    // callers for cubic interpolation

    template <typename T>
    void call_resize_cubic_glob(const PtrStepSz<T>& src, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        BrdReplicate<T> brd(src.rows, src.cols);
        BorderReader< PtrStep<T>, BrdReplicate<T> > brdSrc(src, brd);
        CubicFilter< BorderReader< PtrStep<T>, BrdReplicate<T> > > filteredSrc(brdSrc);

        resize<<<grid, block, 0, stream>>>(filteredSrc, dst, fy, fx);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T>
    void call_resize_cubic_tex(const PtrStepSz<T>& src, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

        if (srcWhole.data == src.data)
        {
            TextureAccessor<T> texSrc = texAccessor(src, 0, 0);
            CubicFilter< TextureAccessor<T> > filteredSrc(texSrc);

            resize<<<grid, block>>>(filteredSrc, dst, fy, fx);
        }
        else
        {
            TextureAccessor<T> texSrc = texAccessor(srcWhole, yoff, xoff);

            BrdReplicate<T> brd(src.rows, src.cols);
            BorderReader<TextureAccessor<T>, BrdReplicate<T> > brdSrc(texSrc, brd);
            CubicFilter< BorderReader<TextureAccessor<T>, BrdReplicate<T> > > filteredSrc(brdSrc);

            resize<<<grid, block>>>(filteredSrc, dst, fy, fx);
        }

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

    // ResizeNearestDispatcher

    template <typename T> struct ResizeNearestDispatcher
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& /*srcWhole*/, int /*yoff*/, int /*xoff*/, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            call_resize_nearest_glob(src, dst, fy, fx, stream);
        }
    };

    template <typename T> struct SelectImplForNearest
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            if (stream)
                call_resize_nearest_glob(src, dst, fy, fx, stream);
            else
            {
                if (fx > 1 || fy > 1)
                    call_resize_nearest_glob(src, dst, fy, fx, 0);
                else
                    call_resize_nearest_tex(src, srcWhole, yoff, xoff, dst, fy, fx);
            }
        }
    };

    template <> struct ResizeNearestDispatcher<uchar> : SelectImplForNearest<uchar> {};
    template <> struct ResizeNearestDispatcher<uchar4> : SelectImplForNearest<uchar4> {};

    template <> struct ResizeNearestDispatcher<ushort> : SelectImplForNearest<ushort> {};
    template <> struct ResizeNearestDispatcher<ushort4> : SelectImplForNearest<ushort4> {};

    template <> struct ResizeNearestDispatcher<short> : SelectImplForNearest<short> {};
    template <> struct ResizeNearestDispatcher<short4> : SelectImplForNearest<short4> {};

    template <> struct ResizeNearestDispatcher<float> : SelectImplForNearest<float> {};
    template <> struct ResizeNearestDispatcher<float4> : SelectImplForNearest<float4> {};

    // ResizeLinearDispatcher

    template <typename T> struct ResizeLinearDispatcher
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& /*srcWhole*/, int /*yoff*/, int /*xoff*/, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            call_resize_linear_glob(src, dst, fy, fx, stream);
        }
    };

    template <typename T> struct SelectImplForLinear
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            if (stream)
                call_resize_linear_glob(src, dst, fy, fx, stream);
            else
            {
                if (fx > 1 || fy > 1)
                    call_resize_linear_glob(src, dst, fy, fx, 0);
                else
                    call_resize_linear_tex(src, srcWhole, yoff, xoff, dst, fy, fx);
            }
        }
    };

    template <> struct ResizeLinearDispatcher<uchar> : SelectImplForLinear<uchar> {};
    template <> struct ResizeLinearDispatcher<uchar4> : SelectImplForLinear<uchar4> {};

    template <> struct ResizeLinearDispatcher<ushort> : SelectImplForLinear<ushort> {};
    template <> struct ResizeLinearDispatcher<ushort4> : SelectImplForLinear<ushort4> {};

    template <> struct ResizeLinearDispatcher<short> : SelectImplForLinear<short> {};
    template <> struct ResizeLinearDispatcher<short4> : SelectImplForLinear<short4> {};

    template <> struct ResizeLinearDispatcher<float> : SelectImplForLinear<float> {};
    template <> struct ResizeLinearDispatcher<float4> : SelectImplForLinear<float4> {};

    // ResizeCubicDispatcher

    template <typename T> struct ResizeCubicDispatcher
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& /*srcWhole*/, int /*yoff*/, int /*xoff*/, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            call_resize_cubic_glob(src, dst, fy, fx, stream);
        }
    };

    template <typename T> struct SelectImplForCubic
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            if (stream)
                call_resize_cubic_glob(src, dst, fy, fx, stream);
            else
                call_resize_cubic_tex(src, srcWhole, yoff, xoff, dst, fy, fx);
        }
    };

    template <> struct ResizeCubicDispatcher<uchar> : SelectImplForCubic<uchar> {};
    template <> struct ResizeCubicDispatcher<uchar4> : SelectImplForCubic<uchar4> {};

    template <> struct ResizeCubicDispatcher<ushort> : SelectImplForCubic<ushort> {};
    template <> struct ResizeCubicDispatcher<ushort4> : SelectImplForCubic<ushort4> {};

    template <> struct ResizeCubicDispatcher<short> : SelectImplForCubic<short> {};
    template <> struct ResizeCubicDispatcher<short4> : SelectImplForCubic<short4> {};

    template <> struct ResizeCubicDispatcher<float> : SelectImplForCubic<float> {};
    template <> struct ResizeCubicDispatcher<float4> : SelectImplForCubic<float4> {};

    // ResizeAreaDispatcher

    template <typename T> struct ResizeAreaDispatcher
    {
        static void call(const PtrStepSz<T>& src, const PtrStepSz<T>&, int, int, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream)
        {
            const int iscale_x = (int) round(fx);
            const int iscale_y = (int) round(fy);

            const dim3 block(32, 8);
            const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

            if (std::abs(fx - iscale_x) < FLT_MIN && std::abs(fy - iscale_y) < FLT_MIN)
            {
                BrdConstant<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdConstant<T> > brdSrc(src, brd);
                IntegerAreaFilter< BorderReader< PtrStep<T>, BrdConstant<T> > > filteredSrc(brdSrc, fx, fy);

                resize_area<<<grid, block, 0, stream>>>(filteredSrc, dst);
            }
            else
            {
                BrdConstant<T> brd(src.rows, src.cols);
                BorderReader< PtrStep<T>, BrdConstant<T> > brdSrc(src, brd);
                AreaFilter< BorderReader< PtrStep<T>, BrdConstant<T> > > filteredSrc(brdSrc, fx, fy);

                resize_area<<<grid, block, 0, stream>>>(filteredSrc, dst);
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    };

    // resize

    template <typename T> void resize(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream)
    {
        typedef void (*func_t)(const PtrStepSz<T>& src, const PtrStepSz<T>& srcWhole, int yoff, int xoff, const PtrStepSz<T>& dst, float fy, float fx, hipStream_t stream);
        static const func_t funcs[4] =
        {
            ResizeNearestDispatcher<T>::call,
            ResizeLinearDispatcher<T>::call,
            ResizeCubicDispatcher<T>::call,
            ResizeAreaDispatcher<T>::call
        };

        // change to linear if area interpolation upscaling
        if (interpolation == 3 && (fx <= 1.f || fy <= 1.f))
            interpolation = 1;

        funcs[interpolation](static_cast< PtrStepSz<T> >(src), static_cast< PtrStepSz<T> >(srcWhole), yoff, xoff, static_cast< PtrStepSz<T> >(dst), fy, fx, stream);
    }

    template void resize<uchar >(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<uchar3>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<uchar4>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);

    template void resize<ushort >(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<ushort3>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<ushort4>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);

    template void resize<short >(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<short3>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<short4>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);

    template void resize<float >(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<float3>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
    template void resize<float4>(const PtrStepSzb& src, const PtrStepSzb& srcWhole, int yoff, int xoff, const PtrStepSzb& dst, float fy, float fx, int interpolation, hipStream_t stream);
}}}

#endif /* CUDA_DISABLER */
