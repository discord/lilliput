#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        template <typename T> __global__ void pyrUp(const PtrStepSz<T> src, PtrStepSz<T> dst)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type sum_t;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            __shared__ sum_t s_srcPatch[10][10];
            __shared__ sum_t s_dstPatch[20][16];

            if (threadIdx.x < 10 && threadIdx.y < 10)
            {
                int srcx = static_cast<int>((blockIdx.x * blockDim.x) / 2 + threadIdx.x) - 1;
                int srcy = static_cast<int>((blockIdx.y * blockDim.y) / 2 + threadIdx.y) - 1;

                srcx = ::abs(srcx);
                srcx = ::min(src.cols - 1, srcx);

                srcy = ::abs(srcy);
                srcy = ::min(src.rows - 1, srcy);

                s_srcPatch[threadIdx.y][threadIdx.x] = saturate_cast<sum_t>(src(srcy, srcx));
            }

            __syncthreads();

            sum_t sum = VecTraits<sum_t>::all(0);

            const int evenFlag = static_cast<int>((threadIdx.x & 1) == 0);
            const int oddFlag  = static_cast<int>((threadIdx.x & 1) != 0);
            const bool eveny = ((threadIdx.y & 1) == 0);
            const int tidx = threadIdx.x;

            if (eveny)
            {
                sum = sum + (evenFlag * 0.0625f) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 2) >> 1)];
                sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx - 1) >> 1)];
                sum = sum + (evenFlag * 0.375f ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx    ) >> 1)];
                sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 1) >> 1)];
                sum = sum + (evenFlag * 0.0625f) * s_srcPatch[1 + (threadIdx.y >> 1)][1 + ((tidx + 2) >> 1)];
            }

            s_dstPatch[2 + threadIdx.y][threadIdx.x] = sum;

            if (threadIdx.y < 2)
            {
                sum = VecTraits<sum_t>::all(0);

                if (eveny)
                {
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[0][1 + ((tidx - 2) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[0][1 + ((tidx - 1) >> 1)];
                    sum = sum + (evenFlag * 0.375f ) * s_srcPatch[0][1 + ((tidx    ) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[0][1 + ((tidx + 1) >> 1)];
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[0][1 + ((tidx + 2) >> 1)];
                }

                s_dstPatch[threadIdx.y][threadIdx.x] = sum;
            }

            if (threadIdx.y > 13)
            {
                sum = VecTraits<sum_t>::all(0);

                if (eveny)
                {
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[9][1 + ((tidx - 2) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[9][1 + ((tidx - 1) >> 1)];
                    sum = sum + (evenFlag * 0.375f ) * s_srcPatch[9][1 + ((tidx    ) >> 1)];
                    sum = sum + ( oddFlag * 0.25f  ) * s_srcPatch[9][1 + ((tidx + 1) >> 1)];
                    sum = sum + (evenFlag * 0.0625f) * s_srcPatch[9][1 + ((tidx + 2) >> 1)];
                }

                s_dstPatch[4 + threadIdx.y][threadIdx.x] = sum;
            }

            __syncthreads();

            sum = VecTraits<sum_t>::all(0);

            const int tidy = threadIdx.y;

            sum = sum + 0.0625f * s_dstPatch[2 + tidy - 2][threadIdx.x];
            sum = sum + 0.25f   * s_dstPatch[2 + tidy - 1][threadIdx.x];
            sum = sum + 0.375f  * s_dstPatch[2 + tidy    ][threadIdx.x];
            sum = sum + 0.25f   * s_dstPatch[2 + tidy + 1][threadIdx.x];
            sum = sum + 0.0625f * s_dstPatch[2 + tidy + 2][threadIdx.x];

            if (x < dst.cols && y < dst.rows)
                dst(y, x) = saturate_cast<T>(4.0f * sum);
        }

        template <typename T> void pyrUp_caller(PtrStepSz<T> src, PtrStepSz<T> dst, hipStream_t stream)
        {
            const dim3 block(16, 16);
            const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

            pyrUp<<<grid, block, 0, stream>>>(src, dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <typename T> void pyrUp_gpu(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
        {
            pyrUp_caller<T>(static_cast< PtrStepSz<T> >(src), static_cast< PtrStepSz<T> >(dst), stream);
        }

        template void pyrUp_gpu<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<uchar2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<uchar3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<uchar4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        //template void pyrUp_gpu<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<char2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<char3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<char4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrUp_gpu<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<ushort2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<ushort3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<ushort4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrUp_gpu<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<short2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<short3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<short4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        //template void pyrUp_gpu<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<int2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<int3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<int4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);

        template void pyrUp_gpu<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        //template void pyrUp_gpu<float2>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<float3>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
        template void pyrUp_gpu<float4>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev

#endif /* CUDA_DISABLER */
