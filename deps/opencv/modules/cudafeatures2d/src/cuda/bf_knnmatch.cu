#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/vec_distance.hpp"
#include "opencv2/core/cuda/datamov_utils.hpp"
#include "opencv2/core/cuda/warp_shuffle.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace bf_knnmatch
    {
        ///////////////////////////////////////////////////////////////////////////////
        // Reduction

        template <int BLOCK_SIZE>
        __device__ void findBestMatch(float& bestDistance1, float& bestDistance2,
                                      int& bestTrainIdx1, int& bestTrainIdx2,
                                      float* s_distance, int* s_trainIdx)
        {
        #if __CUDA_ARCH__ >= 300
            (void) s_distance;
            (void) s_trainIdx;

            float d1, d2;
            int i1, i2;

            #pragma unroll
            for (int i = BLOCK_SIZE / 2; i >= 1; i /= 2)
            {
                d1 = shfl_down(bestDistance1, i, BLOCK_SIZE);
                d2 = shfl_down(bestDistance2, i, BLOCK_SIZE);
                i1 = shfl_down(bestTrainIdx1, i, BLOCK_SIZE);
                i2 = shfl_down(bestTrainIdx2, i, BLOCK_SIZE);

                if (bestDistance1 < d1)
                {
                    if (d1 < bestDistance2)
                    {
                        bestDistance2 = d1;
                        bestTrainIdx2 = i1;
                    }
                }
                else
                {
                    bestDistance2 = bestDistance1;
                    bestTrainIdx2 = bestTrainIdx1;

                    bestDistance1 = d1;
                    bestTrainIdx1 = i1;

                    if (d2 < bestDistance2)
                    {
                        bestDistance2 = d2;
                        bestTrainIdx2 = i2;
                    }
                }
            }
        #else
            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;

            s_distance += threadIdx.y * BLOCK_SIZE;
            s_trainIdx += threadIdx.y * BLOCK_SIZE;

            s_distance[threadIdx.x] = bestDistance1;
            s_trainIdx[threadIdx.x] = bestTrainIdx1;

            __syncthreads();

            if (threadIdx.x == 0)
            {
                #pragma unroll
                for (int i = 0; i < BLOCK_SIZE; ++i)
                {
                    float val = s_distance[i];

                    if (val < myBestDistance1)
                    {
                        myBestDistance2 = myBestDistance1;
                        myBestTrainIdx2 = myBestTrainIdx1;

                        myBestDistance1 = val;
                        myBestTrainIdx1 = s_trainIdx[i];
                    }
                    else if (val < myBestDistance2)
                    {
                        myBestDistance2 = val;
                        myBestTrainIdx2 = s_trainIdx[i];
                    }
                }
            }

            __syncthreads();

            s_distance[threadIdx.x] = bestDistance2;
            s_trainIdx[threadIdx.x] = bestTrainIdx2;

            __syncthreads();

            if (threadIdx.x == 0)
            {
                #pragma unroll
                for (int i = 0; i < BLOCK_SIZE; ++i)
                {
                    float val = s_distance[i];

                    if (val < myBestDistance2)
                    {
                        myBestDistance2 = val;
                        myBestTrainIdx2 = s_trainIdx[i];
                    }
                }
            }

            bestDistance1 = myBestDistance1;
            bestDistance2 = myBestDistance2;

            bestTrainIdx1 = myBestTrainIdx1;
            bestTrainIdx2 = myBestTrainIdx2;
        #endif
        }

        template <int BLOCK_SIZE>
        __device__ void findBestMatch(float& bestDistance1, float& bestDistance2,
                                       int& bestTrainIdx1, int& bestTrainIdx2,
                                       int& bestImgIdx1, int& bestImgIdx2,
                                       float* s_distance, int* s_trainIdx, int* s_imgIdx)
        {
        #if __CUDA_ARCH__ >= 300
            (void) s_distance;
            (void) s_trainIdx;
            (void) s_imgIdx;

            float d1, d2;
            int i1, i2;
            int j1, j2;

            #pragma unroll
            for (int i = BLOCK_SIZE / 2; i >= 1; i /= 2)
            {
                d1 = shfl_down(bestDistance1, i, BLOCK_SIZE);
                d2 = shfl_down(bestDistance2, i, BLOCK_SIZE);
                i1 = shfl_down(bestTrainIdx1, i, BLOCK_SIZE);
                i2 = shfl_down(bestTrainIdx2, i, BLOCK_SIZE);
                j1 = shfl_down(bestImgIdx1, i, BLOCK_SIZE);
                j2 = shfl_down(bestImgIdx2, i, BLOCK_SIZE);

                if (bestDistance1 < d1)
                {
                    if (d1 < bestDistance2)
                    {
                        bestDistance2 = d1;
                        bestTrainIdx2 = i1;
                        bestImgIdx2 = j1;
                    }
                }
                else
                {
                    bestDistance2 = bestDistance1;
                    bestTrainIdx2 = bestTrainIdx1;
                    bestImgIdx2 = bestImgIdx1;

                    bestDistance1 = d1;
                    bestTrainIdx1 = i1;
                    bestImgIdx1 = j1;

                    if (d2 < bestDistance2)
                    {
                        bestDistance2 = d2;
                        bestTrainIdx2 = i2;
                        bestImgIdx2 = j2;
                    }
                }
            }
        #else
            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;
            int myBestImgIdx1 = -1;
            int myBestImgIdx2 = -1;

            s_distance += threadIdx.y * BLOCK_SIZE;
            s_trainIdx += threadIdx.y * BLOCK_SIZE;
            s_imgIdx   += threadIdx.y * BLOCK_SIZE;

            s_distance[threadIdx.x] = bestDistance1;
            s_trainIdx[threadIdx.x] = bestTrainIdx1;
            s_imgIdx[threadIdx.x]   = bestImgIdx1;

            __syncthreads();

            if (threadIdx.x == 0)
            {
                #pragma unroll
                for (int i = 0; i < BLOCK_SIZE; ++i)
                {
                    float val = s_distance[i];

                    if (val < myBestDistance1)
                    {
                        myBestDistance2 = myBestDistance1;
                        myBestTrainIdx2 = myBestTrainIdx1;
                        myBestImgIdx2   = myBestImgIdx1;

                        myBestDistance1 = val;
                        myBestTrainIdx1 = s_trainIdx[i];
                        myBestImgIdx1   = s_imgIdx[i];
                    }
                    else if (val < myBestDistance2)
                    {
                        myBestDistance2 = val;
                        myBestTrainIdx2 = s_trainIdx[i];
                        myBestImgIdx2   = s_imgIdx[i];
                    }
                }
            }

            __syncthreads();

            s_distance[threadIdx.x] = bestDistance2;
            s_trainIdx[threadIdx.x] = bestTrainIdx2;
            s_imgIdx[threadIdx.x]   = bestImgIdx2;

            __syncthreads();

            if (threadIdx.x == 0)
            {
                #pragma unroll
                for (int i = 0; i < BLOCK_SIZE; ++i)
                {
                    float val = s_distance[i];

                    if (val < myBestDistance2)
                    {
                        myBestDistance2 = val;
                        myBestTrainIdx2 = s_trainIdx[i];
                        myBestImgIdx2   = s_imgIdx[i];
                    }
                }
            }

            bestDistance1 = myBestDistance1;
            bestDistance2 = myBestDistance2;

            bestTrainIdx1 = myBestTrainIdx1;
            bestTrainIdx2 = myBestTrainIdx2;

            bestImgIdx1 = myBestImgIdx1;
            bestImgIdx2 = myBestImgIdx2;
        #endif
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match Unrolled Cached

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename T, typename U>
        __device__ void loadQueryToSmem(int queryIdx, const PtrStepSz<T>& query, U* s_query)
        {
            #pragma unroll
            for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;
                s_query[threadIdx.y * MAX_DESC_LEN + loadX] = loadX < query.cols ? query.ptr(::min(queryIdx, query.rows - 1))[loadX] : 0;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __device__ void loopUnrolledCached(int queryIdx, const PtrStepSz<T>& query, int imgIdx, const PtrStepSz<T>& train, const Mask& mask,
                                           typename Dist::value_type* s_query, typename Dist::value_type* s_train,
                                           float& bestDistance1, float& bestDistance2,
                                           int& bestTrainIdx1, int& bestTrainIdx2,
                                           int& bestImgIdx1, int& bestImgIdx2)
        {
            for (int t = 0, endt = (train.rows + BLOCK_SIZE - 1) / BLOCK_SIZE; t < endt; ++t)
            {
                Dist dist;

                #pragma unroll
                for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
                {
                    const int loadX = threadIdx.x + i * BLOCK_SIZE;

                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                    if (loadX < train.cols)
                    {
                        T val;

                        ForceGlob<T>::Load(train.ptr(::min(t * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                        s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                    }

                    __syncthreads();

                    #pragma unroll
                    for (int j = 0; j < BLOCK_SIZE; ++j)
                        dist.reduceIter(s_query[threadIdx.y * MAX_DESC_LEN + i * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                    __syncthreads();
                }

                typename Dist::result_type distVal = dist;

                const int trainIdx = t * BLOCK_SIZE + threadIdx.x;

                if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx))
                {
                    if (distVal < bestDistance1)
                    {
                        bestImgIdx2   = bestImgIdx1;
                        bestDistance2 = bestDistance1;
                        bestTrainIdx2 = bestTrainIdx1;

                        bestImgIdx1   = imgIdx;
                        bestDistance1 = distVal;
                        bestTrainIdx1 = trainIdx;
                    }
                    else if (distVal < bestDistance2)
                    {
                        bestImgIdx2   = imgIdx;
                        bestDistance2 = distVal;
                        bestTrainIdx2 = trainIdx;
                    }
                }
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolledCached(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, int2* bestTrainIdx, float2* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * MAX_DESC_LEN);

            loadQueryToSmem<BLOCK_SIZE, MAX_DESC_LEN>(queryIdx, query, s_query);

            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;

            loopUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, 0, train, mask, s_query, s_train, myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestTrainIdx1, myBestTrainIdx2);

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, s_distance, s_trainIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = make_int2(myBestTrainIdx1, myBestTrainIdx2);
                bestDistance[queryIdx] = make_float2(myBestDistance1, myBestDistance2);
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolledCached(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                                 const PtrStepSz<int2>& trainIdx, const PtrStepSz<float2>& distance,
                                 hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (BLOCK_SIZE * (MAX_DESC_LEN >= BLOCK_SIZE ? MAX_DESC_LEN : BLOCK_SIZE) + BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, trainIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolledCached(const PtrStepSz<T> query, const PtrStepSz<T>* trains, int n, const Mask mask, int2* bestTrainIdx, int2* bestImgIdx, float2* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * MAX_DESC_LEN);

            loadQueryToSmem<BLOCK_SIZE, MAX_DESC_LEN>(queryIdx, query, s_query);

            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;
            int myBestImgIdx1 = -1;
            int myBestImgIdx2 = -1;

            Mask m = mask;

            for (int imgIdx = 0; imgIdx < n; ++imgIdx)
            {
                const PtrStepSz<T> train = trains[imgIdx];
                m.next();
                loopUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, imgIdx, train, m, s_query, s_train, myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestImgIdx1, myBestImgIdx2);
            }

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);
            int* s_imgIdx = (int*)(smem + 2 * BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestImgIdx1, myBestImgIdx2, s_distance, s_trainIdx, s_imgIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = make_int2(myBestTrainIdx1, myBestTrainIdx2);
                bestImgIdx[queryIdx] = make_int2(myBestImgIdx1, myBestImgIdx2);
                bestDistance[queryIdx] = make_float2(myBestDistance1, myBestDistance2);
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolledCached(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                                 const PtrStepSz<int2>& trainIdx, const PtrStepSz<int2>& imgIdx, const PtrStepSz<float2>& distance,
                                 hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (BLOCK_SIZE * (MAX_DESC_LEN >= 2 * BLOCK_SIZE ? MAX_DESC_LEN : 2 * BLOCK_SIZE) + BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, trains, n, mask, trainIdx.data, imgIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match Unrolled

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __device__ void loopUnrolled(int queryIdx, const PtrStepSz<T>& query, int imgIdx, const PtrStepSz<T>& train, const Mask& mask,
                                     typename Dist::value_type* s_query, typename Dist::value_type* s_train,
                                     float& bestDistance1, float& bestDistance2,
                                     int& bestTrainIdx1, int& bestTrainIdx2,
                                     int& bestImgIdx1, int& bestImgIdx2)
        {
            for (int t = 0, endt = (train.rows + BLOCK_SIZE - 1) / BLOCK_SIZE; t < endt; ++t)
            {
                Dist dist;

                #pragma unroll
                for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
                {
                    const int loadX = threadIdx.x + i * BLOCK_SIZE;

                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                    if (loadX < query.cols)
                    {
                        T val;

                        ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                        s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                        ForceGlob<T>::Load(train.ptr(::min(t * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                        s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                    }

                    __syncthreads();

                    #pragma unroll
                    for (int j = 0; j < BLOCK_SIZE; ++j)
                        dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                    __syncthreads();
                }

                typename Dist::result_type distVal = dist;

                const int trainIdx = t * BLOCK_SIZE + threadIdx.x;

                if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx))
                {
                    if (distVal < bestDistance1)
                    {
                        bestImgIdx2   = bestImgIdx1;
                        bestDistance2 = bestDistance1;
                        bestTrainIdx2 = bestTrainIdx1;

                        bestImgIdx1   = imgIdx;
                        bestDistance1 = distVal;
                        bestTrainIdx1 = trainIdx;
                    }
                    else if (distVal < bestDistance2)
                    {
                        bestImgIdx2   = imgIdx;
                        bestDistance2 = distVal;
                        bestTrainIdx2 = trainIdx;
                    }
                }
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolled(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, int2* bestTrainIdx, float2* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;

            loopUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, 0, train, mask, s_query, s_train, myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestTrainIdx1, myBestTrainIdx2);

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, s_distance, s_trainIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = make_int2(myBestTrainIdx1, myBestTrainIdx2);
                bestDistance[queryIdx] = make_float2(myBestDistance1, myBestDistance2);
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                           const PtrStepSz<int2>& trainIdx, const PtrStepSz<float2>& distance,
                           hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, trainIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolled(const PtrStepSz<T> query, const PtrStepSz<T>* trains, int n, const Mask mask, int2* bestTrainIdx, int2* bestImgIdx, float2* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;
            int myBestImgIdx1 = -1;
            int myBestImgIdx2 = -1;

            Mask m = mask;

            for (int imgIdx = 0; imgIdx < n; ++imgIdx)
            {
                const PtrStepSz<T> train = trains[imgIdx];
                m.next();
                loopUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, imgIdx, train, m, s_query, s_train, myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestImgIdx1, myBestImgIdx2);
            }

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);
            int* s_imgIdx = (int*)(smem + 2 * BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestImgIdx1, myBestImgIdx2, s_distance, s_trainIdx, s_imgIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = make_int2(myBestTrainIdx1, myBestTrainIdx2);
                bestImgIdx[queryIdx] = make_int2(myBestImgIdx1, myBestImgIdx2);
                bestDistance[queryIdx] = make_float2(myBestDistance1, myBestDistance2);
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                           const PtrStepSz<int2>& trainIdx, const PtrStepSz<int2>& imgIdx, const PtrStepSz<float2>& distance,
                           hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (3 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, trains, n, mask, trainIdx.data, imgIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __device__ void loop(int queryIdx, const PtrStepSz<T>& query, int imgIdx, const PtrStepSz<T>& train, const Mask& mask,
                             typename Dist::value_type* s_query, typename Dist::value_type* s_train,
                             float& bestDistance1, float& bestDistance2,
                             int& bestTrainIdx1, int& bestTrainIdx2,
                             int& bestImgIdx1, int& bestImgIdx2)
        {
            for (int t = 0, endt = (train.rows + BLOCK_SIZE - 1) / BLOCK_SIZE; t < endt; ++t)
            {
                Dist dist;

                for (int i = 0, endi = (query.cols + BLOCK_SIZE - 1) / BLOCK_SIZE; i < endi; ++i)
                {
                    const int loadX = threadIdx.x + i * BLOCK_SIZE;

                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                    if (loadX < query.cols)
                    {
                        T val;

                        ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                        s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                        ForceGlob<T>::Load(train.ptr(::min(t * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                        s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                    }

                    __syncthreads();

                    #pragma unroll
                    for (int j = 0; j < BLOCK_SIZE; ++j)
                        dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                    __syncthreads();
                }

                typename Dist::result_type distVal = dist;

                const int trainIdx = t * BLOCK_SIZE + threadIdx.x;

                if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx))
                {
                    if (distVal < bestDistance1)
                    {
                        bestImgIdx2   = bestImgIdx1;
                        bestDistance2 = bestDistance1;
                        bestTrainIdx2 = bestTrainIdx1;

                        bestImgIdx1   = imgIdx;
                        bestDistance1 = distVal;
                        bestTrainIdx1 = trainIdx;
                    }
                    else if (distVal < bestDistance2)
                    {
                        bestImgIdx2   = imgIdx;
                        bestDistance2 = distVal;
                        bestTrainIdx2 = trainIdx;
                    }
                }
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __global__ void match(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, int2* bestTrainIdx, float2* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;

            loop<BLOCK_SIZE, Dist>(queryIdx, query, 0, train, mask, s_query, s_train, myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestTrainIdx1, myBestTrainIdx2);

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, s_distance, s_trainIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = make_int2(myBestTrainIdx1, myBestTrainIdx2);
                bestDistance[queryIdx] = make_float2(myBestDistance1, myBestDistance2);
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        void match(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                   const PtrStepSz<int2>& trainIdx, const PtrStepSz<float2>& distance,
                   hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            match<BLOCK_SIZE, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, trainIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __global__ void match(const PtrStepSz<T> query, const PtrStepSz<T>* trains, int n, const Mask mask, int2* bestTrainIdx, int2* bestImgIdx, float2* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            float myBestDistance1 = numeric_limits<float>::max();
            float myBestDistance2 = numeric_limits<float>::max();
            int myBestTrainIdx1 = -1;
            int myBestTrainIdx2 = -1;
            int myBestImgIdx1 = -1;
            int myBestImgIdx2 = -1;

            Mask m = mask;

            for (int imgIdx = 0; imgIdx < n; ++imgIdx)
            {
                const PtrStepSz<T> train = trains[imgIdx];
                m.next();
                loop<BLOCK_SIZE, Dist>(queryIdx, query, imgIdx, train, m, s_query, s_train, myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestImgIdx1, myBestImgIdx2);
            }

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);
            int* s_imgIdx = (int*)(smem + 2 * BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance1, myBestDistance2, myBestTrainIdx1, myBestTrainIdx2, myBestImgIdx1, myBestImgIdx2, s_distance, s_trainIdx, s_imgIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = make_int2(myBestTrainIdx1, myBestTrainIdx2);
                bestImgIdx[queryIdx] = make_int2(myBestImgIdx1, myBestImgIdx2);
                bestDistance[queryIdx] = make_float2(myBestDistance1, myBestDistance2);
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        void match(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                   const PtrStepSz<int2>& trainIdx, const PtrStepSz<int2>& imgIdx, const PtrStepSz<float2>& distance,
                   hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (3 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            match<BLOCK_SIZE, Dist><<<grid, block, smemSize, stream>>>(query, trains, n, mask, trainIdx.data, imgIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // knnMatch 2 dispatcher

        template <typename Dist, typename T, typename Mask>
        void match2Dispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                              const PtrStepSzb& trainIdx, const PtrStepSzb& distance,
                              hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolledCached<16, 64, Dist>(query, train, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolledCached<16, 128, Dist>(query, train, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, train, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            else if (query.cols <= 512)
            {
                matchUnrolled<16, 512, Dist>(query, train, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            else if (query.cols <= 1024)
            {
                matchUnrolled<16, 1024, Dist>(query, train, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }*/
            else
            {
                match<16, Dist>(query, train, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
        }

        template <typename Dist, typename T, typename Mask>
        void match2Dispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                              const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance,
                              hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolledCached<16, 64, Dist>(query, trains, n, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<int2> >(imgIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolledCached<16, 128, Dist>(query, trains, n, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<int2> >(imgIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, trains, n, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<int2> >(imgIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            else if (query.cols <= 512)
            {
                matchUnrolled<16, 512, Dist>(query, trains, n, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<int2> >(imgIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
            else if (query.cols <= 1024)
            {
                matchUnrolled<16, 1024, Dist>(query, trains, n, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<int2> >(imgIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }*/
            else
            {
                match<16, Dist>(query, trains, n, mask, static_cast< PtrStepSz<int2> >(trainIdx), static_cast< PtrStepSz<int2> >(imgIdx), static_cast< PtrStepSz<float2> > (distance), stream);
            }
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Calc distance kernel

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void calcDistanceUnrolled(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, PtrStepf allDist)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
            const int trainIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Dist dist;

            #pragma unroll
            for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;

                if (loadX < query.cols)
                {
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = query.ptr(::min(queryIdx, query.rows - 1))[loadX];
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = train.ptr(::min(blockIdx.x * BLOCK_SIZE + threadIdx.y, train.rows - 1))[loadX];
                }
                else
                {
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < BLOCK_SIZE; ++j)
                    dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                __syncthreads();
            }

            if (queryIdx < query.rows && trainIdx < train.rows)
            {
                float distVal = numeric_limits<float>::max();

                if (mask(queryIdx, trainIdx))
                    distVal = (typename Dist::result_type)dist;

                allDist.ptr(queryIdx)[trainIdx] = distVal;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void calcDistanceUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask, const PtrStepSzf& allDist, hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            calcDistanceUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, allDist);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __global__ void calcDistance(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, PtrStepf allDist)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
            const int trainIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Dist dist;

            for (int i = 0, endi = (query.cols + BLOCK_SIZE - 1) / BLOCK_SIZE; i < endi; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;

                if (loadX < query.cols)
                {
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = query.ptr(::min(queryIdx, query.rows - 1))[loadX];
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = train.ptr(::min(blockIdx.x * BLOCK_SIZE + threadIdx.y, train.rows - 1))[loadX];
                }
                else
                {
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < BLOCK_SIZE; ++j)
                    dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                __syncthreads();
            }

            if (queryIdx < query.rows && trainIdx < train.rows)
            {
                float distVal = numeric_limits<float>::max();

                if (mask(queryIdx, trainIdx))
                    distVal = (typename Dist::result_type)dist;

                allDist.ptr(queryIdx)[trainIdx] = distVal;
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        void calcDistance(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask, const PtrStepSzf& allDist, hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            calcDistance<BLOCK_SIZE, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, allDist);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Calc Distance dispatcher

        template <typename Dist, typename T, typename Mask>
        void calcDistanceDispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                                    const PtrStepSzf& allDist,
                                    hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                calcDistanceUnrolled<16, 64, Dist>(query, train, mask, allDist, stream);
            }
            else if (query.cols <= 128)
            {
                calcDistanceUnrolled<16, 128, Dist>(query, train, mask, allDist, stream);
            }
            /*else if (query.cols <= 256)
            {
                calcDistanceUnrolled<16, 256, Dist>(query, train, mask, allDist, stream);
            }
            else if (query.cols <= 512)
            {
                calcDistanceUnrolled<16, 512, Dist>(query, train, mask, allDist, stream);
            }
            else if (query.cols <= 1024)
            {
                calcDistanceUnrolled<16, 1024, Dist>(query, train, mask, allDist, stream);
            }*/
            else
            {
                calcDistance<16, Dist>(query, train, mask, allDist, stream);
            }
        }

        ///////////////////////////////////////////////////////////////////////////////
        // find knn match kernel

        template <int BLOCK_SIZE>
        __global__ void findBestMatch(PtrStepSzf allDist, int i, PtrStepi trainIdx, PtrStepf distance)
        {
            const int SMEM_SIZE = BLOCK_SIZE > 64 ? BLOCK_SIZE : 64;
            __shared__ float s_dist[SMEM_SIZE];
            __shared__ int s_trainIdx[SMEM_SIZE];

            const int queryIdx = blockIdx.x;

            float* allDistRow = allDist.ptr(queryIdx);

            float dist = numeric_limits<float>::max();
            int bestIdx = -1;

            for (int i = threadIdx.x; i < allDist.cols; i += BLOCK_SIZE)
            {
                float reg = allDistRow[i];
                if (reg < dist)
                {
                    dist = reg;
                    bestIdx = i;
                }
            }

            s_dist[threadIdx.x] = dist;
            s_trainIdx[threadIdx.x] = bestIdx;
            __syncthreads();

            reduceKeyVal<BLOCK_SIZE>(s_dist, dist, s_trainIdx, bestIdx, threadIdx.x, less<float>());

            if (threadIdx.x == 0)
            {
                if (dist < numeric_limits<float>::max())
                {
                    allDistRow[bestIdx] = numeric_limits<float>::max();
                    trainIdx.ptr(queryIdx)[i] = bestIdx;
                    distance.ptr(queryIdx)[i] = dist;
                }
            }
        }

        template <int BLOCK_SIZE>
        void findKnnMatch(int k, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSzf& allDist, hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, 1, 1);
            const dim3 grid(trainIdx.rows, 1, 1);

            for (int i = 0; i < k; ++i)
            {
                findBestMatch<BLOCK_SIZE><<<grid, block, 0, stream>>>(allDist, i, trainIdx, distance);
                cudaSafeCall( hipGetLastError() );
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void findKnnMatchDispatcher(int k, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream)
        {
            findKnnMatch<256>(k, static_cast<PtrStepSzi>(trainIdx), static_cast<PtrStepSzf>(distance), allDist, stream);
        }

        ///////////////////////////////////////////////////////////////////////////////
        // knn match Dispatcher

        template <typename Dist, typename T, typename Mask>
        void matchDispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>& train, int k, const Mask& mask,
            const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist,
            hipStream_t stream)
        {
            if (k == 2)
            {
                match2Dispatcher<Dist>(query, train, mask, trainIdx, distance, stream);
            }
            else
            {
                calcDistanceDispatcher<Dist>(query, train, mask, allDist, stream);
                findKnnMatchDispatcher(k, trainIdx, distance, allDist, stream);
            }
        }

        ///////////////////////////////////////////////////////////////////////////////
        // knn match caller

        template <typename T> void matchL1_gpu(const PtrStepSzb& query, const PtrStepSzb& train, int k, const PtrStepSzb& mask,
            const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist,
            hipStream_t stream)
        {
            if (mask.data)
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), k, SingleMask(mask), trainIdx, distance, allDist, stream);
            else
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), k, WithOutMask(), trainIdx, distance, allDist, stream);
        }

        template void matchL1_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchL1_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchL1_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchL1_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchL1_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchL1_gpu<float >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);

        template <typename T> void matchL2_gpu(const PtrStepSzb& query, const PtrStepSzb& train, int k, const PtrStepSzb& mask,
            const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist,
            hipStream_t stream)
        {
            if (mask.data)
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), k, SingleMask(mask), trainIdx, distance, allDist, stream);
            else
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), k, WithOutMask(), trainIdx, distance, allDist, stream);
        }

        //template void matchL2_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchL2_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchL2_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchL2_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchL2_gpu<float >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const PtrStepSzb& query, const PtrStepSzb& train, int k, const PtrStepSzb& mask,
            const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist,
            hipStream_t stream)
        {
            if (mask.data)
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), k, SingleMask(mask), trainIdx, distance, allDist, stream);
            else
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), k, WithOutMask(), trainIdx, distance, allDist, stream);
        }

        template void matchHamming_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        //template void matchHamming_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);
        template void matchHamming_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, int k, const PtrStepSzb& mask, const PtrStepSzb& trainIdx, const PtrStepSzb& distance, const PtrStepSzf& allDist, hipStream_t stream);

        template <typename T> void match2L1_gpu(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks,
            const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance,
            hipStream_t stream)
        {
            if (masks.data)
                match2Dispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, MaskCollection(masks.data), trainIdx, imgIdx, distance, stream);
            else
                match2Dispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, WithOutMask(), trainIdx, imgIdx, distance,  stream);
        }

        template void match2L1_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2L1_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2L1_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2L1_gpu<short >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2L1_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2L1_gpu<float >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);

        template <typename T> void match2L2_gpu(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks,
            const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance,
            hipStream_t stream)
        {
            if (masks.data)
                match2Dispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, MaskCollection(masks.data), trainIdx, imgIdx, distance, stream);
            else
                match2Dispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, WithOutMask(), trainIdx, imgIdx, distance, stream);
        }

        //template void match2L2_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2L2_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2L2_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2L2_gpu<short >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2L2_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2L2_gpu<float >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);

        template <typename T> void match2Hamming_gpu(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks,
            const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance,
            hipStream_t stream)
        {
            if (masks.data)
                match2Dispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, MaskCollection(masks.data), trainIdx, imgIdx, distance, stream);
            else
                match2Dispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, WithOutMask(), trainIdx, imgIdx, distance, stream);
        }

        template void match2Hamming_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2Hamming_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2Hamming_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        //template void match2Hamming_gpu<short >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
        template void match2Hamming_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzb& trainIdx, const PtrStepSzb& imgIdx, const PtrStepSzb& distance, hipStream_t stream);
    } // namespace bf_knnmatch
}}} // namespace cv { namespace cuda { namespace cudev {


#endif /* CUDA_DISABLER */
