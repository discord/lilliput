#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/version.h>


#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/utility.hpp"
namespace cv { namespace cuda { namespace device
{
    namespace orb
    {
        ////////////////////////////////////////////////////////////////////////////////////////////////////////
        // cull

        int cull_gpu(int* loc, float* response, int size, int n_points, hipStream_t stream)
        {
            thrust::device_ptr<int> loc_ptr(loc);
            thrust::device_ptr<float> response_ptr(response);
#if THRUST_VERSION >= 100800
#if THRUST_VERSION >= 100802
            if (stream)
            {
                thrust::sort_by_key(thrust::cuda::par(ThrustAllocator::getAllocator()).on(stream), response_ptr, response_ptr + size, loc_ptr, thrust::greater<float>());
            }
            else
            {
                thrust::sort_by_key(thrust::cuda::par(ThrustAllocator::getAllocator()), response_ptr, response_ptr + size, loc_ptr, thrust::greater<float>());
            }
#else
            if(stream)
            {
                thrust::sort_by_key(thrust::cuda::par.on(stream), response_ptr, response_ptr + size, loc_ptr, thrust::greater<float>());
            }else
            {
                thrust::sort_by_key(response_ptr, response_ptr + size, loc_ptr, thrust::greater<float>());
            }
#endif
#else
            thrust::sort_by_key(response_ptr, response_ptr + size, loc_ptr, thrust::greater<float>());
#endif
            return n_points;
        }

        ////////////////////////////////////////////////////////////////////////////////////////////////////////
        // HarrisResponses

        __global__ void HarrisResponses(const PtrStepb img, const short2* loc_, float* response, const int npoints, const int blockSize, const float harris_k)
        {
            __shared__ int smem0[8 * 32];
            __shared__ int smem1[8 * 32];
            __shared__ int smem2[8 * 32];

            const int ptidx = blockIdx.x * blockDim.y + threadIdx.y;

            if (ptidx < npoints)
            {
                const short2 loc = loc_[ptidx];

                const int r = blockSize / 2;
                const int x0 = loc.x - r;
                const int y0 = loc.y - r;

                int a = 0, b = 0, c = 0;

                for (int ind = threadIdx.x; ind < blockSize * blockSize; ind += blockDim.x)
                {
                    const int i = ind / blockSize;
                    const int j = ind % blockSize;

                    int Ix = (img(y0 + i, x0 + j + 1) - img(y0 + i, x0 + j - 1)) * 2 +
                        (img(y0 + i - 1, x0 + j + 1) - img(y0 + i - 1, x0 + j - 1)) +
                        (img(y0 + i + 1, x0 + j + 1) - img(y0 + i + 1, x0 + j - 1));

                    int Iy = (img(y0 + i + 1, x0 + j) - img(y0 + i - 1, x0 + j)) * 2 +
                        (img(y0 + i + 1, x0 + j - 1) - img(y0 + i - 1, x0 + j - 1)) +
                        (img(y0 + i + 1, x0 + j + 1) - img(y0 + i - 1, x0 + j + 1));

                    a += Ix * Ix;
                    b += Iy * Iy;
                    c += Ix * Iy;
                }

                int* srow0 = smem0 + threadIdx.y * blockDim.x;
                int* srow1 = smem1 + threadIdx.y * blockDim.x;
                int* srow2 = smem2 + threadIdx.y * blockDim.x;

                plus<int> op;
                reduce<32>(smem_tuple(srow0, srow1, srow2), thrust::tie(a, b, c), threadIdx.x, thrust::make_tuple(op, op, op));

                if (threadIdx.x == 0)
                {
                    float scale = (1 << 2) * blockSize * 255.0f;
                    scale = 1.0f / scale;
                    const float scale_sq_sq = scale * scale * scale * scale;

                    response[ptidx] = ((float)a * b - (float)c * c - harris_k * ((float)a + b) * ((float)a + b)) * scale_sq_sq;
                }
            }
        }

        void HarrisResponses_gpu(PtrStepSzb img, const short2* loc, float* response, const int npoints, int blockSize, float harris_k, hipStream_t stream)
        {
            dim3 block(32, 8);

            dim3 grid;
            grid.x = divUp(npoints, block.y);

            HarrisResponses<<<grid, block, 0, stream>>>(img, loc, response, npoints, blockSize, harris_k);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////////////////////////////////////
        // IC_Angle

        __constant__ int c_u_max[32];

        void loadUMax(const int* u_max, int count)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_u_max), u_max, count * sizeof(int)) );
        }

        __global__ void IC_Angle(const PtrStepb image, const short2* loc_, float* angle, const int npoints, const int half_k)
        {
            __shared__ int smem0[8 * 32];
            __shared__ int smem1[8 * 32];

            int* srow0 = smem0 + threadIdx.y * blockDim.x;
            int* srow1 = smem1 + threadIdx.y * blockDim.x;

            plus<int> op;

            const int ptidx = blockIdx.x * blockDim.y + threadIdx.y;

            if (ptidx < npoints)
            {
                int m_01 = 0, m_10 = 0;

                const short2 loc = loc_[ptidx];

                // Treat the center line differently, v=0
                for (int u = threadIdx.x - half_k; u <= half_k; u += blockDim.x)
                    m_10 += u * image(loc.y, loc.x + u);

                reduce<32>(srow0, m_10, threadIdx.x, op);

                for (int v = 1; v <= half_k; ++v)
                {
                    // Proceed over the two lines
                    int v_sum = 0;
                    int m_sum = 0;
                    const int d = c_u_max[v];

                    for (int u = threadIdx.x - d; u <= d; u += blockDim.x)
                    {
                        int val_plus = image(loc.y + v, loc.x + u);
                        int val_minus = image(loc.y - v, loc.x + u);

                        v_sum += (val_plus - val_minus);
                        m_sum += u * (val_plus + val_minus);
                    }

                    reduce<32>(smem_tuple(srow0, srow1), thrust::tie(v_sum, m_sum), threadIdx.x, thrust::make_tuple(op, op));

                    m_10 += m_sum;
                    m_01 += v * v_sum;
                }

                if (threadIdx.x == 0)
                {
                    float kp_dir = ::atan2f((float)m_01, (float)m_10);
                    kp_dir += (kp_dir < 0) * (2.0f * CV_PI_F);
                    kp_dir *= 180.0f / CV_PI_F;

                    angle[ptidx] = kp_dir;
                }
            }
        }

        void IC_Angle_gpu(PtrStepSzb image, const short2* loc, float* angle, int npoints, int half_k, hipStream_t stream)
        {
            dim3 block(32, 8);

            dim3 grid;
            grid.x = divUp(npoints, block.y);

            IC_Angle<<<grid, block, 0, stream>>>(image, loc, angle, npoints, half_k);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////////////////////////////////////
        // computeOrbDescriptor

        template <int WTA_K> struct OrbDescriptor;

        #define GET_VALUE(idx) \
            img(loc.y + __float2int_rn(pattern_x[idx] * sina + pattern_y[idx] * cosa), \
                loc.x + __float2int_rn(pattern_x[idx] * cosa - pattern_y[idx] * sina))

        template <> struct OrbDescriptor<2>
        {
            __device__ static int calc(const PtrStepb& img, short2 loc, const int* pattern_x, const int* pattern_y, float sina, float cosa, int i)
            {
                pattern_x += 16 * i;
                pattern_y += 16 * i;

                int t0, t1, val;

                t0 = GET_VALUE(0); t1 = GET_VALUE(1);
                val = t0 < t1;

                t0 = GET_VALUE(2); t1 = GET_VALUE(3);
                val |= (t0 < t1) << 1;

                t0 = GET_VALUE(4); t1 = GET_VALUE(5);
                val |= (t0 < t1) << 2;

                t0 = GET_VALUE(6); t1 = GET_VALUE(7);
                val |= (t0 < t1) << 3;

                t0 = GET_VALUE(8); t1 = GET_VALUE(9);
                val |= (t0 < t1) << 4;

                t0 = GET_VALUE(10); t1 = GET_VALUE(11);
                val |= (t0 < t1) << 5;

                t0 = GET_VALUE(12); t1 = GET_VALUE(13);
                val |= (t0 < t1) << 6;

                t0 = GET_VALUE(14); t1 = GET_VALUE(15);
                val |= (t0 < t1) << 7;

                return val;
            }
        };

        template <> struct OrbDescriptor<3>
        {
            __device__ static int calc(const PtrStepb& img, short2 loc, const int* pattern_x, const int* pattern_y, float sina, float cosa, int i)
            {
                pattern_x += 12 * i;
                pattern_y += 12 * i;

                int t0, t1, t2, val;

                t0 = GET_VALUE(0); t1 = GET_VALUE(1); t2 = GET_VALUE(2);
                val = t2 > t1 ? (t2 > t0 ? 2 : 0) : (t1 > t0);

                t0 = GET_VALUE(3); t1 = GET_VALUE(4); t2 = GET_VALUE(5);
                val |= (t2 > t1 ? (t2 > t0 ? 2 : 0) : (t1 > t0)) << 2;

                t0 = GET_VALUE(6); t1 = GET_VALUE(7); t2 = GET_VALUE(8);
                val |= (t2 > t1 ? (t2 > t0 ? 2 : 0) : (t1 > t0)) << 4;

                t0 = GET_VALUE(9); t1 = GET_VALUE(10); t2 = GET_VALUE(11);
                val |= (t2 > t1 ? (t2 > t0 ? 2 : 0) : (t1 > t0)) << 6;

                return val;
            }
        };

        template <> struct OrbDescriptor<4>
        {
            __device__ static int calc(const PtrStepb& img, short2 loc, const int* pattern_x, const int* pattern_y, float sina, float cosa, int i)
            {
                pattern_x += 16 * i;
                pattern_y += 16 * i;

                int t0, t1, t2, t3, k, val;
                int a, b;

                t0 = GET_VALUE(0); t1 = GET_VALUE(1);
                t2 = GET_VALUE(2); t3 = GET_VALUE(3);
                a = 0, b = 2;
                if( t1 > t0 ) t0 = t1, a = 1;
                if( t3 > t2 ) t2 = t3, b = 3;
                k = t0 > t2 ? a : b;
                val = k;

                t0 = GET_VALUE(4); t1 = GET_VALUE(5);
                t2 = GET_VALUE(6); t3 = GET_VALUE(7);
                a = 0, b = 2;
                if( t1 > t0 ) t0 = t1, a = 1;
                if( t3 > t2 ) t2 = t3, b = 3;
                k = t0 > t2 ? a : b;
                val |= k << 2;

                t0 = GET_VALUE(8); t1 = GET_VALUE(9);
                t2 = GET_VALUE(10); t3 = GET_VALUE(11);
                a = 0, b = 2;
                if( t1 > t0 ) t0 = t1, a = 1;
                if( t3 > t2 ) t2 = t3, b = 3;
                k = t0 > t2 ? a : b;
                val |= k << 4;

                t0 = GET_VALUE(12); t1 = GET_VALUE(13);
                t2 = GET_VALUE(14); t3 = GET_VALUE(15);
                a = 0, b = 2;
                if( t1 > t0 ) t0 = t1, a = 1;
                if( t3 > t2 ) t2 = t3, b = 3;
                k = t0 > t2 ? a : b;
                val |= k << 6;

                return val;
            }
        };

        #undef GET_VALUE

        template <int WTA_K>
        __global__ void computeOrbDescriptor(const PtrStepb img, const short2* loc, const float* angle_, const int npoints,
            const int* pattern_x, const int* pattern_y, PtrStepb desc, int dsize)
        {
            const int descidx = blockIdx.x * blockDim.x + threadIdx.x;
            const int ptidx = blockIdx.y * blockDim.y + threadIdx.y;

            if (ptidx < npoints && descidx < dsize)
            {
                float angle = angle_[ptidx];
                angle *= (float)(CV_PI_F / 180.f);

                float sina, cosa;
                ::sincosf(angle, &sina, &cosa);

                desc.ptr(ptidx)[descidx] = OrbDescriptor<WTA_K>::calc(img, loc[ptidx], pattern_x, pattern_y, sina, cosa, descidx);
            }
        }

        void computeOrbDescriptor_gpu(PtrStepb img, const short2* loc, const float* angle, const int npoints,
            const int* pattern_x, const int* pattern_y, PtrStepb desc, int dsize, int WTA_K, hipStream_t stream)
        {
            dim3 block(32, 8);

            dim3 grid;
            grid.x = divUp(dsize, block.x);
            grid.y = divUp(npoints, block.y);

            switch (WTA_K)
            {
            case 2:
                computeOrbDescriptor<2><<<grid, block, 0, stream>>>(img, loc, angle, npoints, pattern_x, pattern_y, desc, dsize);
                break;

            case 3:
                computeOrbDescriptor<3><<<grid, block, 0, stream>>>(img, loc, angle, npoints, pattern_x, pattern_y, desc, dsize);
                break;

            case 4:
                computeOrbDescriptor<4><<<grid, block, 0, stream>>>(img, loc, angle, npoints, pattern_x, pattern_y, desc, dsize);
                break;
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////////////////////////////////////
        // mergeLocation

        __global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
        {
            const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

            if (ptidx < npoints)
            {
                short2 loc = loc_[ptidx];

                x[ptidx] = loc.x * scale;
                y[ptidx] = loc.y * scale;
            }
        }

        void mergeLocation_gpu(const short2* loc, float* x, float* y, int npoints, float scale, hipStream_t stream)
        {
            dim3 block(256);

            dim3 grid;
            grid.x = divUp(npoints, block.x);

            mergeLocation<<<grid, block, 0, stream>>>(loc, x, y, npoints, scale);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}

#endif /* CUDA_DISABLER */
