#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/vec_distance.hpp"
#include "opencv2/core/cuda/datamov_utils.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace bf_radius_match
    {
        ///////////////////////////////////////////////////////////////////////////////
        // Match Unrolled

        template <int BLOCK_SIZE, int MAX_DESC_LEN, bool SAVE_IMG_IDX, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolled(const PtrStepSz<T> query, int imgIdx, const PtrStepSz<T> train, float maxDistance, const Mask mask,
            PtrStepi bestTrainIdx, PtrStepi bestImgIdx, PtrStepf bestDistance, unsigned int* nMatches, int maxCount)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
            const int trainIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Dist dist;

            #pragma unroll
            for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;

                s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                if (loadX < query.cols)
                {
                    T val;

                    ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                    ForceGlob<T>::Load(train.ptr(::min(blockIdx.x * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < BLOCK_SIZE; ++j)
                    dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                __syncthreads();
            }

            float distVal = (typename Dist::result_type)dist;

            if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx) && distVal < maxDistance)
            {
                unsigned int ind = atomicInc(nMatches + queryIdx, (unsigned int) -1);
                if (ind < maxCount)
                {
                    bestTrainIdx.ptr(queryIdx)[ind] = trainIdx;
                    if (SAVE_IMG_IDX) bestImgIdx.ptr(queryIdx)[ind] = imgIdx;
                    bestDistance.ptr(queryIdx)[ind] = distVal;
                }
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>& train, float maxDistance, const Mask& mask,
            const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, false, Dist><<<grid, block, smemSize, stream>>>(query, 0, train, maxDistance, mask,
                trainIdx, PtrStepi(), distance, nMatches.data, trainIdx.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T>
        void matchUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, float maxDistance, const PtrStepSzb* masks,
            const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            for (int i = 0; i < n; ++i)
            {
                const PtrStepSz<T> train = trains[i];

                const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

                if (masks != 0 && masks[i].data)
                {
                    matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, SingleMask(masks[i]),
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                else
                {
                    matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, WithOutMask(),
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                cudaSafeCall( hipGetLastError() );
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match

        template <int BLOCK_SIZE, bool SAVE_IMG_IDX, typename Dist, typename T, typename Mask>
        __global__ void match(const PtrStepSz<T> query, int imgIdx, const PtrStepSz<T> train, float maxDistance, const Mask mask,
            PtrStepi bestTrainIdx, PtrStepi bestImgIdx, PtrStepf bestDistance, unsigned int* nMatches, int maxCount)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
            const int trainIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Dist dist;

            for (int i = 0, endi = (query.cols + BLOCK_SIZE - 1) / BLOCK_SIZE; i < endi; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;

                s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                if (loadX < query.cols)
                {
                    T val;

                    ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                    ForceGlob<T>::Load(train.ptr(::min(blockIdx.x * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                }

                __syncthreads();

                #pragma unroll
                for (int j = 0; j < BLOCK_SIZE; ++j)
                    dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                __syncthreads();
            }

            float distVal = (typename Dist::result_type)dist;

            if (queryIdx < query.rows && trainIdx < train.rows && mask(queryIdx, trainIdx) && distVal < maxDistance)
            {
                unsigned int ind = atomicInc(nMatches + queryIdx, (unsigned int) -1);
                if (ind < maxCount)
                {
                    bestTrainIdx.ptr(queryIdx)[ind] = trainIdx;
                    if (SAVE_IMG_IDX) bestImgIdx.ptr(queryIdx)[ind] = imgIdx;
                    bestDistance.ptr(queryIdx)[ind] = distVal;
                }
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        void match(const PtrStepSz<T>& query, const PtrStepSz<T>& train, float maxDistance, const Mask& mask,
            const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            match<BLOCK_SIZE, false, Dist><<<grid, block, smemSize, stream>>>(query, 0, train, maxDistance, mask,
                trainIdx, PtrStepi(), distance, nMatches.data, trainIdx.cols);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, typename Dist, typename T>
        void match(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, float maxDistance, const PtrStepSzb* masks,
            const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            for (int i = 0; i < n; ++i)
            {
                const PtrStepSz<T> train = trains[i];

                const dim3 grid(divUp(train.rows, BLOCK_SIZE), divUp(query.rows, BLOCK_SIZE));

                if (masks != 0 && masks[i].data)
                {
                    match<BLOCK_SIZE, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, SingleMask(masks[i]),
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                else
                {
                    match<BLOCK_SIZE, true, Dist><<<grid, block, smemSize, stream>>>(query, i, train, maxDistance, WithOutMask(),
                        trainIdx, imgIdx, distance, nMatches.data, trainIdx.cols);
                }
                cudaSafeCall( hipGetLastError() );
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match dispatcher

        template <typename Dist, typename T, typename Mask>
        void matchDispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>& train, float maxDistance, const Mask& mask,
                             const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
                             hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolled<16, 64, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolled<16, 128, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 512)
            {
                matchUnrolled<16, 512, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 1024)
            {
                matchUnrolled<16, 1024, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }*/
            else
            {
                match<16, Dist>(query, train, maxDistance, mask, trainIdx, distance, nMatches, stream);
            }
        }

        template <typename Dist, typename T>
        void matchDispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, float maxDistance, const PtrStepSzb* masks,
                             const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
                             hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolled<16, 64, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolled<16, 128, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 512)
            {
                matchUnrolled<16, 512, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
            else if (query.cols <= 1024)
            {
                matchUnrolled<16, 1024, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }*/
            else
            {
                match<16, Dist>(query, trains, n, maxDistance, masks, trainIdx, imgIdx, distance, nMatches, stream);
            }
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Radius Match caller

        template <typename T> void matchL1_gpu(const PtrStepSzb& query, const PtrStepSzb& train, float maxDistance, const PtrStepSzb& mask,
            const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), maxDistance, SingleMask(mask),
                    trainIdx, distance, nMatches,
                    stream);
            }
            else
            {
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), maxDistance, WithOutMask(),
                    trainIdx, distance, nMatches,
                    stream);
            }
        }

        template void matchL1_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL1_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<float >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);

        template <typename T> void matchL2_gpu(const PtrStepSzb& query, const PtrStepSzb& train, float maxDistance, const PtrStepSzb& mask,
            const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), maxDistance, SingleMask(mask),
                    trainIdx, distance, nMatches,
                    stream);
            }
            else
            {
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), maxDistance, WithOutMask(),
                    trainIdx, distance, nMatches,
                    stream);
            }
        }

        //template void matchL2_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL2_gpu<float >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const PtrStepSzb& query, const PtrStepSzb& train, float maxDistance, const PtrStepSzb& mask,
            const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), maxDistance, SingleMask(mask),
                    trainIdx, distance, nMatches,
                    stream);
            }
            else
            {
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), maxDistance, WithOutMask(),
                    trainIdx, distance, nMatches,
                    stream);
            }
        }

        template void matchHamming_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchHamming_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchHamming_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, float maxDistance, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);

        template <typename T> void matchL1_gpu(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks,
            const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains, n, maxDistance, masks,
                trainIdx, imgIdx, distance, nMatches,
                stream);
        }

        template void matchL1_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL1_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<short >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL1_gpu<float >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);

        template <typename T> void matchL2_gpu(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks,
            const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains, n, maxDistance, masks,
                trainIdx, imgIdx, distance, nMatches,
                stream);
        }

        //template void matchL2_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<short >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchL2_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchL2_gpu<float >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks,
            const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches,
            hipStream_t stream)
        {
            matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains, n, maxDistance, masks,
                trainIdx, imgIdx, distance, nMatches,
                stream);
        }

        template void matchHamming_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        //template void matchHamming_gpu<short >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
        template void matchHamming_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb* trains, int n, float maxDistance, const PtrStepSzb* masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, const PtrStepSz<unsigned int>& nMatches, hipStream_t stream);
    } // namespace bf_radius_match
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
