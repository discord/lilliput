#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/vec_distance.hpp"
#include "opencv2/core/cuda/datamov_utils.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace bf_match
    {
        ///////////////////////////////////////////////////////////////////////////////
        // Reduction

        template <int BLOCK_SIZE>
        __device__ void findBestMatch(float& bestDistance, int& bestTrainIdx, float* s_distance, int* s_trainIdx)
        {
            s_distance += threadIdx.y * BLOCK_SIZE;
            s_trainIdx += threadIdx.y * BLOCK_SIZE;

            reduceKeyVal<BLOCK_SIZE>(s_distance, bestDistance, s_trainIdx, bestTrainIdx, threadIdx.x, less<float>());
        }

        template <int BLOCK_SIZE>
        __device__ void findBestMatch(float& bestDistance, int& bestTrainIdx, int& bestImgIdx, float* s_distance, int* s_trainIdx, int* s_imgIdx)
        {
            s_distance += threadIdx.y * BLOCK_SIZE;
            s_trainIdx += threadIdx.y * BLOCK_SIZE;
            s_imgIdx   += threadIdx.y * BLOCK_SIZE;

            reduceKeyVal<BLOCK_SIZE>(s_distance, bestDistance, smem_tuple(s_trainIdx, s_imgIdx), thrust::tie(bestTrainIdx, bestImgIdx), threadIdx.x, less<float>());
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match Unrolled Cached

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename T, typename U>
        __device__ void loadQueryToSmem(int queryIdx, const PtrStepSz<T>& query, U* s_query)
        {
            #pragma unroll
            for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
            {
                const int loadX = threadIdx.x + i * BLOCK_SIZE;
                s_query[threadIdx.y * MAX_DESC_LEN + loadX] = loadX < query.cols ? query.ptr(::min(queryIdx, query.rows - 1))[loadX] : 0;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __device__ void loopUnrolledCached(int queryIdx, const PtrStepSz<T>& query,volatile int imgIdx, const PtrStepSz<T>& train, const Mask& mask,
                                           typename Dist::value_type* s_query, typename Dist::value_type* s_train,
                                           float& bestDistance, int& bestTrainIdx, int& bestImgIdx)
        {
            for (int t = 0, endt = (train.rows + BLOCK_SIZE - 1) / BLOCK_SIZE; t < endt; ++t)
            {
                Dist dist;

                #pragma unroll
                for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
                {
                    const int loadX = threadIdx.x + i * BLOCK_SIZE;

                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                    if (loadX < train.cols)
                    {
                        T val;

                        ForceGlob<T>::Load(train.ptr(::min(t * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                        s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                    }

                    __syncthreads();

                    #pragma unroll
                    for (int j = 0; j < BLOCK_SIZE; ++j)
                        dist.reduceIter(s_query[threadIdx.y * MAX_DESC_LEN + i * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                    __syncthreads();
                }

                typename Dist::result_type distVal = dist;

                const int trainIdx = t * BLOCK_SIZE + threadIdx.x;

                if (queryIdx < query.rows && trainIdx < train.rows && distVal < bestDistance && mask(queryIdx, trainIdx))
                {
                    bestImgIdx = imgIdx;
                    bestDistance = distVal;
                    bestTrainIdx = trainIdx;
                }
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolledCached(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, int* bestTrainIdx, float* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * MAX_DESC_LEN);

            loadQueryToSmem<BLOCK_SIZE, MAX_DESC_LEN>(queryIdx, query, s_query);

            float myBestDistance = numeric_limits<float>::max();
            int myBestTrainIdx = -1;

            loopUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, 0, train, mask, s_query, s_train, myBestDistance, myBestTrainIdx, myBestTrainIdx);

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance, myBestTrainIdx, s_distance, s_trainIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = myBestTrainIdx;
                bestDistance[queryIdx] = myBestDistance;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolledCached(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                                 const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                                 hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (BLOCK_SIZE * (MAX_DESC_LEN >= BLOCK_SIZE ? MAX_DESC_LEN : BLOCK_SIZE) + BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, trainIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolledCached(const PtrStepSz<T> query, const PtrStepSz<T>* trains, int n, const Mask mask,
                                            int* bestTrainIdx, int* bestImgIdx, float* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * MAX_DESC_LEN);

            loadQueryToSmem<BLOCK_SIZE, MAX_DESC_LEN>(queryIdx, query, s_query);

            float myBestDistance = numeric_limits<float>::max();
            int myBestTrainIdx = -1;
            int myBestImgIdx = -1;

            Mask m = mask;

            for (int imgIdx = 0; imgIdx < n; ++imgIdx)
            {
                const PtrStepSz<T> train = trains[imgIdx];
                m.next();
                loopUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, imgIdx, train, m, s_query, s_train, myBestDistance, myBestTrainIdx, myBestImgIdx);
            }

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);
            int* s_imgIdx = (int*)(smem + 2 * BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance, myBestTrainIdx, myBestImgIdx, s_distance, s_trainIdx, s_imgIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = myBestTrainIdx;
                bestImgIdx[queryIdx] = myBestImgIdx;
                bestDistance[queryIdx] = myBestDistance;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolledCached(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                                 const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                                 hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (BLOCK_SIZE * (MAX_DESC_LEN >= 2 * BLOCK_SIZE ? MAX_DESC_LEN : 2 * BLOCK_SIZE) + BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolledCached<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, trains, n, mask, trainIdx.data, imgIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match Unrolled

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __device__ void loopUnrolled(int queryIdx, const PtrStepSz<T>& query,volatile int imgIdx, const PtrStepSz<T>& train, const Mask& mask,
                                     typename Dist::value_type* s_query, typename Dist::value_type* s_train,
                                     float& bestDistance, int& bestTrainIdx, int& bestImgIdx)
        {
            for (int t = 0, endt = (train.rows + BLOCK_SIZE - 1) / BLOCK_SIZE; t < endt; ++t)
            {
                Dist dist;

                #pragma unroll
                for (int i = 0; i < MAX_DESC_LEN / BLOCK_SIZE; ++i)
                {
                    const int loadX = threadIdx.x + i * BLOCK_SIZE;

                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                    if (loadX < query.cols)
                    {
                        T val;

                        ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                        s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                        ForceGlob<T>::Load(train.ptr(::min(t * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                        s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                    }

                    __syncthreads();

                    #pragma unroll
                    for (int j = 0; j < BLOCK_SIZE; ++j)
                        dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                    __syncthreads();
                }

                typename Dist::result_type distVal = dist;

                const int trainIdx = t * BLOCK_SIZE + threadIdx.x;

                if (queryIdx < query.rows && trainIdx < train.rows && distVal < bestDistance && mask(queryIdx, trainIdx))
                {
                    bestImgIdx = imgIdx;
                    bestDistance = distVal;
                    bestTrainIdx = trainIdx;
                }
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolled(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, int* bestTrainIdx, float* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            float myBestDistance = numeric_limits<float>::max();
            int myBestTrainIdx = -1;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            loopUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, 0, train, mask, s_query, s_train, myBestDistance, myBestTrainIdx, myBestTrainIdx);

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance, myBestTrainIdx, s_distance, s_trainIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = myBestTrainIdx;
                bestDistance[queryIdx] = myBestDistance;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                           const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                           hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, trainIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        __global__ void matchUnrolled(const PtrStepSz<T> query, const PtrStepSz<T>* trains, int n, const Mask mask,
                                      int* bestTrainIdx, int* bestImgIdx, float* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            float myBestDistance = numeric_limits<float>::max();
            int myBestTrainIdx = -1;
            int myBestImgIdx = -1;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Mask m = mask;

            for (int imgIdx = 0; imgIdx < n; ++imgIdx)
            {
                const PtrStepSz<T> train = trains[imgIdx];
                m.next();
                loopUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist>(queryIdx, query, imgIdx, train, m, s_query, s_train, myBestDistance, myBestTrainIdx, myBestImgIdx);
            }

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);
            int* s_imgIdxIdx = (int*)(smem + 2 * BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance, myBestTrainIdx, myBestImgIdx, s_distance, s_trainIdx, s_imgIdxIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = myBestTrainIdx;
                bestImgIdx[queryIdx] = myBestImgIdx;
                bestDistance[queryIdx] = myBestDistance;
            }
        }

        template <int BLOCK_SIZE, int MAX_DESC_LEN, typename Dist, typename T, typename Mask>
        void matchUnrolled(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                           const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                           hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (3 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            matchUnrolled<BLOCK_SIZE, MAX_DESC_LEN, Dist><<<grid, block, smemSize, stream>>>(query, trains, n, mask, trainIdx.data, imgIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __device__ void loop(int queryIdx, const PtrStepSz<T>& query, volatile int imgIdx, const PtrStepSz<T>& train, const Mask& mask,
                             typename Dist::value_type* s_query, typename Dist::value_type* s_train,
                             float& bestDistance, int& bestTrainIdx, int& bestImgIdx)
        {
            for (int t = 0, endt = (train.rows + BLOCK_SIZE - 1) / BLOCK_SIZE; t < endt; ++t)
            {
                Dist dist;

                for (int i = 0, endi = (query.cols + BLOCK_SIZE - 1) / BLOCK_SIZE; i < endi; ++i)
                {
                    const int loadX = threadIdx.x + i * BLOCK_SIZE;

                    s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
                    s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = 0;

                    if (loadX < query.cols)
                    {
                        T val;

                        ForceGlob<T>::Load(query.ptr(::min(queryIdx, query.rows - 1)), loadX, val);
                        s_query[threadIdx.y * BLOCK_SIZE + threadIdx.x] = val;

                        ForceGlob<T>::Load(train.ptr(::min(t * BLOCK_SIZE + threadIdx.y, train.rows - 1)), loadX, val);
                        s_train[threadIdx.x * BLOCK_SIZE + threadIdx.y] = val;
                    }

                    __syncthreads();

                    #pragma unroll
                    for (int j = 0; j < BLOCK_SIZE; ++j)
                        dist.reduceIter(s_query[threadIdx.y * BLOCK_SIZE + j], s_train[j * BLOCK_SIZE + threadIdx.x]);

                    __syncthreads();
                }

                typename Dist::result_type distVal = dist;

                const int trainIdx = t * BLOCK_SIZE + threadIdx.x;

                if (queryIdx < query.rows && trainIdx < train.rows && distVal < bestDistance && mask(queryIdx, trainIdx))
                {
                    bestImgIdx = imgIdx;
                    bestDistance = distVal;
                    bestTrainIdx = trainIdx;
                }
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __global__ void match(const PtrStepSz<T> query, const PtrStepSz<T> train, const Mask mask, int* bestTrainIdx, float* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            float myBestDistance = numeric_limits<float>::max();
            int myBestTrainIdx = -1;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            loop<BLOCK_SIZE, Dist>(queryIdx, query, 0, train, mask, s_query, s_train, myBestDistance, myBestTrainIdx, myBestTrainIdx);

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance, myBestTrainIdx, s_distance, s_trainIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = myBestTrainIdx;
                bestDistance[queryIdx] = myBestDistance;
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        void match(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                   const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                   hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (2 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            match<BLOCK_SIZE, Dist><<<grid, block, smemSize, stream>>>(query, train, mask, trainIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        __global__ void match(const PtrStepSz<T> query, const PtrStepSz<T>* trains, int n, const Mask mask,
                              int* bestTrainIdx, int* bestImgIdx, float* bestDistance)
        {
            extern __shared__ int smem[];

            const int queryIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;

            float myBestDistance = numeric_limits<float>::max();
            int myBestTrainIdx = -1;
            int myBestImgIdx = -1;

            typename Dist::value_type* s_query = (typename Dist::value_type*)(smem);
            typename Dist::value_type* s_train = (typename Dist::value_type*)(smem + BLOCK_SIZE * BLOCK_SIZE);

            Mask m = mask;
            for (int imgIdx = 0; imgIdx < n; ++imgIdx)
            {
                const PtrStepSz<T> train = trains[imgIdx];
                m.next();
                loop<BLOCK_SIZE, Dist>(queryIdx, query, imgIdx, train, m, s_query, s_train, myBestDistance, myBestTrainIdx, myBestImgIdx);
            }

            __syncthreads();

            float* s_distance = (float*)(smem);
            int* s_trainIdx = (int*)(smem + BLOCK_SIZE * BLOCK_SIZE);
            int* s_imgIdxIdx = (int*)(smem + 2 * BLOCK_SIZE * BLOCK_SIZE);

            findBestMatch<BLOCK_SIZE>(myBestDistance, myBestTrainIdx, myBestImgIdx, s_distance, s_trainIdx, s_imgIdxIdx);

            if (queryIdx < query.rows && threadIdx.x == 0)
            {
                bestTrainIdx[queryIdx] = myBestTrainIdx;
                bestImgIdx[queryIdx] = myBestImgIdx;
                bestDistance[queryIdx] = myBestDistance;
            }
        }

        template <int BLOCK_SIZE, typename Dist, typename T, typename Mask>
        void match(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                   const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                   hipStream_t stream)
        {
            const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            const dim3 grid(divUp(query.rows, BLOCK_SIZE));

            const size_t smemSize = (3 * BLOCK_SIZE * BLOCK_SIZE) * sizeof(int);

            match<BLOCK_SIZE, Dist><<<grid, block, smemSize, stream>>>(query, trains, n, mask, trainIdx.data, imgIdx.data, distance.data);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match dispatcher

        template <typename Dist, typename T, typename Mask>
        void matchDispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>& train, const Mask& mask,
                             const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                             hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolledCached<16, 64, Dist>(query, train, mask, trainIdx, distance, stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolledCached<16, 128, Dist>(query, train, mask, trainIdx, distance, stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, train, mask, trainIdx, distance, stream);
            }
            else if (query.cols <= 512)
            {
                matchUnrolled<16, 512, Dist>(query, train, mask, trainIdx, distance, stream);
            }
            else if (query.cols <= 1024)
            {
                matchUnrolled<16, 1024, Dist>(query, train, mask, trainIdx, distance, stream);
            }*/
            else
            {
                match<16, Dist>(query, train, mask, trainIdx, distance, stream);
            }
        }

        template <typename Dist, typename T, typename Mask>
        void matchDispatcher(const PtrStepSz<T>& query, const PtrStepSz<T>* trains, int n, const Mask& mask,
                             const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                             hipStream_t stream)
        {
            if (query.cols <= 64)
            {
                matchUnrolledCached<16, 64, Dist>(query, trains, n, mask, trainIdx, imgIdx, distance, stream);
            }
            else if (query.cols <= 128)
            {
                matchUnrolledCached<16, 128, Dist>(query, trains, n, mask, trainIdx, imgIdx, distance, stream);
            }
            /*else if (query.cols <= 256)
            {
                matchUnrolled<16, 256, Dist>(query, trains, n, mask, trainIdx, imgIdx, distance, stream);
            }
            else if (query.cols <= 512)
            {
                matchUnrolled<16, 512, Dist>(query, trains, n, mask, trainIdx, imgIdx, distance, stream);
            }
            else if (query.cols <= 1024)
            {
                matchUnrolled<16, 1024, Dist>(query, trains, n, mask, trainIdx, imgIdx, distance, stream);
            }*/
            else
            {
                match<16, Dist>(query, trains, n, mask, trainIdx, imgIdx, distance, stream);
            }
        }

        ///////////////////////////////////////////////////////////////////////////////
        // Match caller

        template <typename T> void matchL1_gpu(const PtrStepSzb& query, const PtrStepSzb& train, const PtrStepSzb& mask,
                                               const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                                               hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), SingleMask(mask),
                    trainIdx, distance,
                    stream);
            }
            else
            {
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), WithOutMask(),
                    trainIdx, distance,
                    stream);
            }
        }

        template void matchL1_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL1_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<float >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);

        template <typename T> void matchL2_gpu(const PtrStepSzb& query, const PtrStepSzb& train, const PtrStepSzb& mask,
                                               const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                                               hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), SingleMask(mask),
                    trainIdx, distance,
                    stream);
            }
            else
            {
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), WithOutMask(),
                    trainIdx, distance,
                    stream);
            }
        }

        //template void matchL2_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL2_gpu<float >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const PtrStepSzb& query, const PtrStepSzb& train, const PtrStepSzb& mask,
                                                    const PtrStepSzi& trainIdx, const PtrStepSzf& distance,
                                                    hipStream_t stream)
        {
            if (mask.data)
            {
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), SingleMask(mask),
                    trainIdx, distance,
                    stream);
            }
            else
            {
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), static_cast< PtrStepSz<T> >(train), WithOutMask(),
                    trainIdx, distance,
                    stream);
            }
        }

        template void matchHamming_gpu<uchar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchHamming_gpu<short >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchHamming_gpu<int   >(const PtrStepSzb& queryDescs, const PtrStepSzb& trainDescs, const PtrStepSzb& mask, const PtrStepSzi& trainIdx, const PtrStepSzf& distance, hipStream_t stream);

        template <typename T> void matchL1_gpu(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks,
                                               const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                                                hipStream_t stream)
        {
            if (masks.data)
            {
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, MaskCollection(masks.data),
                    trainIdx, imgIdx, distance,
                    stream);
            }
            else
            {
                matchDispatcher< L1Dist<T> >(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, WithOutMask(),
                    trainIdx, imgIdx, distance,
                    stream);
            }
        }

        template void matchL1_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL1_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<short >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL1_gpu<float >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);

        template <typename T> void matchL2_gpu(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks,
                                               const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                                               hipStream_t stream)
        {
            if (masks.data)
            {
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, MaskCollection(masks.data),
                    trainIdx, imgIdx, distance,
                    stream);
            }
            else
            {
                matchDispatcher<L2Dist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, WithOutMask(),
                    trainIdx, imgIdx, distance,
                    stream);
            }
        }

        //template void matchL2_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<short >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchL2_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchL2_gpu<float >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& maskCollection, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);

        template <typename T> void matchHamming_gpu(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks,
                                                    const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance,
                                                    hipStream_t stream)
        {
            if (masks.data)
            {
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, MaskCollection(masks.data),
                    trainIdx, imgIdx, distance,
                    stream);
            }
            else
            {
                matchDispatcher<HammingDist>(static_cast< PtrStepSz<T> >(query), (const PtrStepSz<T>*)trains.ptr(), trains.cols, WithOutMask(),
                    trainIdx, imgIdx, distance,
                    stream);
            }
        }

        template void matchHamming_gpu<uchar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchHamming_gpu<schar >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchHamming_gpu<ushort>(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        //template void matchHamming_gpu<short >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
        template void matchHamming_gpu<int   >(const PtrStepSzb& query, const PtrStepSzb& trains, const PtrStepSz<PtrStepb>& masks, const PtrStepSzi& trainIdx, const PtrStepSzi& imgIdx, const PtrStepSzf& distance, hipStream_t stream);
    } // namespace bf_match
}}} // namespace cv { namespace cuda { namespace cudev {


#endif /* CUDA_DISABLER */
