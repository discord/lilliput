#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#if defined(HAVE_OPENCV_CUDAARITHM) && defined(HAVE_OPENCV_CUDAWARPING) && defined(HAVE_OPENCV_CUDAFILTERS)

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace btv_l1_cudev
{
    void buildMotionMaps(PtrStepSzf forwardMotionX, PtrStepSzf forwardMotionY,
                         PtrStepSzf backwardMotionX, PtrStepSzf bacwardMotionY,
                         PtrStepSzf forwardMapX, PtrStepSzf forwardMapY,
                         PtrStepSzf backwardMapX, PtrStepSzf backwardMapY);

    template <int cn>
    void upscale(const PtrStepSzb src, PtrStepSzb dst, int scale, hipStream_t stream);

    void diffSign(PtrStepSzf src1, PtrStepSzf src2, PtrStepSzf dst, hipStream_t stream);

    void loadBtvWeights(const float* weights, size_t count);
    template <int cn> void calcBtvRegularization(PtrStepSzb src, PtrStepSzb dst, int ksize);
}

namespace btv_l1_cudev
{
    __global__ void buildMotionMapsKernel(const PtrStepSzf forwardMotionX, const PtrStepf forwardMotionY,
                                          PtrStepf backwardMotionX, PtrStepf backwardMotionY,
                                          PtrStepf forwardMapX, PtrStepf forwardMapY,
                                          PtrStepf backwardMapX, PtrStepf backwardMapY)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= forwardMotionX.cols || y >= forwardMotionX.rows)
            return;

        const float fx = forwardMotionX(y, x);
        const float fy = forwardMotionY(y, x);

        const float bx = backwardMotionX(y, x);
        const float by = backwardMotionY(y, x);

        forwardMapX(y, x) = x + bx;
        forwardMapY(y, x) = y + by;

        backwardMapX(y, x) = x + fx;
        backwardMapY(y, x) = y + fy;
    }

    void buildMotionMaps(PtrStepSzf forwardMotionX, PtrStepSzf forwardMotionY,
                         PtrStepSzf backwardMotionX, PtrStepSzf bacwardMotionY,
                         PtrStepSzf forwardMapX, PtrStepSzf forwardMapY,
                         PtrStepSzf backwardMapX, PtrStepSzf backwardMapY)
    {
        const dim3 block(32, 8);
        const dim3 grid(divUp(forwardMapX.cols, block.x), divUp(forwardMapX.rows, block.y));

        buildMotionMapsKernel<<<grid, block>>>(forwardMotionX, forwardMotionY,
                                               backwardMotionX, bacwardMotionY,
                                               forwardMapX, forwardMapY,
                                               backwardMapX, backwardMapY);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T>
    __global__ void upscaleKernel(const PtrStepSz<T> src, PtrStep<T> dst, const int scale)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= src.cols || y >= src.rows)
            return;

        dst(y * scale, x * scale) = src(y, x);
    }

    template <int cn>
    void upscale(const PtrStepSzb src, PtrStepSzb dst, int scale, hipStream_t stream)
    {
        typedef typename TypeVec<float, cn>::vec_type src_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

        upscaleKernel<src_t><<<grid, block, 0, stream>>>((PtrStepSz<src_t>) src, (PtrStepSz<src_t>) dst, scale);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void upscale<1>(const PtrStepSzb src, PtrStepSzb dst, int scale, hipStream_t stream);
    template void upscale<3>(const PtrStepSzb src, PtrStepSzb dst, int scale, hipStream_t stream);
    template void upscale<4>(const PtrStepSzb src, PtrStepSzb dst, int scale, hipStream_t stream);

    __device__ __forceinline__ float diffSign(float a, float b)
    {
        return a > b ? 1.0f : a < b ? -1.0f : 0.0f;
    }
    __device__ __forceinline__ float3 diffSign(const float3& a, const float3& b)
    {
        return make_float3(
            a.x > b.x ? 1.0f : a.x < b.x ? -1.0f : 0.0f,
            a.y > b.y ? 1.0f : a.y < b.y ? -1.0f : 0.0f,
            a.z > b.z ? 1.0f : a.z < b.z ? -1.0f : 0.0f
        );
    }
    __device__ __forceinline__ float4 diffSign(const float4& a, const float4& b)
    {
        return make_float4(
            a.x > b.x ? 1.0f : a.x < b.x ? -1.0f : 0.0f,
            a.y > b.y ? 1.0f : a.y < b.y ? -1.0f : 0.0f,
            a.z > b.z ? 1.0f : a.z < b.z ? -1.0f : 0.0f,
            0.0f
        );
    }

    struct DiffSign : binary_function<float, float, float>
    {
        __device__ __forceinline__ float operator ()(float a, float b) const
        {
            return diffSign(a, b);
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<btv_l1_cudev::DiffSign> : DefaultTransformFunctorTraits<btv_l1_cudev::DiffSign>
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
}}}

namespace btv_l1_cudev
{
    void diffSign(PtrStepSzf src1, PtrStepSzf src2, PtrStepSzf dst, hipStream_t stream)
    {
        transform(src1, src2, dst, DiffSign(), WithOutMask(), stream);
    }

    __constant__ float c_btvRegWeights[16*16];

    template <typename T>
    __global__ void calcBtvRegularizationKernel(const PtrStepSz<T> src, PtrStep<T> dst, const int ksize)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x + ksize;
        const int y = blockIdx.y * blockDim.y + threadIdx.y + ksize;

        if (y >= src.rows - ksize || x >= src.cols - ksize)
            return;

        const T srcVal = src(y, x);

        T dstVal = VecTraits<T>::all(0);

        for (int m = 0, count = 0; m <= ksize; ++m)
        {
            for (int l = ksize; l + m >= 0; --l, ++count)
                dstVal = dstVal + c_btvRegWeights[count] * (diffSign(srcVal, src(y + m, x + l)) - diffSign(src(y - m, x - l), srcVal));
        }

        dst(y, x) = dstVal;
    }

    void loadBtvWeights(const float* weights, size_t count)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_btvRegWeights), weights, count * sizeof(float)) );
    }

    template <int cn>
    void calcBtvRegularization(PtrStepSzb src, PtrStepSzb dst, int ksize)
    {
        typedef typename TypeVec<float, cn>::vec_type src_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

        calcBtvRegularizationKernel<src_t><<<grid, block>>>((PtrStepSz<src_t>) src, (PtrStepSz<src_t>) dst, ksize);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }

    template void calcBtvRegularization<1>(PtrStepSzb src, PtrStepSzb dst, int ksize);
    template void calcBtvRegularization<3>(PtrStepSzb src, PtrStepSzb dst, int ksize);
    template void calcBtvRegularization<4>(PtrStepSzb src, PtrStepSzb dst, int ksize);
}

#endif
